/*
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "join.h"

size_t int_size = sizeof(int_type);
size_t float_size = sizeof(float_type);

template <typename HeadFlagType>
struct join_head_flag_predicate
        : public thrust::binary_function<HeadFlagType,HeadFlagType,bool>
{
    __host__ __device__
    bool operator()(HeadFlagType left, HeadFlagType right) const
    {
        return !right;
    }
};

template <typename HeadFlagType>
struct join_head_flag_predicate1
        : public thrust::binary_function<HeadFlagType,HeadFlagType,bool>
{
    __host__ __device__
    bool operator()(HeadFlagType left, HeadFlagType right) const
    {
        return (right == 1);
    }
};

struct minus2 : public thrust::unary_function<unsigned int, unsigned int>
{
    __host__ __device__
    unsigned int operator()(unsigned int x) {
        return x-2;
    }
};

struct is_zero : public thrust::unary_function<unsigned int, unsigned int>
{
    __host__ __device__
    unsigned int operator()(unsigned int x) {
        return x == 0;
    }
};




struct join_functor
{

    const unsigned int * lowerbound;
    const unsigned int * upperbound;
    const unsigned int * address;
    unsigned int * output;
    unsigned int * output1;

    join_functor(const unsigned int * _lowerbound, const unsigned int * _upperbound,
                 const unsigned int * _address, unsigned int * _output, unsigned int * _output1):
        lowerbound(_lowerbound), upperbound(_upperbound),
        address(_address), output(_output), output1(_output1) {}

    template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {

        if (upperbound[i] != lowerbound[i]) {
            output[address[i] ] = i;
            output1[address[i] ] = lowerbound[i]+2;
        };

    }
};


unsigned int join(int_type* d_input,int_type* d_values,
          thrust::device_vector<unsigned int>& d_res1, thrust::device_vector<unsigned int>& d_res2,
          unsigned int bRecCount, unsigned int aRecCount, bool left_join, searchEngine_t engine, CuDeviceMem* btree)
{
    thrust::device_ptr<unsigned int> d_output1 = thrust::device_malloc<unsigned int>(bRecCount);
    thrust::device_ptr<int_type> d_i(d_input);
    thrust::device_ptr<int_type> d_v(d_values);


    thrust::counting_iterator<unsigned int, thrust::device_space_tag> begin(0);
    thrust::device_ptr<unsigned int> d_output = thrust::device_malloc<unsigned int>(bRecCount);
		
						
	searchStatus_t status = searchKeys(engine, aRecCount, SEARCH_TYPE_INT64, 
                            		  (hipDeviceptr_t)d_input, SEARCH_ALGO_LOWER_BOUND,
			                          (hipDeviceptr_t)d_values, bRecCount, btree->Handle(),
			                          (hipDeviceptr_t)thrust::raw_pointer_cast(d_output));
			
    thrust::device_ptr<unsigned int> d_output2 = thrust::device_malloc<unsigned int>(bRecCount);

    status = searchKeys(engine, aRecCount, SEARCH_TYPE_INT64, 
	                    (hipDeviceptr_t)d_input, SEARCH_ALGO_UPPER_BOUND,
		                (hipDeviceptr_t)d_values, bRecCount, btree->Handle(),
		                (hipDeviceptr_t)thrust::raw_pointer_cast(d_output1));							

    thrust::transform(d_output1, d_output1+bRecCount, d_output, d_output2, thrust::minus<unsigned int>());
    unsigned int sz =  thrust::reduce(d_output2, d_output2+bRecCount, 0, thrust::plus<unsigned int>());
		
	unsigned int left_sz = 0;
	if (left_join) {
		left_sz = thrust::count(d_output2, d_output2+bRecCount,0);			
	};	
    d_res1.resize(sz + left_sz);
    d_res2.resize(sz);

	if (left_join && left_sz) {
		thrust::copy_if(thrust::make_counting_iterator((unsigned int)0), thrust::make_counting_iterator(bRecCount-1), d_output2, d_res1.begin()+ sz, is_zero() );
	};		
		
    thrust::exclusive_scan(d_output2, d_output2+bRecCount, d_output2);  // addresses

    if (sz != 0 ) {

        thrust::fill(d_res1.begin(), d_res1.begin() + sz, (unsigned int)0);
        thrust::fill(d_res2.begin(), d_res2.end(), (unsigned int)1);

        join_functor ff(thrust::raw_pointer_cast(d_output),
                        thrust::raw_pointer_cast(d_output1),
                        thrust::raw_pointer_cast(d_output2),
                        thrust::raw_pointer_cast(&d_res1[0]),
                        thrust::raw_pointer_cast(&d_res2[0]));


        thrust::for_each(begin, begin + bRecCount, ff);
		
        thrust::inclusive_scan_by_key(d_res1.begin(), d_res1.begin() + sz, d_res1.begin(), d_res1.begin(), join_head_flag_predicate<unsigned int>(), thrust::maximum<unsigned int>()); // in-place scan
        thrust::inclusive_scan_by_key(d_res2.begin(), d_res2.end(), d_res2.begin(), d_res2.begin(), join_head_flag_predicate1<unsigned int>(), thrust::plus<unsigned int>()); // in-place scan
		
        thrust::transform(d_res2.begin(), d_res2.end(), d_res2.begin(), minus2());
		
    }
    thrust::device_free(d_output2);
    thrust::device_free(d_output);    
    thrust::device_free(d_output1);
	return left_sz;
}



