/*
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */


#include <thrust/device_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/binary_search.h>
#include <thrust/reduce.h>
#include <thrust/fill.h>
#include <thrust/scan.h>
#include <thrust/device_ptr.h>

template <typename HeadFlagType>
struct join_head_flag_predicate
        : public thrust::binary_function<HeadFlagType,HeadFlagType,bool>
{
    __host__ __device__
    bool operator()(HeadFlagType left, HeadFlagType right) const
    {
        return !right;
    }
};

template <typename HeadFlagType>
struct join_head_flag_predicate1
        : public thrust::binary_function<HeadFlagType,HeadFlagType,bool>
{
    __host__ __device__
    bool operator()(HeadFlagType left, HeadFlagType right) const
    {
        return (right == 1);
    }
};

struct minus2 : public thrust::unary_function<unsigned int, unsigned int>
{
    __host__ __device__
    unsigned int operator()(unsigned int x) {
        return x-2;
    }
};




struct join_functor
{

    const unsigned int * lowerbound;
    const unsigned int * upperbound;
    const unsigned int * address;
    unsigned int * output;
    unsigned int * output1;

    join_functor(const unsigned int * _lowerbound, const unsigned int * _upperbound,
                 const unsigned int * _address, unsigned int * _output, unsigned int * _output1):
        lowerbound(_lowerbound), upperbound(_upperbound),
        address(_address), output(_output), output1(_output1) {}

    template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {

        //    for(unsigned int j = 0; j < upperbound[i]-lowerbound[i]; j++)
        //  {
        //output[address[i] + j] = i;
        //output1[address[i] + j] = lowerbound[i]+j;
        if (upperbound[i] != lowerbound[i]) {
            output[address[i] ] = i;
            output1[address[i] ] = lowerbound[i]+2;
        };

        //};
    }
};


void join(int_type* d_input,int_type* d_values,
          thrust::device_vector<unsigned int>& d_res1, thrust::device_vector<unsigned int>& d_res2,
          unsigned int bRecCount, unsigned int aRecCount, bool isUnique)
{


    //float time;
    //hipEvent_t start, stop;


//hipEventCreate(&start);
//hipEventCreate(&stop) ;



    thrust::device_ptr<unsigned int> d_output1 = thrust::device_malloc<unsigned int>(bRecCount);

    thrust::device_ptr<int_type> d_i(d_input);
    thrust::device_ptr<int_type> d_v(d_values);

    if (!isUnique) {

        thrust::device_ptr<unsigned int> d_output = thrust::device_malloc<unsigned int>(bRecCount);
        thrust::lower_bound(d_i, d_i+aRecCount,
                            d_v, d_v+bRecCount,
                            d_output);

        thrust::device_ptr<unsigned int> d_output2 = thrust::device_malloc<unsigned int>(bRecCount);

        thrust::upper_bound(d_i, d_i+aRecCount,
                            d_v, d_v+bRecCount,
                            d_output1);



        thrust::transform(d_output1, d_output1+bRecCount, d_output, d_output2, thrust::minus<unsigned int>());

        unsigned int sz =  thrust::reduce(d_output2, d_output2+bRecCount, 0, thrust::plus<unsigned int>());

        //    cout << "join end " << sz << endl;

        thrust::exclusive_scan(d_output2, d_output2+bRecCount, d_output2);  // addresses

        thrust::counting_iterator<unsigned int, thrust::device_space_tag> begin(0);

        d_res1.resize(sz);
        d_res2.resize(sz);

        if (sz != 0 ) {

            thrust::fill(d_res1.begin(), d_res1.end(), (unsigned int)0);
            thrust::fill(d_res2.begin(), d_res2.end(), (unsigned int)1);

            join_functor ff(thrust::raw_pointer_cast(d_output),
                            thrust::raw_pointer_cast(d_output1),
                            thrust::raw_pointer_cast(d_output2),
                            thrust::raw_pointer_cast(&d_res1[0]),
                            thrust::raw_pointer_cast(&d_res2[0]));


            thrust::for_each(begin, begin + bRecCount, ff);


            thrust::inclusive_scan_by_key(d_res1.begin(), d_res1.end(), d_res1.begin(), d_res1.begin(), join_head_flag_predicate<unsigned int>(), thrust::maximum<unsigned int>()); // in-place scan
            thrust::inclusive_scan_by_key(d_res2.begin(), d_res2.end(), d_res2.begin(), d_res2.begin(), join_head_flag_predicate1<unsigned int>(), thrust::plus<unsigned int>()); // in-place scan
            thrust::transform(d_res2.begin(), d_res2.end(), d_res2.begin(), minus2());

        };
        thrust::device_free(d_output2);
        thrust::device_free(d_output);
    }
    else {  // DW style join with unique dimension keys
	
	//hipEventRecord(start, 0) ;

        thrust::binary_search(d_i, d_i+aRecCount,
                              d_v, d_v+bRecCount,
                              d_output1);

      //  hipEventRecord(stop, 0) ;
      //  hipEventSynchronize(stop) ;
      //  hipEventElapsedTime(&time, start, stop);
      //  hipEventRecord(start, 0);

        //printf("Time BinS:  %3.1f ms \n", time);
        	

        unsigned int sz =  thrust::reduce(d_output1, d_output1+bRecCount);
        d_res1.resize(sz);
        d_res2.resize(sz);

        if(sz) {
            thrust::counting_iterator<unsigned int> seq(0);
            thrust::copy_if(seq,seq+bRecCount,d_output1,d_res1.begin(),thrust::identity<unsigned int>());

            thrust::device_ptr<unsigned int> d_output = thrust::device_malloc<unsigned int>(bRecCount);
            thrust::lower_bound(d_i, d_i+aRecCount, d_v, d_v+bRecCount,  d_output);

            thrust::copy_if(d_output,d_output+bRecCount,d_output1,d_res2.begin(),thrust::identity<unsigned int>());
            thrust::device_free(d_output);
        };

        //cout << "fast join end " << sz << endl;
    };
    thrust::device_free(d_output1);
}



