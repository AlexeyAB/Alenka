/*
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */


#include <thrust/device_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/binary_search.h>
#include <thrust/reduce.h>
#include <thrust/fill.h>
#include <thrust/scan.h>
#include <thrust/device_ptr.h>


template <typename HeadFlagType>
struct join_head_flag_predicate
        : public thrust::binary_function<HeadFlagType,HeadFlagType,bool>
{
    __host__ __device__
    bool operator()(HeadFlagType left, HeadFlagType right) const
    {
        return !right;
    }
};

template <typename HeadFlagType>
struct join_head_flag_predicate1
        : public thrust::binary_function<HeadFlagType,HeadFlagType,bool>
{
    __host__ __device__
    bool operator()(HeadFlagType left, HeadFlagType right) const
    {
        return (right == 1);
    }
};

struct minus2 : public thrust::unary_function<unsigned int, unsigned int>
{
  __host__ __device__
  unsigned int operator()(unsigned int x) { return x-2; }
};




struct nzj
{
    __host__ __device__
    bool operator()(const unsigned int x)
    {
        return (x != 0);
    }
};


struct join_functor
{

    const unsigned int * lowerbound;
    const unsigned int * upperbound;
    const unsigned int * address;
    unsigned int * output;
    unsigned int * output1;

    join_functor(const unsigned int * _lowerbound, const unsigned int * _upperbound,
                 const unsigned int * _address, unsigned int * _output, unsigned int * _output1):
        lowerbound(_lowerbound), upperbound(_upperbound),
        address(_address), output(_output), output1(_output1) {}

    template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {

    //    for(unsigned int j = 0; j < upperbound[i]-lowerbound[i]; j++)
      //  {
            //output[address[i] + j] = i;
            //output1[address[i] + j] = lowerbound[i]+j;
			if (upperbound[i] != lowerbound[i]) {
                output[address[i] ] = i;
                output1[address[i] ] = lowerbound[i]+2;
			};	

        //};
    }
};


void join(int_type* d_input,int_type* d_values,
          thrust::device_vector<unsigned int>& d_res1, thrust::device_vector<unsigned int>& d_res2,
          unsigned int bRecCount, unsigned int aRecCount, bool isUnique)
{

	
	//float time;
    //hipEvent_t start, stop;
	
	
 //hipEventCreate(&start);
 //hipEventCreate(&stop) ;
 //hipEventRecord(start, 0) ;


    
    thrust::device_ptr<unsigned int> d_output1 = thrust::device_malloc<unsigned int>(bRecCount);   	
		
    thrust::device_ptr<int_type> d_i(d_input);
    thrust::device_ptr<int_type> d_v(d_values);

    if (!isUnique) {							
	
	    thrust::device_ptr<unsigned int> d_output = thrust::device_malloc<unsigned int>(bRecCount);
        thrust::lower_bound(d_i, d_i+aRecCount,
                            d_v, d_v+bRecCount,
                            d_output);				
	
	    thrust::device_ptr<unsigned int> d_output2 = thrust::device_malloc<unsigned int>(bRecCount);
		
        thrust::upper_bound(d_i, d_i+aRecCount,
                            d_v, d_v+bRecCount,
                            d_output1);
						
				
	
        thrust::transform(d_output1, d_output1+bRecCount, d_output, d_output2, thrust::minus<unsigned int>());	
	
	    unsigned int sz =  thrust::reduce(d_output2, d_output2+bRecCount, 0, thrust::plus<unsigned int>());	

	//    cout << "join end " << sz << endl;	

        thrust::exclusive_scan(d_output2, d_output2+bRecCount, d_output2);  // addresses		
		
        thrust::counting_iterator<unsigned int, thrust::device_space_tag> begin(0);
	
        d_res1.resize(sz);
        d_res2.resize(sz);
	
	    if (sz != 0 ) {
	
	        thrust::fill(d_res1.begin(), d_res1.end(), (unsigned int)0);
	        thrust::fill(d_res2.begin(), d_res2.end(), (unsigned int)1);
		
            join_functor ff(thrust::raw_pointer_cast(d_output),
                            thrust::raw_pointer_cast(d_output1),
                            thrust::raw_pointer_cast(d_output2),
                            thrust::raw_pointer_cast(&d_res1[0]),
                            thrust::raw_pointer_cast(&d_res2[0]));


            thrust::for_each(begin, begin + bRecCount, ff);	
	
	
	        thrust::inclusive_scan_by_key(d_res1.begin(), d_res1.end(), d_res1.begin(), d_res1.begin(), join_head_flag_predicate<unsigned int>(), thrust::maximum<unsigned int>()); // in-place scan
            thrust::inclusive_scan_by_key(d_res2.begin(), d_res2.end(), d_res2.begin(), d_res2.begin(), join_head_flag_predicate1<unsigned int>(), thrust::plus<unsigned int>()); // in-place scan
	        thrust::transform(d_res2.begin(), d_res2.end(), d_res2.begin(), minus2());		
		
	    };	
		thrust::device_free(d_output2);	
		thrust::device_free(d_output);	
	}
    else {  // DW style join with unique dimension keys
  	    
        thrust::binary_search(d_i, d_i+aRecCount,
                              d_v, d_v+bRecCount,
                              d_output1);		
							
	/*hipEventRecord(stop, 0) ;
hipEventSynchronize(stop) ;
 hipEventElapsedTime(&time, start, stop);
 hipEventRecord(start, 0);

printf("Time to generate2:  %3.1f ms \n", time);
		*/					
							
        unsigned int sz =  thrust::reduce(d_output1, d_output1+bRecCount);								
        d_res1.resize(sz);
        d_res2.resize(sz);
		


		
		if(sz) { 		
		    thrust::counting_iterator<unsigned int> seq(0);
		    thrust::copy_if(seq,seq+bRecCount,d_output1,d_res1.begin(),nzj());				
		
	        thrust::device_ptr<unsigned int> d_output = thrust::device_malloc<unsigned int>(bRecCount);
            thrust::lower_bound(d_i, d_i+aRecCount, d_v, d_v+bRecCount,  d_output);						
		
		    thrust::copy_if(d_output,d_output+bRecCount,d_output1,d_res2.begin(),nzj());
			thrust::device_free(d_output);	
		};			

        cout << "fast join end " << sz << endl;
    };		
    
    thrust::device_free(d_output1);	
    

}





void join(CudaChar* d_input,CudaChar* d_values,
          thrust::device_vector<unsigned int>& d_res1, thrust::device_vector<unsigned int>& d_res2)
{

// should be similar to joining numeric columns except that we shoud do it repeatedly for all columns in a varchar variable

    thrust::device_ptr<unsigned int> d_output = thrust::device_malloc<unsigned int>(d_values->mRecCount);
    thrust::device_ptr<unsigned int> d_output1 = thrust::device_malloc<unsigned int>(d_values->mRecCount);
    thrust::device_ptr<unsigned int> d_output2 = thrust::device_malloc<unsigned int>(d_values->mRecCount);


    thrust::device_ptr<char> d_i((d_input->d_columns)[0]);
    thrust::device_ptr<char> d_v((d_values->d_columns)[0]);

    thrust::lower_bound(d_i, d_i + d_input->mRecCount,
                        d_v, d_v+ d_values->mRecCount,
                        d_output);

    thrust::upper_bound(d_i, d_i + d_input->mRecCount,
                        d_v, d_v+ d_values->mRecCount,
                        d_output1);


    thrust::transform(d_output1, d_output1+d_values->mRecCount, d_output, d_output2, thrust::minus<unsigned int>());

    int sz =  thrust::reduce(d_output2, d_output2+d_values->mRecCount, 0);

    thrust::exclusive_scan(d_output2, d_output2+d_values->mRecCount, d_output2);  // addresses

    thrust::counting_iterator<unsigned int, thrust::device_space_tag> begin(0);

    d_res1.resize(sz);
    d_res2.resize(sz);

    join_functor ff(thrust::raw_pointer_cast(d_output),
                           thrust::raw_pointer_cast(d_output1),
                           thrust::raw_pointer_cast(d_output2),
                           thrust::raw_pointer_cast(&d_res1[0]),
                           thrust::raw_pointer_cast(&d_res2[0]));

    thrust::for_each(begin, begin + d_values->mRecCount, ff);

    thrust::device_free(d_output);
    thrust::device_free(d_output1);
    thrust::device_free(d_output2);

// gather and compare the rest of varchar columns

    int cc;

    if(d_input->mColumnCount > d_values->mColumnCount)
        cc = d_input->mColumnCount;
    else
        cc = d_values->mColumnCount;


    thrust::device_ptr<char> d_blanks;
    if(d_input->mColumnCount != d_values->mColumnCount) {
        d_blanks = thrust::device_malloc<char>(sz);
        thrust::sequence(d_blanks, d_blanks+sz, 0, 0);
    };

    int new_sz = sz;


    for(int i=1; i< cc; i++) {
        thrust::device_ptr<char> d_ii;
        if (d_input->mColumnCount >= i+1)
            d_ii = thrust::device_pointer_cast((d_input->d_columns)[i]); // check if columns actually exist
        else
            d_ii = thrust::device_pointer_cast(d_blanks);

        thrust::device_ptr<char> d_vv;
        if (d_values->mColumnCount >= i+1)
            d_vv = thrust::device_pointer_cast((d_values->d_columns)[i]);
        else
            d_vv = thrust::device_pointer_cast(d_blanks);

        thrust::device_ptr<char> d_out = thrust::device_malloc<char>(new_sz);
        thrust::device_ptr<char> d_out1 = thrust::device_malloc<char>(new_sz);
        thrust::device_ptr<unsigned int> v = thrust::device_malloc<unsigned int>(new_sz);

        thrust::gather(d_res2.begin(), d_res2.end(), d_ii, d_out);
        thrust::gather(d_res1.begin(), d_res1.end(), d_vv, d_out1);

        thrust::transform(d_out, d_out+new_sz, d_out1, v, thrust::equal_to<char>());
        new_sz =  thrust::reduce(v, v+new_sz, 0);

        thrust::device_ptr<unsigned int> res1_tmp = thrust::device_malloc<unsigned int>(new_sz);
        thrust::device_ptr<unsigned int> res2_tmp = thrust::device_malloc<unsigned int>(new_sz);

        thrust::copy_if(d_res1.begin(),d_res1.end(), v, res1_tmp, nzj());
        thrust::copy_if(d_res2.begin(),d_res2.end(), v, res2_tmp, nzj());

        d_res1.resize(new_sz);
        d_res2.resize(new_sz);

        thrust::copy(res1_tmp, res1_tmp+new_sz, d_res1.begin());
        thrust::copy(res2_tmp, res2_tmp+new_sz, d_res2.begin());

    };


}
