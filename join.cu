/*
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "join.h"

size_t int_size = sizeof(int_type);
size_t float_size = sizeof(float_type);

template <typename HeadFlagType>
struct join_head_flag_predicate
        : public thrust::binary_function<HeadFlagType,HeadFlagType,bool>
{
    __host__ __device__
    bool operator()(HeadFlagType left, HeadFlagType right) const
    {
        return !right;
    }
};

template <typename HeadFlagType>
struct join_head_flag_predicate1
        : public thrust::binary_function<HeadFlagType,HeadFlagType,bool>
{
    __host__ __device__
    bool operator()(HeadFlagType left, HeadFlagType right) const
    {
        return (right == 1);
    }
};

struct minus2 : public thrust::unary_function<unsigned int, unsigned int>
{
    __host__ __device__
    unsigned int operator()(unsigned int x) {
        return x-2;
    }
};

struct is_zero : public thrust::unary_function<unsigned int, unsigned int>
{
    __host__ __device__
    unsigned int operator()(unsigned int x) {
        return x == 0;
    }
};




struct join_functor
{

    const unsigned int * lowerbound;
    const unsigned int * upperbound;
    const unsigned int * address;
    unsigned int * output;
    unsigned int * output1;

    join_functor(const unsigned int * _lowerbound, const unsigned int * _upperbound,
                 const unsigned int * _address, unsigned int * _output, unsigned int * _output1):
        lowerbound(_lowerbound), upperbound(_upperbound),
        address(_address), output(_output), output1(_output1) {}

    template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {

        if (upperbound[i] != lowerbound[i]) {
            output[address[i] ] = i;
            output1[address[i] ] = lowerbound[i]+2;
        };

    }
};


unsigned int join(int_type* right,int_type* left,
          thrust::device_vector<unsigned int>& d_res1, thrust::device_vector<unsigned int>& d_res2,
          unsigned int cnt_l, unsigned int cnt_r, bool left_join)
{
    thrust::device_ptr<unsigned int> d_output1 = thrust::device_malloc<unsigned int>(cnt_l);
    thrust::device_ptr<int_type> d_i(right);
    thrust::device_ptr<int_type> d_v(left);	
	
    thrust::counting_iterator<unsigned int, thrust::device_space_tag> begin(0);
    thrust::device_ptr<unsigned int> d_output = thrust::device_malloc<unsigned int>(cnt_l);
		
	/*searchStatus_t status = searchKeys(engine, cnt_r, SEARCH_TYPE_INT64, 
                            		  (hipDeviceptr_t)right, SEARCH_ALGO_LOWER_BOUND,
			                          (hipDeviceptr_t)left, cnt_l, btree->Handle(),
			                          (hipDeviceptr_t)thrust::raw_pointer_cast(d_output));
	*/		
	        thrust::lower_bound(d_i, d_i+cnt_r,
                            d_v, d_v+cnt_l,
                            d_output);


    			
    thrust::device_ptr<unsigned int> d_output2 = thrust::device_malloc<unsigned int>(cnt_l);

    /*status = searchKeys(engine, cnt_r, SEARCH_TYPE_INT64, 
	                    (hipDeviceptr_t)right, SEARCH_ALGO_UPPER_BOUND,
		                (hipDeviceptr_t)left, cnt_l, btree->Handle(),
		                (hipDeviceptr_t)thrust::raw_pointer_cast(d_output1));		
	*/					
	
       thrust::upper_bound(d_i, d_i+cnt_r,
                            d_v, d_v+cnt_l,
                            d_output1);
		

    thrust::transform(d_output1, d_output1+cnt_l, d_output, d_output2, thrust::minus<unsigned int>());
    unsigned int sz =  thrust::reduce(d_output2, d_output2+cnt_l, 0, thrust::plus<unsigned int>());
		
	unsigned int left_sz = 0;
	if (left_join) {
		left_sz = thrust::count(d_output2, d_output2+cnt_l,0);			
	};	
    d_res1.resize(sz + left_sz);
    d_res2.resize(sz);

	if (left_join && left_sz) {
		thrust::copy_if(thrust::make_counting_iterator((unsigned int)0), thrust::make_counting_iterator(cnt_l-1), d_output2, d_res1.begin()+ sz, is_zero() );
	};		
		
    thrust::exclusive_scan(d_output2, d_output2+cnt_l, d_output2);  // addresses

    if (sz != 0 ) {

        thrust::fill(d_res1.begin(), d_res1.begin() + sz, (unsigned int)0);
        thrust::fill(d_res2.begin(), d_res2.end(), (unsigned int)1);

        join_functor ff(thrust::raw_pointer_cast(d_output),
                        thrust::raw_pointer_cast(d_output1),
                        thrust::raw_pointer_cast(d_output2),
                        thrust::raw_pointer_cast(&d_res1[0]),
                        thrust::raw_pointer_cast(&d_res2[0]));


        thrust::for_each(begin, begin + cnt_l, ff);
		
        thrust::inclusive_scan_by_key(d_res1.begin(), d_res1.begin() + sz, d_res1.begin(), d_res1.begin(), join_head_flag_predicate<unsigned int>(), thrust::maximum<unsigned int>()); // in-place scan
        thrust::inclusive_scan_by_key(d_res2.begin(), d_res2.end(), d_res2.begin(), d_res2.begin(), join_head_flag_predicate1<unsigned int>(), thrust::plus<unsigned int>()); // in-place scan
		
        thrust::transform(d_res2.begin(), d_res2.end(), d_res2.begin(), minus2());
		
    }
    thrust::device_free(d_output2);
    thrust::device_free(d_output);    
    thrust::device_free(d_output1);
	return left_sz;
}



