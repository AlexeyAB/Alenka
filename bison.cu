#include "hip/hip_runtime.h"

/* A Bison parser, made by GNU Bison 2.4.1.  */

/* Skeleton implementation for Bison's Yacc-like parsers in C
   
      Copyright (C) 1984, 1989, 1990, 2000, 2001, 2002, 2003, 2004, 2005, 2006
   Free Software Foundation, Inc.
   
   This program is free software: you can redistribute it and/or modify
   it under the terms of the GNU General Public License as published by
   the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.
   
   This program is distributed in the hope that it will be useful,
   but WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU General Public License for more details.
   
   You should have received a copy of the GNU General Public License
   along with this program.  If not, see <http://www.gnu.org/licenses/>.  */

/* As a special exception, you may create a larger work that contains
   part or all of the Bison parser skeleton and distribute that work
   under terms of your choice, so long as that work isn't itself a
   parser generator using the skeleton or a modified version thereof
   as a parser skeleton.  Alternatively, if you modify or redistribute
   the parser skeleton itself, you may (at your option) remove this
   special exception, which will cause the skeleton and the resulting
   Bison output files to be licensed under the GNU General Public
   License without this special exception.
   
   This special exception was added by the Free Software Foundation in
   version 2.2 of Bison.  */

/* C LALR(1) parser skeleton written by Richard Stallman, by
   simplifying the original so-called "semantic" parser.  */

/* All symbols defined below should begin with yy or YY, to avoid
   infringing on user name space.  This should be done even for local
   variables, as they might otherwise be expanded by user macros.
   There are some unavoidable exceptions within include files to
   define necessary library symbols; they are noted "INFRINGES ON
   USER NAME SPACE" below.  */

/* Identify Bison output.  */
#define YYBISON 1

/* Bison version.  */
#define YYBISON_VERSION "2.4.1"

/* Skeleton name.  */
#define YYSKELETON_NAME "yacc.c"

/* Pure parsers.  */
#define YYPURE 0

/* Push parsers.  */
#define YYPUSH 0

/* Pull parsers.  */
#define YYPULL 1

/* Using locations.  */
#define YYLSP_NEEDED 0



/* Copy the first part of user declarations.  */

/* Line 189 of yacc.c  */
#line 17 "bison.y"


#include "lex.yy.c"
#include "cm.h"

    void clean_queues();
    void order_inplace(CudaSet* a, stack<string> exe_type);
    void yyerror(char *s, ...);
    void emit(char *s, ...);
    void emit_mul();
    void emit_add();
    void emit_minus();
    void emit_distinct();
    void emit_div();
    void emit_and();
    void emit_eq();
    void emit_or();
    void emit_cmp(int val);
    void emit_var(char *s, int c, char *f);
    void emit_var_asc(char *s);
    void emit_var_desc(char *s);
    void emit_name(char *name);
    void emit_count();
    void emit_sum();
    void emit_average();
    void emit_min();
    void emit_max();
    void emit_string(char *str);
    void emit_number(int_type val);
    void emit_float(float_type val);
    void emit_decimal(float_type val);
    void emit_sel_name(char* name);
    void emit_limit(int val);
    void emit_union(char *s, char *f1, char *f2);
    void emit_varchar(char *s, int c, char *f, int d);
    void emit_load(char *s, char *f, int d, char* sep);
    void emit_load_binary(char *s, char *f, int d);
    void emit_store(char *s, char *f, char* sep);
    void emit_store_binary(char *s, char *f, char* sep);
    void emit_store_binary(char *s, char *f);
    void emit_filter(char *s, char *f, int e);
    void emit_order(char *s, char *f, int e, int ll = 0);
    void emit_group(char *s, char *f, int e);
    void emit_select(char *s, char *f, int ll);
    void emit_join(char *s, char *j1, int grp);
    void emit_join_tab(char *s, bool left);
    void emit_distinct();



/* Line 189 of yacc.c  */
#line 124 "bison.cu"

/* Enabling traces.  */
#ifndef YYDEBUG
# define YYDEBUG 0
#endif

/* Enabling verbose error messages.  */
#ifdef YYERROR_VERBOSE
# undef YYERROR_VERBOSE
# define YYERROR_VERBOSE 1
#else
# define YYERROR_VERBOSE 0
#endif

/* Enabling the token table.  */
#ifndef YYTOKEN_TABLE
# define YYTOKEN_TABLE 0
#endif


/* Tokens.  */
#ifndef YYTOKENTYPE
# define YYTOKENTYPE
   /* Put the tokens into the symbol table, so that GDB and other debuggers
      know about them.  */
   enum yytokentype {
     FILENAME = 258,
     NAME = 259,
     STRING = 260,
     INTNUM = 261,
     DECIMAL1 = 262,
     BOOL1 = 263,
     APPROXNUM = 264,
     USERVAR = 265,
     ASSIGN = 266,
     EQUAL = 267,
     OR = 268,
     XOR = 269,
     AND = 270,
     DISTINCT = 271,
     REGEXP = 272,
     LIKE = 273,
     IS = 274,
     IN = 275,
     NOT = 276,
     BETWEEN = 277,
     COMPARISON = 278,
     SHIFT = 279,
     MOD = 280,
     UMINUS = 281,
     LOAD = 282,
     STREAM = 283,
     FILTER = 284,
     BY = 285,
     JOIN = 286,
     STORE = 287,
     INTO = 288,
     GROUP = 289,
     FROM = 290,
     SELECT = 291,
     AS = 292,
     ORDER = 293,
     ASC = 294,
     DESC = 295,
     COUNT = 296,
     USING = 297,
     SUM = 298,
     AVG = 299,
     MIN = 300,
     MAX = 301,
     LIMIT = 302,
     ON = 303,
     BINARY = 304,
     LEFT = 305
   };
#endif



#if ! defined YYSTYPE && ! defined YYSTYPE_IS_DECLARED
typedef union YYSTYPE
{

/* Line 214 of yacc.c  */
#line 67 "bison.y"

    int intval;
    float floatval;
    char *strval;
    int subtok;



/* Line 214 of yacc.c  */
#line 219 "bison.cu"
} YYSTYPE;
# define YYSTYPE_IS_TRIVIAL 1
# define yystype YYSTYPE /* obsolescent; will be withdrawn */
# define YYSTYPE_IS_DECLARED 1
#endif


/* Copy the second part of user declarations.  */


/* Line 264 of yacc.c  */
#line 231 "bison.cu"

#ifdef short
# undef short
#endif

#ifdef YYTYPE_UINT8
typedef YYTYPE_UINT8 yytype_uint8;
#else
typedef unsigned char yytype_uint8;
#endif

#ifdef YYTYPE_INT8
typedef YYTYPE_INT8 yytype_int8;
#elif (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
typedef signed char yytype_int8;
#else
typedef short int yytype_int8;
#endif

#ifdef YYTYPE_UINT16
typedef YYTYPE_UINT16 yytype_uint16;
#else
typedef unsigned short int yytype_uint16;
#endif

#ifdef YYTYPE_INT16
typedef YYTYPE_INT16 yytype_int16;
#else
typedef short int yytype_int16;
#endif

#ifndef YYSIZE_T
# ifdef __SIZE_TYPE__
#  define YYSIZE_T __SIZE_TYPE__
# elif defined size_t
#  define YYSIZE_T size_t
# elif ! defined YYSIZE_T && (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
#  include <stddef.h> /* INFRINGES ON USER NAME SPACE */
#  define YYSIZE_T size_t
# else
#  define YYSIZE_T unsigned int
# endif
#endif

#define YYSIZE_MAXIMUM ((YYSIZE_T) -1)

#ifndef YY_
# if YYENABLE_NLS
#  if ENABLE_NLS
#   include <libintl.h> /* INFRINGES ON USER NAME SPACE */
#   define YY_(msgid) dgettext ("bison-runtime", msgid)
#  endif
# endif
# ifndef YY_
#  define YY_(msgid) msgid
# endif
#endif

/* Suppress unused-variable warnings by "using" E.  */
#if ! defined lint || defined __GNUC__
# define YYUSE(e) ((void) (e))
#else
# define YYUSE(e) /* empty */
#endif

/* Identity function, used to suppress warnings about constant conditions.  */
#ifndef lint
# define YYID(n) (n)
#else
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static int
YYID (int yyi)
#else
static int
YYID (yyi)
    int yyi;
#endif
{
  return yyi;
}
#endif

#if ! defined yyoverflow || YYERROR_VERBOSE

/* The parser invokes alloca or malloc; define the necessary symbols.  */

# ifdef YYSTACK_USE_ALLOCA
#  if YYSTACK_USE_ALLOCA
#   ifdef __GNUC__
#    define YYSTACK_ALLOC __builtin_alloca
#   elif defined __BUILTIN_VA_ARG_INCR
#    include <alloca.h> /* INFRINGES ON USER NAME SPACE */
#   elif defined _AIX
#    define YYSTACK_ALLOC __alloca
#   elif defined _MSC_VER
#    include <malloc.h> /* INFRINGES ON USER NAME SPACE */
#    define alloca _alloca
#   else
#    define YYSTACK_ALLOC alloca
#    if ! defined _ALLOCA_H && ! defined _STDLIB_H && (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
#     include <stdlib.h> /* INFRINGES ON USER NAME SPACE */
#     ifndef _STDLIB_H
#      define _STDLIB_H 1
#     endif
#    endif
#   endif
#  endif
# endif

# ifdef YYSTACK_ALLOC
   /* Pacify GCC's `empty if-body' warning.  */
#  define YYSTACK_FREE(Ptr) do { /* empty */; } while (YYID (0))
#  ifndef YYSTACK_ALLOC_MAXIMUM
    /* The OS might guarantee only one guard page at the bottom of the stack,
       and a page size can be as small as 4096 bytes.  So we cannot safely
       invoke alloca (N) if N exceeds 4096.  Use a slightly smaller number
       to allow for a few compiler-allocated temporary stack slots.  */
#   define YYSTACK_ALLOC_MAXIMUM 4032 /* reasonable circa 2006 */
#  endif
# else
#  define YYSTACK_ALLOC YYMALLOC
#  define YYSTACK_FREE YYFREE
#  ifndef YYSTACK_ALLOC_MAXIMUM
#   define YYSTACK_ALLOC_MAXIMUM YYSIZE_MAXIMUM
#  endif
#  if (defined __cplusplus && ! defined _STDLIB_H \
       && ! ((defined YYMALLOC || defined malloc) \
	     && (defined YYFREE || defined free)))
#   include <stdlib.h> /* INFRINGES ON USER NAME SPACE */
#   ifndef _STDLIB_H
#    define _STDLIB_H 1
#   endif
#  endif
#  ifndef YYMALLOC
#   define YYMALLOC malloc
#   if ! defined malloc && ! defined _STDLIB_H && (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
void *malloc (YYSIZE_T); /* INFRINGES ON USER NAME SPACE */
#   endif
#  endif
#  ifndef YYFREE
#   define YYFREE free
#   if ! defined free && ! defined _STDLIB_H && (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
void free (void *); /* INFRINGES ON USER NAME SPACE */
#   endif
#  endif
# endif
#endif /* ! defined yyoverflow || YYERROR_VERBOSE */


#if (! defined yyoverflow \
     && (! defined __cplusplus \
	 || (defined YYSTYPE_IS_TRIVIAL && YYSTYPE_IS_TRIVIAL)))

/* A type that is properly aligned for any stack member.  */
union yyalloc
{
  yytype_int16 yyss_alloc;
  YYSTYPE yyvs_alloc;
};

/* The size of the maximum gap between one aligned stack and the next.  */
# define YYSTACK_GAP_MAXIMUM (sizeof (union yyalloc) - 1)

/* The size of an array large to enough to hold all stacks, each with
   N elements.  */
# define YYSTACK_BYTES(N) \
     ((N) * (sizeof (yytype_int16) + sizeof (YYSTYPE)) \
      + YYSTACK_GAP_MAXIMUM)

/* Copy COUNT objects from FROM to TO.  The source and destination do
   not overlap.  */
# ifndef YYCOPY
#  if defined __GNUC__ && 1 < __GNUC__
#   define YYCOPY(To, From, Count) \
      __builtin_memcpy (To, From, (Count) * sizeof (*(From)))
#  else
#   define YYCOPY(To, From, Count)		\
      do					\
	{					\
	  YYSIZE_T yyi;				\
	  for (yyi = 0; yyi < (Count); yyi++)	\
	    (To)[yyi] = (From)[yyi];		\
	}					\
      while (YYID (0))
#  endif
# endif

/* Relocate STACK from its old location to the new one.  The
   local variables YYSIZE and YYSTACKSIZE give the old and new number of
   elements in the stack, and YYPTR gives the new location of the
   stack.  Advance YYPTR to a properly aligned location for the next
   stack.  */
# define YYSTACK_RELOCATE(Stack_alloc, Stack)				\
    do									\
      {									\
	YYSIZE_T yynewbytes;						\
	YYCOPY (&yyptr->Stack_alloc, Stack, yysize);			\
	Stack = &yyptr->Stack_alloc;					\
	yynewbytes = yystacksize * sizeof (*Stack) + YYSTACK_GAP_MAXIMUM; \
	yyptr += yynewbytes / sizeof (*yyptr);				\
      }									\
    while (YYID (0))

#endif

/* YYFINAL -- State number of the termination state.  */
#define YYFINAL  8
/* YYLAST -- Last index in YYTABLE.  */
#define YYLAST   457

/* YYNTOKENS -- Number of terminals.  */
#define YYNTOKENS  68
/* YYNNTS -- Number of nonterminals.  */
#define YYNNTS  13
/* YYNRULES -- Number of rules.  */
#define YYNRULES  64
/* YYNRULES -- Number of states.  */
#define YYNSTATES  161

/* YYTRANSLATE(YYLEX) -- Bison symbol number corresponding to YYLEX.  */
#define YYUNDEFTOK  2
#define YYMAXUTOK   305

#define YYTRANSLATE(YYX)						\
  ((unsigned int) (YYX) <= YYMAXUTOK ? yytranslate[YYX] : YYUNDEFTOK)

/* YYTRANSLATE[YYLEX] -- Bison symbol number corresponding to YYLEX.  */
static const yytype_uint8 yytranslate[] =
{
       0,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,    21,     2,     2,     2,    32,    26,     2,
      61,    62,    30,    28,    67,    29,    63,    31,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,    66,    60,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,    34,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,    64,    25,    65,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     1,     2,     3,     4,
       5,     6,     7,     8,     9,    10,    11,    12,    13,    14,
      15,    16,    17,    18,    19,    20,    22,    23,    24,    27,
      33,    35,    36,    37,    38,    39,    40,    41,    42,    43,
      44,    45,    46,    47,    48,    49,    50,    51,    52,    53,
      54,    55,    56,    57,    58,    59
};

#if YYDEBUG
/* YYPRHS[YYN] -- Index of the first RHS symbol of rule number YYN in
   YYRHS.  */
static const yytype_uint16 yyprhs[] =
{
       0,     0,     3,     6,    10,    12,    20,    33,    43,    49,
      56,    65,    75,    82,    84,    88,    90,    92,    94,    96,
      98,   100,   110,   117,   120,   123,   128,   133,   138,   143,
     148,   151,   155,   159,   163,   167,   171,   175,   179,   183,
     187,   191,   195,   198,   201,   205,   211,   215,   219,   224,
     225,   229,   233,   239,   241,   245,   247,   251,   252,   254,
     257,   262,   268,   274,   275
};

/* YYRHS -- A `-1'-separated list of the rules' RHS.  */
static const yytype_int8 yyrhs[] =
{
      69,     0,    -1,    70,    60,    -1,    69,    70,    60,    -1,
      71,    -1,     4,    11,    45,    74,    44,     4,    73,    -1,
       4,    11,    36,     3,    51,    61,     3,    62,    46,    61,
      75,    62,    -1,     4,    11,    36,     3,    58,    46,    61,
      75,    62,    -1,     4,    11,    38,     4,    78,    -1,     4,
      11,    47,     4,    39,    77,    -1,     4,    11,    45,    74,
      44,     4,    79,    73,    -1,    41,     4,    42,     3,    51,
      61,     3,    62,    80,    -1,    41,     4,    42,     3,    80,
      58,    -1,     4,    -1,     4,    63,     4,    -1,    10,    -1,
       5,    -1,     6,    -1,     9,    -1,     7,    -1,     8,    -1,
       4,    64,     6,    65,    66,     4,    61,     6,    62,    -1,
       4,    64,     6,    65,    66,     4,    -1,     4,    48,    -1,
       4,    49,    -1,    50,    61,    72,    62,    -1,    52,    61,
      72,    62,    -1,    53,    61,    72,    62,    -1,    54,    61,
      72,    62,    -1,    55,    61,    72,    62,    -1,    16,    72,
      -1,    72,    28,    72,    -1,    72,    29,    72,    -1,    72,
      30,    72,    -1,    72,    31,    72,    -1,    72,    32,    72,
      -1,    72,    33,    72,    -1,    72,    15,    72,    -1,    72,
      12,    72,    -1,    72,    13,    72,    -1,    72,    14,    72,
      -1,    72,    27,    72,    -1,    22,    72,    -1,    21,    72,
      -1,    72,    24,    72,    -1,    72,    24,    61,    71,    62,
      -1,    61,    72,    62,    -1,    72,    19,     8,    -1,    72,
      19,    22,     8,    -1,    -1,    43,    39,    76,    -1,    72,
      46,     4,    -1,    74,    67,    72,    46,     4,    -1,    72,
      -1,    75,    67,    72,    -1,    72,    -1,    72,    67,    76,
      -1,    -1,    76,    -1,    39,    72,    -1,    40,     4,    57,
      72,    -1,    59,    40,     4,    57,    72,    -1,    40,     4,
      57,    72,    79,    -1,    -1,    56,     6,    -1
};

/* YYRLINE[YYN] -- source line where rule number YYN was defined.  */
static const yytype_uint8 yyrline[] =
{
       0,   140,   140,   141,   145,   148,   150,   152,   154,   156,
     158,   160,   162,   167,   168,   169,   170,   171,   172,   173,
     174,   175,   176,   177,   178,   179,   180,   181,   182,   183,
     184,   188,   189,   190,   191,   192,   193,   195,   196,   197,
     198,   199,   200,   201,   202,   204,   205,   209,   210,   213,
     216,   220,   221,   225,   226,   230,   231,   234,   236,   239,
     242,   243,   244,   246,   249
};
#endif

#if YYDEBUG || YYERROR_VERBOSE || YYTOKEN_TABLE
/* YYTNAME[SYMBOL-NUM] -- String name of the symbol SYMBOL-NUM.
   First, the terminals, then, starting at YYNTOKENS, nonterminals.  */
static const char *const yytname[] =
{
  "$end", "error", "$undefined", "FILENAME", "NAME", "STRING", "INTNUM",
  "DECIMAL1", "BOOL1", "APPROXNUM", "USERVAR", "ASSIGN", "EQUAL", "OR",
  "XOR", "AND", "DISTINCT", "REGEXP", "LIKE", "IS", "IN", "'!'", "NOT",
  "BETWEEN", "COMPARISON", "'|'", "'&'", "SHIFT", "'+'", "'-'", "'*'",
  "'/'", "'%'", "MOD", "'^'", "UMINUS", "LOAD", "STREAM", "FILTER", "BY",
  "JOIN", "STORE", "INTO", "GROUP", "FROM", "SELECT", "AS", "ORDER", "ASC",
  "DESC", "COUNT", "USING", "SUM", "AVG", "MIN", "MAX", "LIMIT", "ON",
  "BINARY", "LEFT", "';'", "'('", "')'", "'.'", "'{'", "'}'", "':'", "','",
  "$accept", "stmt_list", "stmt", "select_stmt", "expr", "opt_group_list",
  "expr_list", "load_list", "val_list", "opt_val_list", "opt_where",
  "join_list", "opt_limit", 0
};
#endif

# ifdef YYPRINT
/* YYTOKNUM[YYLEX-NUM] -- Internal token number corresponding to
   token YYLEX-NUM.  */
static const yytype_uint16 yytoknum[] =
{
       0,   256,   257,   258,   259,   260,   261,   262,   263,   264,
     265,   266,   267,   268,   269,   270,   271,   272,   273,   274,
     275,    33,   276,   277,   278,   124,    38,   279,    43,    45,
      42,    47,    37,   280,    94,   281,   282,   283,   284,   285,
     286,   287,   288,   289,   290,   291,   292,   293,   294,   295,
     296,   297,   298,   299,   300,   301,   302,   303,   304,   305,
      59,    40,    41,    46,   123,   125,    58,    44
};
# endif

/* YYR1[YYN] -- Symbol number of symbol that rule YYN derives.  */
static const yytype_uint8 yyr1[] =
{
       0,    68,    69,    69,    70,    71,    71,    71,    71,    71,
      71,    71,    71,    72,    72,    72,    72,    72,    72,    72,
      72,    72,    72,    72,    72,    72,    72,    72,    72,    72,
      72,    72,    72,    72,    72,    72,    72,    72,    72,    72,
      72,    72,    72,    72,    72,    72,    72,    72,    72,    73,
      73,    74,    74,    75,    75,    76,    76,    77,    77,    78,
      79,    79,    79,    80,    80
};

/* YYR2[YYN] -- Number of symbols composing right hand side of rule YYN.  */
static const yytype_uint8 yyr2[] =
{
       0,     2,     2,     3,     1,     7,    12,     9,     5,     6,
       8,     9,     6,     1,     3,     1,     1,     1,     1,     1,
       1,     9,     6,     2,     2,     4,     4,     4,     4,     4,
       2,     3,     3,     3,     3,     3,     3,     3,     3,     3,
       3,     3,     2,     2,     3,     5,     3,     3,     4,     0,
       3,     3,     5,     1,     3,     1,     3,     0,     1,     2,
       4,     5,     5,     0,     2
};

/* YYDEFACT[STATE-NAME] -- Default rule to reduce with in state
   STATE-NUM when YYTABLE doesn't specify something else to do.  Zero
   means the default is an error.  */
static const yytype_uint8 yydefact[] =
{
       0,     0,     0,     0,     0,     4,     0,     0,     1,     0,
       2,     0,     0,     0,     0,     0,     3,     0,     0,    13,
      16,    17,    19,    20,    18,    15,     0,     0,     0,     0,
       0,     0,     0,     0,     0,     0,     0,     0,    63,     0,
       0,     0,     8,    23,    24,     0,     0,    30,    43,    42,
       0,     0,     0,     0,     0,     0,     0,     0,     0,     0,
       0,     0,     0,     0,     0,     0,     0,     0,     0,     0,
       0,     0,    57,     0,     0,     0,     0,     0,    59,    14,
       0,     0,     0,     0,     0,     0,    46,    38,    39,    40,
      37,    47,     0,     0,    44,    41,    31,    32,    33,    34,
      35,    36,    51,    49,     0,    55,    58,     9,     0,    64,
      12,     0,     0,     0,    25,    26,    27,    28,    29,    48,
      13,     0,     0,     0,     0,     5,    49,     0,     0,     0,
       0,    53,     0,     0,    45,     0,     0,     0,    10,    52,
      56,    63,     0,     7,     0,    22,     0,    50,     0,    11,
       0,    54,     0,    60,     0,     0,     0,    62,    61,     6,
      21
};

/* YYDEFGOTO[NTERM-NUM].  */
static const yytype_int16 yydefgoto[] =
{
      -1,     3,     4,     5,   105,   125,    36,   132,   106,   107,
      42,   126,    75
};

/* YYPACT[STATE-NUM] -- Index in YYTABLE of the portion describing
   STATE-NUM.  */
#define YYPACT_NINF -126
static const yytype_int16 yypact[] =
{
       6,    -3,     5,     1,   -44,  -126,    59,   -18,  -126,   -34,
    -126,    53,    64,   113,    65,    54,  -126,   -39,    31,    37,
    -126,  -126,  -126,  -126,  -126,  -126,   113,   113,   113,    33,
      41,    67,    71,    75,   113,   337,   -42,    76,    27,    77,
      78,   113,  -126,  -126,  -126,   122,   133,   401,   411,   411,
     113,   113,   113,   113,   113,   182,   113,   113,   113,   113,
      -2,   138,   113,   113,   113,   113,   113,   113,   113,   136,
     145,   113,   113,    89,   146,    93,   150,    94,   386,  -126,
      91,   204,   226,   248,   270,   292,  -126,   386,     3,   154,
     401,  -126,   149,    55,   418,   424,    81,    81,  -126,  -126,
    -126,  -126,  -126,   -36,   363,    60,  -126,  -126,   155,  -126,
    -126,    99,   113,    96,  -126,  -126,  -126,  -126,  -126,  -126,
      18,   102,   166,   132,   135,  -126,   129,   172,   113,   115,
     134,   386,    36,   175,  -126,   141,   113,   185,  -126,  -126,
    -126,   144,   142,  -126,   113,   143,   113,  -126,   148,  -126,
     113,   386,   196,   315,   113,    63,   158,  -126,   386,  -126,
    -126
};

/* YYPGOTO[NTERM-NUM].  */
static const yytype_int16 yypgoto[] =
{
    -126,  -126,   205,   114,   -13,    95,  -126,    72,  -125,  -126,
    -126,    73,    83
};

/* YYTABLE[YYPACT[STATE-NUM]].  What to do in state STATE-NUM.  If
   positive, shift that token.  If negative, reduce the rule which
   number is the opposite.  If zero, do what YYDEFACT says.
   If YYTABLE_NINF, syntax error.  */
#define YYTABLE_NINF -1
static const yytype_uint8 yytable[] =
{
      35,     8,    70,   140,   122,     1,    91,   123,     6,     7,
       1,   147,    39,    47,    48,    49,    10,    58,    59,    40,
      92,    55,    60,   124,    15,    71,    16,    61,    78,     6,
      62,    63,    64,    65,    66,    67,    68,    81,    82,    83,
      84,    85,     2,    87,    88,    89,    90,     2,    94,    95,
      96,    97,    98,    99,   100,   101,    17,    38,   104,   120,
      20,    21,    22,    23,    24,    25,    43,    44,    18,    37,
      41,    26,    56,    57,    58,    59,    27,    28,    73,    60,
      55,    45,    46,    74,    61,    43,    44,    62,    63,    64,
      65,    66,    67,    68,    50,    11,     2,    12,   143,   131,
      45,    46,    51,   144,    13,    29,    14,    30,    31,    32,
      33,    65,    66,    67,    68,    72,    34,    19,    20,    21,
      22,    23,    24,    25,    77,   159,    79,   128,    52,    26,
     144,   151,    53,   153,    27,    28,    54,   131,    76,    80,
     102,   158,    19,    20,    21,    22,    23,    24,    25,   103,
     108,   110,   109,   111,    26,   112,   113,   119,   129,    27,
      28,   130,   133,    29,   134,    30,    31,    32,    33,    59,
     135,   136,   123,    60,    34,   137,   139,   141,    61,   145,
     142,    62,    63,    64,    65,    66,    67,    68,    29,   148,
      30,    31,    32,    33,    56,    57,    58,    59,   146,    93,
      74,    60,   156,   150,   152,   154,    61,   121,     9,    62,
      63,    64,    65,    66,    67,    68,    56,    57,    58,    59,
     160,   138,   155,    60,   149,     0,   157,     0,    61,     0,
       0,    62,    63,    64,    65,    66,    67,    68,    56,    57,
      58,    59,     0,     0,    86,    60,     0,     0,     0,     0,
      61,     0,     0,    62,    63,    64,    65,    66,    67,    68,
      56,    57,    58,    59,     0,     0,   114,    60,     0,     0,
       0,     0,    61,     0,     0,    62,    63,    64,    65,    66,
      67,    68,    56,    57,    58,    59,     0,     0,   115,    60,
       0,     0,     0,     0,    61,     0,     0,    62,    63,    64,
      65,    66,    67,    68,    56,    57,    58,    59,     0,     0,
     116,    60,     0,     0,     0,     0,    61,     0,     0,    62,
      63,    64,    65,    66,    67,    68,     0,    56,    57,    58,
      59,     0,   117,     0,    60,     0,     0,     0,     0,    61,
       0,     0,    62,    63,    64,    65,    66,    67,    68,    56,
      57,    58,    59,     0,   118,   122,    60,     0,     0,     0,
       0,    61,     0,     0,    62,    63,    64,    65,    66,    67,
      68,     0,     0,     0,   124,    56,    57,    58,    59,     0,
       0,     0,    60,    69,     0,     0,     0,    61,     0,     0,
      62,    63,    64,    65,    66,    67,    68,     0,    56,    57,
      58,    59,     0,     0,     0,    60,     0,     0,     0,   127,
      61,     0,     0,    62,    63,    64,    65,    66,    67,    68,
      60,     0,     0,     0,     0,    61,     0,     0,    62,    63,
      64,    65,    66,    67,    68,    61,     0,     0,    62,    63,
      64,    65,    66,    67,    68,    62,    63,    64,    65,    66,
      67,    68,    63,    64,    65,    66,    67,    68
};

static const yytype_int16 yycheck[] =
{
      13,     0,    44,   128,    40,     4,     8,    43,    11,     4,
       4,   136,    51,    26,    27,    28,    60,    14,    15,    58,
      22,    34,    19,    59,    42,    67,    60,    24,    41,    11,
      27,    28,    29,    30,    31,    32,    33,    50,    51,    52,
      53,    54,    41,    56,    57,    58,    59,    41,    61,    62,
      63,    64,    65,    66,    67,    68,     3,     3,    71,     4,
       5,     6,     7,     8,     9,    10,    48,    49,     4,     4,
      39,    16,    12,    13,    14,    15,    21,    22,    51,    19,
      93,    63,    64,    56,    24,    48,    49,    27,    28,    29,
      30,    31,    32,    33,    61,    36,    41,    38,    62,   112,
      63,    64,    61,    67,    45,    50,    47,    52,    53,    54,
      55,    30,    31,    32,    33,    39,    61,     4,     5,     6,
       7,     8,     9,    10,    46,    62,     4,    67,    61,    16,
      67,   144,    61,   146,    21,    22,    61,   150,    61,     6,
       4,   154,     4,     5,     6,     7,     8,     9,    10,     4,
      61,    58,     6,     3,    16,    61,    65,     8,     3,    21,
      22,    62,    66,    50,    62,    52,    53,    54,    55,    15,
       4,    39,    43,    19,    61,    40,     4,    62,    24,     4,
      46,    27,    28,    29,    30,    31,    32,    33,    50,     4,
      52,    53,    54,    55,    12,    13,    14,    15,    57,    61,
      56,    19,     6,    61,    61,    57,    24,    93,     3,    27,
      28,    29,    30,    31,    32,    33,    12,    13,    14,    15,
      62,   126,   150,    19,   141,    -1,   153,    -1,    24,    -1,
      -1,    27,    28,    29,    30,    31,    32,    33,    12,    13,
      14,    15,    -1,    -1,    62,    19,    -1,    -1,    -1,    -1,
      24,    -1,    -1,    27,    28,    29,    30,    31,    32,    33,
      12,    13,    14,    15,    -1,    -1,    62,    19,    -1,    -1,
      -1,    -1,    24,    -1,    -1,    27,    28,    29,    30,    31,
      32,    33,    12,    13,    14,    15,    -1,    -1,    62,    19,
      -1,    -1,    -1,    -1,    24,    -1,    -1,    27,    28,    29,
      30,    31,    32,    33,    12,    13,    14,    15,    -1,    -1,
      62,    19,    -1,    -1,    -1,    -1,    24,    -1,    -1,    27,
      28,    29,    30,    31,    32,    33,    -1,    12,    13,    14,
      15,    -1,    62,    -1,    19,    -1,    -1,    -1,    -1,    24,
      -1,    -1,    27,    28,    29,    30,    31,    32,    33,    12,
      13,    14,    15,    -1,    62,    40,    19,    -1,    -1,    -1,
      -1,    24,    -1,    -1,    27,    28,    29,    30,    31,    32,
      33,    -1,    -1,    -1,    59,    12,    13,    14,    15,    -1,
      -1,    -1,    19,    46,    -1,    -1,    -1,    24,    -1,    -1,
      27,    28,    29,    30,    31,    32,    33,    -1,    12,    13,
      14,    15,    -1,    -1,    -1,    19,    -1,    -1,    -1,    46,
      24,    -1,    -1,    27,    28,    29,    30,    31,    32,    33,
      19,    -1,    -1,    -1,    -1,    24,    -1,    -1,    27,    28,
      29,    30,    31,    32,    33,    24,    -1,    -1,    27,    28,
      29,    30,    31,    32,    33,    27,    28,    29,    30,    31,
      32,    33,    28,    29,    30,    31,    32,    33
};

/* YYSTOS[STATE-NUM] -- The (internal number of the) accessing
   symbol of state STATE-NUM.  */
static const yytype_uint8 yystos[] =
{
       0,     4,    41,    69,    70,    71,    11,     4,     0,    70,
      60,    36,    38,    45,    47,    42,    60,     3,     4,     4,
       5,     6,     7,     8,     9,    10,    16,    21,    22,    50,
      52,    53,    54,    55,    61,    72,    74,     4,     3,    51,
      58,    39,    78,    48,    49,    63,    64,    72,    72,    72,
      61,    61,    61,    61,    61,    72,    12,    13,    14,    15,
      19,    24,    27,    28,    29,    30,    31,    32,    33,    46,
      44,    67,    39,    51,    56,    80,    61,    46,    72,     4,
       6,    72,    72,    72,    72,    72,    62,    72,    72,    72,
      72,     8,    22,    61,    72,    72,    72,    72,    72,    72,
      72,    72,     4,     4,    72,    72,    76,    77,    61,     6,
      58,     3,    61,    65,    62,    62,    62,    62,    62,     8,
       4,    71,    40,    43,    59,    73,    79,    46,    67,     3,
      62,    72,    75,    66,    62,     4,    39,    40,    73,     4,
      76,    62,    46,    62,    67,     4,    57,    76,     4,    80,
      61,    72,    61,    72,    57,    75,     6,    79,    72,    62,
      62
};

#define yyerrok		(yyerrstatus = 0)
#define yyclearin	(yychar = YYEMPTY)
#define YYEMPTY		(-2)
#define YYEOF		0

#define YYACCEPT	goto yyacceptlab
#define YYABORT		goto yyabortlab
#define YYERROR		goto yyerrorlab


/* Like YYERROR except do call yyerror.  This remains here temporarily
   to ease the transition to the new meaning of YYERROR, for GCC.
   Once GCC version 2 has supplanted version 1, this can go.  */

#define YYFAIL		goto yyerrlab

#define YYRECOVERING()  (!!yyerrstatus)

#define YYBACKUP(Token, Value)					\
do								\
  if (yychar == YYEMPTY && yylen == 1)				\
    {								\
      yychar = (Token);						\
      yylval = (Value);						\
      yytoken = YYTRANSLATE (yychar);				\
      YYPOPSTACK (1);						\
      goto yybackup;						\
    }								\
  else								\
    {								\
      yyerror (YY_("syntax error: cannot back up")); \
      YYERROR;							\
    }								\
while (YYID (0))


#define YYTERROR	1
#define YYERRCODE	256


/* YYLLOC_DEFAULT -- Set CURRENT to span from RHS[1] to RHS[N].
   If N is 0, then set CURRENT to the empty location which ends
   the previous symbol: RHS[0] (always defined).  */

#define YYRHSLOC(Rhs, K) ((Rhs)[K])
#ifndef YYLLOC_DEFAULT
# define YYLLOC_DEFAULT(Current, Rhs, N)				\
    do									\
      if (YYID (N))                                                    \
	{								\
	  (Current).first_line   = YYRHSLOC (Rhs, 1).first_line;	\
	  (Current).first_column = YYRHSLOC (Rhs, 1).first_column;	\
	  (Current).last_line    = YYRHSLOC (Rhs, N).last_line;		\
	  (Current).last_column  = YYRHSLOC (Rhs, N).last_column;	\
	}								\
      else								\
	{								\
	  (Current).first_line   = (Current).last_line   =		\
	    YYRHSLOC (Rhs, 0).last_line;				\
	  (Current).first_column = (Current).last_column =		\
	    YYRHSLOC (Rhs, 0).last_column;				\
	}								\
    while (YYID (0))
#endif


/* YY_LOCATION_PRINT -- Print the location on the stream.
   This macro was not mandated originally: define only if we know
   we won't break user code: when these are the locations we know.  */

#ifndef YY_LOCATION_PRINT
# if YYLTYPE_IS_TRIVIAL
#  define YY_LOCATION_PRINT(File, Loc)			\
     fprintf (File, "%d.%d-%d.%d",			\
	      (Loc).first_line, (Loc).first_column,	\
	      (Loc).last_line,  (Loc).last_column)
# else
#  define YY_LOCATION_PRINT(File, Loc) ((void) 0)
# endif
#endif


/* YYLEX -- calling `yylex' with the right arguments.  */

#ifdef YYLEX_PARAM
# define YYLEX yylex (YYLEX_PARAM)
#else
# define YYLEX yylex ()
#endif

/* Enable debugging if requested.  */
#if YYDEBUG

# ifndef YYFPRINTF
#  include <stdio.h> /* INFRINGES ON USER NAME SPACE */
#  define YYFPRINTF fprintf
# endif

# define YYDPRINTF(Args)			\
do {						\
  if (yydebug)					\
    YYFPRINTF Args;				\
} while (YYID (0))

# define YY_SYMBOL_PRINT(Title, Type, Value, Location)			  \
do {									  \
  if (yydebug)								  \
    {									  \
      YYFPRINTF (stderr, "%s ", Title);					  \
      yy_symbol_print (stderr,						  \
		  Type, Value); \
      YYFPRINTF (stderr, "\n");						  \
    }									  \
} while (YYID (0))


/*--------------------------------.
| Print this symbol on YYOUTPUT.  |
`--------------------------------*/

/*ARGSUSED*/
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yy_symbol_value_print (FILE *yyoutput, int yytype, YYSTYPE const * const yyvaluep)
#else
static void
yy_symbol_value_print (yyoutput, yytype, yyvaluep)
    FILE *yyoutput;
    int yytype;
    YYSTYPE const * const yyvaluep;
#endif
{
  if (!yyvaluep)
    return;
# ifdef YYPRINT
  if (yytype < YYNTOKENS)
    YYPRINT (yyoutput, yytoknum[yytype], *yyvaluep);
# else
  YYUSE (yyoutput);
# endif
  switch (yytype)
    {
      default:
	break;
    }
}


/*--------------------------------.
| Print this symbol on YYOUTPUT.  |
`--------------------------------*/

#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yy_symbol_print (FILE *yyoutput, int yytype, YYSTYPE const * const yyvaluep)
#else
static void
yy_symbol_print (yyoutput, yytype, yyvaluep)
    FILE *yyoutput;
    int yytype;
    YYSTYPE const * const yyvaluep;
#endif
{
  if (yytype < YYNTOKENS)
    YYFPRINTF (yyoutput, "token %s (", yytname[yytype]);
  else
    YYFPRINTF (yyoutput, "nterm %s (", yytname[yytype]);

  yy_symbol_value_print (yyoutput, yytype, yyvaluep);
  YYFPRINTF (yyoutput, ")");
}

/*------------------------------------------------------------------.
| yy_stack_print -- Print the state stack from its BOTTOM up to its |
| TOP (included).                                                   |
`------------------------------------------------------------------*/

#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yy_stack_print (yytype_int16 *yybottom, yytype_int16 *yytop)
#else
static void
yy_stack_print (yybottom, yytop)
    yytype_int16 *yybottom;
    yytype_int16 *yytop;
#endif
{
  YYFPRINTF (stderr, "Stack now");
  for (; yybottom <= yytop; yybottom++)
    {
      int yybot = *yybottom;
      YYFPRINTF (stderr, " %d", yybot);
    }
  YYFPRINTF (stderr, "\n");
}

# define YY_STACK_PRINT(Bottom, Top)				\
do {								\
  if (yydebug)							\
    yy_stack_print ((Bottom), (Top));				\
} while (YYID (0))


/*------------------------------------------------.
| Report that the YYRULE is going to be reduced.  |
`------------------------------------------------*/

#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yy_reduce_print (YYSTYPE *yyvsp, int yyrule)
#else
static void
yy_reduce_print (yyvsp, yyrule)
    YYSTYPE *yyvsp;
    int yyrule;
#endif
{
  int yynrhs = yyr2[yyrule];
  int yyi;
  unsigned long int yylno = yyrline[yyrule];
  YYFPRINTF (stderr, "Reducing stack by rule %d (line %lu):\n",
	     yyrule - 1, yylno);
  /* The symbols being reduced.  */
  for (yyi = 0; yyi < yynrhs; yyi++)
    {
      YYFPRINTF (stderr, "   $%d = ", yyi + 1);
      yy_symbol_print (stderr, yyrhs[yyprhs[yyrule] + yyi],
		       &(yyvsp[(yyi + 1) - (yynrhs)])
		       		       );
      YYFPRINTF (stderr, "\n");
    }
}

# define YY_REDUCE_PRINT(Rule)		\
do {					\
  if (yydebug)				\
    yy_reduce_print (yyvsp, Rule); \
} while (YYID (0))

/* Nonzero means print parse trace.  It is left uninitialized so that
   multiple parsers can coexist.  */
int yydebug;
#else /* !YYDEBUG */
# define YYDPRINTF(Args)
# define YY_SYMBOL_PRINT(Title, Type, Value, Location)
# define YY_STACK_PRINT(Bottom, Top)
# define YY_REDUCE_PRINT(Rule)
#endif /* !YYDEBUG */


/* YYINITDEPTH -- initial size of the parser's stacks.  */
#ifndef	YYINITDEPTH
# define YYINITDEPTH 200
#endif

/* YYMAXDEPTH -- maximum size the stacks can grow to (effective only
   if the built-in stack extension method is used).

   Do not make this value too large; the results are undefined if
   YYSTACK_ALLOC_MAXIMUM < YYSTACK_BYTES (YYMAXDEPTH)
   evaluated with infinite-precision integer arithmetic.  */

#ifndef YYMAXDEPTH
# define YYMAXDEPTH 10000
#endif



#if YYERROR_VERBOSE

# ifndef yystrlen
#  if defined __GLIBC__ && defined _STRING_H
#   define yystrlen strlen
#  else
/* Return the length of YYSTR.  */
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static YYSIZE_T
yystrlen (const char *yystr)
#else
static YYSIZE_T
yystrlen (yystr)
    const char *yystr;
#endif
{
  YYSIZE_T yylen;
  for (yylen = 0; yystr[yylen]; yylen++)
    continue;
  return yylen;
}
#  endif
# endif

# ifndef yystpcpy
#  if defined __GLIBC__ && defined _STRING_H && defined _GNU_SOURCE
#   define yystpcpy stpcpy
#  else
/* Copy YYSRC to YYDEST, returning the address of the terminating '\0' in
   YYDEST.  */
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static char *
yystpcpy (char *yydest, const char *yysrc)
#else
static char *
yystpcpy (yydest, yysrc)
    char *yydest;
    const char *yysrc;
#endif
{
  char *yyd = yydest;
  const char *yys = yysrc;

  while ((*yyd++ = *yys++) != '\0')
    continue;

  return yyd - 1;
}
#  endif
# endif

# ifndef yytnamerr
/* Copy to YYRES the contents of YYSTR after stripping away unnecessary
   quotes and backslashes, so that it's suitable for yyerror.  The
   heuristic is that double-quoting is unnecessary unless the string
   contains an apostrophe, a comma, or backslash (other than
   backslash-backslash).  YYSTR is taken from yytname.  If YYRES is
   null, do not copy; instead, return the length of what the result
   would have been.  */
static YYSIZE_T
yytnamerr (char *yyres, const char *yystr)
{
  if (*yystr == '"')
    {
      YYSIZE_T yyn = 0;
      char const *yyp = yystr;

      for (;;)
	switch (*++yyp)
	  {
	  case '\'':
	  case ',':
	    goto do_not_strip_quotes;

	  case '\\':
	    if (*++yyp != '\\')
	      goto do_not_strip_quotes;
	    /* Fall through.  */
	  default:
	    if (yyres)
	      yyres[yyn] = *yyp;
	    yyn++;
	    break;

	  case '"':
	    if (yyres)
	      yyres[yyn] = '\0';
	    return yyn;
	  }
    do_not_strip_quotes: ;
    }

  if (! yyres)
    return yystrlen (yystr);

  return yystpcpy (yyres, yystr) - yyres;
}
# endif

/* Copy into YYRESULT an error message about the unexpected token
   YYCHAR while in state YYSTATE.  Return the number of bytes copied,
   including the terminating null byte.  If YYRESULT is null, do not
   copy anything; just return the number of bytes that would be
   copied.  As a special case, return 0 if an ordinary "syntax error"
   message will do.  Return YYSIZE_MAXIMUM if overflow occurs during
   size calculation.  */
static YYSIZE_T
yysyntax_error (char *yyresult, int yystate, int yychar)
{
  int yyn = yypact[yystate];

  if (! (YYPACT_NINF < yyn && yyn <= YYLAST))
    return 0;
  else
    {
      int yytype = YYTRANSLATE (yychar);
      YYSIZE_T yysize0 = yytnamerr (0, yytname[yytype]);
      YYSIZE_T yysize = yysize0;
      YYSIZE_T yysize1;
      int yysize_overflow = 0;
      enum { YYERROR_VERBOSE_ARGS_MAXIMUM = 5 };
      char const *yyarg[YYERROR_VERBOSE_ARGS_MAXIMUM];
      int yyx;

# if 0
      /* This is so xgettext sees the translatable formats that are
	 constructed on the fly.  */
      YY_("syntax error, unexpected %s");
      YY_("syntax error, unexpected %s, expecting %s");
      YY_("syntax error, unexpected %s, expecting %s or %s");
      YY_("syntax error, unexpected %s, expecting %s or %s or %s");
      YY_("syntax error, unexpected %s, expecting %s or %s or %s or %s");
# endif
      char *yyfmt;
      char const *yyf;
      static char const yyunexpected[] = "syntax error, unexpected %s";
      static char const yyexpecting[] = ", expecting %s";
      static char const yyor[] = " or %s";
      char yyformat[sizeof yyunexpected
		    + sizeof yyexpecting - 1
		    + ((YYERROR_VERBOSE_ARGS_MAXIMUM - 2)
		       * (sizeof yyor - 1))];
      char const *yyprefix = yyexpecting;

      /* Start YYX at -YYN if negative to avoid negative indexes in
	 YYCHECK.  */
      int yyxbegin = yyn < 0 ? -yyn : 0;

      /* Stay within bounds of both yycheck and yytname.  */
      int yychecklim = YYLAST - yyn + 1;
      int yyxend = yychecklim < YYNTOKENS ? yychecklim : YYNTOKENS;
      int yycount = 1;

      yyarg[0] = yytname[yytype];
      yyfmt = yystpcpy (yyformat, yyunexpected);

      for (yyx = yyxbegin; yyx < yyxend; ++yyx)
	if (yycheck[yyx + yyn] == yyx && yyx != YYTERROR)
	  {
	    if (yycount == YYERROR_VERBOSE_ARGS_MAXIMUM)
	      {
		yycount = 1;
		yysize = yysize0;
		yyformat[sizeof yyunexpected - 1] = '\0';
		break;
	      }
	    yyarg[yycount++] = yytname[yyx];
	    yysize1 = yysize + yytnamerr (0, yytname[yyx]);
	    yysize_overflow |= (yysize1 < yysize);
	    yysize = yysize1;
	    yyfmt = yystpcpy (yyfmt, yyprefix);
	    yyprefix = yyor;
	  }

      yyf = YY_(yyformat);
      yysize1 = yysize + yystrlen (yyf);
      yysize_overflow |= (yysize1 < yysize);
      yysize = yysize1;

      if (yysize_overflow)
	return YYSIZE_MAXIMUM;

      if (yyresult)
	{
	  /* Avoid sprintf, as that infringes on the user's name space.
	     Don't have undefined behavior even if the translation
	     produced a string with the wrong number of "%s"s.  */
	  char *yyp = yyresult;
	  int yyi = 0;
	  while ((*yyp = *yyf) != '\0')
	    {
	      if (*yyp == '%' && yyf[1] == 's' && yyi < yycount)
		{
		  yyp += yytnamerr (yyp, yyarg[yyi++]);
		  yyf += 2;
		}
	      else
		{
		  yyp++;
		  yyf++;
		}
	    }
	}
      return yysize;
    }
}
#endif /* YYERROR_VERBOSE */


/*-----------------------------------------------.
| Release the memory associated to this symbol.  |
`-----------------------------------------------*/

/*ARGSUSED*/
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yydestruct (const char *yymsg, int yytype, YYSTYPE *yyvaluep)
#else
static void
yydestruct (yymsg, yytype, yyvaluep)
    const char *yymsg;
    int yytype;
    YYSTYPE *yyvaluep;
#endif
{
  YYUSE (yyvaluep);

  if (!yymsg)
    yymsg = "Deleting";
  YY_SYMBOL_PRINT (yymsg, yytype, yyvaluep, yylocationp);

  switch (yytype)
    {

      default:
	break;
    }
}

/* Prevent warnings from -Wmissing-prototypes.  */
#ifdef YYPARSE_PARAM
#if defined __STDC__ || defined __cplusplus
int yyparse (void *YYPARSE_PARAM);
#else
int yyparse ();
#endif
#else /* ! YYPARSE_PARAM */
#if defined __STDC__ || defined __cplusplus
int yyparse (void);
#else
int yyparse ();
#endif
#endif /* ! YYPARSE_PARAM */


/* The lookahead symbol.  */
int yychar;

/* The semantic value of the lookahead symbol.  */
YYSTYPE yylval;

/* Number of syntax errors so far.  */
int yynerrs;



/*-------------------------.
| yyparse or yypush_parse.  |
`-------------------------*/

#ifdef YYPARSE_PARAM
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
int
yyparse (void *YYPARSE_PARAM)
#else
int
yyparse (YYPARSE_PARAM)
    void *YYPARSE_PARAM;
#endif
#else /* ! YYPARSE_PARAM */
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
int
yyparse (void)
#else
int
yyparse ()

#endif
#endif
{


    int yystate;
    /* Number of tokens to shift before error messages enabled.  */
    int yyerrstatus;

    /* The stacks and their tools:
       `yyss': related to states.
       `yyvs': related to semantic values.

       Refer to the stacks thru separate pointers, to allow yyoverflow
       to reallocate them elsewhere.  */

    /* The state stack.  */
    yytype_int16 yyssa[YYINITDEPTH];
    yytype_int16 *yyss;
    yytype_int16 *yyssp;

    /* The semantic value stack.  */
    YYSTYPE yyvsa[YYINITDEPTH];
    YYSTYPE *yyvs;
    YYSTYPE *yyvsp;

    YYSIZE_T yystacksize;

  int yyn;
  int yyresult;
  /* Lookahead token as an internal (translated) token number.  */
  int yytoken;
  /* The variables used to return semantic value and location from the
     action routines.  */
  YYSTYPE yyval;

#if YYERROR_VERBOSE
  /* Buffer for error messages, and its allocated size.  */
  char yymsgbuf[128];
  char *yymsg = yymsgbuf;
  YYSIZE_T yymsg_alloc = sizeof yymsgbuf;
#endif

#define YYPOPSTACK(N)   (yyvsp -= (N), yyssp -= (N))

  /* The number of symbols on the RHS of the reduced rule.
     Keep to zero when no symbol should be popped.  */
  int yylen = 0;

  yytoken = 0;
  yyss = yyssa;
  yyvs = yyvsa;
  yystacksize = YYINITDEPTH;

  YYDPRINTF ((stderr, "Starting parse\n"));

  yystate = 0;
  yyerrstatus = 0;
  yynerrs = 0;
  yychar = YYEMPTY; /* Cause a token to be read.  */

  /* Initialize stack pointers.
     Waste one element of value and location stack
     so that they stay on the same level as the state stack.
     The wasted elements are never initialized.  */
  yyssp = yyss;
  yyvsp = yyvs;

  goto yysetstate;

/*------------------------------------------------------------.
| yynewstate -- Push a new state, which is found in yystate.  |
`------------------------------------------------------------*/
 yynewstate:
  /* In all cases, when you get here, the value and location stacks
     have just been pushed.  So pushing a state here evens the stacks.  */
  yyssp++;

 yysetstate:
  *yyssp = yystate;

  if (yyss + yystacksize - 1 <= yyssp)
    {
      /* Get the current used size of the three stacks, in elements.  */
      YYSIZE_T yysize = yyssp - yyss + 1;

#ifdef yyoverflow
      {
	/* Give user a chance to reallocate the stack.  Use copies of
	   these so that the &'s don't force the real ones into
	   memory.  */
	YYSTYPE *yyvs1 = yyvs;
	yytype_int16 *yyss1 = yyss;

	/* Each stack pointer address is followed by the size of the
	   data in use in that stack, in bytes.  This used to be a
	   conditional around just the two extra args, but that might
	   be undefined if yyoverflow is a macro.  */
	yyoverflow (YY_("memory exhausted"),
		    &yyss1, yysize * sizeof (*yyssp),
		    &yyvs1, yysize * sizeof (*yyvsp),
		    &yystacksize);

	yyss = yyss1;
	yyvs = yyvs1;
      }
#else /* no yyoverflow */
# ifndef YYSTACK_RELOCATE
      goto yyexhaustedlab;
# else
      /* Extend the stack our own way.  */
      if (YYMAXDEPTH <= yystacksize)
	goto yyexhaustedlab;
      yystacksize *= 2;
      if (YYMAXDEPTH < yystacksize)
	yystacksize = YYMAXDEPTH;

      {
	yytype_int16 *yyss1 = yyss;
	union yyalloc *yyptr =
	  (union yyalloc *) YYSTACK_ALLOC (YYSTACK_BYTES (yystacksize));
	if (! yyptr)
	  goto yyexhaustedlab;
	YYSTACK_RELOCATE (yyss_alloc, yyss);
	YYSTACK_RELOCATE (yyvs_alloc, yyvs);
#  undef YYSTACK_RELOCATE
	if (yyss1 != yyssa)
	  YYSTACK_FREE (yyss1);
      }
# endif
#endif /* no yyoverflow */

      yyssp = yyss + yysize - 1;
      yyvsp = yyvs + yysize - 1;

      YYDPRINTF ((stderr, "Stack size increased to %lu\n",
		  (unsigned long int) yystacksize));

      if (yyss + yystacksize - 1 <= yyssp)
	YYABORT;
    }

  YYDPRINTF ((stderr, "Entering state %d\n", yystate));

  if (yystate == YYFINAL)
    YYACCEPT;

  goto yybackup;

/*-----------.
| yybackup.  |
`-----------*/
yybackup:

  /* Do appropriate processing given the current state.  Read a
     lookahead token if we need one and don't already have one.  */

  /* First try to decide what to do without reference to lookahead token.  */
  yyn = yypact[yystate];
  if (yyn == YYPACT_NINF)
    goto yydefault;

  /* Not known => get a lookahead token if don't already have one.  */

  /* YYCHAR is either YYEMPTY or YYEOF or a valid lookahead symbol.  */
  if (yychar == YYEMPTY)
    {
      YYDPRINTF ((stderr, "Reading a token: "));
      yychar = YYLEX;
    }

  if (yychar <= YYEOF)
    {
      yychar = yytoken = YYEOF;
      YYDPRINTF ((stderr, "Now at end of input.\n"));
    }
  else
    {
      yytoken = YYTRANSLATE (yychar);
      YY_SYMBOL_PRINT ("Next token is", yytoken, &yylval, &yylloc);
    }

  /* If the proper action on seeing token YYTOKEN is to reduce or to
     detect an error, take that action.  */
  yyn += yytoken;
  if (yyn < 0 || YYLAST < yyn || yycheck[yyn] != yytoken)
    goto yydefault;
  yyn = yytable[yyn];
  if (yyn <= 0)
    {
      if (yyn == 0 || yyn == YYTABLE_NINF)
	goto yyerrlab;
      yyn = -yyn;
      goto yyreduce;
    }

  /* Count tokens shifted since error; after three, turn off error
     status.  */
  if (yyerrstatus)
    yyerrstatus--;

  /* Shift the lookahead token.  */
  YY_SYMBOL_PRINT ("Shifting", yytoken, &yylval, &yylloc);

  /* Discard the shifted token.  */
  yychar = YYEMPTY;

  yystate = yyn;
  *++yyvsp = yylval;

  goto yynewstate;


/*-----------------------------------------------------------.
| yydefault -- do the default action for the current state.  |
`-----------------------------------------------------------*/
yydefault:
  yyn = yydefact[yystate];
  if (yyn == 0)
    goto yyerrlab;
  goto yyreduce;


/*-----------------------------.
| yyreduce -- Do a reduction.  |
`-----------------------------*/
yyreduce:
  /* yyn is the number of a rule to reduce with.  */
  yylen = yyr2[yyn];

  /* If YYLEN is nonzero, implement the default value of the action:
     `$$ = $1'.

     Otherwise, the following line sets YYVAL to garbage.
     This behavior is undocumented and Bison
     users should not rely upon it.  Assigning to YYVAL
     unconditionally makes the parser a bit smaller, and it avoids a
     GCC warning that YYVAL may be used uninitialized.  */
  yyval = yyvsp[1-yylen];


  YY_REDUCE_PRINT (yyn);
  switch (yyn)
    {
        case 4:

/* Line 1455 of yacc.c  */
#line 145 "bison.y"
    { emit("STMT"); ;}
    break;

  case 5:

/* Line 1455 of yacc.c  */
#line 149 "bison.y"
    { emit_select((yyvsp[(1) - (7)].strval), (yyvsp[(6) - (7)].strval), (yyvsp[(7) - (7)].intval)); ;}
    break;

  case 6:

/* Line 1455 of yacc.c  */
#line 151 "bison.y"
    {  emit_load((yyvsp[(1) - (12)].strval), (yyvsp[(4) - (12)].strval), (yyvsp[(11) - (12)].intval), (yyvsp[(7) - (12)].strval)); ;}
    break;

  case 7:

/* Line 1455 of yacc.c  */
#line 153 "bison.y"
    {  emit_load_binary((yyvsp[(1) - (9)].strval), (yyvsp[(4) - (9)].strval), (yyvsp[(8) - (9)].intval)); ;}
    break;

  case 8:

/* Line 1455 of yacc.c  */
#line 155 "bison.y"
    {  emit_filter((yyvsp[(1) - (5)].strval), (yyvsp[(4) - (5)].strval), (yyvsp[(5) - (5)].intval));;}
    break;

  case 9:

/* Line 1455 of yacc.c  */
#line 157 "bison.y"
    {  emit_order((yyvsp[(1) - (6)].strval), (yyvsp[(4) - (6)].strval), (yyvsp[(6) - (6)].intval));;}
    break;

  case 10:

/* Line 1455 of yacc.c  */
#line 159 "bison.y"
    { emit_join((yyvsp[(1) - (8)].strval),(yyvsp[(6) - (8)].strval),(yyvsp[(7) - (8)].intval)); ;}
    break;

  case 11:

/* Line 1455 of yacc.c  */
#line 161 "bison.y"
    { emit_store((yyvsp[(2) - (9)].strval),(yyvsp[(4) - (9)].strval),(yyvsp[(7) - (9)].strval)); ;}
    break;

  case 12:

/* Line 1455 of yacc.c  */
#line 163 "bison.y"
    { emit_store_binary((yyvsp[(2) - (6)].strval),(yyvsp[(4) - (6)].strval)); ;}
    break;

  case 13:

/* Line 1455 of yacc.c  */
#line 167 "bison.y"
    { emit_name((yyvsp[(1) - (1)].strval)); ;}
    break;

  case 14:

/* Line 1455 of yacc.c  */
#line 168 "bison.y"
    { emit("FIELDNAME %s.%s", (yyvsp[(1) - (3)].strval), (yyvsp[(3) - (3)].strval)); ;}
    break;

  case 15:

/* Line 1455 of yacc.c  */
#line 169 "bison.y"
    { emit("USERVAR %s", (yyvsp[(1) - (1)].strval)); ;}
    break;

  case 16:

/* Line 1455 of yacc.c  */
#line 170 "bison.y"
    { emit_string((yyvsp[(1) - (1)].strval)); ;}
    break;

  case 17:

/* Line 1455 of yacc.c  */
#line 171 "bison.y"
    { emit_number((yyvsp[(1) - (1)].intval)); ;}
    break;

  case 18:

/* Line 1455 of yacc.c  */
#line 172 "bison.y"
    { emit_float((yyvsp[(1) - (1)].floatval)); ;}
    break;

  case 19:

/* Line 1455 of yacc.c  */
#line 173 "bison.y"
    { emit_decimal((yyvsp[(1) - (1)].intval)); ;}
    break;

  case 20:

/* Line 1455 of yacc.c  */
#line 174 "bison.y"
    { emit("BOOL %d", (yyvsp[(1) - (1)].intval)); ;}
    break;

  case 21:

/* Line 1455 of yacc.c  */
#line 175 "bison.y"
    { emit_varchar((yyvsp[(1) - (9)].strval), (yyvsp[(3) - (9)].intval), (yyvsp[(6) - (9)].strval), (yyvsp[(8) - (9)].intval));;}
    break;

  case 22:

/* Line 1455 of yacc.c  */
#line 176 "bison.y"
    { emit_var((yyvsp[(1) - (6)].strval), (yyvsp[(3) - (6)].intval), (yyvsp[(6) - (6)].strval));;}
    break;

  case 23:

/* Line 1455 of yacc.c  */
#line 177 "bison.y"
    { emit_var_asc((yyvsp[(1) - (2)].strval));;}
    break;

  case 24:

/* Line 1455 of yacc.c  */
#line 178 "bison.y"
    { emit_var_desc((yyvsp[(1) - (2)].strval));;}
    break;

  case 25:

/* Line 1455 of yacc.c  */
#line 179 "bison.y"
    { emit_count(); ;}
    break;

  case 26:

/* Line 1455 of yacc.c  */
#line 180 "bison.y"
    { emit_sum(); ;}
    break;

  case 27:

/* Line 1455 of yacc.c  */
#line 181 "bison.y"
    { emit_average(); ;}
    break;

  case 28:

/* Line 1455 of yacc.c  */
#line 182 "bison.y"
    { emit_min(); ;}
    break;

  case 29:

/* Line 1455 of yacc.c  */
#line 183 "bison.y"
    { emit_max(); ;}
    break;

  case 30:

/* Line 1455 of yacc.c  */
#line 184 "bison.y"
    { emit_distinct(); ;}
    break;

  case 31:

/* Line 1455 of yacc.c  */
#line 188 "bison.y"
    { emit_add(); ;}
    break;

  case 32:

/* Line 1455 of yacc.c  */
#line 189 "bison.y"
    { emit_minus(); ;}
    break;

  case 33:

/* Line 1455 of yacc.c  */
#line 190 "bison.y"
    { emit_mul(); ;}
    break;

  case 34:

/* Line 1455 of yacc.c  */
#line 191 "bison.y"
    { emit_div(); ;}
    break;

  case 35:

/* Line 1455 of yacc.c  */
#line 192 "bison.y"
    { emit("MOD"); ;}
    break;

  case 36:

/* Line 1455 of yacc.c  */
#line 193 "bison.y"
    { emit("MOD"); ;}
    break;

  case 37:

/* Line 1455 of yacc.c  */
#line 195 "bison.y"
    { emit_and(); ;}
    break;

  case 38:

/* Line 1455 of yacc.c  */
#line 196 "bison.y"
    { emit_eq(); ;}
    break;

  case 39:

/* Line 1455 of yacc.c  */
#line 197 "bison.y"
    { emit_or(); ;}
    break;

  case 40:

/* Line 1455 of yacc.c  */
#line 198 "bison.y"
    { emit("XOR"); ;}
    break;

  case 41:

/* Line 1455 of yacc.c  */
#line 199 "bison.y"
    { emit("SHIFT %s", (yyvsp[(2) - (3)].subtok)==1?"left":"right"); ;}
    break;

  case 42:

/* Line 1455 of yacc.c  */
#line 200 "bison.y"
    { emit("NOT"); ;}
    break;

  case 43:

/* Line 1455 of yacc.c  */
#line 201 "bison.y"
    { emit("NOT"); ;}
    break;

  case 44:

/* Line 1455 of yacc.c  */
#line 202 "bison.y"
    { emit_cmp((yyvsp[(2) - (3)].subtok)); ;}
    break;

  case 45:

/* Line 1455 of yacc.c  */
#line 204 "bison.y"
    { emit("CMPSELECT %d", (yyvsp[(2) - (5)].subtok)); ;}
    break;

  case 46:

/* Line 1455 of yacc.c  */
#line 205 "bison.y"
    {emit("EXPR");;}
    break;

  case 47:

/* Line 1455 of yacc.c  */
#line 209 "bison.y"
    { emit("ISBOOL %d", (yyvsp[(3) - (3)].intval)); ;}
    break;

  case 48:

/* Line 1455 of yacc.c  */
#line 210 "bison.y"
    { emit("ISBOOL %d", (yyvsp[(4) - (4)].intval)); emit("NOT"); ;}
    break;

  case 49:

/* Line 1455 of yacc.c  */
#line 213 "bison.y"
    { /* nil */
    (yyval.intval) = 0;
;}
    break;

  case 50:

/* Line 1455 of yacc.c  */
#line 216 "bison.y"
    { (yyval.intval) = (yyvsp[(3) - (3)].intval);}
    break;

  case 51:

/* Line 1455 of yacc.c  */
#line 220 "bison.y"
    { (yyval.intval) = 1; emit_sel_name((yyvsp[(3) - (3)].strval));;}
    break;

  case 52:

/* Line 1455 of yacc.c  */
#line 221 "bison.y"
    { (yyval.intval) = (yyvsp[(1) - (5)].intval) + 1; emit_sel_name((yyvsp[(5) - (5)].strval));;}
    break;

  case 53:

/* Line 1455 of yacc.c  */
#line 225 "bison.y"
    { (yyval.intval) = 1; ;}
    break;

  case 54:

/* Line 1455 of yacc.c  */
#line 226 "bison.y"
    {(yyval.intval) = (yyvsp[(1) - (3)].intval) + 1; ;}
    break;

  case 55:

/* Line 1455 of yacc.c  */
#line 230 "bison.y"
    { (yyval.intval) = 1; ;}
    break;

  case 56:

/* Line 1455 of yacc.c  */
#line 231 "bison.y"
    { (yyval.intval) = 1 + (yyvsp[(3) - (3)].intval); ;}
    break;

  case 57:

/* Line 1455 of yacc.c  */
#line 234 "bison.y"
    { /* nil */
    (yyval.intval) = 0
;}
    break;

  case 59:

/* Line 1455 of yacc.c  */
#line 239 "bison.y"
    { emit("FILTER BY"); ;}
    break;

  case 60:

/* Line 1455 of yacc.c  */
#line 242 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(2) - (4)].strval), 0);;}
    break;

  case 61:

/* Line 1455 of yacc.c  */
#line 243 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (5)].strval), 1);;}
    break;

  case 62:

/* Line 1455 of yacc.c  */
#line 244 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(2) - (5)].strval), 0); ;}
    break;

  case 63:

/* Line 1455 of yacc.c  */
#line 246 "bison.y"
    { /* nil */
    (yyval.intval) = 0
;}
    break;

  case 64:

/* Line 1455 of yacc.c  */
#line 249 "bison.y"
    { emit_limit((yyvsp[(2) - (2)].intval)); ;}
    break;



/* Line 1455 of yacc.c  */
#line 2045 "bison.cu"
      default: break;
    }
  YY_SYMBOL_PRINT ("-> $$ =", yyr1[yyn], &yyval, &yyloc);

  YYPOPSTACK (yylen);
  yylen = 0;
  YY_STACK_PRINT (yyss, yyssp);

  *++yyvsp = yyval;

  /* Now `shift' the result of the reduction.  Determine what state
     that goes to, based on the state we popped back to and the rule
     number reduced by.  */

  yyn = yyr1[yyn];

  yystate = yypgoto[yyn - YYNTOKENS] + *yyssp;
  if (0 <= yystate && yystate <= YYLAST && yycheck[yystate] == *yyssp)
    yystate = yytable[yystate];
  else
    yystate = yydefgoto[yyn - YYNTOKENS];

  goto yynewstate;


/*------------------------------------.
| yyerrlab -- here on detecting error |
`------------------------------------*/
yyerrlab:
  /* If not already recovering from an error, report this error.  */
  if (!yyerrstatus)
    {
      ++yynerrs;
#if ! YYERROR_VERBOSE
      yyerror (YY_("syntax error"));
#else
      {
	YYSIZE_T yysize = yysyntax_error (0, yystate, yychar);
	if (yymsg_alloc < yysize && yymsg_alloc < YYSTACK_ALLOC_MAXIMUM)
	  {
	    YYSIZE_T yyalloc = 2 * yysize;
	    if (! (yysize <= yyalloc && yyalloc <= YYSTACK_ALLOC_MAXIMUM))
	      yyalloc = YYSTACK_ALLOC_MAXIMUM;
	    if (yymsg != yymsgbuf)
	      YYSTACK_FREE (yymsg);
	    yymsg = (char *) YYSTACK_ALLOC (yyalloc);
	    if (yymsg)
	      yymsg_alloc = yyalloc;
	    else
	      {
		yymsg = yymsgbuf;
		yymsg_alloc = sizeof yymsgbuf;
	      }
	  }

	if (0 < yysize && yysize <= yymsg_alloc)
	  {
	    (void) yysyntax_error (yymsg, yystate, yychar);
	    yyerror (yymsg);
	  }
	else
	  {
	    yyerror (YY_("syntax error"));
	    if (yysize != 0)
	      goto yyexhaustedlab;
	  }
      }
#endif
    }



  if (yyerrstatus == 3)
    {
      /* If just tried and failed to reuse lookahead token after an
	 error, discard it.  */

      if (yychar <= YYEOF)
	{
	  /* Return failure if at end of input.  */
	  if (yychar == YYEOF)
	    YYABORT;
	}
      else
	{
	  yydestruct ("Error: discarding",
		      yytoken, &yylval);
	  yychar = YYEMPTY;
	}
    }

  /* Else will try to reuse lookahead token after shifting the error
     token.  */
  goto yyerrlab1;


/*---------------------------------------------------.
| yyerrorlab -- error raised explicitly by YYERROR.  |
`---------------------------------------------------*/
yyerrorlab:

  /* Pacify compilers like GCC when the user code never invokes
     YYERROR and the label yyerrorlab therefore never appears in user
     code.  */
  if (/*CONSTCOND*/ 0)
     goto yyerrorlab;

  /* Do not reclaim the symbols of the rule which action triggered
     this YYERROR.  */
  YYPOPSTACK (yylen);
  yylen = 0;
  YY_STACK_PRINT (yyss, yyssp);
  yystate = *yyssp;
  goto yyerrlab1;


/*-------------------------------------------------------------.
| yyerrlab1 -- common code for both syntax error and YYERROR.  |
`-------------------------------------------------------------*/
yyerrlab1:
  yyerrstatus = 3;	/* Each real token shifted decrements this.  */

  for (;;)
    {
      yyn = yypact[yystate];
      if (yyn != YYPACT_NINF)
	{
	  yyn += YYTERROR;
	  if (0 <= yyn && yyn <= YYLAST && yycheck[yyn] == YYTERROR)
	    {
	      yyn = yytable[yyn];
	      if (0 < yyn)
		break;
	    }
	}

      /* Pop the current state because it cannot handle the error token.  */
      if (yyssp == yyss)
	YYABORT;


      yydestruct ("Error: popping",
		  yystos[yystate], yyvsp);
      YYPOPSTACK (1);
      yystate = *yyssp;
      YY_STACK_PRINT (yyss, yyssp);
    }

  *++yyvsp = yylval;


  /* Shift the error token.  */
  YY_SYMBOL_PRINT ("Shifting", yystos[yyn], yyvsp, yylsp);

  yystate = yyn;
  goto yynewstate;


/*-------------------------------------.
| yyacceptlab -- YYACCEPT comes here.  |
`-------------------------------------*/
yyacceptlab:
  yyresult = 0;
  goto yyreturn;

/*-----------------------------------.
| yyabortlab -- YYABORT comes here.  |
`-----------------------------------*/
yyabortlab:
  yyresult = 1;
  goto yyreturn;

#if !defined(yyoverflow) || YYERROR_VERBOSE
/*-------------------------------------------------.
| yyexhaustedlab -- memory exhaustion comes here.  |
`-------------------------------------------------*/
yyexhaustedlab:
  yyerror (YY_("memory exhausted"));
  yyresult = 2;
  /* Fall through.  */
#endif

yyreturn:
  if (yychar != YYEMPTY)
     yydestruct ("Cleanup: discarding lookahead",
		 yytoken, &yylval);
  /* Do not reclaim the symbols of the rule which action triggered
     this YYABORT or YYACCEPT.  */
  YYPOPSTACK (yylen);
  YY_STACK_PRINT (yyss, yyssp);
  while (yyssp != yyss)
    {
      yydestruct ("Cleanup: popping",
		  yystos[*yyssp], yyvsp);
      YYPOPSTACK (1);
    }
#ifndef yyoverflow
  if (yyss != yyssa)
    YYSTACK_FREE (yyss);
#endif
#if YYERROR_VERBOSE
  if (yymsg != yymsgbuf)
    YYSTACK_FREE (yymsg);
#endif
  /* Make sure YYID is used.  */
  return YYID (yyresult);
}



/* Line 1675 of yacc.c  */
#line 252 "bison.y"


#include "filter.h"
#include "select.h"
#include "merge.h"
#include "zone_map.h"
#include "atof.h"
#include "sorts.cu"
#include <limits>
#include "cudpp_src_2.0/include/cudpp_hash.h"


size_t int_size = sizeof(int_type);
size_t float_size = sizeof(float_type);

FILE *file_pointer;
queue<string> namevars;
queue<string> typevars;
queue<int> sizevars;
queue<int> cols;

queue<unsigned int> j_col_count;
unsigned int sel_count = 0;
unsigned int join_cnt = 0;
unsigned int distinct_cnt = 0;
int join_col_cnt = 0;
stack<string> op_join;
bool left_join;

unsigned int statement_count = 0;
map<string,unsigned int> stat;
bool scan_state = 0;
string separator, f_file;
unsigned int int_col_count;
CUDPPHandle theCudpp;

using namespace thrust::placeholders;


void emit_name(char *name)
{
    op_type.push("NAME");
    op_value.push(name);
}

void emit_limit(int val)
{
    op_nums.push(val);
}


void emit_string(char *str)
{   // remove the float_type quotes
    string sss(str,1, strlen(str)-2);
    op_type.push("STRING");
    op_value.push(sss);
}


void emit_number(int_type val)
{
    op_type.push("NUMBER");
    op_nums.push(val);
}

void emit_float(float_type val)
{
    op_type.push("FLOAT");
    op_nums_f.push(val);
}

void emit_decimal(float_type val)
{
    op_type.push("DECIMAL");
    op_nums_f.push(val);
}



void emit_mul()
{
    op_type.push("MUL");
}

void emit_add()
{
    op_type.push("ADD");
}

void emit_div()
{
    op_type.push("DIV");
}

void emit_and()
{
    op_type.push("AND");
    join_col_cnt++;
}

void emit_eq()
{
    op_type.push("JOIN");
}

void emit_distinct()
{
    op_type.push("DISTINCT");
    distinct_cnt++;
}

void emit_or()
{
    op_type.push("OR");
}


void emit_minus()
{
    op_type.push("MINUS");
}

void emit_cmp(int val)
{
    op_type.push("CMP");
    op_nums.push(val);
}

void emit(char *s, ...)
{


}

void emit_var(char *s, int c, char *f)
{
    namevars.push(s);
    typevars.push(f);
    sizevars.push(0);
    cols.push(c);
}

void emit_var_asc(char *s)
{
    op_type.push(s);
    op_value.push("ASC");
}

void emit_var_desc(char *s)
{
    op_type.push(s);
    op_value.push("DESC");
}


void emit_varchar(char *s, int c, char *f, int d)
{
    namevars.push(s);
    typevars.push(f);
    sizevars.push(d);
    cols.push(c);
}

void emit_sel_name(char *s)
{
    op_type.push("emit sel_name");
    op_value.push(s);
    sel_count++;
}

void emit_count()
{
    op_type.push("COUNT");
}

void emit_sum()
{
    op_type.push("SUM");
}


void emit_average()
{
    op_type.push("AVG");
}

void emit_min()
{
    op_type.push("MIN");
}

void emit_max()
{
    op_type.push("MAX");
}

void emit_join_tab(char *s, bool left)
{
    op_join.push(s);
    left_join = left;
};




void order_inplace(CudaSet* a, stack<string> exe_type, set<string> field_names, unsigned int segment)
{
    //std::clock_t start1 = std::clock();
    unsigned int sz = a->mRecCount;
    thrust::device_ptr<unsigned int> permutation = thrust::device_malloc<unsigned int>(sz);
    thrust::sequence(permutation, permutation+sz,0,1);

    unsigned int* raw_ptr = thrust::raw_pointer_cast(permutation);
    void* temp;
    // find the largest mRecSize of all data sources exe_type.top()
    unsigned int maxSize = 0;
    for (set<string>::iterator it=field_names.begin(); it!=field_names.end(); ++it) {
        CudaSet *t = varNames[setMap[*it]];
        if(t->mRecCount > maxSize)
            maxSize = t->mRecCount;
    };


    unsigned int max_c = max_char(a);

    if(max_c > float_size)
        CUDA_SAFE_CALL(hipMalloc((void **) &temp, maxSize*max_c));
    else
        CUDA_SAFE_CALL(hipMalloc((void **) &temp, maxSize*float_size));

    unsigned int str_count = 0;


    for(int i=0; !exe_type.empty(); ++i, exe_type.pop()) {
        int colInd = (a->columnNames).find(exe_type.top())->second;
        if (a->type[colInd] == 0)
            update_permutation(a->d_columns_int[a->type_index[colInd]], raw_ptr, sz, "ASC", (int_type*)temp);
        else if (a->type[colInd] == 1)
            update_permutation(a->d_columns_float[a->type_index[colInd]], raw_ptr, sz,"ASC", (float_type*)temp);
        else {
            // use int col int_col_count
            update_permutation(a->d_columns_int[int_col_count+str_count], raw_ptr, sz, "ASC", (int_type*)temp);
            str_count++;
        };
    };

    str_count = 0;

    for (set<string>::iterator it=field_names.begin(); it!=field_names.end(); ++it) {
        int i = a->columnNames[*it];
        if (a->type[i] == 0)
            apply_permutation(a->d_columns_int[a->type_index[i]], raw_ptr, sz, (int_type*)temp);
        else if (a->type[i] == 1)
            apply_permutation(a->d_columns_float[a->type_index[i]], raw_ptr, sz, (float_type*)temp);
        else {
            apply_permutation_char(a->d_columns_char[a->type_index[i]], raw_ptr, sz, (char*)temp, a->char_size[a->type_index[i]]);
            apply_permutation(a->d_columns_int[int_col_count + str_count], raw_ptr, sz, (int_type*)temp);
            str_count++;
        };
    };


    hipFree(temp);
    thrust::device_free(permutation);

}

int hh = 0;

void emit_join(char *s, char *j1, int grp)
{

    string j2 = op_join.top();
    op_join.pop();

    statement_count++;
    if (scan_state == 0) {
        if (stat.find(j1) == stat.end()) {
            cout << "Join : couldn't find variable " << j1 << endl;
            exit(1);
        };
        if (stat.find(j2) == stat.end()) {
            cout << "Join : couldn't find variable " << j2 << endl;
            exit(1);
        };
        stat[s] = statement_count;
        stat[j1] = statement_count;
        stat[j2] = statement_count;
        return;
    };


    if(varNames.find(j1) == varNames.end() || varNames.find(j2) == varNames.end()) {
        clean_queues();
        return;
    };

    CudaSet* left = varNames.find(j1)->second;
    CudaSet* right = varNames.find(j2)->second;

    queue<string> op_sel;
    queue<string> op_sel_as;
    for(int i=0; i < sel_count; i++) {
        op_sel.push(op_value.front());
        op_value.pop();
        op_sel_as.push(op_value.front());
        op_value.pop();
    };

    string f1 = op_value.front();
    op_value.pop();
    string f2 = op_value.front();
    op_value.pop();

    cout << "JOIN " << s <<  " " <<  getFreeMem() <<  endl;
    //cout << "join col count " << join_col_cnt << endl;

    queue<string> op_v1(op_value);
    while(op_v1.size() ) {
        op_v1.pop();
        grp++;
    };


    std::clock_t start1 = std::clock();
    CudaSet* c = new CudaSet(right,left,0,op_sel, op_sel_as);

    if (left->mRecCount == 0 || right->mRecCount == 0) {
        c = new CudaSet(left,right,0, op_sel, op_sel_as);
        varNames[s] = c;
        clean_queues();
        return;
    };

    unsigned int colInd1, colInd2;
    string tmpstr;
    if (left->columnNames.find(f1) != left->columnNames.end()) {
        colInd1 = (left->columnNames).find(f1)->second;
        if (right->columnNames.find(f2) != right->columnNames.end()) {
            colInd2 = (right->columnNames).find(f2)->second;
        }
        else {
            cout << "Couldn't find column " << f2 << endl;
            exit(0);
        };
    }
    else if (right->columnNames.find(f1) != right->columnNames.end()) {
        colInd2 = (right->columnNames).find(f1)->second;
        tmpstr = f1;
        f1 = f2;
        if (left->columnNames.find(f2) != left->columnNames.end()) {
            colInd1 = (left->columnNames).find(f2)->second;
            f2 = tmpstr;
        }
        else {
            cout << "Couldn't find column " << f2 << endl;
            exit(0);
        };
    }
    else {
        cout << "Couldn't find column " << f1 << endl;
        exit(0);
    };


    if (!((left->type[colInd1] == 0 && right->type[colInd2]  == 0) || (left->type[colInd1] == 2 && right->type[colInd2]  == 2)
            || (left->type[colInd1] == 1 && right->type[colInd2]  == 1 && left->decimal[colInd1] && right->decimal[colInd2]))) {
        cout << "Joins on floats are not supported " << endl;
        exit(0);
    };
    bool decimal_join = 0;
    if (left->type[colInd1] == 1 && right->type[colInd2]  == 1)
        decimal_join = 1;

    set<string> field_names;
    stack<string> exe_type;
    exe_type.push(f2);
    field_names.insert(f2);

    bool str_join = 0;
    //if join is on strings then add integer columns to left and right tables and modify colInd1 and colInd2

    if (right->type[colInd2]  == 2) {
        str_join = 1;
        right->d_columns_int.push_back(thrust::device_vector<int_type>());
        for(unsigned int i = 0; i < right->segCount; i++) {
            right->add_hashed_strings(f2, i, right->d_columns_int.size()-1);
        };
    };

    // need to allocate all right columns
    queue<string> cc;
    unsigned int rcount;
    curr_segment = 10000000;


    queue<string> op_vd(op_value);
    queue<string> op_alt(op_sel);
    unsigned int jc = join_col_cnt;
    while(jc) {
        jc--;
        op_vd.pop();
        op_alt.push(op_vd.front());
        op_vd.pop();
    };
    unsigned int cnt_r = load_queue(op_alt, right, str_join, f2, rcount);

    if(str_join) {
        colInd2 = right->mColumnCount+1;
        right->type_index[colInd2] = right->d_columns_int.size()-1;
    };


    //here we need to make sure that right column is ordered. If not then we order it and keep the permutation
    bool sorted;
    if(!decimal_join)
        sorted = thrust::is_sorted(right->d_columns_int[right->type_index[colInd2]].begin(), right->d_columns_int[right->type_index[colInd2]].begin() + cnt_r);
    else
        sorted = thrust::is_sorted(right->d_columns_float[right->type_index[colInd2]].begin(), right->d_columns_float[right->type_index[colInd2]].begin() + cnt_r);


    if(!sorted) {

        queue<string> ss(op_sel);
        thrust::device_vector<unsigned int> v(cnt_r);
        thrust::sequence(v.begin(),v.end(),0,1);

        unsigned int max_c	= max_char(right);
        unsigned int mm;
        if(max_c > 8)
            mm = (max_c/8) + 1;
        else
            mm = 1;

        thrust::device_ptr<int_type> d_tmp = thrust::device_malloc<int_type>(cnt_r*mm);
        thrust::sort_by_key(right->d_columns_int[right->type_index[colInd2]].begin(), right->d_columns_int[right->type_index[colInd2]].begin() + cnt_r, v.begin());

        //for(unsigned int i = 0; i < right->mColumnCount; i++) {
        unsigned int i;
        while(!ss.empty()) {
            if (right->columnNames.find(ss.front()) != right->columnNames.end()) {
                i = right->columnNames[ss.front()];

                if(i != colInd2) {
                    if(right->type[i] == 0) {
                        thrust::gather(v.begin(), v.end(), right->d_columns_int[right->type_index[i]].begin(), d_tmp);
                        thrust::copy(d_tmp, d_tmp + cnt_r, right->d_columns_int[right->type_index[i]].begin());
                    }
                    else if(right->type[i] == 1) {
                        thrust::gather(v.begin(), v.end(), right->d_columns_float[right->type_index[i]].begin(), d_tmp);
                        thrust::copy(d_tmp, d_tmp + cnt_r, right->d_columns_float[right->type_index[i]].begin());
                    }
                    else {
                        str_gather(thrust::raw_pointer_cast(v.data()), cnt_r, (void*)right->d_columns_char[right->type_index[i]], (void*) thrust::raw_pointer_cast(d_tmp), right->char_size[right->type_index[i]]);
                        hipMemcpy( (void*)right->d_columns_char[right->type_index[i]], (void*) thrust::raw_pointer_cast(d_tmp), cnt_r*right->char_size[right->type_index[i]], hipMemcpyDeviceToDevice);
                    };
                };
            };
            ss.pop();
        };
        thrust::device_free(d_tmp);
    };

    bool v64bit;
    if(right->d_columns_int[right->type_index[colInd2]][cnt_r-1] > std::numeric_limits<unsigned int>::max())
        v64bit = 1;
    else
        v64bit = 0;


    while(!cc.empty())
        cc.pop();

    if (left->type[colInd1]  == 2) {
        left->d_columns_int.push_back(thrust::device_vector<int_type>());
        //colInd1 = left->mColumnCount+1;
        //left->type_index[colInd1] = left->d_columns_int.size()-1;
    }
    else {
        cc.push(f1);
        allocColumns(left, cc);
    };

    thrust::device_vector<unsigned int> d_res1;
    thrust::device_vector<unsigned int> d_res2;
    unsigned int cnt_l, res_count, tot_count = 0, offset = 0, k = 0;

    queue<string> lc(cc);
    curr_segment = 10000000;
    CUDPPResult result;

    CUDPPHandle hash_table_handle;
    CUDPPHashTableConfig config;
    config.type = CUDPP_MULTIVALUE_HASH_TABLE;
    config.kInputSize = cnt_r;
    config.space_usage = 1.5f;

    cout << "creating table with " << cnt_r << " " << getFreeMem()  << endl;
    result = cudppHashTable(theCudpp, &hash_table_handle, &config);

    if (result == CUDPP_SUCCESS)
        cout << "hash tables created " << getFreeMem() << endl;

    unsigned int tt;
    if(left->maxRecs > rcount)
        tt = left->maxRecs;
    else
        tt = rcount;

    thrust::device_ptr<unsigned int> d_r = thrust::device_malloc<unsigned int>(tt);
    thrust::device_vector<unsigned int> v(cnt_r);
    thrust::sequence(v.begin(),v.end(),0,1);

    thrust::copy(right->d_columns_int[right->type_index[colInd2]].begin(), right->d_columns_int[right->type_index[colInd2]].begin() + cnt_r,
                 d_r);
    result = cudppHashInsert(hash_table_handle, thrust::raw_pointer_cast(d_r),
                             thrust::raw_pointer_cast(v.data()), cnt_r);

    if (result == CUDPP_SUCCESS)
        cout << "hash table inserted " << getFreeMem() << endl;

    thrust::device_ptr<uint2> res = thrust::device_malloc<uint2>(left->maxRecs);
	
	
    for (unsigned int i = 0; i < left->segCount; i++) {

        cout << "segment " << i << " " << getFreeMem() << endl;
        cnt_l = 0;

		
        if (left->type[colInd1]  != 2) {
            copyColumns(left, lc, i, cnt_l);
        }
        else {
            left->add_hashed_strings(f1, i, left->d_columns_int.size());
        };

        if(left->prm.empty()) {
            //copy all records
            cnt_l = left->mRecCount;
        }
        else {
            cnt_l = left->prm_count[i];
        };
		
				
        if (cnt_l) {

            unsigned int idx;
            if(!str_join)
                idx = left->type_index[colInd1];
            else
                idx = left->d_columns_int.size()-1;

            unsigned int left_sz;
            if(decimal_join) {
                thrust::transform(left->d_columns_float[idx].begin(), left->d_columns_float[idx].begin() + cnt_l,
                                  d_r, float_to_int_lower());
            }
            else {
                thrust::copy(left->d_columns_int[idx].begin(), left->d_columns_int[idx].begin() + cnt_l,
                             d_r);
            };

			cout << "joining " << cnt_l << " with " << cnt_r << endl;
            result = cudppHashRetrieve(hash_table_handle, thrust::raw_pointer_cast(d_r),
                                       thrust::raw_pointer_cast(res), cnt_l);
            if (result != CUDPP_SUCCESS)
                cout << "Failed retrieve " << endl;

            uint2 rr = thrust::reduce(res, res+cnt_l, make_uint2(0,0), Uint2Sum());
            res_count = rr.y;
            d_res1.resize(res_count);
            d_res2.resize(res_count);

            if(res_count) {
                thrust::counting_iterator<unsigned int> begin(0);
                uint2_split ff(thrust::raw_pointer_cast(res),thrust::raw_pointer_cast(d_r));
                thrust::for_each(begin, begin + cnt_l, ff);

                thrust::exclusive_scan(d_r, d_r+cnt_l, d_r );  // addresses
                join_functor1 ff1(thrust::raw_pointer_cast(res),
                                  thrust::raw_pointer_cast(d_r),
                                  thrust::raw_pointer_cast(d_res1.data()),
                                  thrust::raw_pointer_cast(d_res2.data()));
                thrust::for_each(begin, begin + cnt_l, ff1);
				if(v64bit) {// need to check the upper 32 bits
				    thrust::device_ptr<bool> d_add = thrust::device_malloc<bool>(d_res1.size());
					if(decimal_join) {
					    thrust::permutation_iterator<ElementIterator_float,IndexIterator> iter_left(left->d_columns_float[idx].begin(), d_res1.begin());
						thrust::permutation_iterator<ElementIterator_float,IndexIterator> iter_right(right->d_columns_float[right->type_index[colInd2]].begin(), d_res2.begin());						
						thrust::transform(iter_left, iter_left+d_res2.size(), iter_right, d_add, float_upper_equal_to());						
					}
					else {
					    thrust::permutation_iterator<ElementIterator_int,IndexIterator> iter_left(left->d_columns_int[idx].begin(), d_res1.begin());
						thrust::permutation_iterator<ElementIterator_int,IndexIterator> iter_right(right->d_columns_int[right->type_index[colInd2]].begin(), d_res2.begin());						
						thrust::transform(iter_left, iter_left+d_res2.size(), iter_right, d_add, int_upper_equal_to());
					};	
                    unsigned int new_cnt = thrust::count(d_add, d_add+d_res1.size(), 1);
                    thrust::stable_partition(d_res1.begin(), d_res1.begin() + d_res2.size(), d_add, thrust::identity<unsigned int>());
                    thrust::stable_partition(d_res2.begin(), d_res2.end(), d_add, thrust::identity<unsigned int>());

                    thrust::device_free(d_add);
                    d_res2.resize(new_cnt);
                    d_res1.resize(new_cnt);
					
				};

            };


            // check if the join is a multicolumn join
            while(join_col_cnt) {

                join_col_cnt--;
                string f3 = op_value.front();
                op_value.pop();
                string f4 = op_value.front();
                op_value.pop();

                queue<string> rc;
                rc.push(f3);

                allocColumns(left, rc);
                copyColumns(left, rc, i, cnt_l);
                rc.pop();
				thrust::device_ptr<bool> d_add = thrust::device_malloc<bool>(d_res1.size());

                if (d_res1.size() && d_res2.size()) {
                    unsigned int colInd3 = (left->columnNames).find(f3)->second;
                    unsigned int colInd4 = (right->columnNames).find(f4)->second;                    

                    if (left->type[colInd3] == 1 && right->type[colInd4]  == 1) {

                        if(right->d_columns_float[right->type_index[colInd4]].size() == 0)
                            unsigned int cnt_r = load_queue(rc, right, 0, f4, rcount);

                        thrust::device_ptr<int_type> d_l = thrust::device_malloc<int_type>(d_res1.size());
                        thrust::permutation_iterator<ElementIterator_float,IndexIterator> iter_left(left->d_columns_float[left->type_index[colInd3]].begin(), d_res1.begin());
                        thrust::transform(iter_left, iter_left+d_res1.size(), d_l, float_to_long());

                        thrust::device_ptr<int_type> d_r = thrust::device_malloc<int_type>(d_res1.size());
                        thrust::permutation_iterator<ElementIterator_float,IndexIterator> iter_right(right->d_columns_float[right->type_index[colInd4]].begin(), d_res2.begin());
                        thrust::transform(iter_right, iter_right+d_res2.size(), d_r, float_to_long());
                        thrust::transform(d_l, d_l+d_res1.size(), d_r, d_add, thrust::equal_to<int_type>());
                    }
                    else {
                        thrust::permutation_iterator<ElementIterator_int,IndexIterator> iter_left(left->d_columns_int[left->type_index[colInd3]].begin(), d_res1.begin());
                        thrust::permutation_iterator<ElementIterator_int,IndexIterator> iter_right(right->d_columns_int[right->type_index[colInd4]].begin(), d_res2.begin());
                        thrust::transform(iter_left, iter_left+d_res2.size(), iter_right, d_add, thrust::equal_to<int_type>());
                    };

                    unsigned int new_cnt = thrust::count(d_add, d_add+d_res1.size(), 1);
                    thrust::stable_partition(d_res1.begin(), d_res1.begin() + d_res2.size(), d_add, thrust::identity<unsigned int>());
                    thrust::stable_partition(d_res2.begin(), d_res2.end(), d_add, thrust::identity<unsigned int>());

                    d_res2.resize(new_cnt);
                    thrust::device_free(d_add);
                    if(!left_join) {
                        d_res1.resize(new_cnt);
                    }
                    else {
                        left_sz = d_res1.size() - d_res2.size();
                    };
                };
            };

            res_count = d_res1.size();
            tot_count = tot_count + res_count;
            //cout << "res " << res_count << endl;

            std::clock_t start5 = std::clock();
            if(res_count) {

                offset = c->mRecCount;
                if(i == 0)
                    c->reserve(res_count*(left->segCount+1));
                c->resize(res_count);
                queue<string> op_sel1(op_sel);
                unsigned int colInd, c_colInd;
                if(left->segCount == 1) {
                    thrust::device_free(d_r);
                    thrust::device_free(res);
                };

                while(!op_sel1.empty()) {

                    while(!cc.empty())
                        cc.pop();

                    cc.push(op_sel1.front());
                    c_colInd = c->columnNames[op_sel1.front()];

                    if(left->columnNames.find(op_sel1.front()) !=  left->columnNames.end()) {
                        // copy field's segment to device, gather it and copy to the host
                        unsigned int colInd = left->columnNames[op_sel1.front()];
                        reset_offsets();

                        allocColumns(left, cc);
                        copyColumns(left, cc, i, k);

                        //gather
                        if(left->type[colInd] == 0) {
                            thrust::permutation_iterator<ElementIterator_int,IndexIterator> iter(left->d_columns_int[left->type_index[colInd]].begin(), d_res1.begin());
                            thrust::copy(iter, iter + res_count, c->h_columns_int[c->type_index[c_colInd]].begin() + offset);
                        }
                        else if(left->type[colInd] == 1) {
                            thrust::permutation_iterator<ElementIterator_float,IndexIterator> iter(left->d_columns_float[left->type_index[colInd]].begin(), d_res1.begin());
                            thrust::copy(iter, iter + res_count, c->h_columns_float[c->type_index[c_colInd]].begin() + offset);
                        }
                        else { //strings
                            thrust::device_ptr<char> d_tmp = thrust::device_malloc<char>(res_count*left->char_size[left->type_index[colInd]]);
                            str_gather(thrust::raw_pointer_cast(d_res1.data()), res_count, (void*)left->d_columns_char[left->type_index[colInd]],
                                       (void*) thrust::raw_pointer_cast(d_tmp), left->char_size[left->type_index[colInd]]);

                            hipMemcpy( (void*)&c->h_columns_char[c->type_index[c_colInd]][offset*c->char_size[c->type_index[c_colInd]]], (void*) thrust::raw_pointer_cast(d_tmp),
                                        c->char_size[c->type_index[c_colInd]] * res_count, hipMemcpyDeviceToHost);
                            thrust::device_free(d_tmp);
                        }
                        left->deAllocColumnOnDevice(colInd);

                    }
                    else if(right->columnNames.find(op_sel1.front()) !=  right->columnNames.end()) {
                        colInd = right->columnNames[op_sel1.front()];

                        //gather
                        if(right->type[colInd] == 0) {
                            thrust::permutation_iterator<ElementIterator_int,IndexIterator> iter(right->d_columns_int[right->type_index[colInd]].begin(), d_res2.begin());
                            thrust::copy(iter, iter + d_res2.size(), c->h_columns_int[c->type_index[c_colInd]].begin() + offset);
                            if(left_join && left_sz) {
                                thrust::fill(c->h_columns_int[c->type_index[c_colInd]].begin() + offset + d_res2.size(),
                                             c->h_columns_int[c->type_index[c_colInd]].begin() + offset + d_res2.size() + left_sz,
                                             0);
                            };
                        }
                        else if(right->type[colInd] == 1) {
                            thrust::permutation_iterator<ElementIterator_float,IndexIterator> iter(right->d_columns_float[right->type_index[colInd]].begin(), d_res2.begin());
                            thrust::copy(iter, iter + d_res2.size(), c->h_columns_float[c->type_index[c_colInd]].begin() + offset);
                            if(left_join && left_sz) {
                                thrust::fill(c->h_columns_float[c->type_index[c_colInd]].begin() + offset + d_res2.size(),
                                             c->h_columns_float[c->type_index[c_colInd]].begin() + offset + d_res2.size() + left_sz,
                                             0);
                            };
                        }
                        else { //strings

                            thrust::device_ptr<char> d_tmp = thrust::device_malloc<char>(d_res2.size()*right->char_size[right->type_index[colInd]]);

                            str_gather(thrust::raw_pointer_cast(d_res2.data()), d_res2.size(), (void*)right->d_columns_char[right->type_index[colInd]],
                                       (void*) thrust::raw_pointer_cast(d_tmp), right->char_size[right->type_index[colInd]]);

                            hipMemcpy( (void*)(c->h_columns_char[c->type_index[c_colInd]] + offset*c->char_size[c->type_index[c_colInd]]), (void*) thrust::raw_pointer_cast(d_tmp),
                                        c->char_size[c->type_index[c_colInd]] * d_res2.size(), hipMemcpyDeviceToHost);
                            if(left_join && left_sz) {
                                memset((void*)(c->h_columns_char[c->type_index[c_colInd]] + (d_res2.size() + offset)*c->char_size[c->type_index[c_colInd]]), 0,
                                       left_sz*c->char_size[c->type_index[c_colInd]]);
                            };
                            thrust::device_free(d_tmp);
                        }
                    }
                    else {
                        cout << "Couldn't find field " << op_sel1.front() << endl;
                        exit(0);
                    };
                    op_sel1.pop();
                };
            };
        };
    };

    d_res1.resize(0);
    d_res1.shrink_to_fit();
    d_res2.resize(0);
    d_res2.shrink_to_fit();


    left->deAllocOnDevice();
    right->deAllocOnDevice();
    c->deAllocOnDevice();
    cudppDestroyHashTable(theCudpp, hash_table_handle);

    unsigned int i = 0;
    while(!col_aliases.empty()) {
        c->columnNames[col_aliases.front()] = i;
        col_aliases.pop();
        i++;
    };

    varNames[s] = c;
    c->mRecCount = tot_count;
    c->maxRecs = tot_count;
    for ( map<string,int>::iterator it=c->columnNames.begin() ; it != c->columnNames.end(); ++it )
        setMap[(*it).first] = s;

    clean_queues();

    if(stat[s] == statement_count) {
        c->free();
        varNames.erase(s);
    };

    if(stat[j1] == statement_count) {
        left->free();
        varNames.erase(j1);
    };

    if(stat[j2] == statement_count && (strcmp(j1,j2.c_str()) != 0)) {
        right->free();
        varNames.erase(j2);
    };

    std::cout<< "join time " <<  ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) <<'\n';
}


void order_on_host(CudaSet *a, CudaSet* b, queue<string> names, stack<string> exe_type, stack<string> exe_value)
{
    unsigned int tot = 0;
    if(!a->not_compressed) { //compressed
        allocColumns(a, names);

        unsigned int c = 0;
        if(a->prm_count.size())	{
            for(unsigned int i = 0; i < a->prm.size(); i++)
                c = c + a->prm_count[i];
        }
        else
            c = a->mRecCount;
        a->mRecCount = 0;
        a->resize(c);

        unsigned int cnt = 0;
        for(unsigned int i = 0; i < a->segCount; i++) {
            copyColumns(a, names, (a->segCount - i) - 1, cnt);	//uses segment 1 on a host	to copy data from a file to gpu
            if (a->mRecCount) {
                a->CopyToHost((c - tot) - a->mRecCount, a->mRecCount);
                tot = tot + a->mRecCount;
            };
        };
    }
    else
        tot = a->mRecCount;

    b->resize(tot); //resize host arrays
    a->mRecCount = tot;

    unsigned int* permutation = new unsigned int[a->mRecCount];
    thrust::sequence(permutation, permutation + a->mRecCount);

    unsigned int maxSize =  a->mRecCount;
    char* temp;
    unsigned int max_c = max_char(a);

    if(max_c > float_size)
        temp = new char[maxSize*max_c];
    else
        temp = new char[maxSize*float_size];

    // sort on host

    for(int i=0; !exe_type.empty(); ++i, exe_type.pop(),exe_value.pop()) {
        int colInd = (a->columnNames).find(exe_type.top())->second;

        if ((a->type)[colInd] == 0)
            update_permutation_host(a->h_columns_int[a->type_index[colInd]].data(), permutation, a->mRecCount, exe_value.top(), (int_type*)temp);
        else if ((a->type)[colInd] == 1)
            update_permutation_host(a->h_columns_float[a->type_index[colInd]].data(), permutation, a->mRecCount,exe_value.top(), (float_type*)temp);
        else {
            update_permutation_char_host(a->h_columns_char[a->type_index[colInd]], permutation, a->mRecCount, exe_value.top(), b->h_columns_char[b->type_index[colInd]], a->char_size[a->type_index[colInd]]);
        };
    };

    for (unsigned int i = 0; i < a->mColumnCount; i++) {
        if ((a->type)[i] == 0) {
            apply_permutation_host(a->h_columns_int[a->type_index[i]].data(), permutation, a->mRecCount, b->h_columns_int[b->type_index[i]].data());
        }
        else if ((a->type)[i] == 1)
            apply_permutation_host(a->h_columns_float[a->type_index[i]].data(), permutation, a->mRecCount, b->h_columns_float[b->type_index[i]].data());
        else {
            apply_permutation_char_host(a->h_columns_char[a->type_index[i]], permutation, a->mRecCount, b->h_columns_char[b->type_index[i]], a->char_size[a->type_index[i]]);
        };
    };
	
    delete [] temp;
    delete [] permutation;
}



void emit_order(char *s, char *f, int e, int ll)
{
    if(ll == 0)
        statement_count++;

    if (scan_state == 0 && ll == 0) {
        if (stat.find(f) == stat.end()) {
            cout << "Order : couldn't find variable " << f << endl;
            exit(1);
        };
        stat[s] = statement_count;
        stat[f] = statement_count;
        return;
    };

    if(varNames.find(f) == varNames.end() ) {
        clean_queues();
        return;
    };

    CudaSet* a = varNames.find(f)->second;


    if (a->mRecCount == 0)	{
        if(varNames.find(s) == varNames.end())
            varNames[s] = new CudaSet(0,1);
        else {
            CudaSet* c = varNames.find(s)->second;
            c->mRecCount = 0;
        };
        return;
    };

    stack<string> exe_type, exe_value;

    cout << "order: " << s << " " << f << endl;;


    for(int i=0; !op_type.empty(); ++i, op_type.pop(),op_value.pop()) {
        if ((op_type.front()).compare("NAME") == 0) {
            exe_type.push(op_value.front());
            exe_value.push("ASC");
        }
        else {
            exe_type.push(op_type.front());
            exe_value.push(op_value.front());
        };
    };

    stack<string> tp(exe_type);
    queue<string> op_vx;
    while (!tp.empty()) {
        op_vx.push(tp.top());
        tp.pop();
    };

    queue<string> names;
    for ( map<string,int>::iterator it=a->columnNames.begin() ; it != a->columnNames.end(); ++it )
        names.push((*it).first);

    CudaSet *b = a->copyDeviceStruct();

    //lets find out if our data set fits into a GPU
    size_t mem_available = getFreeMem();
    size_t rec_size = 0;
    for(unsigned int i = 0; i < a->mColumnCount; i++) {
        if(a->type[i] == 0)
            rec_size = rec_size + int_size;
        else if(a->type[i] == 1)
            rec_size = rec_size + float_size;
        else
            rec_size = rec_size + a->char_size[a->type_index[i]];
    };
    bool fits;
    if (rec_size*a->mRecCount > (mem_available/2)) // doesn't fit into a GPU
        fits = 0;
    else fits = 1;

    if(!fits) {
        order_on_host(a, b, names, exe_type, exe_value);
    }
    else {
        // initialize permutation to [0, 1, 2, ... ,N-1]
        thrust::device_ptr<unsigned int> permutation = thrust::device_malloc<unsigned int>(a->mRecCount);
        thrust::sequence(permutation, permutation+(a->mRecCount));

        unsigned int* raw_ptr = thrust::raw_pointer_cast(permutation);

        unsigned int maxSize =  a->mRecCount;
        void* temp;
        unsigned int max_c = max_char(a);

        if(max_c > float_size)
            CUDA_SAFE_CALL(hipMalloc((void **) &temp, maxSize*max_c));
        else
            CUDA_SAFE_CALL(hipMalloc((void **) &temp, maxSize*float_size));

        varNames[setMap[exe_type.top()]]->oldRecCount = varNames[setMap[exe_type.top()]]->mRecCount;


        unsigned int rcount;

        a->mRecCount = load_queue(names, a, 1, op_vx.front(), rcount);

        varNames[setMap[exe_type.top()]]->mRecCount = varNames[setMap[exe_type.top()]]->oldRecCount;
        unsigned int str_count = 0;


        for(int i=0; !exe_type.empty(); ++i, exe_type.pop(),exe_value.pop()) {
            int colInd = (a->columnNames).find(exe_type.top())->second;
            if ((a->type)[colInd] == 0)
                update_permutation(a->d_columns_int[a->type_index[colInd]], raw_ptr, a->mRecCount, exe_value.top(), (int_type*)temp);
            else if ((a->type)[colInd] == 1)
                update_permutation(a->d_columns_float[a->type_index[colInd]], raw_ptr, a->mRecCount,exe_value.top(), (float_type*)temp);
            else {
                update_permutation_char(a->d_columns_char[a->type_index[colInd]], raw_ptr, a->mRecCount, exe_value.top(), (char*)temp, a->char_size[a->type_index[colInd]]);
                //update_permutation(a->d_columns_int[int_col_count+str_count], raw_ptr, a->mRecCount, exe_value.top(), (int_type*)temp);
                str_count++;
            };
        };

        b->resize(a->mRecCount); //resize host arrays
        b->mRecCount = a->mRecCount;
        str_count = 0;

        for (unsigned int i = 0; i < a->mColumnCount; i++) {
            if ((a->type)[i] == 0)
                apply_permutation(a->d_columns_int[a->type_index[i]], raw_ptr, a->mRecCount, (int_type*)temp);
            else if ((a->type)[i] == 1)
                apply_permutation(a->d_columns_float[a->type_index[i]], raw_ptr, a->mRecCount, (float_type*)temp);
            else {
                apply_permutation_char(a->d_columns_char[a->type_index[i]], raw_ptr, a->mRecCount, (char*)temp, a->char_size[a->type_index[i]]);
                str_count++;
            };
        };

        for(unsigned int i = 0; i < a->mColumnCount; i++) {
            switch(a->type[i]) {
            case 0 :
                thrust::copy(a->d_columns_int[a->type_index[i]].begin(), a->d_columns_int[a->type_index[i]].begin() + a->mRecCount, b->h_columns_int[b->type_index[i]].begin());
                break;
            case 1 :
                thrust::copy(a->d_columns_float[a->type_index[i]].begin(), a->d_columns_float[a->type_index[i]].begin() + a->mRecCount, b->h_columns_float[b->type_index[i]].begin());
                break;
            default :
                hipMemcpy(b->h_columns_char[b->type_index[i]], a->d_columns_char[a->type_index[i]], a->char_size[a->type_index[i]]*a->mRecCount, hipMemcpyDeviceToHost);
            }
        };

        b->deAllocOnDevice();
        a->deAllocOnDevice();


        thrust::device_free(permutation);
        hipFree(temp);
    };

    varNames[s] = b;
    b->segCount = 1;
    b->not_compressed = 1;

    if(stat[f] == statement_count && !a->keep) {
        a->free();
        varNames.erase(f);
    };
}


void emit_select(char *s, char *f, int ll)
{
    statement_count++;
    if (scan_state == 0) {
        if (stat.find(f) == stat.end()) {
            cout << "Select : couldn't find variable " << f << endl;
            exit(1);
        };
        stat[s] = statement_count;
        stat[f] = statement_count;
        return;
    };


    if(varNames.find(f) == varNames.end()) {
        clean_queues();
        return;
    };



    queue<string> op_v1(op_value);
    while(op_v1.size() > ll)
        op_v1.pop();


    stack<string> op_v2;
    queue<string> op_v3;

    for(int i=0; i < ll; ++i) {
        op_v2.push(op_v1.front());
        op_v3.push(op_v1.front());
        op_v1.pop();
    };


    CudaSet *a;
    a = varNames.find(f)->second;


    if(a->mRecCount == 0) {
        CudaSet *c;
        c = new CudaSet(0,1);
        varNames[s] = c;
        clean_queues();
        return;
    };

    cout << "SELECT " << s << " " << f << endl;
    //cout << "free mem " << getFreeMem() << endl;
    std::clock_t start1 = std::clock();

    // here we need to determine the column count and composition

    queue<string> op_v(op_value);
    queue<string> op_vx;
    set<string> field_names;
    map<string,string> aliases;
    string tt;

    for(int i=0; !op_v.empty(); ++i, op_v.pop()) {
        if(a->columnNames.find(op_v.front()) != a->columnNames.end()) {
            field_names.insert(op_v.front());
            if(aliases.count(op_v.front()) == 0 && aliases.size() < ll) {
                tt = op_v.front();
                op_v.pop();
                aliases[tt] = op_v.front();
            };
        };
    };


    for (set<string>::iterator it=field_names.begin(); it!=field_names.end(); ++it)  {
        op_vx.push(*it);
    };

    // find out how many columns a new set will have
    queue<string> op_t(op_type);
    int_type col_count = 0;

    for(int i=0; !op_t.empty(); ++i, op_t.pop())
        if((op_t.front()).compare("emit sel_name") == 0)
            col_count++;

    CudaSet *b, *c;

    curr_segment = 10000000;
    allocColumns(a, op_vx);

    unsigned int cycle_count;
    if(!a->prm.empty())
        cycle_count = varNames[setMap[op_value.front()]]->segCount;
    else
        cycle_count = a->segCount;

    unsigned long long int ol_count = a->mRecCount;
	unsigned int cnt;
    //varNames[setMap[op_value.front()]]->oldRecCount = varNames[setMap[op_value.front()]]->mRecCount;
    a->oldRecCount = a->mRecCount;
    b = new CudaSet(0, col_count);
    bool b_set = 0, c_set = 0;

    unsigned int long long tmp_size = a->mRecCount;
    if(a->segCount > 1)
        tmp_size = a->maxRecs;

    boost::unordered_map<long long int, unsigned int> mymap; //this is where we keep the hashes of the records
    vector<thrust::device_vector<int_type> > distinct_val; //keeps array of DISTINCT values for every key
    vector<thrust::device_vector<int_type> > distinct_hash; //keeps array of DISTINCT values for every key
    vector<thrust::device_vector<int_type> > distinct_tmp;

    for(unsigned int i = 0; i < distinct_cnt; i++) {
        distinct_tmp.push_back(thrust::device_vector<int_type>(tmp_size));
        distinct_val.push_back(thrust::device_vector<int_type>());
        distinct_hash.push_back(thrust::device_vector<int_type>());
    };

// find out how many string columns we have. Add int_type columns to store string hashes for sort/groupby ops.
    stack<string> op_s = op_v2;
    int_col_count = a->d_columns_int.size();

    while(!op_s.empty()) {
        int colInd = (a->columnNames).find(op_s.top())->second;
        if (a->type[colInd] == 2) {
            a->d_columns_int.push_back(thrust::device_vector<int_type>());
        };
        op_s.pop();
    };

    unsigned int s_cnt;
    bool one_liner;

    for(unsigned int i = 0; i < cycle_count; i++) {          // MAIN CYCLE
        cout << "cycle " << i << " select mem " << getFreeMem() << endl;

        reset_offsets();
        op_s = op_v2;
        s_cnt = 0;

        while(!op_s.empty()) {

            int colInd = (a->columnNames).find(op_s.top())->second;
            if (a->type[colInd] == 2) {
                a->d_columns_int[int_col_count + s_cnt].resize(0);
                a->add_hashed_strings(op_s.top(), i, int_col_count + s_cnt);
                s_cnt++;
            };
            op_s.pop();
        };

        cnt = 0;
        copyColumns(a, op_vx, i, cnt);

        if(a->mRecCount) {
            if (ll != 0) {
                order_inplace(a,op_v2,field_names,i);
                a->GroupBy(op_v2, int_col_count);
            };


            for(unsigned int z = int_col_count; z < a->d_columns_int.size()-1; z++)
                a->d_columns_int[z].resize(0);

            select(op_type,op_value,op_nums, op_nums_f,a,b, distinct_tmp, one_liner);

            if(!b_set) {
                for ( map<string,int>::iterator it=b->columnNames.begin() ; it != b->columnNames.end(); ++it )
                    setMap[(*it).first] = s;
                b_set = 1;
                unsigned int old_cnt = b->mRecCount;
                b->mRecCount = 0;
                b->resize(varNames[setMap[op_vx.front()]]->maxRecs);
                b->mRecCount = old_cnt;
            };

            if (!c_set) {
                c = new CudaSet(0, col_count);
                create_c(c,b);
                c_set = 1;
            };

            if (ll != 0 && cycle_count > 1  ) {
                add(c,b,op_v3, mymap, aliases, distinct_tmp, distinct_val, distinct_hash, a);
            }
            else {
                //copy b to c
                unsigned int c_offset = c->mRecCount;
                c->resize(b->mRecCount);
                for(unsigned int j=0; j < b->mColumnCount; j++) {
                    if (b->type[j] == 0) {
                        thrust::copy(b->d_columns_int[b->type_index[j]].begin(), b->d_columns_int[b->type_index[j]].begin() + b->mRecCount, c->h_columns_int[c->type_index[j]].begin() + c_offset);
                    }
                    else if (b->type[j] == 1) {
                        thrust::copy(b->d_columns_float[b->type_index[j]].begin(), b->d_columns_float[b->type_index[j]].begin() + b->mRecCount, c->h_columns_float[c->type_index[j]].begin() + c_offset);
                    }
                    else {
                        hipMemcpy((void*)(thrust::raw_pointer_cast(c->h_columns_char[c->type_index[j]] + b->char_size[b->type_index[j]]*c_offset)), (void*)thrust::raw_pointer_cast(b->d_columns_char[b->type_index[j]]),
                                   b->char_size[b->type_index[j]] * b->mRecCount, hipMemcpyDeviceToHost);
                    };
                };

            };
        };
    };

    a->mRecCount = ol_count;
    a->mRecCount = a->oldRecCount;
    a->deAllocOnDevice();
    b->deAllocOnDevice();

    if (ll != 0) {
        count_avg(c, mymap, distinct_hash);
    }
    else {
        if(one_liner) {
            count_simple(c);
        };
    };

    reset_offsets();
    c->maxRecs = c->mRecCount;
    c->name = s;
    c->keep = 1;

    for ( map<string,int>::iterator it=c->columnNames.begin() ; it != c->columnNames.end(); ++it ) {
        setMap[(*it).first] = s;
    };

    cout << "final select " << c->mRecCount << endl;
    clean_queues();

    varNames[s] = c;
    b->free();
    varNames[s]->keep = 1;

    if(stat[s] == statement_count) {
        varNames[s]->free();
        varNames.erase(s);
    };

    if(stat[f] == statement_count && a->keep == 0) {
        a->free();
        varNames.erase(f);
    };
    std::cout<< "select time " <<  ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) <<'\n';
}


void emit_filter(char *s, char *f, int e)
{
    statement_count++;
    if (scan_state == 0) {
        if (stat.find(f) == stat.end()) {
            cout << "Filter : couldn't find variable " << f << endl;
            exit(1);
        };
        stat[s] = statement_count;
        stat[f] = statement_count;
        clean_queues();
        return;
    };

    if(varNames.find(f) == varNames.end()) {
        clean_queues();
        return;
    };

    CudaSet *a, *b;

    a = varNames.find(f)->second;
    a->name = f;
    std::clock_t start1 = std::clock();

    if(a->mRecCount == 0) {
        b = new CudaSet(0,1);
    }
    else {
        cout << "FILTER " << s << " " << f << " " << getFreeMem() << endl;

        b = a->copyDeviceStruct();
        b->name = s;

        unsigned int cycle_count = 1, cnt = 0;
        allocColumns(a, op_value);

        varNames[setMap[op_value.front()]]->oldRecCount = varNames[setMap[op_value.front()]]->mRecCount;

        if(a->segCount != 1)
            cycle_count = varNames[setMap[op_value.front()]]->segCount;

        oldCount = a->mRecCount;
        thrust::device_vector<unsigned int> p(a->maxRecs);

        for(unsigned int i = 0; i < cycle_count; i++) {
            map_check = zone_map_check(op_type,op_value,op_nums, op_nums_f, a, i);
            cout << "MAP CHECK " << map_check << endl;
            reset_offsets();
            if(map_check == 'R') {
                copyColumns(a, op_value, i, cnt);
                filter(op_type,op_value,op_nums, op_nums_f,a, b, i, p);
            }
            else  {
                setPrm(a,b,map_check,i);
            }
        };
        a->mRecCount = oldCount;
        varNames[setMap[op_value.front()]]->mRecCount = varNames[setMap[op_value.front()]]->oldRecCount;
        cout << "filter is finished " << b->mRecCount << " " << getFreeMem()  << endl;
        a->deAllocOnDevice();
    };

    clean_queues();

    if (varNames.count(s) > 0)
        varNames[s]->free();

    varNames[s] = b;

    if(stat[s] == statement_count) {
        b->free();
        varNames.erase(s);
    };
    if(stat[f] == statement_count && !a->keep) {
        //a->free();
        //varNames.erase(f);
    };
    std::cout<< "filter time " <<  ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) << " " << getFreeMem() << '\n';
}

void emit_store(char *s, char *f, char* sep)
{
    statement_count++;
    if (scan_state == 0) {
        if (stat.find(s) == stat.end()) {
            cout << "Store : couldn't find variable " << s << endl;
            exit(1);
        };
        stat[s] = statement_count;
        return;
    };

    if(varNames.find(s) == varNames.end())
        return;

    CudaSet* a = varNames.find(s)->second;
    cout << "STORE: " << s << " " << f << " " << sep << endl;

    int limit = 0;
    if(!op_nums.empty()) {
        limit = op_nums.front();
        op_nums.pop();
    };

    a->Store(f,sep, limit, 0);

    if(stat[s] == statement_count  && a->keep == 0) {
        a->free();
        varNames.erase(s);
    };
};


void emit_store_binary(char *s, char *f)
{
    statement_count++;
    if (scan_state == 0) {
        if (stat.find(s) == stat.end()) {
            cout << "Store : couldn't find variable " << s << endl;
            exit(1);
        };
        stat[s] = statement_count;
        return;
    };

    if(varNames.find(s) == varNames.end())
        return;

    CudaSet* a = varNames.find(s)->second;

    if(stat[f] == statement_count)
        a->deAllocOnDevice();

    printf("STORE: %s %s \n", s, f);

    int limit = 0;
    if(!op_nums.empty()) {
        limit = op_nums.front();
        op_nums.pop();
    };
    total_count = 0;
    total_segments = 0;
    fact_file_loaded = 0;

    while(!fact_file_loaded)	{
        cout << "LOADING " << f_file << " " << separator << endl;
        if(a->text_source)
            fact_file_loaded = a->LoadBigFile(f_file.c_str(), separator.c_str());
        a->Store(f,"", limit, 1);
    };

    if(stat[f] == statement_count && !a->keep) {
        a->free();
        varNames.erase(s);
    };

};


void emit_load_binary(char *s, char *f, int d)
{
    statement_count++;
    if (scan_state == 0) {
        stat[s] = statement_count;
        return;
    };

    printf("BINARY LOAD: %s %s \n", s, f);

    CudaSet *a;
    unsigned int segCount, maxRecs;
    char f1[100];
    strcpy(f1, f);
    strcat(f1,".");
    char col_pos[3];
    itoaa(cols.front(),col_pos);
    strcat(f1,col_pos);
    strcat(f1,".header");

    FILE* ff = fopen(f1, "rb");
    fread((char *)&totalRecs, 8, 1, ff);
    fread((char *)&segCount, 4, 1, ff);
    fread((char *)&maxRecs, 4, 1, ff);
    fclose(ff);

    queue<string> names(namevars);
    while(!names.empty()) {
        setMap[names.front()] = s;
        names.pop();
    };

    a = new CudaSet(namevars, typevars, sizevars, cols,totalRecs, f);
    a->segCount = segCount;
    a->maxRecs = maxRecs;
    a->keep = 1;
    varNames[s] = a;

    if(stat[s] == statement_count )  {
        a->free();
        varNames.erase(s);
    };
}





void emit_load(char *s, char *f, int d, char* sep)
{
    statement_count++;
    if (scan_state == 0) {
        stat[s] = statement_count;
        return;
    };

    printf("LOAD: %s %s %d  %s \n", s, f, d, sep);

    CudaSet *a;

    a = new CudaSet(namevars, typevars, sizevars, cols, process_count);
    a->mRecCount = 0;
    a->resize(process_count);
    a->keep = true;
    a->not_compressed = 1;

    string separator1(sep);
    separator = separator1;
    string ff(f);
    f_file = ff;
    a->maxRecs = a->mRecCount;
    a->segCount = 0;
    varNames[s] = a;

    if(stat[s] == statement_count)  {
        a->free();
        varNames.erase(s);
    };
}



void yyerror(char *s, ...)
{
    extern int yylineno;
    va_list ap;
    va_start(ap, s);

    fprintf(stderr, "%d: error: ", yylineno);
    vfprintf(stderr, s, ap);
    fprintf(stderr, "\n");
}

void clean_queues()
{
    while(!op_type.empty()) op_type.pop();
    while(!op_value.empty()) op_value.pop();
    while(!op_join.empty()) op_join.pop();
    while(!op_nums.empty()) op_nums.pop();
    while(!op_nums_f.empty()) op_nums_f.pop();
    while(!j_col_count.empty()) j_col_count.pop();
    while(!namevars.empty()) namevars.pop();
    while(!typevars.empty()) typevars.pop();
    while(!sizevars.empty()) sizevars.pop();
    while(!cols.empty()) cols.pop();

    sel_count = 0;
    join_cnt = 0;
    join_col_cnt = 0;
    distinct_cnt = 0;
    reset_offsets();
}



int main(int ac, char **av)
{
    extern FILE *yyin;
    //hipDeviceProp_t deviceProp;

    //hipGetDeviceProperties(&deviceProp, 0);
    //if (!deviceProp.canMapHostMemory)
    //    cout << "Device 0 cannot map host memory" << endl;

    //hipSetDeviceFlags(hipDeviceMapHost);
    cudppCreate(&theCudpp);

    long long int r30 = RAND_MAX*rand()+rand();
    long long int s30 = RAND_MAX*rand()+rand();
    long long int t4  = rand() & 0xf;

    hash_seed = (r30 << 34) + (s30 << 4) + t4;

    if (ac == 1) {
        cout << "Usage : alenka -l process_count script.sql" << endl;
        exit(1);
    };

    if(strcmp(av[1],"-l") == 0) {
        process_count = atoff(av[2]);
        cout << "Process count = " << process_count << endl;
    }
    else {
        process_count = 6200000;
        cout << "Process count = 6200000 " << endl;
    };

    if((yyin = fopen(av[ac-1], "r")) == NULL) {
        perror(av[ac-1]);
        exit(1);
    };

    if(yyparse()) {
        printf("SQL scan parse failed\n");
        exit(1);
    };

    scan_state = 1;

    std::clock_t start1 = std::clock();
    statement_count = 0;
    clean_queues();

    if(ac > 1 && (yyin = fopen(av[ac-1], "r")) == NULL) {
        perror(av[1]);
        exit(1);
    }

    PROC_FLUSH_BUF ( yyin );
    statement_count = 0;

    if(!yyparse())
        cout << "SQL scan parse worked" << endl;
    else
        cout << "SQL scan parse failed" << endl;

    if(alloced_sz)
        hipFree(alloced_tmp);

    fclose(yyin);
    std::cout<< "cycle time " <<  ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) <<'\n';
    cudppDestroy(theCudpp);

}



