#include "hip/hip_runtime.h"

/* A Bison parser, made by GNU Bison 2.4.1.  */

/* Skeleton implementation for Bison's Yacc-like parsers in C
   
      Copyright (C) 1984, 1989, 1990, 2000, 2001, 2002, 2003, 2004, 2005, 2006
   Free Software Foundation, Inc.
   
   This program is free software: you can redistribute it and/or modify
   it under the terms of the GNU General Public License as published by
   the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.
   
   This program is distributed in the hope that it will be useful,
   but WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU General Public License for more details.
   
   You should have received a copy of the GNU General Public License
   along with this program.  If not, see <http://www.gnu.org/licenses/>.  */

/* As a special exception, you may create a larger work that contains
   part or all of the Bison parser skeleton and distribute that work
   under terms of your choice, so long as that work isn't itself a
   parser generator using the skeleton or a modified version thereof
   as a parser skeleton.  Alternatively, if you modify or redistribute
   the parser skeleton itself, you may (at your option) remove this
   special exception, which will cause the skeleton and the resulting
   Bison output files to be licensed under the GNU General Public
   License without this special exception.
   
   This special exception was added by the Free Software Foundation in
   version 2.2 of Bison.  */

/* C LALR(1) parser skeleton written by Richard Stallman, by
   simplifying the original so-called "semantic" parser.  */

/* All symbols defined below should begin with yy or YY, to avoid
   infringing on user name space.  This should be done even for local
   variables, as they might otherwise be expanded by user macros.
   There are some unavoidable exceptions within include files to
   define necessary library symbols; they are noted "INFRINGES ON
   USER NAME SPACE" below.  */

/* Identify Bison output.  */
#define YYBISON 1

/* Bison version.  */
#define YYBISON_VERSION "2.4.1"

/* Skeleton name.  */
#define YYSKELETON_NAME "yacc.c"

/* Pure parsers.  */
#define YYPURE 0

/* Push parsers.  */
#define YYPUSH 0

/* Pull parsers.  */
#define YYPULL 1

/* Using locations.  */
#define YYLSP_NEEDED 0



/* Copy the first part of user declarations.  */

/* Line 189 of yacc.c  */
#line 17 "bison.y"


#include "lex.yy.c"
#include "cm.cu"


    void clean_queues();
    void order_inplace(CudaSet* a, stack<string> exe_type);
    void yyerror(char *s, ...);
    void emit(char *s, ...);
    void emit_mul();
    void emit_add();
    void emit_minus();
    void emit_div();
    void emit_and();
    void emit_eq();
    void emit_or();
    void emit_cmp(int val);
    void emit_var(char *s, int c, char *f);
    void emit_var_asc(char *s);
    void emit_var_desc(char *s);
    void emit_name(char *name);
    void emit_count();
    void emit_sum();
    void emit_average();
    void emit_min();
    void emit_max();
    void emit_string(char *str);
    void emit_number(int_type val);
    void emit_float(float_type val);
    void emit_decimal(float_type val);
    void emit_sel_name(char* name);
    void emit_limit(int val);
    void emit_union(char *s, char *f1, char *f2);
    void emit_varchar(char *s, int c, char *f, int d);
    void emit_load(char *s, char *f, int d, char* sep);
    void emit_load_binary(char *s, char *f, int d);
    void emit_store(char *s, char *f, char* sep);
    void emit_store_binary(char *s, char *f, char* sep);
    void emit_store_binary(char *s, char *f);
    void emit_filter(char *s, char *f, int e);
    void emit_order(char *s, char *f, int e, int ll = 0);
    void emit_group(char *s, char *f, int e);
    void emit_select(char *s, char *f, int ll);
    void emit_join(char *s, char *j1);
    void emit_join_tab(char *s);
    void emit_distinct(char *s, char *f);



/* Line 189 of yacc.c  */
#line 124 "bison.cu"

/* Enabling traces.  */
#ifndef YYDEBUG
# define YYDEBUG 0
#endif

/* Enabling verbose error messages.  */
#ifdef YYERROR_VERBOSE
# undef YYERROR_VERBOSE
# define YYERROR_VERBOSE 1
#else
# define YYERROR_VERBOSE 0
#endif

/* Enabling the token table.  */
#ifndef YYTOKEN_TABLE
# define YYTOKEN_TABLE 0
#endif


/* Tokens.  */
#ifndef YYTOKENTYPE
# define YYTOKENTYPE
   /* Put the tokens into the symbol table, so that GDB and other debuggers
      know about them.  */
   enum yytokentype {
     FILENAME = 258,
     NAME = 259,
     STRING = 260,
     INTNUM = 261,
     DECIMAL1 = 262,
     BOOL1 = 263,
     APPROXNUM = 264,
     USERVAR = 265,
     ASSIGN = 266,
     EQUAL = 267,
     OR = 268,
     XOR = 269,
     AND = 270,
     REGEXP = 271,
     LIKE = 272,
     IS = 273,
     IN = 274,
     NOT = 275,
     BETWEEN = 276,
     COMPARISON = 277,
     SHIFT = 278,
     MOD = 279,
     UMINUS = 280,
     LOAD = 281,
     STREAM = 282,
     FILTER = 283,
     BY = 284,
     JOIN = 285,
     STORE = 286,
     INTO = 287,
     GROUP = 288,
     FROM = 289,
     SELECT = 290,
     AS = 291,
     ORDER = 292,
     ASC = 293,
     DESC = 294,
     COUNT = 295,
     USING = 296,
     SUM = 297,
     AVG = 298,
     MIN = 299,
     MAX = 300,
     LIMIT = 301,
     ON = 302,
     BINARY = 303
   };
#endif



#if ! defined YYSTYPE && ! defined YYSTYPE_IS_DECLARED
typedef union YYSTYPE
{

/* Line 214 of yacc.c  */
#line 67 "bison.y"

    int intval;
    float floatval;
    char *strval;
    int subtok;



/* Line 214 of yacc.c  */
#line 217 "bison.cu"
} YYSTYPE;
# define YYSTYPE_IS_TRIVIAL 1
# define yystype YYSTYPE /* obsolescent; will be withdrawn */
# define YYSTYPE_IS_DECLARED 1
#endif


/* Copy the second part of user declarations.  */


/* Line 264 of yacc.c  */
#line 229 "bison.cu"

#ifdef short
# undef short
#endif

#ifdef YYTYPE_UINT8
typedef YYTYPE_UINT8 yytype_uint8;
#else
typedef unsigned char yytype_uint8;
#endif

#ifdef YYTYPE_INT8
typedef YYTYPE_INT8 yytype_int8;
#elif (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
typedef signed char yytype_int8;
#else
typedef short int yytype_int8;
#endif

#ifdef YYTYPE_UINT16
typedef YYTYPE_UINT16 yytype_uint16;
#else
typedef unsigned short int yytype_uint16;
#endif

#ifdef YYTYPE_INT16
typedef YYTYPE_INT16 yytype_int16;
#else
typedef short int yytype_int16;
#endif

#ifndef YYSIZE_T
# ifdef __SIZE_TYPE__
#  define YYSIZE_T __SIZE_TYPE__
# elif defined size_t
#  define YYSIZE_T size_t
# elif ! defined YYSIZE_T && (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
#  include <stddef.h> /* INFRINGES ON USER NAME SPACE */
#  define YYSIZE_T size_t
# else
#  define YYSIZE_T unsigned int
# endif
#endif

#define YYSIZE_MAXIMUM ((YYSIZE_T) -1)

#ifndef YY_
# if YYENABLE_NLS
#  if ENABLE_NLS
#   include <libintl.h> /* INFRINGES ON USER NAME SPACE */
#   define YY_(msgid) dgettext ("bison-runtime", msgid)
#  endif
# endif
# ifndef YY_
#  define YY_(msgid) msgid
# endif
#endif

/* Suppress unused-variable warnings by "using" E.  */
#if ! defined lint || defined __GNUC__
# define YYUSE(e) ((void) (e))
#else
# define YYUSE(e) /* empty */
#endif

/* Identity function, used to suppress warnings about constant conditions.  */
#ifndef lint
# define YYID(n) (n)
#else
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static int
YYID (int yyi)
#else
static int
YYID (yyi)
    int yyi;
#endif
{
  return yyi;
}
#endif

#if ! defined yyoverflow || YYERROR_VERBOSE

/* The parser invokes alloca or malloc; define the necessary symbols.  */

# ifdef YYSTACK_USE_ALLOCA
#  if YYSTACK_USE_ALLOCA
#   ifdef __GNUC__
#    define YYSTACK_ALLOC __builtin_alloca
#   elif defined __BUILTIN_VA_ARG_INCR
#    include <alloca.h> /* INFRINGES ON USER NAME SPACE */
#   elif defined _AIX
#    define YYSTACK_ALLOC __alloca
#   elif defined _MSC_VER
#    include <malloc.h> /* INFRINGES ON USER NAME SPACE */
#    define alloca _alloca
#   else
#    define YYSTACK_ALLOC alloca
#    if ! defined _ALLOCA_H && ! defined _STDLIB_H && (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
#     include <stdlib.h> /* INFRINGES ON USER NAME SPACE */
#     ifndef _STDLIB_H
#      define _STDLIB_H 1
#     endif
#    endif
#   endif
#  endif
# endif

# ifdef YYSTACK_ALLOC
   /* Pacify GCC's `empty if-body' warning.  */
#  define YYSTACK_FREE(Ptr) do { /* empty */; } while (YYID (0))
#  ifndef YYSTACK_ALLOC_MAXIMUM
    /* The OS might guarantee only one guard page at the bottom of the stack,
       and a page size can be as small as 4096 bytes.  So we cannot safely
       invoke alloca (N) if N exceeds 4096.  Use a slightly smaller number
       to allow for a few compiler-allocated temporary stack slots.  */
#   define YYSTACK_ALLOC_MAXIMUM 4032 /* reasonable circa 2006 */
#  endif
# else
#  define YYSTACK_ALLOC YYMALLOC
#  define YYSTACK_FREE YYFREE
#  ifndef YYSTACK_ALLOC_MAXIMUM
#   define YYSTACK_ALLOC_MAXIMUM YYSIZE_MAXIMUM
#  endif
#  if (defined __cplusplus && ! defined _STDLIB_H \
       && ! ((defined YYMALLOC || defined malloc) \
	     && (defined YYFREE || defined free)))
#   include <stdlib.h> /* INFRINGES ON USER NAME SPACE */
#   ifndef _STDLIB_H
#    define _STDLIB_H 1
#   endif
#  endif
#  ifndef YYMALLOC
#   define YYMALLOC malloc
#   if ! defined malloc && ! defined _STDLIB_H && (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
void *malloc (YYSIZE_T); /* INFRINGES ON USER NAME SPACE */
#   endif
#  endif
#  ifndef YYFREE
#   define YYFREE free
#   if ! defined free && ! defined _STDLIB_H && (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
void free (void *); /* INFRINGES ON USER NAME SPACE */
#   endif
#  endif
# endif
#endif /* ! defined yyoverflow || YYERROR_VERBOSE */


#if (! defined yyoverflow \
     && (! defined __cplusplus \
	 || (defined YYSTYPE_IS_TRIVIAL && YYSTYPE_IS_TRIVIAL)))

/* A type that is properly aligned for any stack member.  */
union yyalloc
{
  yytype_int16 yyss_alloc;
  YYSTYPE yyvs_alloc;
};

/* The size of the maximum gap between one aligned stack and the next.  */
# define YYSTACK_GAP_MAXIMUM (sizeof (union yyalloc) - 1)

/* The size of an array large to enough to hold all stacks, each with
   N elements.  */
# define YYSTACK_BYTES(N) \
     ((N) * (sizeof (yytype_int16) + sizeof (YYSTYPE)) \
      + YYSTACK_GAP_MAXIMUM)

/* Copy COUNT objects from FROM to TO.  The source and destination do
   not overlap.  */
# ifndef YYCOPY
#  if defined __GNUC__ && 1 < __GNUC__
#   define YYCOPY(To, From, Count) \
      __builtin_memcpy (To, From, (Count) * sizeof (*(From)))
#  else
#   define YYCOPY(To, From, Count)		\
      do					\
	{					\
	  YYSIZE_T yyi;				\
	  for (yyi = 0; yyi < (Count); yyi++)	\
	    (To)[yyi] = (From)[yyi];		\
	}					\
      while (YYID (0))
#  endif
# endif

/* Relocate STACK from its old location to the new one.  The
   local variables YYSIZE and YYSTACKSIZE give the old and new number of
   elements in the stack, and YYPTR gives the new location of the
   stack.  Advance YYPTR to a properly aligned location for the next
   stack.  */
# define YYSTACK_RELOCATE(Stack_alloc, Stack)				\
    do									\
      {									\
	YYSIZE_T yynewbytes;						\
	YYCOPY (&yyptr->Stack_alloc, Stack, yysize);			\
	Stack = &yyptr->Stack_alloc;					\
	yynewbytes = yystacksize * sizeof (*Stack) + YYSTACK_GAP_MAXIMUM; \
	yyptr += yynewbytes / sizeof (*yyptr);				\
      }									\
    while (YYID (0))

#endif

/* YYFINAL -- State number of the termination state.  */
#define YYFINAL  8
/* YYLAST -- Last index in YYTABLE.  */
#define YYLAST   446

/* YYNTOKENS -- Number of terminals.  */
#define YYNTOKENS  66
/* YYNNTS -- Number of nonterminals.  */
#define YYNNTS  13
/* YYNRULES -- Number of rules.  */
#define YYNRULES  62
/* YYNRULES -- Number of states.  */
#define YYNSTATES  153

/* YYTRANSLATE(YYLEX) -- Bison symbol number corresponding to YYLEX.  */
#define YYUNDEFTOK  2
#define YYMAXUTOK   303

#define YYTRANSLATE(YYX)						\
  ((unsigned int) (YYX) <= YYMAXUTOK ? yytranslate[YYX] : YYUNDEFTOK)

/* YYTRANSLATE[YYLEX] -- Bison symbol number corresponding to YYLEX.  */
static const yytype_uint8 yytranslate[] =
{
       0,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,    20,     2,     2,     2,    31,    25,     2,
      59,    60,    29,    27,    65,    28,    61,    30,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,    64,    58,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,    33,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,    62,    24,    63,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     1,     2,     3,     4,
       5,     6,     7,     8,     9,    10,    11,    12,    13,    14,
      15,    16,    17,    18,    19,    21,    22,    23,    26,    32,
      34,    35,    36,    37,    38,    39,    40,    41,    42,    43,
      44,    45,    46,    47,    48,    49,    50,    51,    52,    53,
      54,    55,    56,    57
};

#if YYDEBUG
/* YYPRHS[YYN] -- Index of the first RHS symbol of rule number YYN in
   YYRHS.  */
static const yytype_uint16 yyprhs[] =
{
       0,     0,     3,     6,    10,    12,    20,    33,    43,    49,
      56,    64,    74,    81,    83,    87,    89,    91,    93,    95,
      97,    99,   109,   116,   119,   122,   127,   132,   137,   142,
     147,   151,   155,   159,   163,   167,   171,   175,   179,   183,
     187,   191,   194,   197,   201,   207,   211,   215,   220,   221,
     225,   229,   235,   237,   241,   243,   247,   248,   250,   253,
     258,   264,   265
};

/* YYRHS -- A `-1'-separated list of the rules' RHS.  */
static const yytype_int8 yyrhs[] =
{
      67,     0,    -1,    68,    58,    -1,    67,    68,    58,    -1,
      69,    -1,     4,    11,    44,    72,    43,     4,    71,    -1,
       4,    11,    35,     3,    50,    59,     3,    60,    45,    59,
      73,    60,    -1,     4,    11,    35,     3,    57,    45,    59,
      73,    60,    -1,     4,    11,    37,     4,    76,    -1,     4,
      11,    46,     4,    38,    75,    -1,     4,    11,    44,    72,
      43,     4,    77,    -1,    40,     4,    41,     3,    50,    59,
       3,    60,    78,    -1,    40,     4,    41,     3,    78,    57,
      -1,     4,    -1,     4,    61,     4,    -1,    10,    -1,     5,
      -1,     6,    -1,     9,    -1,     7,    -1,     8,    -1,     4,
      62,     6,    63,    64,     4,    59,     6,    60,    -1,     4,
      62,     6,    63,    64,     4,    -1,     4,    47,    -1,     4,
      48,    -1,    49,    59,    70,    60,    -1,    51,    59,    70,
      60,    -1,    52,    59,    70,    60,    -1,    53,    59,    70,
      60,    -1,    54,    59,    70,    60,    -1,    70,    27,    70,
      -1,    70,    28,    70,    -1,    70,    29,    70,    -1,    70,
      30,    70,    -1,    70,    31,    70,    -1,    70,    32,    70,
      -1,    70,    15,    70,    -1,    70,    12,    70,    -1,    70,
      13,    70,    -1,    70,    14,    70,    -1,    70,    26,    70,
      -1,    21,    70,    -1,    20,    70,    -1,    70,    23,    70,
      -1,    70,    23,    59,    69,    60,    -1,    59,    70,    60,
      -1,    70,    18,     8,    -1,    70,    18,    21,     8,    -1,
      -1,    42,    38,    74,    -1,    70,    45,     4,    -1,    72,
      65,    70,    45,     4,    -1,    70,    -1,    73,    65,    70,
      -1,    70,    -1,    70,    65,    74,    -1,    -1,    74,    -1,
      38,    70,    -1,    39,     4,    56,    70,    -1,    39,     4,
      56,    70,    77,    -1,    -1,    55,     6,    -1
};

/* YYRLINE[YYN] -- source line where rule number YYN was defined.  */
static const yytype_uint8 yyrline[] =
{
       0,   137,   137,   138,   142,   145,   147,   149,   151,   153,
     155,   157,   159,   164,   165,   166,   167,   168,   169,   170,
     171,   172,   173,   174,   175,   176,   177,   178,   179,   180,
     184,   185,   186,   187,   188,   189,   191,   192,   193,   194,
     195,   196,   197,   198,   200,   201,   205,   206,   209,   212,
     216,   217,   221,   222,   226,   227,   230,   232,   235,   238,
     239,   241,   244
};
#endif

#if YYDEBUG || YYERROR_VERBOSE || YYTOKEN_TABLE
/* YYTNAME[SYMBOL-NUM] -- String name of the symbol SYMBOL-NUM.
   First, the terminals, then, starting at YYNTOKENS, nonterminals.  */
static const char *const yytname[] =
{
  "$end", "error", "$undefined", "FILENAME", "NAME", "STRING", "INTNUM",
  "DECIMAL1", "BOOL1", "APPROXNUM", "USERVAR", "ASSIGN", "EQUAL", "OR",
  "XOR", "AND", "REGEXP", "LIKE", "IS", "IN", "'!'", "NOT", "BETWEEN",
  "COMPARISON", "'|'", "'&'", "SHIFT", "'+'", "'-'", "'*'", "'/'", "'%'",
  "MOD", "'^'", "UMINUS", "LOAD", "STREAM", "FILTER", "BY", "JOIN",
  "STORE", "INTO", "GROUP", "FROM", "SELECT", "AS", "ORDER", "ASC", "DESC",
  "COUNT", "USING", "SUM", "AVG", "MIN", "MAX", "LIMIT", "ON", "BINARY",
  "';'", "'('", "')'", "'.'", "'{'", "'}'", "':'", "','", "$accept",
  "stmt_list", "stmt", "select_stmt", "expr", "opt_group_list",
  "expr_list", "load_list", "val_list", "opt_val_list", "opt_where",
  "join_list", "opt_limit", 0
};
#endif

# ifdef YYPRINT
/* YYTOKNUM[YYLEX-NUM] -- Internal token number corresponding to
   token YYLEX-NUM.  */
static const yytype_uint16 yytoknum[] =
{
       0,   256,   257,   258,   259,   260,   261,   262,   263,   264,
     265,   266,   267,   268,   269,   270,   271,   272,   273,   274,
      33,   275,   276,   277,   124,    38,   278,    43,    45,    42,
      47,    37,   279,    94,   280,   281,   282,   283,   284,   285,
     286,   287,   288,   289,   290,   291,   292,   293,   294,   295,
     296,   297,   298,   299,   300,   301,   302,   303,    59,    40,
      41,    46,   123,   125,    58,    44
};
# endif

/* YYR1[YYN] -- Symbol number of symbol that rule YYN derives.  */
static const yytype_uint8 yyr1[] =
{
       0,    66,    67,    67,    68,    69,    69,    69,    69,    69,
      69,    69,    69,    70,    70,    70,    70,    70,    70,    70,
      70,    70,    70,    70,    70,    70,    70,    70,    70,    70,
      70,    70,    70,    70,    70,    70,    70,    70,    70,    70,
      70,    70,    70,    70,    70,    70,    70,    70,    71,    71,
      72,    72,    73,    73,    74,    74,    75,    75,    76,    77,
      77,    78,    78
};

/* YYR2[YYN] -- Number of symbols composing right hand side of rule YYN.  */
static const yytype_uint8 yyr2[] =
{
       0,     2,     2,     3,     1,     7,    12,     9,     5,     6,
       7,     9,     6,     1,     3,     1,     1,     1,     1,     1,
       1,     9,     6,     2,     2,     4,     4,     4,     4,     4,
       3,     3,     3,     3,     3,     3,     3,     3,     3,     3,
       3,     2,     2,     3,     5,     3,     3,     4,     0,     3,
       3,     5,     1,     3,     1,     3,     0,     1,     2,     4,
       5,     0,     2
};

/* YYDEFACT[STATE-NAME] -- Default rule to reduce with in state
   STATE-NUM when YYTABLE doesn't specify something else to do.  Zero
   means the default is an error.  */
static const yytype_uint8 yydefact[] =
{
       0,     0,     0,     0,     0,     4,     0,     0,     1,     0,
       2,     0,     0,     0,     0,     0,     3,     0,     0,    13,
      16,    17,    19,    20,    18,    15,     0,     0,     0,     0,
       0,     0,     0,     0,     0,     0,     0,    61,     0,     0,
       0,     8,    23,    24,     0,     0,    42,    41,     0,     0,
       0,     0,     0,     0,     0,     0,     0,     0,     0,     0,
       0,     0,     0,     0,     0,     0,     0,     0,     0,     0,
      56,     0,     0,     0,     0,     0,    58,    14,     0,     0,
       0,     0,     0,     0,    45,    37,    38,    39,    36,    46,
       0,     0,    43,    40,    30,    31,    32,    33,    34,    35,
      50,    48,     0,    54,    57,     9,     0,    62,    12,     0,
       0,     0,    25,    26,    27,    28,    29,    47,    13,     0,
       0,     0,     5,    10,     0,     0,     0,     0,    52,     0,
       0,    44,     0,     0,    51,    55,    61,     0,     7,     0,
      22,     0,    49,    11,     0,    53,     0,    59,     0,     0,
      60,     6,    21
};

/* YYDEFGOTO[NTERM-NUM].  */
static const yytype_int16 yydefgoto[] =
{
      -1,     3,     4,     5,   103,   122,    35,   129,   104,   105,
      41,   123,    73
};

/* YYPACT[STATE-NUM] -- Index in YYTABLE of the portion describing
   STATE-NUM.  */
#define YYPACT_NINF -122
static const yytype_int16 yypact[] =
{
      14,    -3,     7,     5,   -34,  -122,    50,    23,  -122,    28,
    -122,    52,    61,    62,    77,    85,  -122,   -35,    51,   -45,
    -122,  -122,  -122,  -122,  -122,  -122,    62,    62,    33,    36,
      44,    49,    58,    62,   300,   -42,    71,   -29,    59,    65,
      62,  -122,  -122,  -122,   115,   114,     2,     2,    62,    62,
      62,    62,    62,   171,    62,    62,    62,    62,    -2,   128,
      62,    62,    62,    62,    62,    62,    62,   118,   119,    62,
      62,    66,   121,    67,   126,    84,   364,  -122,    81,   192,
     214,   235,   257,   278,  -122,   364,   383,   401,   142,  -122,
     122,    53,   408,   414,    69,    69,  -122,  -122,  -122,  -122,
    -122,   -32,   321,   127,  -122,  -122,   143,  -122,  -122,    87,
      62,    88,  -122,  -122,  -122,  -122,  -122,  -122,    29,    91,
     157,   124,  -122,  -122,   159,    62,   104,   130,   364,    15,
     162,  -122,   111,    62,  -122,  -122,   123,   117,  -122,    62,
     129,    62,  -122,  -122,    62,   364,   184,   342,    19,   131,
    -122,  -122,  -122
};

/* YYPGOTO[NTERM-NUM].  */
static const yytype_int16 yypgoto[] =
{
    -122,  -122,   190,   105,   -13,  -122,  -122,    64,  -121,  -122,
    -122,    48,    73
};

/* YYTABLE[YYPACT[STATE-NUM]].  What to do in state STATE-NUM.  If
   positive, shift that token.  If negative, reduce the rule which
   number is the opposite.  If zero, do what YYDEFACT says.
   If YYTABLE_NINF, syntax error.  */
#define YYTABLE_NINF -1
static const yytype_uint8 yytable[] =
{
      34,    68,    42,    43,   135,     8,    89,   120,     6,     1,
     121,     7,   142,    46,    47,    38,    44,    45,     1,    90,
      53,    71,    39,    69,    10,    59,    72,    76,    60,    61,
      62,    63,    64,    65,    66,    79,    80,    81,    82,    83,
       6,    85,    86,    87,    88,     2,    92,    93,    94,    95,
      96,    97,    98,    99,     2,    17,   102,   118,    20,    21,
      22,    23,    24,    25,    15,    18,    19,    20,    21,    22,
      23,    24,    25,    26,    27,   138,    42,    43,    53,   151,
     139,    36,    26,    27,   139,    11,    16,    12,    37,    40,
      44,    45,    48,     2,    13,    49,    14,   128,    63,    64,
      65,    66,    28,    50,    29,    30,    31,    32,    51,    70,
      75,    28,    33,    29,    30,    31,    32,    52,    74,    77,
      78,    33,   100,   101,   108,   106,   145,   107,   147,   109,
     117,   128,    19,    20,    21,    22,    23,    24,    25,    54,
      55,    56,    57,   110,   111,    58,   126,   127,    26,    27,
      59,   131,   130,    60,    61,    62,    63,    64,    65,    66,
      58,   132,   133,   134,   136,    59,   140,   141,    60,    61,
      62,    63,    64,    65,    66,   137,   144,    28,    72,    29,
      30,    31,    32,    54,    55,    56,    57,    91,   146,    58,
     149,   152,   125,     9,    59,   150,   119,    60,    61,    62,
      63,    64,    65,    66,    54,    55,    56,    57,   148,   143,
      58,     0,     0,     0,     0,    59,     0,     0,    60,    61,
      62,    63,    64,    65,    66,     0,    54,    55,    56,    57,
       0,    84,    58,     0,     0,     0,     0,    59,     0,     0,
      60,    61,    62,    63,    64,    65,    66,    54,    55,    56,
      57,     0,   112,    58,     0,     0,     0,     0,    59,     0,
       0,    60,    61,    62,    63,    64,    65,    66,     0,    54,
      55,    56,    57,     0,   113,    58,     0,     0,     0,     0,
      59,     0,     0,    60,    61,    62,    63,    64,    65,    66,
      54,    55,    56,    57,     0,   114,    58,     0,     0,     0,
       0,    59,     0,     0,    60,    61,    62,    63,    64,    65,
      66,     0,    54,    55,    56,    57,     0,   115,    58,     0,
       0,     0,     0,    59,     0,     0,    60,    61,    62,    63,
      64,    65,    66,    54,    55,    56,    57,     0,   116,    58,
       0,     0,     0,     0,    59,    67,     0,    60,    61,    62,
      63,    64,    65,    66,    54,    55,    56,    57,     0,     0,
      58,     0,     0,     0,     0,    59,   124,     0,    60,    61,
      62,    63,    64,    65,    66,     0,    54,    55,    56,    57,
       0,   120,    58,     0,     0,     0,     0,    59,     0,     0,
      60,    61,    62,    63,    64,    65,    66,    56,    57,     0,
       0,    58,     0,     0,     0,     0,    59,     0,     0,    60,
      61,    62,    63,    64,    65,    66,    57,     0,     0,    58,
       0,     0,     0,     0,    59,     0,     0,    60,    61,    62,
      63,    64,    65,    66,    60,    61,    62,    63,    64,    65,
      66,    61,    62,    63,    64,    65,    66
};

static const yytype_int16 yycheck[] =
{
      13,    43,    47,    48,   125,     0,     8,    39,    11,     4,
      42,     4,   133,    26,    27,    50,    61,    62,     4,    21,
      33,    50,    57,    65,    58,    23,    55,    40,    26,    27,
      28,    29,    30,    31,    32,    48,    49,    50,    51,    52,
      11,    54,    55,    56,    57,    40,    59,    60,    61,    62,
      63,    64,    65,    66,    40,     3,    69,     4,     5,     6,
       7,     8,     9,    10,    41,     4,     4,     5,     6,     7,
       8,     9,    10,    20,    21,    60,    47,    48,    91,    60,
      65,     4,    20,    21,    65,    35,    58,    37,     3,    38,
      61,    62,    59,    40,    44,    59,    46,   110,    29,    30,
      31,    32,    49,    59,    51,    52,    53,    54,    59,    38,
      45,    49,    59,    51,    52,    53,    54,    59,    59,     4,
       6,    59,     4,     4,    57,    59,   139,     6,   141,     3,
       8,   144,     4,     5,     6,     7,     8,     9,    10,    12,
      13,    14,    15,    59,    63,    18,     3,    60,    20,    21,
      23,    60,    64,    26,    27,    28,    29,    30,    31,    32,
      18,     4,    38,     4,    60,    23,     4,    56,    26,    27,
      28,    29,    30,    31,    32,    45,    59,    49,    55,    51,
      52,    53,    54,    12,    13,    14,    15,    59,    59,    18,
       6,    60,    65,     3,    23,   147,    91,    26,    27,    28,
      29,    30,    31,    32,    12,    13,    14,    15,   144,   136,
      18,    -1,    -1,    -1,    -1,    23,    -1,    -1,    26,    27,
      28,    29,    30,    31,    32,    -1,    12,    13,    14,    15,
      -1,    60,    18,    -1,    -1,    -1,    -1,    23,    -1,    -1,
      26,    27,    28,    29,    30,    31,    32,    12,    13,    14,
      15,    -1,    60,    18,    -1,    -1,    -1,    -1,    23,    -1,
      -1,    26,    27,    28,    29,    30,    31,    32,    -1,    12,
      13,    14,    15,    -1,    60,    18,    -1,    -1,    -1,    -1,
      23,    -1,    -1,    26,    27,    28,    29,    30,    31,    32,
      12,    13,    14,    15,    -1,    60,    18,    -1,    -1,    -1,
      -1,    23,    -1,    -1,    26,    27,    28,    29,    30,    31,
      32,    -1,    12,    13,    14,    15,    -1,    60,    18,    -1,
      -1,    -1,    -1,    23,    -1,    -1,    26,    27,    28,    29,
      30,    31,    32,    12,    13,    14,    15,    -1,    60,    18,
      -1,    -1,    -1,    -1,    23,    45,    -1,    26,    27,    28,
      29,    30,    31,    32,    12,    13,    14,    15,    -1,    -1,
      18,    -1,    -1,    -1,    -1,    23,    45,    -1,    26,    27,
      28,    29,    30,    31,    32,    -1,    12,    13,    14,    15,
      -1,    39,    18,    -1,    -1,    -1,    -1,    23,    -1,    -1,
      26,    27,    28,    29,    30,    31,    32,    14,    15,    -1,
      -1,    18,    -1,    -1,    -1,    -1,    23,    -1,    -1,    26,
      27,    28,    29,    30,    31,    32,    15,    -1,    -1,    18,
      -1,    -1,    -1,    -1,    23,    -1,    -1,    26,    27,    28,
      29,    30,    31,    32,    26,    27,    28,    29,    30,    31,
      32,    27,    28,    29,    30,    31,    32
};

/* YYSTOS[STATE-NUM] -- The (internal number of the) accessing
   symbol of state STATE-NUM.  */
static const yytype_uint8 yystos[] =
{
       0,     4,    40,    67,    68,    69,    11,     4,     0,    68,
      58,    35,    37,    44,    46,    41,    58,     3,     4,     4,
       5,     6,     7,     8,     9,    10,    20,    21,    49,    51,
      52,    53,    54,    59,    70,    72,     4,     3,    50,    57,
      38,    76,    47,    48,    61,    62,    70,    70,    59,    59,
      59,    59,    59,    70,    12,    13,    14,    15,    18,    23,
      26,    27,    28,    29,    30,    31,    32,    45,    43,    65,
      38,    50,    55,    78,    59,    45,    70,     4,     6,    70,
      70,    70,    70,    70,    60,    70,    70,    70,    70,     8,
      21,    59,    70,    70,    70,    70,    70,    70,    70,    70,
       4,     4,    70,    70,    74,    75,    59,     6,    57,     3,
      59,    63,    60,    60,    60,    60,    60,     8,     4,    69,
      39,    42,    71,    77,    45,    65,     3,    60,    70,    73,
      64,    60,     4,    38,     4,    74,    60,    45,    60,    65,
       4,    56,    74,    78,    59,    70,    59,    70,    73,     6,
      77,    60,    60
};

#define yyerrok		(yyerrstatus = 0)
#define yyclearin	(yychar = YYEMPTY)
#define YYEMPTY		(-2)
#define YYEOF		0

#define YYACCEPT	goto yyacceptlab
#define YYABORT		goto yyabortlab
#define YYERROR		goto yyerrorlab


/* Like YYERROR except do call yyerror.  This remains here temporarily
   to ease the transition to the new meaning of YYERROR, for GCC.
   Once GCC version 2 has supplanted version 1, this can go.  */

#define YYFAIL		goto yyerrlab

#define YYRECOVERING()  (!!yyerrstatus)

#define YYBACKUP(Token, Value)					\
do								\
  if (yychar == YYEMPTY && yylen == 1)				\
    {								\
      yychar = (Token);						\
      yylval = (Value);						\
      yytoken = YYTRANSLATE (yychar);				\
      YYPOPSTACK (1);						\
      goto yybackup;						\
    }								\
  else								\
    {								\
      yyerror (YY_("syntax error: cannot back up")); \
      YYERROR;							\
    }								\
while (YYID (0))


#define YYTERROR	1
#define YYERRCODE	256


/* YYLLOC_DEFAULT -- Set CURRENT to span from RHS[1] to RHS[N].
   If N is 0, then set CURRENT to the empty location which ends
   the previous symbol: RHS[0] (always defined).  */

#define YYRHSLOC(Rhs, K) ((Rhs)[K])
#ifndef YYLLOC_DEFAULT
# define YYLLOC_DEFAULT(Current, Rhs, N)				\
    do									\
      if (YYID (N))                                                    \
	{								\
	  (Current).first_line   = YYRHSLOC (Rhs, 1).first_line;	\
	  (Current).first_column = YYRHSLOC (Rhs, 1).first_column;	\
	  (Current).last_line    = YYRHSLOC (Rhs, N).last_line;		\
	  (Current).last_column  = YYRHSLOC (Rhs, N).last_column;	\
	}								\
      else								\
	{								\
	  (Current).first_line   = (Current).last_line   =		\
	    YYRHSLOC (Rhs, 0).last_line;				\
	  (Current).first_column = (Current).last_column =		\
	    YYRHSLOC (Rhs, 0).last_column;				\
	}								\
    while (YYID (0))
#endif


/* YY_LOCATION_PRINT -- Print the location on the stream.
   This macro was not mandated originally: define only if we know
   we won't break user code: when these are the locations we know.  */

#ifndef YY_LOCATION_PRINT
# if YYLTYPE_IS_TRIVIAL
#  define YY_LOCATION_PRINT(File, Loc)			\
     fprintf (File, "%d.%d-%d.%d",			\
	      (Loc).first_line, (Loc).first_column,	\
	      (Loc).last_line,  (Loc).last_column)
# else
#  define YY_LOCATION_PRINT(File, Loc) ((void) 0)
# endif
#endif


/* YYLEX -- calling `yylex' with the right arguments.  */

#ifdef YYLEX_PARAM
# define YYLEX yylex (YYLEX_PARAM)
#else
# define YYLEX yylex ()
#endif

/* Enable debugging if requested.  */
#if YYDEBUG

# ifndef YYFPRINTF
#  include <stdio.h> /* INFRINGES ON USER NAME SPACE */
#  define YYFPRINTF fprintf
# endif

# define YYDPRINTF(Args)			\
do {						\
  if (yydebug)					\
    YYFPRINTF Args;				\
} while (YYID (0))

# define YY_SYMBOL_PRINT(Title, Type, Value, Location)			  \
do {									  \
  if (yydebug)								  \
    {									  \
      YYFPRINTF (stderr, "%s ", Title);					  \
      yy_symbol_print (stderr,						  \
		  Type, Value); \
      YYFPRINTF (stderr, "\n");						  \
    }									  \
} while (YYID (0))


/*--------------------------------.
| Print this symbol on YYOUTPUT.  |
`--------------------------------*/

/*ARGSUSED*/
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yy_symbol_value_print (FILE *yyoutput, int yytype, YYSTYPE const * const yyvaluep)
#else
static void
yy_symbol_value_print (yyoutput, yytype, yyvaluep)
    FILE *yyoutput;
    int yytype;
    YYSTYPE const * const yyvaluep;
#endif
{
  if (!yyvaluep)
    return;
# ifdef YYPRINT
  if (yytype < YYNTOKENS)
    YYPRINT (yyoutput, yytoknum[yytype], *yyvaluep);
# else
  YYUSE (yyoutput);
# endif
  switch (yytype)
    {
      default:
	break;
    }
}


/*--------------------------------.
| Print this symbol on YYOUTPUT.  |
`--------------------------------*/

#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yy_symbol_print (FILE *yyoutput, int yytype, YYSTYPE const * const yyvaluep)
#else
static void
yy_symbol_print (yyoutput, yytype, yyvaluep)
    FILE *yyoutput;
    int yytype;
    YYSTYPE const * const yyvaluep;
#endif
{
  if (yytype < YYNTOKENS)
    YYFPRINTF (yyoutput, "token %s (", yytname[yytype]);
  else
    YYFPRINTF (yyoutput, "nterm %s (", yytname[yytype]);

  yy_symbol_value_print (yyoutput, yytype, yyvaluep);
  YYFPRINTF (yyoutput, ")");
}

/*------------------------------------------------------------------.
| yy_stack_print -- Print the state stack from its BOTTOM up to its |
| TOP (included).                                                   |
`------------------------------------------------------------------*/

#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yy_stack_print (yytype_int16 *yybottom, yytype_int16 *yytop)
#else
static void
yy_stack_print (yybottom, yytop)
    yytype_int16 *yybottom;
    yytype_int16 *yytop;
#endif
{
  YYFPRINTF (stderr, "Stack now");
  for (; yybottom <= yytop; yybottom++)
    {
      int yybot = *yybottom;
      YYFPRINTF (stderr, " %d", yybot);
    }
  YYFPRINTF (stderr, "\n");
}

# define YY_STACK_PRINT(Bottom, Top)				\
do {								\
  if (yydebug)							\
    yy_stack_print ((Bottom), (Top));				\
} while (YYID (0))


/*------------------------------------------------.
| Report that the YYRULE is going to be reduced.  |
`------------------------------------------------*/

#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yy_reduce_print (YYSTYPE *yyvsp, int yyrule)
#else
static void
yy_reduce_print (yyvsp, yyrule)
    YYSTYPE *yyvsp;
    int yyrule;
#endif
{
  int yynrhs = yyr2[yyrule];
  int yyi;
  unsigned long int yylno = yyrline[yyrule];
  YYFPRINTF (stderr, "Reducing stack by rule %d (line %lu):\n",
	     yyrule - 1, yylno);
  /* The symbols being reduced.  */
  for (yyi = 0; yyi < yynrhs; yyi++)
    {
      YYFPRINTF (stderr, "   $%d = ", yyi + 1);
      yy_symbol_print (stderr, yyrhs[yyprhs[yyrule] + yyi],
		       &(yyvsp[(yyi + 1) - (yynrhs)])
		       		       );
      YYFPRINTF (stderr, "\n");
    }
}

# define YY_REDUCE_PRINT(Rule)		\
do {					\
  if (yydebug)				\
    yy_reduce_print (yyvsp, Rule); \
} while (YYID (0))

/* Nonzero means print parse trace.  It is left uninitialized so that
   multiple parsers can coexist.  */
int yydebug;
#else /* !YYDEBUG */
# define YYDPRINTF(Args)
# define YY_SYMBOL_PRINT(Title, Type, Value, Location)
# define YY_STACK_PRINT(Bottom, Top)
# define YY_REDUCE_PRINT(Rule)
#endif /* !YYDEBUG */


/* YYINITDEPTH -- initial size of the parser's stacks.  */
#ifndef	YYINITDEPTH
# define YYINITDEPTH 200
#endif

/* YYMAXDEPTH -- maximum size the stacks can grow to (effective only
   if the built-in stack extension method is used).

   Do not make this value too large; the results are undefined if
   YYSTACK_ALLOC_MAXIMUM < YYSTACK_BYTES (YYMAXDEPTH)
   evaluated with infinite-precision integer arithmetic.  */

#ifndef YYMAXDEPTH
# define YYMAXDEPTH 10000
#endif



#if YYERROR_VERBOSE

# ifndef yystrlen
#  if defined __GLIBC__ && defined _STRING_H
#   define yystrlen strlen
#  else
/* Return the length of YYSTR.  */
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static YYSIZE_T
yystrlen (const char *yystr)
#else
static YYSIZE_T
yystrlen (yystr)
    const char *yystr;
#endif
{
  YYSIZE_T yylen;
  for (yylen = 0; yystr[yylen]; yylen++)
    continue;
  return yylen;
}
#  endif
# endif

# ifndef yystpcpy
#  if defined __GLIBC__ && defined _STRING_H && defined _GNU_SOURCE
#   define yystpcpy stpcpy
#  else
/* Copy YYSRC to YYDEST, returning the address of the terminating '\0' in
   YYDEST.  */
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static char *
yystpcpy (char *yydest, const char *yysrc)
#else
static char *
yystpcpy (yydest, yysrc)
    char *yydest;
    const char *yysrc;
#endif
{
  char *yyd = yydest;
  const char *yys = yysrc;

  while ((*yyd++ = *yys++) != '\0')
    continue;

  return yyd - 1;
}
#  endif
# endif

# ifndef yytnamerr
/* Copy to YYRES the contents of YYSTR after stripping away unnecessary
   quotes and backslashes, so that it's suitable for yyerror.  The
   heuristic is that double-quoting is unnecessary unless the string
   contains an apostrophe, a comma, or backslash (other than
   backslash-backslash).  YYSTR is taken from yytname.  If YYRES is
   null, do not copy; instead, return the length of what the result
   would have been.  */
static YYSIZE_T
yytnamerr (char *yyres, const char *yystr)
{
  if (*yystr == '"')
    {
      YYSIZE_T yyn = 0;
      char const *yyp = yystr;

      for (;;)
	switch (*++yyp)
	  {
	  case '\'':
	  case ',':
	    goto do_not_strip_quotes;

	  case '\\':
	    if (*++yyp != '\\')
	      goto do_not_strip_quotes;
	    /* Fall through.  */
	  default:
	    if (yyres)
	      yyres[yyn] = *yyp;
	    yyn++;
	    break;

	  case '"':
	    if (yyres)
	      yyres[yyn] = '\0';
	    return yyn;
	  }
    do_not_strip_quotes: ;
    }

  if (! yyres)
    return yystrlen (yystr);

  return yystpcpy (yyres, yystr) - yyres;
}
# endif

/* Copy into YYRESULT an error message about the unexpected token
   YYCHAR while in state YYSTATE.  Return the number of bytes copied,
   including the terminating null byte.  If YYRESULT is null, do not
   copy anything; just return the number of bytes that would be
   copied.  As a special case, return 0 if an ordinary "syntax error"
   message will do.  Return YYSIZE_MAXIMUM if overflow occurs during
   size calculation.  */
static YYSIZE_T
yysyntax_error (char *yyresult, int yystate, int yychar)
{
  int yyn = yypact[yystate];

  if (! (YYPACT_NINF < yyn && yyn <= YYLAST))
    return 0;
  else
    {
      int yytype = YYTRANSLATE (yychar);
      YYSIZE_T yysize0 = yytnamerr (0, yytname[yytype]);
      YYSIZE_T yysize = yysize0;
      YYSIZE_T yysize1;
      int yysize_overflow = 0;
      enum { YYERROR_VERBOSE_ARGS_MAXIMUM = 5 };
      char const *yyarg[YYERROR_VERBOSE_ARGS_MAXIMUM];
      int yyx;

# if 0
      /* This is so xgettext sees the translatable formats that are
	 constructed on the fly.  */
      YY_("syntax error, unexpected %s");
      YY_("syntax error, unexpected %s, expecting %s");
      YY_("syntax error, unexpected %s, expecting %s or %s");
      YY_("syntax error, unexpected %s, expecting %s or %s or %s");
      YY_("syntax error, unexpected %s, expecting %s or %s or %s or %s");
# endif
      char *yyfmt;
      char const *yyf;
      static char const yyunexpected[] = "syntax error, unexpected %s";
      static char const yyexpecting[] = ", expecting %s";
      static char const yyor[] = " or %s";
      char yyformat[sizeof yyunexpected
		    + sizeof yyexpecting - 1
		    + ((YYERROR_VERBOSE_ARGS_MAXIMUM - 2)
		       * (sizeof yyor - 1))];
      char const *yyprefix = yyexpecting;

      /* Start YYX at -YYN if negative to avoid negative indexes in
	 YYCHECK.  */
      int yyxbegin = yyn < 0 ? -yyn : 0;

      /* Stay within bounds of both yycheck and yytname.  */
      int yychecklim = YYLAST - yyn + 1;
      int yyxend = yychecklim < YYNTOKENS ? yychecklim : YYNTOKENS;
      int yycount = 1;

      yyarg[0] = yytname[yytype];
      yyfmt = yystpcpy (yyformat, yyunexpected);

      for (yyx = yyxbegin; yyx < yyxend; ++yyx)
	if (yycheck[yyx + yyn] == yyx && yyx != YYTERROR)
	  {
	    if (yycount == YYERROR_VERBOSE_ARGS_MAXIMUM)
	      {
		yycount = 1;
		yysize = yysize0;
		yyformat[sizeof yyunexpected - 1] = '\0';
		break;
	      }
	    yyarg[yycount++] = yytname[yyx];
	    yysize1 = yysize + yytnamerr (0, yytname[yyx]);
	    yysize_overflow |= (yysize1 < yysize);
	    yysize = yysize1;
	    yyfmt = yystpcpy (yyfmt, yyprefix);
	    yyprefix = yyor;
	  }

      yyf = YY_(yyformat);
      yysize1 = yysize + yystrlen (yyf);
      yysize_overflow |= (yysize1 < yysize);
      yysize = yysize1;

      if (yysize_overflow)
	return YYSIZE_MAXIMUM;

      if (yyresult)
	{
	  /* Avoid sprintf, as that infringes on the user's name space.
	     Don't have undefined behavior even if the translation
	     produced a string with the wrong number of "%s"s.  */
	  char *yyp = yyresult;
	  int yyi = 0;
	  while ((*yyp = *yyf) != '\0')
	    {
	      if (*yyp == '%' && yyf[1] == 's' && yyi < yycount)
		{
		  yyp += yytnamerr (yyp, yyarg[yyi++]);
		  yyf += 2;
		}
	      else
		{
		  yyp++;
		  yyf++;
		}
	    }
	}
      return yysize;
    }
}
#endif /* YYERROR_VERBOSE */


/*-----------------------------------------------.
| Release the memory associated to this symbol.  |
`-----------------------------------------------*/

/*ARGSUSED*/
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yydestruct (const char *yymsg, int yytype, YYSTYPE *yyvaluep)
#else
static void
yydestruct (yymsg, yytype, yyvaluep)
    const char *yymsg;
    int yytype;
    YYSTYPE *yyvaluep;
#endif
{
  YYUSE (yyvaluep);

  if (!yymsg)
    yymsg = "Deleting";
  YY_SYMBOL_PRINT (yymsg, yytype, yyvaluep, yylocationp);

  switch (yytype)
    {

      default:
	break;
    }
}

/* Prevent warnings from -Wmissing-prototypes.  */
#ifdef YYPARSE_PARAM
#if defined __STDC__ || defined __cplusplus
int yyparse (void *YYPARSE_PARAM);
#else
int yyparse ();
#endif
#else /* ! YYPARSE_PARAM */
#if defined __STDC__ || defined __cplusplus
int yyparse (void);
#else
int yyparse ();
#endif
#endif /* ! YYPARSE_PARAM */


/* The lookahead symbol.  */
int yychar;

/* The semantic value of the lookahead symbol.  */
YYSTYPE yylval;

/* Number of syntax errors so far.  */
int yynerrs;



/*-------------------------.
| yyparse or yypush_parse.  |
`-------------------------*/

#ifdef YYPARSE_PARAM
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
int
yyparse (void *YYPARSE_PARAM)
#else
int
yyparse (YYPARSE_PARAM)
    void *YYPARSE_PARAM;
#endif
#else /* ! YYPARSE_PARAM */
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
int
yyparse (void)
#else
int
yyparse ()

#endif
#endif
{


    int yystate;
    /* Number of tokens to shift before error messages enabled.  */
    int yyerrstatus;

    /* The stacks and their tools:
       `yyss': related to states.
       `yyvs': related to semantic values.

       Refer to the stacks thru separate pointers, to allow yyoverflow
       to reallocate them elsewhere.  */

    /* The state stack.  */
    yytype_int16 yyssa[YYINITDEPTH];
    yytype_int16 *yyss;
    yytype_int16 *yyssp;

    /* The semantic value stack.  */
    YYSTYPE yyvsa[YYINITDEPTH];
    YYSTYPE *yyvs;
    YYSTYPE *yyvsp;

    YYSIZE_T yystacksize;

  int yyn;
  int yyresult;
  /* Lookahead token as an internal (translated) token number.  */
  int yytoken;
  /* The variables used to return semantic value and location from the
     action routines.  */
  YYSTYPE yyval;

#if YYERROR_VERBOSE
  /* Buffer for error messages, and its allocated size.  */
  char yymsgbuf[128];
  char *yymsg = yymsgbuf;
  YYSIZE_T yymsg_alloc = sizeof yymsgbuf;
#endif

#define YYPOPSTACK(N)   (yyvsp -= (N), yyssp -= (N))

  /* The number of symbols on the RHS of the reduced rule.
     Keep to zero when no symbol should be popped.  */
  int yylen = 0;

  yytoken = 0;
  yyss = yyssa;
  yyvs = yyvsa;
  yystacksize = YYINITDEPTH;

  YYDPRINTF ((stderr, "Starting parse\n"));

  yystate = 0;
  yyerrstatus = 0;
  yynerrs = 0;
  yychar = YYEMPTY; /* Cause a token to be read.  */

  /* Initialize stack pointers.
     Waste one element of value and location stack
     so that they stay on the same level as the state stack.
     The wasted elements are never initialized.  */
  yyssp = yyss;
  yyvsp = yyvs;

  goto yysetstate;

/*------------------------------------------------------------.
| yynewstate -- Push a new state, which is found in yystate.  |
`------------------------------------------------------------*/
 yynewstate:
  /* In all cases, when you get here, the value and location stacks
     have just been pushed.  So pushing a state here evens the stacks.  */
  yyssp++;

 yysetstate:
  *yyssp = yystate;

  if (yyss + yystacksize - 1 <= yyssp)
    {
      /* Get the current used size of the three stacks, in elements.  */
      YYSIZE_T yysize = yyssp - yyss + 1;

#ifdef yyoverflow
      {
	/* Give user a chance to reallocate the stack.  Use copies of
	   these so that the &'s don't force the real ones into
	   memory.  */
	YYSTYPE *yyvs1 = yyvs;
	yytype_int16 *yyss1 = yyss;

	/* Each stack pointer address is followed by the size of the
	   data in use in that stack, in bytes.  This used to be a
	   conditional around just the two extra args, but that might
	   be undefined if yyoverflow is a macro.  */
	yyoverflow (YY_("memory exhausted"),
		    &yyss1, yysize * sizeof (*yyssp),
		    &yyvs1, yysize * sizeof (*yyvsp),
		    &yystacksize);

	yyss = yyss1;
	yyvs = yyvs1;
      }
#else /* no yyoverflow */
# ifndef YYSTACK_RELOCATE
      goto yyexhaustedlab;
# else
      /* Extend the stack our own way.  */
      if (YYMAXDEPTH <= yystacksize)
	goto yyexhaustedlab;
      yystacksize *= 2;
      if (YYMAXDEPTH < yystacksize)
	yystacksize = YYMAXDEPTH;

      {
	yytype_int16 *yyss1 = yyss;
	union yyalloc *yyptr =
	  (union yyalloc *) YYSTACK_ALLOC (YYSTACK_BYTES (yystacksize));
	if (! yyptr)
	  goto yyexhaustedlab;
	YYSTACK_RELOCATE (yyss_alloc, yyss);
	YYSTACK_RELOCATE (yyvs_alloc, yyvs);
#  undef YYSTACK_RELOCATE
	if (yyss1 != yyssa)
	  YYSTACK_FREE (yyss1);
      }
# endif
#endif /* no yyoverflow */

      yyssp = yyss + yysize - 1;
      yyvsp = yyvs + yysize - 1;

      YYDPRINTF ((stderr, "Stack size increased to %lu\n",
		  (unsigned long int) yystacksize));

      if (yyss + yystacksize - 1 <= yyssp)
	YYABORT;
    }

  YYDPRINTF ((stderr, "Entering state %d\n", yystate));

  if (yystate == YYFINAL)
    YYACCEPT;

  goto yybackup;

/*-----------.
| yybackup.  |
`-----------*/
yybackup:

  /* Do appropriate processing given the current state.  Read a
     lookahead token if we need one and don't already have one.  */

  /* First try to decide what to do without reference to lookahead token.  */
  yyn = yypact[yystate];
  if (yyn == YYPACT_NINF)
    goto yydefault;

  /* Not known => get a lookahead token if don't already have one.  */

  /* YYCHAR is either YYEMPTY or YYEOF or a valid lookahead symbol.  */
  if (yychar == YYEMPTY)
    {
      YYDPRINTF ((stderr, "Reading a token: "));
      yychar = YYLEX;
    }

  if (yychar <= YYEOF)
    {
      yychar = yytoken = YYEOF;
      YYDPRINTF ((stderr, "Now at end of input.\n"));
    }
  else
    {
      yytoken = YYTRANSLATE (yychar);
      YY_SYMBOL_PRINT ("Next token is", yytoken, &yylval, &yylloc);
    }

  /* If the proper action on seeing token YYTOKEN is to reduce or to
     detect an error, take that action.  */
  yyn += yytoken;
  if (yyn < 0 || YYLAST < yyn || yycheck[yyn] != yytoken)
    goto yydefault;
  yyn = yytable[yyn];
  if (yyn <= 0)
    {
      if (yyn == 0 || yyn == YYTABLE_NINF)
	goto yyerrlab;
      yyn = -yyn;
      goto yyreduce;
    }

  /* Count tokens shifted since error; after three, turn off error
     status.  */
  if (yyerrstatus)
    yyerrstatus--;

  /* Shift the lookahead token.  */
  YY_SYMBOL_PRINT ("Shifting", yytoken, &yylval, &yylloc);

  /* Discard the shifted token.  */
  yychar = YYEMPTY;

  yystate = yyn;
  *++yyvsp = yylval;

  goto yynewstate;


/*-----------------------------------------------------------.
| yydefault -- do the default action for the current state.  |
`-----------------------------------------------------------*/
yydefault:
  yyn = yydefact[yystate];
  if (yyn == 0)
    goto yyerrlab;
  goto yyreduce;


/*-----------------------------.
| yyreduce -- Do a reduction.  |
`-----------------------------*/
yyreduce:
  /* yyn is the number of a rule to reduce with.  */
  yylen = yyr2[yyn];

  /* If YYLEN is nonzero, implement the default value of the action:
     `$$ = $1'.

     Otherwise, the following line sets YYVAL to garbage.
     This behavior is undocumented and Bison
     users should not rely upon it.  Assigning to YYVAL
     unconditionally makes the parser a bit smaller, and it avoids a
     GCC warning that YYVAL may be used uninitialized.  */
  yyval = yyvsp[1-yylen];


  YY_REDUCE_PRINT (yyn);
  switch (yyn)
    {
        case 4:

/* Line 1455 of yacc.c  */
#line 142 "bison.y"
    { emit("STMT"); ;}
    break;

  case 5:

/* Line 1455 of yacc.c  */
#line 146 "bison.y"
    { emit_select((yyvsp[(1) - (7)].strval), (yyvsp[(6) - (7)].strval), (yyvsp[(7) - (7)].intval)); ;}
    break;

  case 6:

/* Line 1455 of yacc.c  */
#line 148 "bison.y"
    {  emit_load((yyvsp[(1) - (12)].strval), (yyvsp[(4) - (12)].strval), (yyvsp[(11) - (12)].intval), (yyvsp[(7) - (12)].strval)); ;}
    break;

  case 7:

/* Line 1455 of yacc.c  */
#line 150 "bison.y"
    {  emit_load_binary((yyvsp[(1) - (9)].strval), (yyvsp[(4) - (9)].strval), (yyvsp[(8) - (9)].intval)); ;}
    break;

  case 8:

/* Line 1455 of yacc.c  */
#line 152 "bison.y"
    {  emit_filter((yyvsp[(1) - (5)].strval), (yyvsp[(4) - (5)].strval), (yyvsp[(5) - (5)].intval));;}
    break;

  case 9:

/* Line 1455 of yacc.c  */
#line 154 "bison.y"
    {  emit_order((yyvsp[(1) - (6)].strval), (yyvsp[(4) - (6)].strval), (yyvsp[(6) - (6)].intval));;}
    break;

  case 10:

/* Line 1455 of yacc.c  */
#line 156 "bison.y"
    { emit_join((yyvsp[(1) - (7)].strval),(yyvsp[(6) - (7)].strval)); ;}
    break;

  case 11:

/* Line 1455 of yacc.c  */
#line 158 "bison.y"
    { emit_store((yyvsp[(2) - (9)].strval),(yyvsp[(4) - (9)].strval),(yyvsp[(7) - (9)].strval)); ;}
    break;

  case 12:

/* Line 1455 of yacc.c  */
#line 160 "bison.y"
    { emit_store_binary((yyvsp[(2) - (6)].strval),(yyvsp[(4) - (6)].strval)); ;}
    break;

  case 13:

/* Line 1455 of yacc.c  */
#line 164 "bison.y"
    { emit_name((yyvsp[(1) - (1)].strval)); ;}
    break;

  case 14:

/* Line 1455 of yacc.c  */
#line 165 "bison.y"
    { emit("FIELDNAME %s.%s", (yyvsp[(1) - (3)].strval), (yyvsp[(3) - (3)].strval)); ;}
    break;

  case 15:

/* Line 1455 of yacc.c  */
#line 166 "bison.y"
    { emit("USERVAR %s", (yyvsp[(1) - (1)].strval)); ;}
    break;

  case 16:

/* Line 1455 of yacc.c  */
#line 167 "bison.y"
    { emit_string((yyvsp[(1) - (1)].strval)); ;}
    break;

  case 17:

/* Line 1455 of yacc.c  */
#line 168 "bison.y"
    { emit_number((yyvsp[(1) - (1)].intval)); ;}
    break;

  case 18:

/* Line 1455 of yacc.c  */
#line 169 "bison.y"
    { emit_float((yyvsp[(1) - (1)].floatval)); ;}
    break;

  case 19:

/* Line 1455 of yacc.c  */
#line 170 "bison.y"
    { emit_decimal((yyvsp[(1) - (1)].intval)); ;}
    break;

  case 20:

/* Line 1455 of yacc.c  */
#line 171 "bison.y"
    { emit("BOOL %d", (yyvsp[(1) - (1)].intval)); ;}
    break;

  case 21:

/* Line 1455 of yacc.c  */
#line 172 "bison.y"
    { emit_varchar((yyvsp[(1) - (9)].strval), (yyvsp[(3) - (9)].intval), (yyvsp[(6) - (9)].strval), (yyvsp[(8) - (9)].intval));;}
    break;

  case 22:

/* Line 1455 of yacc.c  */
#line 173 "bison.y"
    { emit_var((yyvsp[(1) - (6)].strval), (yyvsp[(3) - (6)].intval), (yyvsp[(6) - (6)].strval));;}
    break;

  case 23:

/* Line 1455 of yacc.c  */
#line 174 "bison.y"
    { emit_var_asc((yyvsp[(1) - (2)].strval));;}
    break;

  case 24:

/* Line 1455 of yacc.c  */
#line 175 "bison.y"
    { emit_var_desc((yyvsp[(1) - (2)].strval));;}
    break;

  case 25:

/* Line 1455 of yacc.c  */
#line 176 "bison.y"
    { emit_count(); ;}
    break;

  case 26:

/* Line 1455 of yacc.c  */
#line 177 "bison.y"
    { emit_sum(); ;}
    break;

  case 27:

/* Line 1455 of yacc.c  */
#line 178 "bison.y"
    { emit_average(); ;}
    break;

  case 28:

/* Line 1455 of yacc.c  */
#line 179 "bison.y"
    { emit_min(); ;}
    break;

  case 29:

/* Line 1455 of yacc.c  */
#line 180 "bison.y"
    { emit_max(); ;}
    break;

  case 30:

/* Line 1455 of yacc.c  */
#line 184 "bison.y"
    { emit_add(); ;}
    break;

  case 31:

/* Line 1455 of yacc.c  */
#line 185 "bison.y"
    { emit_minus(); ;}
    break;

  case 32:

/* Line 1455 of yacc.c  */
#line 186 "bison.y"
    { emit_mul(); ;}
    break;

  case 33:

/* Line 1455 of yacc.c  */
#line 187 "bison.y"
    { emit_div(); ;}
    break;

  case 34:

/* Line 1455 of yacc.c  */
#line 188 "bison.y"
    { emit("MOD"); ;}
    break;

  case 35:

/* Line 1455 of yacc.c  */
#line 189 "bison.y"
    { emit("MOD"); ;}
    break;

  case 36:

/* Line 1455 of yacc.c  */
#line 191 "bison.y"
    { emit_and(); ;}
    break;

  case 37:

/* Line 1455 of yacc.c  */
#line 192 "bison.y"
    { emit_eq(); ;}
    break;

  case 38:

/* Line 1455 of yacc.c  */
#line 193 "bison.y"
    { emit_or(); ;}
    break;

  case 39:

/* Line 1455 of yacc.c  */
#line 194 "bison.y"
    { emit("XOR"); ;}
    break;

  case 40:

/* Line 1455 of yacc.c  */
#line 195 "bison.y"
    { emit("SHIFT %s", (yyvsp[(2) - (3)].subtok)==1?"left":"right"); ;}
    break;

  case 41:

/* Line 1455 of yacc.c  */
#line 196 "bison.y"
    { emit("NOT"); ;}
    break;

  case 42:

/* Line 1455 of yacc.c  */
#line 197 "bison.y"
    { emit("NOT"); ;}
    break;

  case 43:

/* Line 1455 of yacc.c  */
#line 198 "bison.y"
    { emit_cmp((yyvsp[(2) - (3)].subtok)); ;}
    break;

  case 44:

/* Line 1455 of yacc.c  */
#line 200 "bison.y"
    { emit("CMPSELECT %d", (yyvsp[(2) - (5)].subtok)); ;}
    break;

  case 45:

/* Line 1455 of yacc.c  */
#line 201 "bison.y"
    {emit("EXPR");;}
    break;

  case 46:

/* Line 1455 of yacc.c  */
#line 205 "bison.y"
    { emit("ISBOOL %d", (yyvsp[(3) - (3)].intval)); ;}
    break;

  case 47:

/* Line 1455 of yacc.c  */
#line 206 "bison.y"
    { emit("ISBOOL %d", (yyvsp[(4) - (4)].intval)); emit("NOT"); ;}
    break;

  case 48:

/* Line 1455 of yacc.c  */
#line 209 "bison.y"
    { /* nil */
    (yyval.intval) = 0;
;}
    break;

  case 49:

/* Line 1455 of yacc.c  */
#line 212 "bison.y"
    { (yyval.intval) = (yyvsp[(3) - (3)].intval);}
    break;

  case 50:

/* Line 1455 of yacc.c  */
#line 216 "bison.y"
    { (yyval.intval) = 1; emit_sel_name((yyvsp[(3) - (3)].strval));;}
    break;

  case 51:

/* Line 1455 of yacc.c  */
#line 217 "bison.y"
    { (yyval.intval) = (yyvsp[(1) - (5)].intval) + 1; emit_sel_name((yyvsp[(5) - (5)].strval));;}
    break;

  case 52:

/* Line 1455 of yacc.c  */
#line 221 "bison.y"
    { (yyval.intval) = 1; ;}
    break;

  case 53:

/* Line 1455 of yacc.c  */
#line 222 "bison.y"
    {(yyval.intval) = (yyvsp[(1) - (3)].intval) + 1; ;}
    break;

  case 54:

/* Line 1455 of yacc.c  */
#line 226 "bison.y"
    { (yyval.intval) = 1; ;}
    break;

  case 55:

/* Line 1455 of yacc.c  */
#line 227 "bison.y"
    { (yyval.intval) = 1 + (yyvsp[(3) - (3)].intval); ;}
    break;

  case 56:

/* Line 1455 of yacc.c  */
#line 230 "bison.y"
    { /* nil */
    (yyval.intval) = 0
;}
    break;

  case 58:

/* Line 1455 of yacc.c  */
#line 235 "bison.y"
    { emit("FILTER BY"); ;}
    break;

  case 59:

/* Line 1455 of yacc.c  */
#line 238 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(2) - (4)].strval));;}
    break;

  case 60:

/* Line 1455 of yacc.c  */
#line 239 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(2) - (5)].strval)); ;}
    break;

  case 61:

/* Line 1455 of yacc.c  */
#line 241 "bison.y"
    { /* nil */
    (yyval.intval) = 0
;}
    break;

  case 62:

/* Line 1455 of yacc.c  */
#line 244 "bison.y"
    { emit_limit((yyvsp[(2) - (2)].intval)); ;}
    break;



/* Line 1455 of yacc.c  */
#line 2023 "bison.cu"
      default: break;
    }
  YY_SYMBOL_PRINT ("-> $$ =", yyr1[yyn], &yyval, &yyloc);

  YYPOPSTACK (yylen);
  yylen = 0;
  YY_STACK_PRINT (yyss, yyssp);

  *++yyvsp = yyval;

  /* Now `shift' the result of the reduction.  Determine what state
     that goes to, based on the state we popped back to and the rule
     number reduced by.  */

  yyn = yyr1[yyn];

  yystate = yypgoto[yyn - YYNTOKENS] + *yyssp;
  if (0 <= yystate && yystate <= YYLAST && yycheck[yystate] == *yyssp)
    yystate = yytable[yystate];
  else
    yystate = yydefgoto[yyn - YYNTOKENS];

  goto yynewstate;


/*------------------------------------.
| yyerrlab -- here on detecting error |
`------------------------------------*/
yyerrlab:
  /* If not already recovering from an error, report this error.  */
  if (!yyerrstatus)
    {
      ++yynerrs;
#if ! YYERROR_VERBOSE
      yyerror (YY_("syntax error"));
#else
      {
	YYSIZE_T yysize = yysyntax_error (0, yystate, yychar);
	if (yymsg_alloc < yysize && yymsg_alloc < YYSTACK_ALLOC_MAXIMUM)
	  {
	    YYSIZE_T yyalloc = 2 * yysize;
	    if (! (yysize <= yyalloc && yyalloc <= YYSTACK_ALLOC_MAXIMUM))
	      yyalloc = YYSTACK_ALLOC_MAXIMUM;
	    if (yymsg != yymsgbuf)
	      YYSTACK_FREE (yymsg);
	    yymsg = (char *) YYSTACK_ALLOC (yyalloc);
	    if (yymsg)
	      yymsg_alloc = yyalloc;
	    else
	      {
		yymsg = yymsgbuf;
		yymsg_alloc = sizeof yymsgbuf;
	      }
	  }

	if (0 < yysize && yysize <= yymsg_alloc)
	  {
	    (void) yysyntax_error (yymsg, yystate, yychar);
	    yyerror (yymsg);
	  }
	else
	  {
	    yyerror (YY_("syntax error"));
	    if (yysize != 0)
	      goto yyexhaustedlab;
	  }
      }
#endif
    }



  if (yyerrstatus == 3)
    {
      /* If just tried and failed to reuse lookahead token after an
	 error, discard it.  */

      if (yychar <= YYEOF)
	{
	  /* Return failure if at end of input.  */
	  if (yychar == YYEOF)
	    YYABORT;
	}
      else
	{
	  yydestruct ("Error: discarding",
		      yytoken, &yylval);
	  yychar = YYEMPTY;
	}
    }

  /* Else will try to reuse lookahead token after shifting the error
     token.  */
  goto yyerrlab1;


/*---------------------------------------------------.
| yyerrorlab -- error raised explicitly by YYERROR.  |
`---------------------------------------------------*/
yyerrorlab:

  /* Pacify compilers like GCC when the user code never invokes
     YYERROR and the label yyerrorlab therefore never appears in user
     code.  */
  if (/*CONSTCOND*/ 0)
     goto yyerrorlab;

  /* Do not reclaim the symbols of the rule which action triggered
     this YYERROR.  */
  YYPOPSTACK (yylen);
  yylen = 0;
  YY_STACK_PRINT (yyss, yyssp);
  yystate = *yyssp;
  goto yyerrlab1;


/*-------------------------------------------------------------.
| yyerrlab1 -- common code for both syntax error and YYERROR.  |
`-------------------------------------------------------------*/
yyerrlab1:
  yyerrstatus = 3;	/* Each real token shifted decrements this.  */

  for (;;)
    {
      yyn = yypact[yystate];
      if (yyn != YYPACT_NINF)
	{
	  yyn += YYTERROR;
	  if (0 <= yyn && yyn <= YYLAST && yycheck[yyn] == YYTERROR)
	    {
	      yyn = yytable[yyn];
	      if (0 < yyn)
		break;
	    }
	}

      /* Pop the current state because it cannot handle the error token.  */
      if (yyssp == yyss)
	YYABORT;


      yydestruct ("Error: popping",
		  yystos[yystate], yyvsp);
      YYPOPSTACK (1);
      yystate = *yyssp;
      YY_STACK_PRINT (yyss, yyssp);
    }

  *++yyvsp = yylval;


  /* Shift the error token.  */
  YY_SYMBOL_PRINT ("Shifting", yystos[yyn], yyvsp, yylsp);

  yystate = yyn;
  goto yynewstate;


/*-------------------------------------.
| yyacceptlab -- YYACCEPT comes here.  |
`-------------------------------------*/
yyacceptlab:
  yyresult = 0;
  goto yyreturn;

/*-----------------------------------.
| yyabortlab -- YYABORT comes here.  |
`-----------------------------------*/
yyabortlab:
  yyresult = 1;
  goto yyreturn;

#if !defined(yyoverflow) || YYERROR_VERBOSE
/*-------------------------------------------------.
| yyexhaustedlab -- memory exhaustion comes here.  |
`-------------------------------------------------*/
yyexhaustedlab:
  yyerror (YY_("memory exhausted"));
  yyresult = 2;
  /* Fall through.  */
#endif

yyreturn:
  if (yychar != YYEMPTY)
     yydestruct ("Cleanup: discarding lookahead",
		 yytoken, &yylval);
  /* Do not reclaim the symbols of the rule which action triggered
     this YYABORT or YYACCEPT.  */
  YYPOPSTACK (yylen);
  YY_STACK_PRINT (yyss, yyssp);
  while (yyssp != yyss)
    {
      yydestruct ("Cleanup: popping",
		  yystos[*yyssp], yyvsp);
      YYPOPSTACK (1);
    }
#ifndef yyoverflow
  if (yyss != yyssa)
    YYSTACK_FREE (yyss);
#endif
#if YYERROR_VERBOSE
  if (yymsg != yymsgbuf)
    YYSTACK_FREE (yymsg);
#endif
  /* Make sure YYID is used.  */
  return YYID (yyresult);
}



/* Line 1675 of yacc.c  */
#line 247 "bison.y"



#include "filter.cu"
#include "select.cu"
#include "merge.cu"
#include "zone_map.cu"

FILE *file_pointer;
queue<string> namevars;
queue<string> typevars;
queue<int> sizevars;
queue<int> cols;

queue<unsigned int> j_col_count;
unsigned int sel_count = 0;
unsigned int join_cnt = 0;
int join_col_cnt = 0;
unsigned int eqq = 0;
stack<string> op_join;

unsigned int statement_count = 0;
map<string,unsigned int> stat;
bool scan_state = 0;
string separator, f_file;


CUDPPHandle theCudpp;

using namespace thrust::placeholders;


void emit_name(char *name)
{
    op_type.push("NAME");
    op_value.push(name);
}

void emit_limit(int val)
{
    op_nums.push(val);
}


void emit_string(char *str)
{   // remove the float_type quotes
    string sss(str,1, strlen(str)-2);
    op_type.push("STRING");
    op_value.push(sss);
}


void emit_number(int_type val)
{
    op_type.push("NUMBER");
    op_nums.push(val);
}

void emit_float(float_type val)
{
    op_type.push("FLOAT");
    op_nums_f.push(val);
}

void emit_decimal(float_type val)
{
    op_type.push("DECIMAL");
    op_nums_f.push(val);
}



void emit_mul()
{
    op_type.push("MUL");
}

void emit_add()
{
    op_type.push("ADD");
}

void emit_div()
{
    op_type.push("DIV");
}

void emit_and()
{
    op_type.push("AND");
    if (join_col_cnt == -1)
        join_col_cnt++;
    join_col_cnt++;
    eqq = 0;
}

void emit_eq()
{
    //op_type.push("JOIN");
    eqq++;
    join_cnt++;
    if(eqq == join_col_cnt+1) {
        j_col_count.push(join_col_cnt+1);
        join_col_cnt = -1;
    }
    else if (join_col_cnt == -1 )
        j_col_count.push(1);

}



void emit_or()
{
    op_type.push("OR");
}


void emit_minus()
{
    op_type.push("MINUS");
}

void emit_cmp(int val)
{
    op_type.push("CMP");
    op_nums.push(val);
}




void emit(char *s, ...)
{


}


void emit_var(char *s, int c, char *f)
{
    namevars.push(s);
    typevars.push(f);
    sizevars.push(0);
    cols.push(c);
}

void emit_var_asc(char *s)
{
    op_type.push(s);
    op_value.push("ASC");
}

void emit_var_desc(char *s)
{
    op_type.push(s);
    op_value.push("DESC");
}


void emit_varchar(char *s, int c, char *f, int d)
{
    namevars.push(s);
    typevars.push(f);
    sizevars.push(d);
    cols.push(c);
}

void emit_sel_name(char *s)
{
    op_type.push("emit sel_name");
    op_value.push(s);
    sel_count++;
}

void emit_count()
{
    op_type.push("COUNT");
}

void emit_sum()
{
    op_type.push("SUM");
}


void emit_average()
{
    op_type.push("AVG");
}

void emit_min()
{
    op_type.push("MIN");
}

void emit_max()
{
    op_type.push("MAX");
}

void emit_join_tab(char *s)
{
    op_join.push(s);
};




void order_inplace(CudaSet* a, stack<string> exe_type, set<string> field_names, unsigned int segment)
{
    unsigned int sz = a->mRecCount;
    thrust::device_ptr<unsigned int> permutation = thrust::device_malloc<unsigned int>(sz);
    thrust::sequence(permutation, permutation+sz,0,1);


    unsigned int* raw_ptr = thrust::raw_pointer_cast(permutation);
    void* temp;
    // find the largest mRecSize of all data sources exe_type.top()
    unsigned int maxSize = 0;
    for (set<string>::iterator it=field_names.begin(); it!=field_names.end(); ++it) {
        CudaSet *t = varNames[setMap[*it]];
        //cout << "MAX of " << setMap[*it] << " = " << t->mRecCount << endl;
        if(t->mRecCount > maxSize)
            maxSize = t->mRecCount;
    };

    //cout << "max size " << maxSize << endl;
    //cout << "sort alloc " << maxSize << endl;
    //cout << "order mem " << getFreeMem() << endl;
    CUDA_SAFE_CALL(hipMalloc((void **) &temp, maxSize*float_size));

    for(int i=0; !exe_type.empty(); ++i, exe_type.pop()) {
        int colInd = (a->columnNames).find(exe_type.top())->second;
        if ((a->type)[colInd] == 0)
            update_permutation(a->d_columns_int[a->type_index[colInd]], raw_ptr, sz, "ASC", (int_type*)temp);
        else if ((a->type)[colInd] == 1)
            update_permutation(a->d_columns_float[a->type_index[colInd]], raw_ptr, sz,"ASC", (float_type*)temp);
        else {
            CudaChar* c = a->h_columns_cuda_char[a->type_index[colInd]];
            for(int j=(c->mColumnCount)-1; j>=0 ; j--)
                update_permutation((c->d_columns)[j], raw_ptr, sz, "ASC", (char*)temp);
        };
    };
	

    for (set<string>::iterator it=field_names.begin(); it!=field_names.end(); ++it) {
        int i = a->columnNames[*it];
        if ((a->type)[i] == 0)
            apply_permutation(a->d_columns_int[a->type_index[i]], raw_ptr, sz, (int_type*)temp);
        else if ((a->type)[i] == 1)
            apply_permutation(a->d_columns_float[a->type_index[i]], raw_ptr, sz, (float_type*)temp);
        else {
            CudaChar* c = a->h_columns_cuda_char[a->type_index[i]];
            for(int j=(c->mColumnCount)-1; j>=0 ; j--)
                apply_permutation((c->d_columns)[j], raw_ptr, sz, (char*)temp);
        };
    };

    hipFree(temp);
    thrust::device_free(permutation);
	
}




void emit_join(char *s, char *j1)
{

    string j2 = op_join.top();
    op_join.pop();

    statement_count++;
    if (scan_state == 0) {
        if (stat.find(j1) == stat.end()) {
            cout << "Join : couldn't find variable " << j1 << endl;
            exit(1);
        };
        if (stat.find(j2) == stat.end()) {
            cout << "Join : couldn't find variable " << j2 << endl;
            exit(1);
        };
        stat[s] = statement_count;
        stat[j1] = statement_count;
        stat[j2] = statement_count;
        return;
    };
 

    if(varNames.find(j1) == varNames.end() || varNames.find(j2) == varNames.end()) {
        clean_queues();
        return;
    };

    CudaSet* left = varNames.find(j1)->second;
    CudaSet* right = varNames.find(j2)->second;
	
    queue<string> op_sel;
    queue<string> op_sel_as;
    for(int i=0; i < sel_count; i++) {
        op_sel.push(op_value.front());
        op_value.pop();
        op_sel_as.push(op_value.front());
        op_value.pop();
    };

    string f1 = op_value.front();
    op_value.pop();
    string f2 = op_value.front();
    op_value.pop();

    cout << "JOIN " << s <<  " " <<  getFreeMem() <<  endl;

    std::clock_t start1 = std::clock();
    CudaSet* c = new CudaSet(right,left,0,op_sel, op_sel_as);	

    if (left->mRecCount == 0 || right->mRecCount == 0) {
        c = new CudaSet(left,right,0, op_sel, op_sel_as);        
        varNames[s] = c;
        clean_queues();
        return;
    };

    unsigned int colInd1 = (left->columnNames).find(f1)->second;
    unsigned int colInd2 = (right->columnNames).find(f2)->second;
	
	if ((left->type)[colInd1] != 0 || (right->type)[colInd2]  != 0) {
	    cout << "Right now only integer joins are supported " << endl;
		exit(0);
	};	

    set<string> field_names;
    stack<string> exe_type;
    exe_type.push(f2);
    field_names.insert(f2);

    // need to allocate all right columns	
    queue<string> cc;
	queue<string> c1(op_sel);;
	
	while(!c1.empty()) {	
        if(right->columnNames.find(c1.front()) !=  right->columnNames.end()) {
		    if(f2 != c1.front())
                cc.push(c1.front());
		};	
		c1.pop();		
	};	
    cc.push(f2);	

	if(right->prm.size())
        allocColumns(right, cc);	
	
    unsigned int rcount;
    if(!right->prm.empty()) {
 	    rcount = std::accumulate(right->prm_count.begin(), right->prm_count.end(), 0 );
    }
    else
        rcount = right->mRecCount;
	//cout << "rcount = " << rcount << endl;	
	
	queue<string> ct(cc);
	while(!ct.empty()) {	
	    right->allocColumnOnDevice(right->columnNames[ct.front()], rcount);
		ct.pop();		
	};	

	
    //thrust::device_ptr<unsigned int> rr = thrust::device_malloc<unsigned int>(rcount);		
	//right->allocColumnOnDevice(colInd2, rcount);	
	unsigned int cnt_r = 0;	
	   
	if(right->prm.size() == 0) {
       //copy all records	    
	   for(unsigned int i = 0; i < right->mColumnCount; i++)
	       right->CopyColumnToGpu(i);
		   cnt_r = right->mRecCount;
    }	
	else { 
	    //copy and gather all records					
         for(unsigned int i = 0; i < right->segCount; i++) {		 
             //gatherJoin(right, f2, i, cnt_r);	
             copyColumns(right, cc, i, cnt_r);			  			 	 
			 cnt_r = cnt_r + right->prm_count[i];
		 };				
	};
	
	
	unsigned int tt;
    if(left->maxRecs > rcount)	
	    tt = left->maxRecs;
	else
        tt = rcount;
		
	thrust::device_ptr<int_type> d_tmp = thrust::device_malloc<int_type>(tt);			
	
	//here we need to make sure that rr is ordered. If not then we order it and keep the permutation	
	bool sorted = thrust::is_sorted(right->d_columns_int[right->type_index[colInd2]].begin(), right->d_columns_int[right->type_index[colInd2]].begin() + cnt_r);
	
    thrust::device_vector<unsigned int> v(cnt_r);
	thrust::sequence(v.begin(),v.end(),0,1);
    	
	if(!sorted) {
	    thrust::sort_by_key(right->d_columns_int[right->type_index[colInd2]].begin(), right->d_columns_int[right->type_index[colInd2]].begin() + cnt_r, v.begin());
		for(unsigned int i = 0; i < right->mColumnCount; i++) {
		    if(i != colInd2) {
			    if(right->type[i] == 0) {
			        thrust::gather(v.begin(), v.end(), right->d_columns_int[right->type_index[i]].begin(), d_tmp);
				    thrust::copy(d_tmp, d_tmp + cnt_r, right->d_columns_int[right->type_index[i]].begin());					
				}
			    else if(right->type[i] == 1) {			
			        thrust::gather(v.begin(), v.end(), right->d_columns_float[right->type_index[i]].begin(), d_tmp);
				    thrust::copy(d_tmp, d_tmp + cnt_r, right->d_columns_float[right->type_index[i]].begin());
				}                				
			};	
		};
		thrust::sequence(v.begin(),v.end(),0,1);
	};
	thrust::device_free(d_tmp);		
	
	while(!cc.empty())
        cc.pop();
	
    cc.push(f1);
    allocColumns(left, cc);	

    //cout << "successfully loaded l && r " << cnt_l << " " << cnt_r << " " << getFreeMem() << endl;
	
    thrust::device_vector<unsigned int> d_res1;
    thrust::device_vector<unsigned int> d_res2;
    
	thrust::device_ptr<uint2> res = thrust::device_malloc<uint2>(left->maxRecs);
	
	unsigned int cnt_l, res_count, tot_count = 0, offset = 0, k = 0;
    void* g;
    CUDA_SAFE_CALL(hipMalloc((void **) &g, left->maxRecs*int_size));

	//thrust::device_ptr<int_type> g = thrust::device_malloc<int_type>(left->maxRecs);
	queue<string> lc(cc);
	curr_segment = 10000000;	
	CUDPPResult result;
	
	// now for 64bit values we need to create several HashTables where each of them will keep a certain range of values
	// lets find out how many tables we need
	int_type max_val = right->d_columns_int[right->type_index[colInd2]][rcount-1];   
    unsigned int tab_count = (max_val / std::numeric_limits<unsigned int>::max()) + 1;	
    vector<CUDPPHandle> tabs;
    vector<unsigned int> tab_nums;
	unsigned int v_offset = 0;
	int_type min_v, max_v;
	thrust::device_ptr<unsigned int> d_r = thrust::device_malloc<unsigned int>(tt);			
	
	for(unsigned int i = 0; i < tab_count; i ++) {

	    // find out rcount
		min_v = i*std::numeric_limits<unsigned int>::max();
		max_v =  min_v + std::numeric_limits<unsigned int>::max();
      		
		unsigned int loc_count = thrust::count_if(right->d_columns_int[right->type_index[colInd2]].begin(), right->d_columns_int[right->type_index[colInd2]].begin() + rcount,
                                            	  _1 > min_v && _1 <= max_v );
        CUDPPHandle hash_table_handle;
        CUDPPHashTableConfig config;
        config.type = CUDPP_MULTIVALUE_HASH_TABLE;
        config.kInputSize = loc_count;
        config.space_usage = 1.5f;
		
        //cout << "creating table with " << loc_count << " " << getFreeMem()  << endl;		
	    result = cudppHashTable(theCudpp, &hash_table_handle, &config);
        //if (result == CUDPP_SUCCESS)
        //    cout << "hash table created " << getFreeMem() << endl;
					
        //cout << "INSERT " <<  " " << loc_count << " " << getFreeMem() << endl;	

		if(i != 0)				
		    thrust::transform(right->d_columns_int[right->type_index[colInd2]].begin() + v_offset, right->d_columns_int[right->type_index[colInd2]].begin() + v_offset + loc_count,
                              d_r, _1 - i*std::numeric_limits<unsigned int>::max());					
		else
	        thrust::copy(right->d_columns_int[right->type_index[colInd2]].begin() + v_offset, right->d_columns_int[right->type_index[colInd2]].begin() + v_offset + loc_count, d_r);	
	
        result = cudppHashInsert(hash_table_handle, thrust::raw_pointer_cast(d_r),
                                 thrust::raw_pointer_cast(v.data() + v_offset), loc_count);								 
							 
        //if (result == CUDPP_SUCCESS)
        //    cout << "hash table inserted " << getFreeMem() << endl;		
			
		v_offset = v_offset + loc_count;	
		tabs.push_back(hash_table_handle);	
		tab_nums.push_back(loc_count);
	};		
	
	//thrust::device_ptr<int_type> d_trans = thrust::device_malloc<int_type>(tt);			
	
    for (unsigned int i = 0; i < left->segCount; i++) {		
	    
		cout << "segment " << i << " " << getFreeMem() << endl;				
		cnt_l = 0;
		copyColumns(left, lc, i, cnt_l);
        if(left->prm.size() == 0) {
           //copy all records	    
		    cnt_l = left->mRecCount;
        }			
		else {				    	 		
			cnt_l = left->prm_count[i];
		};			
		
		if (cnt_l) { 					        
			
			
			unsigned int off = 0;
			for(unsigned int j = 0; j < tab_count; j ++) {
			
				
				if(j)
				    off = off + tab_nums[j-1];
				
				thrust::device_vector<unsigned int> tc(1);
				tc[0] = j;
			    //when copying to d_r need to make sure to set non-relevant values to zero otherwise they will get truncated to relevant values
				thrust::counting_iterator<unsigned int, thrust::device_space_tag> begin(0);
                trans_int t(thrust::raw_pointer_cast(tc.data()),thrust::raw_pointer_cast(left->d_columns_int[left->type_index[colInd1]].data()), thrust::raw_pointer_cast(d_r));
                thrust::for_each(begin, begin + cnt_l, t);		
					
			
			    result = cudppHashRetrieve(tabs[j], thrust::raw_pointer_cast(d_r),
                                           thrust::raw_pointer_cast(res), cnt_l);
			    if (result != CUDPP_SUCCESS)						   
			        cout << "Failed retrieve " << endl;					


	
		        uint2 rr = thrust::reduce(res, res+cnt_l, make_uint2(0,0), Uint2Sum());		
			    res_count = rr.y;

                if(res_count) {		 				

                
                    uint2_split ff(thrust::raw_pointer_cast(res),thrust::raw_pointer_cast(d_r));
                    thrust::for_each(begin, begin + cnt_l, ff);		
		
		            thrust::exclusive_scan(d_r, d_r+cnt_l, d_r );  // addresses	
		
				    tot_count = tot_count + res_count;
                    d_res1.resize(res_count);
                    d_res2.resize(res_count);				
			
                    join_functor ff1(thrust::raw_pointer_cast(res),
                                     thrust::raw_pointer_cast(d_r),
	     			        		 thrust::raw_pointer_cast(d_res1.data()),
		    			        	 thrust::raw_pointer_cast(d_res2.data()));
                    thrust::for_each(begin, begin + cnt_l, ff1);
					
					thrust::transform(d_res2.begin(), d_res2.end(), d_res2.begin(), _1 + off);		
				

	
				    offset = c->mRecCount;
			        c->resize(res_count);				
			
		            queue<string> op_sel1(op_sel);					
                    while(!op_sel1.empty()) {

	                    while(!cc.empty())
                            cc.pop();

                        cc.push(op_sel1.front());
				
                        if(left->columnNames.find(op_sel1.front()) !=  left->columnNames.end()) {
						    // copy field's segment to device, gather it and copy to the host  
					        unsigned int colInd = left->columnNames[op_sel1.front()];	
                            allocColumns(left, cc);						
					        copyColumns(left, cc, i, k);
					       //gather	   
					       if(left->type[colInd] == 0) {
					           thrust::device_ptr<int_type> d_col((int_type*)g);
					           thrust::gather(d_res1.begin(), d_res1.begin() + res_count, left->d_columns_int[left->type_index[colInd]].begin(), d_col);
					           thrust::copy(d_col, d_col + res_count, c->h_columns_int[c->type_index[c->columnNames[op_sel1.front()]]].begin() + offset);								   
					       }	   
					       else if(left->type[colInd] == 1) {
					           thrust::device_ptr<float_type> d_col((float_type*)g);
					           thrust::gather(d_res1.begin(), d_res1.begin() + res_count, left->d_columns_float[left->type_index[colInd]].begin(), d_col);						   
					           thrust::copy(d_col, d_col + res_count, c->h_columns_float[c->type_index[c->columnNames[op_sel1.front()]]].begin() + offset);												   						   
					       };	   					   

					    }
                        else {
						    unsigned int colInd = right->columnNames[op_sel1.front()];		
					       //gather	   					   
					       if(right->type[colInd] == 0) {
					           thrust::device_ptr<int_type> d_col((int_type*)g);
					           thrust::gather(d_res2.begin(), d_res2.begin() + res_count, right->d_columns_int[right->type_index[colInd]].begin(), d_col);						   
					           thrust::copy(d_col, d_col + res_count, c->h_columns_int[c->type_index[c->columnNames[op_sel1.front()]]].begin() + offset);
					       }
					       else if(right->type[colInd] == 1) {
					           thrust::device_ptr<float_type> d_col((float_type*)g);
					           thrust::gather(d_res2.begin(), d_res2.begin() + res_count, right->d_columns_float[right->type_index[colInd]].begin(), d_col);
					           thrust::copy(d_col, d_col + res_count, c->h_columns_float[c->type_index[c->columnNames[op_sel1.front()]]].begin() + offset);
					       };					   
					    };
                        op_sel1.pop();		  
                    };	
				};	
			};			
        };
    };

	for(unsigned int i = 0; i < tab_count; i ++) 
	    cudppDestroyHashTable(theCudpp, tabs[i]);   
	thrust::device_free(res);				
	hipFree(g);
	thrust::device_free(d_r);		
    d_res1.resize(0);
    d_res1.shrink_to_fit();
    d_res2.resize(0);
    d_res2.shrink_to_fit();
	
	

    cout << "join final end " << tot_count << "  " << getFreeMem() << endl;
		
    left->deAllocOnDevice();
    right->deAllocOnDevice();

    varNames[s] = c;
	c->mRecCount = tot_count; 
    clean_queues();


    if(stat[s] == statement_count) {
        c->free();
        varNames.erase(s);
    };

    if(stat[j1] == statement_count) {
        left->free();
        varNames.erase(j1);
    };

    if(stat[j2] == statement_count && (strcmp(j1,j2.c_str()) != 0)) {
        right->free();
        varNames.erase(j2);
    };

	//exit(0);
    std::cout<< "join time " <<  ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) <<'\n';		

}




void emit_order(char *s, char *f, int e, int ll)
{
    if(ll == 0)
        statement_count++;

    if (scan_state == 0 && ll == 0) {
        if (stat.find(f) == stat.end()) {
            cout << "Order : couldn't find variable " << f << endl;
            exit(1);
        };
        stat[s] = statement_count;
        stat[f] = statement_count;
        return;
    };

    if(varNames.find(f) == varNames.end() ) {
        clean_queues();
        return;
    };


    CudaSet* a = varNames.find(f)->second;


    if (a->mRecCount == 0)	{
        if(varNames.find(s) == varNames.end())
            varNames[s] = new CudaSet(0,1);
        else {
            CudaSet* c = varNames.find(s)->second;
            c->mRecCount = 0;
        };
        return;
    };

    stack<string> exe_type, exe_value;

    cout << "order: " << s << " " << f << endl;;


    for(int i=0; !op_type.empty(); ++i, op_type.pop(),op_value.pop()) {
        if ((op_type.front()).compare("NAME") == 0) {
            exe_type.push(op_value.front());
            exe_value.push("ASC");
        }
        else {
            exe_type.push(op_type.front());
            exe_value.push(op_value.front());
        };
    };

    // initialize permutation to [0, 1, 2, ... ,N-1]

    thrust::device_ptr<unsigned int> permutation = thrust::device_malloc<unsigned int>(a->mRecCount);
    thrust::sequence(permutation, permutation+(a->mRecCount));

    unsigned int* raw_ptr = thrust::raw_pointer_cast(permutation);
    CudaSet *b = a->copyStruct(a->mRecCount);

    // find the largest mRecSize of all data sources

    stack<string> tp(exe_type);
    queue<string> op_vx;
    while (!tp.empty()) {
        op_vx.push(tp.top());
        tp.pop();
    };
	
    unsigned int maxSize =  a->mRecCount, cnt = 0;

    void* temp;
    CUDA_SAFE_CALL(hipMalloc((void **) &temp, maxSize*float_size));

    varNames[setMap[exe_type.top()]]->oldRecCount = varNames[setMap[exe_type.top()]]->mRecCount;
    allocColumns(a, op_vx);
    copyColumns(a, op_vx, 0, cnt);

    varNames[setMap[exe_type.top()]]->mRecCount = varNames[setMap[exe_type.top()]]->oldRecCount;

    for(int i=0; !exe_type.empty(); ++i, exe_type.pop(),exe_value.pop()) {
        int colInd = (a->columnNames).find(exe_type.top())->second;

        if ((a->type)[colInd] == 0)
            update_permutation(a->d_columns_int[a->type_index[colInd]], raw_ptr, a->mRecCount, exe_value.top(), (int_type*)temp);
        else if ((a->type)[colInd] == 1)
            update_permutation(a->d_columns_float[a->type_index[colInd]], raw_ptr, a->mRecCount,exe_value.top(), (float_type*)temp);
        else {
            CudaChar* c = a->h_columns_cuda_char[a->type_index[colInd]];
            for(int j=(c->mColumnCount)-1; j>=0 ; j--)
                update_permutation((c->d_columns)[j], raw_ptr, a->mRecCount, exe_value.top(), (char*)temp);
        };
    };

    // gather a's prm  to b's prm
    thrust::device_vector<unsigned int> p(a->mRecCount);
    if(a->prm.size() != 0) {
	
        thrust::device_vector<unsigned int> p_a(a->mRecCount);
        b->prm.push_back(new unsigned int[a->mRecCount]);
        b->prm_count.push_back(a->mRecCount);
		b->prm_index.push_back('R');
        hipMemcpy((void**)(thrust::raw_pointer_cast(p_a.data())), (void**)a->prm[0], 4*a->mRecCount, hipMemcpyHostToDevice);
        thrust::gather(permutation, permutation+a->mRecCount, p_a.begin(), p.begin());
        hipMemcpy((void**)b->prm[0], (void**)(thrust::raw_pointer_cast(p.data())), 4*a->mRecCount, hipMemcpyDeviceToHost);
    }
    else {
        b->prm.push_back(new unsigned int[a->mRecCount]);
        b->prm_count.push_back(a->mRecCount);
		b->prm_index.push_back('R');
        thrust::copy(permutation, permutation+a->mRecCount, p.begin());
        hipMemcpy((void**)b->prm[0], (void**)(thrust::raw_pointer_cast(p.data())), 4*a->mRecCount, hipMemcpyDeviceToHost);
    };

    b->deAllocOnDevice();
    a->deAllocOnDevice();


    thrust::device_free(permutation);
    hipFree(temp);

    varNames[s] = b;
    b->segCount = 1;

    if (a->fact_table == 1)
        b->fact_table = 1;
    else
        b->fact_table = 0;

    if(stat[f] == statement_count && !a->keep) {
        a->free();
        varNames.erase(f);
    };
}


void emit_select(char *s, char *f, int ll)
{
    statement_count++;
    if (scan_state == 0) {
        if (stat.find(f) == stat.end()) {
            cout << "Select : couldn't find variable " << f << endl;
            exit(1);
        };
        stat[s] = statement_count;
        stat[f] = statement_count;
        return;
    };


    if(varNames.find(f) == varNames.end()) {
        clean_queues();
        return;
    };



    queue<string> op_v1(op_value);
    while(op_v1.size() > ll)
        op_v1.pop();


    stack<string> op_v2;
    queue<string> op_v3;

    for(int i=0; i < ll; ++i) {
        op_v2.push(op_v1.front());
        op_v3.push(op_v1.front());
        op_v1.pop();
    };


    CudaSet *a;
    a = varNames.find(f)->second;


    if(a->mRecCount == 0) {
        CudaSet *c;
        c = new CudaSet(0,1);
        varNames[s] = c;
        clean_queues();
        return;
    };

    cout << "SELECT " << s << " " << f << endl;
    std::clock_t start1 = std::clock();

    // here we need to determine the column count and composition

    queue<string> op_v(op_value);
    queue<string> op_vx;
    set<string> field_names;
    map<string,string> aliases;
    string tt;

    for(int i=0; !op_v.empty(); ++i, op_v.pop()) {
        if(a->columnNames.find(op_v.front()) != a->columnNames.end()) {
            field_names.insert(op_v.front());
            if(aliases.count(op_v.front()) == 0 && aliases.size() < ll) {
                tt = op_v.front();
                op_v.pop();
                aliases[tt] = op_v.front();
            };

        };
    };


    for (set<string>::iterator it=field_names.begin(); it!=field_names.end(); ++it)  {
        op_vx.push(*it);
    };


    // find out how many columns a new set will have
    queue<string> op_t(op_type);
    int_type col_count = 0;

    for(int i=0; !op_t.empty(); ++i, op_t.pop())
        if((op_t.front()).compare("emit sel_name") == 0)
            col_count++;


    CudaSet* b, *c;

	curr_segment = 10000000;
    allocColumns(a, op_vx);

	unsigned int cycle_count = 1;
	if(a->prm.size())
        cycle_count = varNames[setMap[op_value.front()]]->segCount;
     	

    unsigned int ol_count = a->mRecCount, cnt;
    varNames[setMap[op_value.front()]]->oldRecCount = varNames[setMap[op_value.front()]]->mRecCount;
	
	
    for(unsigned int i = 0; i < cycle_count; i++) {          // MAIN CYCLE
        cout << "cycle " << i << " select mem " << getFreeMem() << endl;

        if(i == 0)
            b = new CudaSet(0, col_count);			

			cnt = 0;
            copyColumns(a, op_vx, i, cnt);			

            if (ll != 0) {
                order_inplace(a,op_v2,field_names,i);
                a->GroupBy(op_v3);
            };
            select(op_type,op_value,op_nums, op_nums_f,a,b, a->mRecCount);

        if(i == 0) {
            for ( map<string,int>::iterator it=b->columnNames.begin() ; it != b->columnNames.end(); ++it )
                setMap[(*it).first] = s;
        };

        if (ll != 0) {
            if (i == 0) {
                c = new CudaSet(b->mRecCount, col_count);
                c->fact_table = 1;
                c->segCount = 1;
            }
            else {
                c->resize(b->mRecCount);
			};	
            add(c,b,op_v3);
        };
    };
    a->mRecCount = ol_count;
    varNames[setMap[op_value.front()]]->mRecCount = varNames[setMap[op_value.front()]]->oldRecCount;

    if(stat[f] == statement_count) {
        a->deAllocOnDevice();
    };


    if (ll != 0) {
        CudaSet *r = merge(c,op_v3, op_v2, aliases);
        c->free();
        c = r;
    };


    c->maxRecs = c->mRecCount;
    c->name = s;
    c->keep = 1;

    for ( map<string,int>::iterator it=c->columnNames.begin() ; it != c->columnNames.end(); ++it ) {
        setMap[(*it).first] = s;
    };

    cout << "final select " << c->mRecCount << endl;

    clean_queues();

    if (ll != 0) {
        varNames[s] = c;
        b->free();
    }
    else
        varNames[s] = b;

    varNames[s]->keep = 1;

    if(stat[s] == statement_count) {
        varNames[s]->free();
        varNames.erase(s);
    };

    if(stat[f] == statement_count && a->keep == 0) {
        a->free();
        varNames.erase(f);
    };
    std::cout<< "select time " <<  ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) <<'\n';
}


void emit_filter(char *s, char *f, int e)
{
    statement_count++;
    if (scan_state == 0) {
        if (stat.find(f) == stat.end()) {
            cout << "Filter : couldn't find variable " << f << endl;
            exit(1);
        };
        stat[s] = statement_count;
        stat[f] = statement_count;
        clean_queues();
        return;
    };


    if(varNames.find(f) == varNames.end()) {
        clean_queues();
        return;
    };

    CudaSet *a, *b;

    a = varNames.find(f)->second;
    a->name = f;
    std::clock_t start1 = std::clock();

    if(a->mRecCount == 0) {
        b = new CudaSet(0,1);
    }
    else {
        cout << "FILTER " << s << " " << f << " " << getFreeMem() << endl;
		

        b = a->copyDeviceStruct();
        b->name = s;

        unsigned int cycle_count = 1, cnt = 0;
        allocColumns(a, op_value);
		
        varNames[setMap[op_value.front()]]->oldRecCount = varNames[setMap[op_value.front()]]->mRecCount;

        if(a->segCount != 1)
            cycle_count = varNames[setMap[op_value.front()]]->segCount;
        
		oldCount = a->mRecCount;
        thrust::device_vector<unsigned int> p(a->maxRecs);


        for(unsigned int i = 0; i < cycle_count; i++) {		 
		    cout << "cycle " << i << endl;
        	map_check = zone_map_check(op_type,op_value,op_nums, op_nums_f, a, i);
	        cout << "MAP CHECK " << map_check << endl;		
            if(map_check == 'R') {			
                copyColumns(a, op_value, i, cnt);
                filter(op_type,op_value,op_nums, op_nums_f,a, b, i, p);
			}
            else  {		
				setPrm(a,b,map_check,i);
			}
        };
		a->mRecCount = oldCount;
        varNames[setMap[op_value.front()]]->mRecCount = varNames[setMap[op_value.front()]]->oldRecCount;
        cout << "filter is finished " << b->mRecCount << " " << getFreeMem()  << endl;             
        a->deAllocOnDevice();
    };

    clean_queues();

    if (varNames.count(s) > 0)
        varNames[s]->free();

    varNames[s] = b;

    if(stat[s] == statement_count) {
        b->free();
        varNames.erase(s);
    };
    if(stat[f] == statement_count && !a->keep) {
        a->free();
        varNames.erase(f);
    };
    std::cout<< "filter time " <<  ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) << " " << getFreeMem() << '\n';
}

void emit_store(char *s, char *f, char* sep)
{
    statement_count++;
    if (scan_state == 0) {
        if (stat.find(s) == stat.end()) {
            cout << "Store : couldn't find variable " << s << endl;
            exit(1);
        };
        stat[s] = statement_count;
        return;
    };


    if(varNames.find(s) == varNames.end())
        return;

    CudaSet* a = varNames.find(s)->second;

    cout << "STORE: " << s << " " << f << " " << sep << endl;


    int limit = 0;
    if(!op_nums.empty()) {
        limit = op_nums.front();
        op_nums.pop();
    };

    a->Store(f,sep, limit, 0);

    if(stat[s] == statement_count  && a->keep == 0) {
        a->free();
        varNames.erase(s);
    };


};


void emit_store_binary(char *s, char *f)
{
    statement_count++;
    if (scan_state == 0) {
        if (stat.find(s) == stat.end()) {
            cout << "Store : couldn't find variable " << s << endl;
            exit(1);
        };
        stat[s] = statement_count;
        return;
    };


    if(varNames.find(s) == varNames.end())
        return;

    CudaSet* a = varNames.find(s)->second;

    if(stat[f] == statement_count)
        a->deAllocOnDevice();


    printf("STORE: %s %s \n", s, f);

    int limit = 0;
    if(!op_nums.empty()) {
        limit = op_nums.front();
        op_nums.pop();
    };

    fact_file_loaded = 0;
    while(!fact_file_loaded)	{
        cout << "LOADING " << f_file << " " << separator << endl;
        fact_file_loaded = a->LoadBigFile(f_file.c_str(), separator.c_str());
        //cout << "STORING " << f << " " << limit << endl;
        a->Store(f,"", limit, 1);
    };

    if(stat[f] == statement_count && !a->keep) {
        a->free();
        varNames.erase(s);
    };

};



void emit_load_binary(char *s, char *f, int d)
{
    statement_count++;
    if (scan_state == 0) {
        stat[s] = statement_count;
        return;
    };

    printf("BINARY LOAD: %s %s \n", s, f);

    CudaSet *a;
    unsigned int segCount, maxRecs;
    char f1[100];
    strcpy(f1, f);
    strcat(f1,".");
    char col_pos[3];
    itoaa(cols.front(),col_pos);
    strcat(f1,col_pos);

    FILE* ff = fopen(f1, "rb");
    fseeko(ff, -16, SEEK_END);
    fread((char *)&totalRecs, 8, 1, ff);
    fread((char *)&segCount, 4, 1, ff);
    fread((char *)&maxRecs, 4, 1, ff);
    fclose(ff);

    queue<string> names(namevars);
    while(!names.empty()) {
        setMap[names.front()] = s;
        names.pop();
    };

    a = new CudaSet(namevars, typevars, sizevars, cols,totalRecs, f);
    a->segCount = segCount;
    a->maxRecs = maxRecs;
    a->keep = 1;
    varNames[s] = a;

    if(stat[s] == statement_count )  {
        a->free();
        varNames.erase(s);
    };
}





void emit_load(char *s, char *f, int d, char* sep)
{
    statement_count++;
    if (scan_state == 0) {
        stat[s] = statement_count;
        return;
    };


    printf("LOAD: %s %s %d  %s \n", s, f, d, sep);

    CudaSet *a;

    a = new CudaSet(namevars, typevars, sizevars, cols, process_count);
    a->mRecCount = 0;
    a->resize(process_count);
    a->keep = true;
    a->fact_table = 1;
    //a->LoadBigFile(f, sep);
    string separator1(sep);
    separator = separator1;
    string ff(f);
    f_file = ff;
    a->maxRecs = a->mRecCount;
    a->segCount = 0;
    varNames[s] = a;

    if(stat[s] == statement_count)  {
        a->free();
        varNames.erase(s);
    };

}



void yyerror(char *s, ...)
{
    extern int yylineno;
    va_list ap;
    va_start(ap, s);

    fprintf(stderr, "%d: error: ", yylineno);
    vfprintf(stderr, s, ap);
    fprintf(stderr, "\n");
}

void clean_queues()
{
    while(!op_type.empty()) op_type.pop();
    while(!op_value.empty()) op_value.pop();
    while(!op_join.empty()) op_join.pop();
    while(!op_nums.empty()) op_nums.pop();
    while(!op_nums_f.empty()) op_nums_f.pop();
    while(!j_col_count.empty()) j_col_count.pop();
    while(!namevars.empty()) namevars.pop();
    while(!typevars.empty()) typevars.pop();
    while(!sizevars.empty()) sizevars.pop();
    while(!cols.empty()) cols.pop();

    sel_count = 0;
    join_cnt = 0;
    join_col_cnt = -1;
    eqq = 0;
}



int main(int ac, char **av)
{
    extern FILE *yyin;
    hipDeviceProp_t deviceProp;

    hipGetDeviceProperties(&deviceProp, 0);
    if (!deviceProp.canMapHostMemory)
        cout << "Device 0 cannot map host memory" << endl;

    hipSetDeviceFlags(hipDeviceMapHost);
    cudppCreate(&theCudpp);

    if (ac == 1) {
        cout << "Usage : alenka -l process_count script.sql" << endl;
        exit(1);
    };

    if(strcmp(av[1],"-l") == 0) {
        process_count = atoff(av[2]);
        cout << "Process count = " << process_count << endl;
    }
    else {
        process_count = 6200000;
        cout << "Process count = 6200000 " << endl;
    };

    if((yyin = fopen(av[ac-1], "r")) == NULL) {
        perror(av[ac-1]);
        exit(1);
    };

    if(yyparse()) {
        printf("SQL scan parse failed\n");
        exit(1);
    };
    fclose(yyin);

    scan_state = 1;

    std::clock_t start1 = std::clock();
    statement_count = 0;
    clean_queues();

    if(ac > 1 && (yyin = fopen(av[ac-1], "r")) == NULL) {
        perror(av[1]);
        exit(1);
    }

    PROC_FLUSH_BUF ( yyin );
    statement_count = 0;

    if(!yyparse())
        cout << "SQL scan parse worked" << endl;
    else
        cout << "SQL scan parse failed" << endl;

    fclose(yyin);
    std::cout<< "cycle time " <<  ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) <<'\n';
    cudppDestroy(theCudpp);

}


