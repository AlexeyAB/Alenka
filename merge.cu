#include "hip/hip_runtime.h"
/*
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */


#include "cm.h"

head_flag_predicate<bool> binary_pred_l;

void add(CudaSet* c, CudaSet* b, queue<string> op_v3)
{
    if (c->columnNames.empty()) {
        // create d_columns and h_columns
        
        map<int,string> columnNames1;
		map<string,int>::iterator it;

        for (  it=b->columnNames.begin() ; it != b->columnNames.end(); ++it ) {
            c->columnNames[(*it).first] = (*it).second;			
            columnNames1[(*it).second] = (*it).first;
        };

        string f_name;
        c->grp_type = new unsigned int[c->mColumnCount];

        for(unsigned int i=0; i < b->mColumnCount; i++) {
            c->cols[i] = b->cols[i];
            c->type[i] = b->type[i];
            c->grp_type[i] = b->grp_type[i];

            if (b->type[i] == 0) {
                c->h_columns_int.push_back(thrust::host_vector<int_type>(c->mRecCount+1));
                c->d_columns_int.push_back(thrust::device_vector<int_type>());
                c->type_index[i] = c->h_columns_int.size()-1;
            }
            else if (b->type[i] == 1) {
                c->h_columns_float.push_back(thrust::host_vector<float_type>(c->mRecCount+1));
                c->d_columns_float.push_back(thrust::device_vector<float_type>());
                c->type_index[i] = c->h_columns_float.size()-1;
            }
            else {
                c->h_columns_cuda_char.push_back(new CudaChar((b->h_columns_cuda_char[b->type_index[i]])->mColumnCount, c->mRecCount));
                c->type_index[i] = c->h_columns_cuda_char.size()-1;
            };
        };
    }
    // append b to c
    for(unsigned int i=0; i < b->mColumnCount; i++) {
        if (b->type[i] == 0 )
            thrust::copy(b->d_columns_int[b->type_index[i]].begin(), b->d_columns_int[b->type_index[i]].begin() + b->mRecCount,
                         c->h_columns_int[c->type_index[i]].begin() + (c->mRecCount-b->mRecCount));
        else if (b->type[i] == 1 )
            thrust::copy(b->d_columns_float[b->type_index[i]].begin(), b->d_columns_float[b->type_index[i]].begin() + b->mRecCount,
                         c->h_columns_float[c->type_index[i]].begin() + (c->mRecCount-b->mRecCount));
        else { //CudaChar
            CudaChar *s = b->h_columns_cuda_char[b->type_index[i]];
            CudaChar *d = c->h_columns_cuda_char[c->type_index[i]];
            for(unsigned int j=0; j < s->mColumnCount; j++)
                thrust::copy(s->d_columns[j].begin(), s->d_columns[j].begin() + b->mRecCount, d->h_columns[j].begin()+(c->mRecCount-b->mRecCount));
        };
    };
}


void order_inplace(CudaSet* a, stack<string> exe_type, map<string,string> aliases)
{
    thrust::device_ptr<unsigned int> permutation = thrust::device_malloc<unsigned int>(a->mRecCount);
    thrust::sequence(permutation, permutation+(a->mRecCount));

    void* temp;
    CUDA_SAFE_CALL(hipMalloc((void **) &temp, a->mRecCount*float_size));
    unsigned int* raw_ptr = thrust::raw_pointer_cast(permutation);

    for(int i=0; !exe_type.empty(); ++i, exe_type.pop()) {

        if ((a->columnNames).find(aliases[exe_type.top()]) ==  a->columnNames.end()) {
            cout << "Sort couldn't find field " << exe_type.top() << endl;
            exit(1);
        };

        int colInd = (a->columnNames).find(aliases[exe_type.top()])->second;

        if(!a->onDevice(colInd) && a->type[colInd] < 2) {
            a->allocColumnOnDevice(colInd,a->mRecCount);
            a->CopyColumnToGpu(colInd, 0, a->mRecCount);
        };

        if ((a->type)[colInd] == 0)
            update_permutation(a->d_columns_int[a->type_index[colInd]], raw_ptr, a->mRecCount, "ASC", (int_type*)temp);
        else if ((a->type)[colInd] == 1)
            update_permutation(a->d_columns_float[a->type_index[colInd]], raw_ptr, a->mRecCount,"ASC", (float_type*)temp);
        else {
            CudaChar* c = a->h_columns_cuda_char[a->type_index[colInd]];
            thrust::device_ptr<char> tmp = thrust::device_malloc<char>(a->mRecCount);

            for(int j=(c->mColumnCount)-1; j>=0 ; j--) {
                c->d_columns[j].resize(a->mRecCount);
                thrust::copy(c->h_columns[j].begin(), c->h_columns[j].begin() + a->mRecCount, c->d_columns[j].begin());
                update_permutation(c->d_columns[j], raw_ptr, a->mRecCount, "ASC", thrust::raw_pointer_cast(tmp));
                c->d_columns[j].resize(0);
                c->d_columns[j].shrink_to_fit();
            };
            thrust::device_free(tmp);
        };
        a->deAllocColumnOnDevice(colInd);
    };


    for(unsigned int i=0; i < a->mColumnCount; i++) {
        if(!a->onDevice(i) && a->type[i] < 2) {
            a->allocColumnOnDevice(i,a->mRecCount);
            a->CopyColumnToGpu(i, 0, a->mRecCount);
        };

        if (a->type[i] == 0)
            apply_permutation(a->d_columns_int[a->type_index[i]], raw_ptr, a->mRecCount, (int_type*)temp);
        else if (a->type[i] == 1)
            apply_permutation(a->d_columns_float[a->type_index[i]], raw_ptr, a->mRecCount, (float_type*)temp);
        else {
            CudaChar* c = a->h_columns_cuda_char[a->type_index[i]];
            for(int j=(c->mColumnCount)-1; j>=0 ; j--) {
                c->d_columns[j].resize(c->mRecCount);
                thrust::copy(c->h_columns[j].begin(), c->h_columns[j].begin() + c->mRecCount, c->d_columns[j].begin());
                apply_permutation(c->d_columns[j], raw_ptr, a->mRecCount, (char*)temp);
                thrust::copy(c->d_columns[j].begin(), c->d_columns[j].begin() + c->mRecCount, c->h_columns[j].begin());
                c->d_columns[j].resize(0);
                c->d_columns[j].shrink_to_fit();
            };
        };
        if (a->type[i] != 2)
            a->CopyColumnToHost(i);
        a->deAllocColumnOnDevice(i);
    };
    thrust::device_free(permutation);
    hipFree(temp);
}



CudaSet* merge(CudaSet* c, queue<string> op_v3, stack<string> op_v2, map<string,string> aliases)
{
    int countIndex;
    int avg_index = -1;

    for(unsigned int i = 0; i < c->mColumnCount; i++) {
        if(c->grp_type[i] == 0) // COUNT
            countIndex = i;
        else if(c->grp_type[i] == 1) // AVG
            avg_index = i;
    };

    CudaSet *r = c->copyStruct(c->mRecCount);

    r->mRecCount = 0;

    if (c->mRecCount != 0) {
        order_inplace(c,op_v2, aliases);		
	
		//change op_v3 to aliases
		queue<string> op;
		for(int i = 0; i < op_v3.size(); op_v3.pop())
		    op.push(aliases[op_v3.front()]);		
		
        c->GroupBy(op);		

        thrust::device_ptr<bool> d_grp(c->grp);

        for(unsigned int j=0; j < c->mColumnCount; j++) {
            c->allocColumnOnDevice(j, c->mRecCount);
            c->CopyColumnToGpu(j, 0, c->mRecCount);

            if (c->grp_type[j] == 3) {	      	  	  // non-grouped columns
                if (c->type[j] == 0) {
                    thrust::device_ptr<int_type> diff = thrust::device_malloc<int_type>(c->grp_count);
                    thrust::copy_if(c->d_columns_int[c->type_index[j]].begin(), c->d_columns_int[c->type_index[j]].begin() + (c->mRecCount), d_grp, diff, nz<bool>());
                    thrust::copy(diff, diff+c->grp_count, r->h_columns_int[r->type_index[j]].begin() + r->mRecCount);
                    thrust::device_free(diff);
                }
                else if (c->type[j] == 1) {
                    thrust::device_ptr<float_type> diff = thrust::device_malloc<float_type>(c->grp_count);
                    thrust::copy_if(c->d_columns_float[c->type_index[j]].begin(), c->d_columns_float[c->type_index[j]].begin() + (c->mRecCount), d_grp, diff, nz<bool>());
                    thrust::copy(diff, diff+c->grp_count, r->h_columns_float[r->type_index[j]].begin() + r->mRecCount);
                    thrust::device_free(diff);
                }
                else if (c->type[j] == 2) {
                    CudaChar *cc = c->h_columns_cuda_char[c->type_index[j]];
                    CudaChar *rr = r->h_columns_cuda_char[r->type_index[j]];
                    thrust::device_ptr<char> diff = thrust::device_malloc<char>(c->grp_count);

                    for(unsigned int k=0; k < (cc->mColumnCount); k++) {
                        thrust::copy_if(cc->d_columns[k].begin(),cc->d_columns[k].begin() + c->mRecCount, d_grp, diff, nz<bool>());
                        thrust::copy(diff,diff+c->grp_count,rr->h_columns[k].begin() + r->mRecCount);
                    };
                    thrust::device_free(diff);
                }
            }
            else if (c->grp_type[j] == 2 || c->grp_type[j] == 1) {  // sum and avg

                if (c->type[j] == 0) {
                    thrust::reduce_by_key(d_grp, d_grp+(c->mRecCount), c->d_columns_int[c->type_index[j]].begin(),
                                          thrust::make_discard_iterator(), c->d_columns_int[c->type_index[j]].begin(),
                                          binary_pred_l,thrust::plus<int_type>());
                    thrust::copy(c->d_columns_int[c->type_index[j]].begin(), c->d_columns_int[c->type_index[j]].begin() + c->grp_count, r->h_columns_int[r->type_index[j]].begin() + r->mRecCount);
                }
                else if (c->type[j] == 1) {
                    thrust::reduce_by_key(d_grp, d_grp+(c->mRecCount), c->d_columns_float[c->type_index[j]].begin(),
                                          thrust::make_discard_iterator(), c->d_columns_float[c->type_index[j]].begin(),
                                          binary_pred_l,thrust::plus<float_type>());
                    thrust::copy(c->d_columns_float[c->type_index[j]].begin(), c->d_columns_float[c->type_index[j]].begin() + c->grp_count, r->h_columns_float[r->type_index[j]].begin() + r->mRecCount);
                }
            }
            else if (c->grp_type[j] == 0) {  // count
                thrust::reduce_by_key(d_grp, d_grp+(c->mRecCount), c->d_columns_int[c->type_index[j]].begin(),
                                      thrust::make_discard_iterator(), c->d_columns_int[c->type_index[j]].begin(),
                                      binary_pred_l,thrust::plus<int_type>());
                thrust::copy(c->d_columns_int[c->type_index[j]].begin(), c->d_columns_int[c->type_index[j]].begin() + c->grp_count, r->h_columns_int[r->type_index[j]].begin() + r->mRecCount);
            }
            else if(c->grp_type[j] == 4) {  // min
                if (c->type[j] == 0) {
                    thrust::reduce_by_key(d_grp, d_grp+(c->mRecCount), c->d_columns_int[c->type_index[j]].begin(),
                                          thrust::make_discard_iterator(), c->d_columns_int[c->type_index[j]].begin(),
                                          binary_pred_l,thrust::minimum<int_type>());
                    thrust::copy(c->d_columns_int[c->type_index[j]].begin(), c->d_columns_int[c->type_index[j]].begin() + c->grp_count, r->h_columns_int[r->type_index[j]].begin() + r->mRecCount);
                }
                else if (c->type[j] == 1) {
                    thrust::reduce_by_key(d_grp, d_grp+(c->mRecCount), c->d_columns_float[c->type_index[j]].begin(),
                                          thrust::make_discard_iterator(), c->d_columns_float[c->type_index[j]].begin(),
                                          binary_pred_l,thrust::minimum<float_type>());
                    thrust::copy(c->d_columns_float[c->type_index[j]].begin(), c->d_columns_float[c->type_index[j]].begin() + c->grp_count, r->h_columns_float[r->type_index[j]].begin() + r->mRecCount);
                };
            }
            else if(c->grp_type[j] == 5) {  // max
                if (c->type[j] == 0) {
                    thrust::reduce_by_key(d_grp, d_grp+(c->mRecCount), c->d_columns_int[c->type_index[j]].begin(),
                                          thrust::make_discard_iterator(), c->d_columns_int[c->type_index[j]].begin(),
                                          binary_pred_l,thrust::maximum<int_type>());
                    thrust::copy(c->d_columns_int[c->type_index[j]].begin(), c->d_columns_int[c->type_index[j]].begin() + c->grp_count, r->h_columns_int[r->type_index[j]].begin() + r->mRecCount);
                }
                else if (c->type[j] == 1) {
                    thrust::reduce_by_key(d_grp, d_grp+(c->mRecCount), c->d_columns_float[c->type_index[j]].begin(),
                                          thrust::make_discard_iterator(), c->d_columns_float[c->type_index[j]].begin(),
                                          binary_pred_l,thrust::maximum<float_type>());
                    thrust::copy(c->d_columns_float[c->type_index[j]].begin(), c->d_columns_float[c->type_index[j]].begin() + c->grp_count, r->h_columns_float[r->type_index[j]].begin() + r->mRecCount);
                };
            };
            c->deAllocColumnOnDevice(j);
        };

        if (avg_index != -1) {
            thrust::device_ptr<float_type> count_d = thrust::device_malloc<float_type>(c->grp_count);

            r->allocColumnOnDevice(countIndex,c->grp_count);
            r->CopyColumnToGpu(countIndex, r->mRecCount, c->grp_count);
            thrust::transform(r->d_columns_int[r->type_index[countIndex]].begin(), r->d_columns_int[r->type_index[countIndex]].begin() + c->grp_count, count_d, long_to_float_type());
            r->deAllocColumnOnDevice(countIndex);

            for(unsigned int k = 0; k < c->mColumnCount; k++)	{
                if(c->grp_type[k] == 1) {   // AVG

                    r->allocColumnOnDevice(k, c->grp_count);
                    r->CopyColumnToGpu(k, r->mRecCount, c->grp_count);
                    unsigned int idx;

                    if (c->type[k] == 0 ) { // int

                        //create a float column k

                        r->h_columns_float.push_back(thrust::host_vector<float_type>(c->grp_count));
                        r->d_columns_float.push_back(thrust::device_vector<float_type>(c->grp_count));
                        idx = r->h_columns_float.size()-1;

                        thrust::transform(r->d_columns_int[r->type_index[k]].begin(), r->d_columns_int[r->type_index[k]].begin() + c->grp_count, count_d,
                                          r->d_columns_float[idx].begin(), div_long_to_float_type());
                        r->type[k] = 1;
                        //dealloc k on device and host
                        r->d_columns_int[r->type_index[k]].resize(0);
                        r->d_columns_int[r->type_index[k]].shrink_to_fit();
                        r->h_columns_int[r->type_index[k]].resize(0);
                        r->h_columns_int[r->type_index[k]].shrink_to_fit();
                        r->type_index[k] = idx;
                    }
                    else               // float
                        thrust::transform(r->d_columns_float[r->type_index[k]].begin(), r->d_columns_float[r->type_index[k]].begin() + c->grp_count, count_d, r->d_columns_float[r->type_index[k]].begin(), thrust::divides<float_type>());
                    r->CopyColumnToHost(k, r->mRecCount, c->grp_count);
                    r->deAllocColumnOnDevice(k);
                };
            };
            thrust::device_free(count_d);
        };
        r->mRecCount = r->mRecCount + c->grp_count;
    };

    r->segCount = 1;
    r->maxRecs = r->mRecCount;
    return r;

};



