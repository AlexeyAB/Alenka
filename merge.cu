/*
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <unordered_map>
#include "cm.h"

head_flag_predicate<bool> binary_pred_l;

int myPow(long long int x, long long int p)
{
  if (p == 0) return 1;
  if (p == 1) return x;

  int tmp = myPow(x, p/2);
  if (p%2 == 0) return tmp * tmp;
  else return x * tmp * tmp;
}

void add(CudaSet* c, CudaSet* b, queue<string> op_v3, std::unordered_map<long long int, unsigned int>& mymap, map<string,string> aliases)
{
    if (c->columnNames.empty()) {
        // create d_columns and h_columns

        map<string,int>::iterator it;

        for (  it=b->columnNames.begin() ; it != b->columnNames.end(); ++it ) {
            c->columnNames[(*it).first] = (*it).second;
        };

        c->grp_type = new unsigned int[c->mColumnCount];

        for(unsigned int i=0; i < b->mColumnCount; i++) {
            c->cols[i] = b->cols[i];
            c->type[i] = b->type[i];
            c->grp_type[i] = b->grp_type[i];

            if (b->type[i] == 0) {
                c->h_columns_int.push_back(thrust::host_vector<int_type>());
            }
            else if (b->type[i] == 1) {
                c->h_columns_float.push_back(thrust::host_vector<float_type>());
            }
            else {
				c->h_columns_char.push_back(NULL); 
				c->char_size.push_back(b->char_size[b->type_index[i]]);
            };
			c->type_index[i] = b->type_index[i];
        };
    }
	
	
	std::unordered_map<long long int, unsigned int>::const_iterator got;	 	  
	 
     b->CopyToHost(0, b->mRecCount);
	 
   
	// store in a variable c only unique records
	// we have do it on a host because the hash table for the expected set sizes(~5 bln records) won't fit into a GPU memory
	// gonna be kinda on the slow side 
	
	
	long long int *b_hash = new long long int[b->mRecCount];
	long long int res, loc;
	unsigned int idx;
	for(unsigned int i = 0; i < b->mRecCount; i++) {
	     queue<string> op_v(op_v3);		 
		 res = 0;
	     for(unsigned int z = 0; z < op_v.size(); op_v.pop()) {	
		     idx = b->columnNames[aliases[op_v.front()]];
		     if(b->type[b->columnNames[aliases[op_v.front()]]] == 0) {  //int
			     loc = MurmurHash64A(thrust::raw_pointer_cast(b->h_columns_int[b->type_index[idx]].data()) + i, int_size, hash_seed)/2;
			 }
			 else if(b->type[b->columnNames[aliases[op_v.front()]]] == 2) {  //string
			     loc = MurmurHash64A(b->h_columns_char[b->type_index[idx]] + i*b->char_size[b->type_index[idx]], b->char_size[b->type_index[idx]], hash_seed);
			 }
			 else {  //float
			     cout << "Group by on float is not supported !!! " << endl;
				 exit(0);
			 };	
             res = res + myPow(loc, z+1);
		 };
	     b_hash[i] = res;
	};
	
	
	 //resize c
	 unsigned int cnt = 0;
	 for(unsigned int i = 0; i < b->mRecCount; i++) {
	     got = mymap.find(b_hash[i]);
	     if(got == mymap.end())
		     cnt++;
	 };
	 unsigned int old_cnt = c->mRecCount;
	 if(cnt)
	     c->resize(cnt);
	
	
	// now lets add to c those records that are not already there and update those that are there
	for(unsigned int i = 0; i < b->mRecCount; i++) {
	    queue<string> op_v(op_v3);
		
	    got = mymap.find(b_hash[i]);		
	    if(got == mymap.end()) {	//not found, need to insert
	//	cout << "insert " << b_hash[i] << endl;
		    mymap[b_hash[i]] = old_cnt;			
			for(unsigned int j=0; j < b->mColumnCount; j++) {	
			    
			    if(b->type[j] == 0) {  //int
					 c->h_columns_int[c->type_index[j]][old_cnt] = b->h_columns_int[b->type_index[j]][i];
				}
				else if(b->type[j] == 1) {  //float
 				    c->h_columns_float[c->type_index[j]][old_cnt] = b->h_columns_float[b->type_index[j]][i];
				}
				else if(b->type[j] == 2) {  //string
					hipMemcpy(c->h_columns_char[c->type_index[j]] + old_cnt*b->char_size[b->type_index[j]], b->h_columns_char[b->type_index[j]] + i*b->char_size[b->type_index[j]],
					           b->char_size[b->type_index[j]], hipMemcpyHostToHost);
				};			
			};		
            old_cnt++;			
		}
		else { //need to update
//		    cout << "update " << b_hash[i] << endl;
		    for(unsigned int j=0; j < b->mColumnCount; j++) {	
			    
			    if (c->grp_type[j] == 2 || c->grp_type[j] == 1 || c->grp_type[j] == 0) {  // SUM || AVG || COUNT
				    if (c->type[j] == 0) {
					   c->h_columns_int[c->type_index[j]][got->second] +=  b->h_columns_int[b->type_index[j]][i];
					}
					else {
					    c->h_columns_float[c->type_index[j]][got->second] += b->h_columns_float[b->type_index[j]][i];
					};
				}
				else if(c->grp_type[j] == 4) {  // MIN
				    if (c->type[j] == 0) {
					     if (c->h_columns_int[c->type_index[j]][got->second] >  b->h_columns_int[b->type_index[j]][i])
					       c->h_columns_int[c->type_index[j]][got->second] =  b->h_columns_int[b->type_index[j]][i];
					}
					else {
					    if (c->h_columns_float[c->type_index[j]][got->second] > b->h_columns_float[b->type_index[j]][i])
						    c->h_columns_float[c->type_index[j]][got->second] = b->h_columns_float[b->type_index[j]][i];
					};				
                } 			
				else if(c->grp_type[j] == 5) {  // MAX
				    if (c->type[j] == 0) {
					     if (c->h_columns_int[c->type_index[j]][got->second] <  b->h_columns_int[b->type_index[j]][i])
					       c->h_columns_int[c->type_index[j]][got->second] =  b->h_columns_int[b->type_index[j]][i];
					}
					else {
					    if (c->h_columns_float[c->type_index[j]][got->second] < b->h_columns_float[b->type_index[j]][i])
						    c->h_columns_float[c->type_index[j]][got->second] = b->h_columns_float[b->type_index[j]][i];
					};				
                } 							
			};
		};		
	};	
	delete [] b_hash;
}





void count_avg(CudaSet* c)
{
    int countIndex;

    for(unsigned int i = 0; i < c->mColumnCount; i++) {
        if(c->grp_type[i] == 0) // COUNT
            countIndex = i;
    };    
	
    if (c->mRecCount != 0) {    
    	
        for(unsigned int k = 0; k < c->mColumnCount; k++)	{
            if(c->grp_type[k] == 1) {   // AVG
            
                if (c->type[k] == 0 ) { // int
                    //create a float column k						
                    c->h_columns_float.push_back(thrust::host_vector<float_type>(c->mRecCount));                    
                    unsigned int idx = c->h_columns_float.size()-1;

				    for(unsigned int z = 0; z < c->mRecCount; z++) {
					    c->h_columns_float[idx][z] =  ((float_type)c->h_columns_int[c->type_index[k]][z]) / (float_type)c->h_columns_int[c->type_index[countIndex]][z];										  
					};				  
                    c->type[k] = 1;
                    c->h_columns_int[c->type_index[k]].resize(0);
                    c->h_columns_int[c->type_index[k]].shrink_to_fit();
                    c->type_index[k] = idx;
                }
                else {              // float
					for(unsigned int z = 0; z < c->mRecCount; z++) {
					    c->h_columns_float[c->type_index[k]][z] =  c->h_columns_float[c->type_index[k]][z] / (float_type)c->h_columns_int[c->type_index[countIndex]][z];										  
					};				  
				};	
            };
        };
    };

    c->segCount = 1;
    c->maxRecs = c->mRecCount;
};

