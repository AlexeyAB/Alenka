#include "hip/hip_runtime.h"
/*
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */


#include "cm.h"

head_flag_predicate<bool> binary_pred_l;

void add(CudaSet* c, CudaSet* b, queue<string> op_v3)
{
    if (c->columnNames.empty()) {
        // create d_columns and h_columns

        map<string,int>::iterator it;

        for (  it=b->columnNames.begin() ; it != b->columnNames.end(); ++it ) {
            c->columnNames[(*it).first] = (*it).second;
        };

        c->grp_type = new unsigned int[c->mColumnCount];

        for(unsigned int i=0; i < b->mColumnCount; i++) {
            c->cols[i] = b->cols[i];
            c->type[i] = b->type[i];
            c->grp_type[i] = b->grp_type[i];

            if (b->type[i] == 0) {
                c->h_columns_int.push_back(thrust::host_vector<int_type>(c->mRecCount+1));
                c->d_columns_int.push_back(thrust::device_vector<int_type>());
                c->type_index[i] = c->h_columns_int.size()-1;
            }
            else if (b->type[i] == 1) {
                c->h_columns_float.push_back(thrust::host_vector<float_type>(c->mRecCount+1));
                c->d_columns_float.push_back(thrust::device_vector<float_type>());
                c->type_index[i] = c->h_columns_float.size()-1;
            }
            else {
				c->h_columns_char.push_back(new char[b->char_size[b->type_index[i]]*c->mRecCount]); 
				c->d_columns_char.push_back(NULL);
                c->type_index[i] = c->h_columns_char.size()-1;
				c->char_size.push_back(b->char_size[b->type_index[i]]);
            };
        };
    }
    // append b to c
    for(unsigned int i=0; i < b->mColumnCount; i++) {
        if (b->type[i] == 0 )
            thrust::copy(b->d_columns_int[b->type_index[i]].begin(), b->d_columns_int[b->type_index[i]].begin() + b->mRecCount,
                         c->h_columns_int[c->type_index[i]].begin() + (c->mRecCount-b->mRecCount));
        else if (b->type[i] == 1 )
            thrust::copy(b->d_columns_float[b->type_index[i]].begin(), b->d_columns_float[b->type_index[i]].begin() + b->mRecCount,
                         c->h_columns_float[c->type_index[i]].begin() + (c->mRecCount-b->mRecCount));
        else { //Char

			hipMemcpy((void*)&c->h_columns_char[c->type_index[i]][(c->mRecCount-b->mRecCount) * b->char_size[b->type_index[i]]], 
			           (void*)b->d_columns_char[b->type_index[i]], b->mRecCount * b->char_size[b->type_index[i]], hipMemcpyDeviceToHost);
        };
    };
}


void order_inplace(CudaSet* a, stack<string> exe_type, map<string,string> aliases)
{
    thrust::device_ptr<unsigned int> permutation = thrust::device_malloc<unsigned int>(a->mRecCount);
    thrust::sequence(permutation, permutation+(a->mRecCount));

    void* temp;
	unsigned int max_char = 0;
	
	for(unsigned int i = 0; i < a->char_size.size(); i++)
	    if (a->char_size[a->type_index[i]] > max_char)
		    max_char = a->char_size[a->type_index[i]];
		
    if(max_char > float_size)	
	    CUDA_SAFE_CALL(hipMalloc((void **) &temp, a->mRecCount*max_char));
	else	
        CUDA_SAFE_CALL(hipMalloc((void **) &temp, a->mRecCount*float_size));
	
	
    //CUDA_SAFE_CALL(hipMalloc((void **) &temp, a->mRecCount*float_size));
    unsigned int* raw_ptr = thrust::raw_pointer_cast(permutation);

    for(int i=0; !exe_type.empty(); ++i, exe_type.pop()) {

        if ((a->columnNames).find(aliases[exe_type.top()]) ==  a->columnNames.end()) {
            cout << "Sort couldn't find field " << exe_type.top() << endl;
            exit(1);
        };

        int colInd = (a->columnNames).find(aliases[exe_type.top()])->second;

        if(!a->onDevice(colInd)) {
            a->allocColumnOnDevice(colInd,a->mRecCount);
            a->CopyColumnToGpu(colInd, 0, a->mRecCount);
        };

        if ((a->type)[colInd] == 0)
            update_permutation(a->d_columns_int[a->type_index[colInd]], raw_ptr, a->mRecCount, "ASC", (int_type*)temp);
        else if ((a->type)[colInd] == 1)
            update_permutation(a->d_columns_float[a->type_index[colInd]], raw_ptr, a->mRecCount,"ASC", (float_type*)temp);
        else {
		    update_permutation_char(a->d_columns_char[a->type_index[colInd]], raw_ptr, a->mRecCount, "ASC", (char*)temp, a->char_size[a->type_index[colInd]]);	
        };
    };


    for(unsigned int i=0; i < a->mColumnCount; i++) {
        if(!a->onDevice(i)) {
            a->allocColumnOnDevice(i,a->mRecCount);
            a->CopyColumnToGpu(i, 0, a->mRecCount);
        };

        if (a->type[i] == 0)
            apply_permutation(a->d_columns_int[a->type_index[i]], raw_ptr, a->mRecCount, (int_type*)temp);
        else if (a->type[i] == 1)
            apply_permutation(a->d_columns_float[a->type_index[i]], raw_ptr, a->mRecCount, (float_type*)temp);
        else {
		    apply_permutation_char(a->d_columns_char[a->type_index[i]], raw_ptr, a->mRecCount, (char*)temp, a->char_size[a->type_index[i]]);
        };
        a->CopyColumnToHost(i);
        a->deAllocColumnOnDevice(i);		
    };
    thrust::device_free(permutation);
    hipFree(temp);
}



CudaSet* merge(CudaSet* c, queue<string> op_v3, stack<string> op_v2, map<string,string> aliases)
{
    int countIndex;
    int avg_index = -1;
	

    for(unsigned int i = 0; i < c->mColumnCount; i++) {
        if(c->grp_type[i] == 0) // COUNT
            countIndex = i;
        else if(c->grp_type[i] == 1) // AVG
            avg_index = i;
    };

    CudaSet *r = c->copyDeviceStruct();
	if(!c->mRecCount) 
	    return r; 
	r->resize(c->mRecCount);

    r->mRecCount = 0;

    if (c->mRecCount != 0) {
        order_inplace(c,op_v2, aliases);
		

        //change op_v3 to aliases
        queue<string> op;
        for(int i = 0; i < op_v3.size(); op_v3.pop())
            op.push(aliases[op_v3.front()]);

        c->GroupBy(op);

        thrust::device_ptr<bool> d_grp(c->grp);
		
        for(unsigned int j=0; j < c->mColumnCount; j++) {
            c->allocColumnOnDevice(j, c->mRecCount);
            c->CopyColumnToGpu(j, 0, c->mRecCount);

            if (c->grp_type[j] == 3) {	      	  	  // non-grouped columns
                if (c->type[j] == 0) {
                    thrust::device_ptr<int_type> diff = thrust::device_malloc<int_type>(c->grp_count);
                    thrust::copy_if(c->d_columns_int[c->type_index[j]].begin(), c->d_columns_int[c->type_index[j]].begin() + (c->mRecCount), d_grp, diff, thrust::identity<bool>());
                    thrust::copy(diff, diff+c->grp_count, r->h_columns_int[r->type_index[j]].begin() + r->mRecCount);
                    thrust::device_free(diff);
                }
                else if (c->type[j] == 1) {
                    thrust::device_ptr<float_type> diff = thrust::device_malloc<float_type>(c->grp_count);
                    thrust::copy_if(c->d_columns_float[c->type_index[j]].begin(), c->d_columns_float[c->type_index[j]].begin() + (c->mRecCount), d_grp, diff, thrust::identity<bool>());
                    thrust::copy(diff, diff+c->grp_count, r->h_columns_float[r->type_index[j]].begin() + r->mRecCount);
                    thrust::device_free(diff);
                }
                else if (c->type[j] == 2) {
                    thrust::device_ptr<char> diff = thrust::device_malloc<char>(c->grp_count*c->char_size[c->type_index[j]]);				
					str_copy_if(c->d_columns_char[c->type_index[j]], c->mRecCount, (char*)thrust::raw_pointer_cast(diff), d_grp, c->char_size[c->type_index[j]]);						
					hipMemcpy((void*)&(r->h_columns_char[r->type_index[j]][r->mRecCount]), 
         			           (void*)thrust::raw_pointer_cast(diff), c->grp_count*c->char_size[c->type_index[j]], hipMemcpyDeviceToHost);

                    thrust::device_free(diff);

                }
            }
            else if (c->grp_type[j] == 2 || c->grp_type[j] == 1) {  // sum and avg

                if (c->type[j] == 0) {
                    thrust::reduce_by_key(d_grp, d_grp+(c->mRecCount), c->d_columns_int[c->type_index[j]].begin(),
                                          thrust::make_discard_iterator(), c->d_columns_int[c->type_index[j]].begin(),
                                          binary_pred_l,thrust::plus<int_type>());
                    thrust::copy(c->d_columns_int[c->type_index[j]].begin(), c->d_columns_int[c->type_index[j]].begin() + c->grp_count, r->h_columns_int[r->type_index[j]].begin() + r->mRecCount);
                }
                else if (c->type[j] == 1) {
                    thrust::reduce_by_key(d_grp, d_grp+(c->mRecCount), c->d_columns_float[c->type_index[j]].begin(),
                                          thrust::make_discard_iterator(), c->d_columns_float[c->type_index[j]].begin(),
                                          binary_pred_l,thrust::plus<float_type>());
                    thrust::copy(c->d_columns_float[c->type_index[j]].begin(), c->d_columns_float[c->type_index[j]].begin() + c->grp_count, r->h_columns_float[r->type_index[j]].begin() + r->mRecCount);
                }
            }
            else if (c->grp_type[j] == 0) {  // count
                thrust::reduce_by_key(d_grp, d_grp+(c->mRecCount), c->d_columns_int[c->type_index[j]].begin(),
                                      thrust::make_discard_iterator(), c->d_columns_int[c->type_index[j]].begin(),
                                      binary_pred_l,thrust::plus<int_type>());
                thrust::copy(c->d_columns_int[c->type_index[j]].begin(), c->d_columns_int[c->type_index[j]].begin() + c->grp_count, r->h_columns_int[r->type_index[j]].begin() + r->mRecCount);
            }
            else if(c->grp_type[j] == 4) {  // min
                if (c->type[j] == 0) {
                    thrust::reduce_by_key(d_grp, d_grp+(c->mRecCount), c->d_columns_int[c->type_index[j]].begin(),
                                          thrust::make_discard_iterator(), c->d_columns_int[c->type_index[j]].begin(),
                                          binary_pred_l,thrust::minimum<int_type>());
                    thrust::copy(c->d_columns_int[c->type_index[j]].begin(), c->d_columns_int[c->type_index[j]].begin() + c->grp_count, r->h_columns_int[r->type_index[j]].begin() + r->mRecCount);
                }
                else if (c->type[j] == 1) {
                    thrust::reduce_by_key(d_grp, d_grp+(c->mRecCount), c->d_columns_float[c->type_index[j]].begin(),
                                          thrust::make_discard_iterator(), c->d_columns_float[c->type_index[j]].begin(),
                                          binary_pred_l,thrust::minimum<float_type>());
                    thrust::copy(c->d_columns_float[c->type_index[j]].begin(), c->d_columns_float[c->type_index[j]].begin() + c->grp_count, r->h_columns_float[r->type_index[j]].begin() + r->mRecCount);
                };
            }
            else if(c->grp_type[j] == 5) {  // max
                if (c->type[j] == 0) {
                    thrust::reduce_by_key(d_grp, d_grp+(c->mRecCount), c->d_columns_int[c->type_index[j]].begin(),
                                          thrust::make_discard_iterator(), c->d_columns_int[c->type_index[j]].begin(),
                                          binary_pred_l,thrust::maximum<int_type>());
                    thrust::copy(c->d_columns_int[c->type_index[j]].begin(), c->d_columns_int[c->type_index[j]].begin() + c->grp_count, r->h_columns_int[r->type_index[j]].begin() + r->mRecCount);
                }
                else if (c->type[j] == 1) {
                    thrust::reduce_by_key(d_grp, d_grp+(c->mRecCount), c->d_columns_float[c->type_index[j]].begin(),
                                          thrust::make_discard_iterator(), c->d_columns_float[c->type_index[j]].begin(),
                                          binary_pred_l,thrust::maximum<float_type>());
                    thrust::copy(c->d_columns_float[c->type_index[j]].begin(), c->d_columns_float[c->type_index[j]].begin() + c->grp_count, r->h_columns_float[r->type_index[j]].begin() + r->mRecCount);
                };
            };
            c->deAllocColumnOnDevice(j);
        };

        if (avg_index != -1) {
            thrust::device_ptr<float_type> count_d = thrust::device_malloc<float_type>(c->grp_count);

            r->allocColumnOnDevice(countIndex,c->grp_count);
            r->CopyColumnToGpu(countIndex, r->mRecCount, c->grp_count);
            thrust::transform(r->d_columns_int[r->type_index[countIndex]].begin(), r->d_columns_int[r->type_index[countIndex]].begin() + c->grp_count, count_d, long_to_float_type());
            r->deAllocColumnOnDevice(countIndex);

            for(unsigned int k = 0; k < c->mColumnCount; k++)	{
                if(c->grp_type[k] == 1) {   // AVG

                    r->allocColumnOnDevice(k, c->grp_count);
                    r->CopyColumnToGpu(k, r->mRecCount, c->grp_count);                    

                    if (c->type[k] == 0 ) { // int

                        //create a float column k						

                        r->h_columns_float.push_back(thrust::host_vector<float_type>(c->grp_count));
                        r->d_columns_float.push_back(thrust::device_vector<float_type>(c->grp_count));
                        unsigned int idx = r->h_columns_float.size()-1;

                        thrust::transform(r->d_columns_int[r->type_index[k]].begin(), r->d_columns_int[r->type_index[k]].begin() + c->grp_count, count_d,
                                          r->d_columns_float[idx].begin(), div_long_to_float_type());
                        r->type[k] = 1;
                        //dealloc k on device and host
                        r->d_columns_int[r->type_index[k]].resize(0);
                        r->d_columns_int[r->type_index[k]].shrink_to_fit();
                        r->h_columns_int[r->type_index[k]].resize(0);
                        r->h_columns_int[r->type_index[k]].shrink_to_fit();
                        r->type_index[k] = idx;
                    }
                    else               // float
                        thrust::transform(r->d_columns_float[r->type_index[k]].begin(), r->d_columns_float[r->type_index[k]].begin() + c->grp_count, count_d, r->d_columns_float[r->type_index[k]].begin(), thrust::divides<float_type>());
                    r->CopyColumnToHost(k, r->mRecCount, c->grp_count);
                    r->deAllocColumnOnDevice(k);
                };
            };
            thrust::device_free(count_d);
        };
        r->mRecCount = r->mRecCount + c->grp_count;
    };

    r->segCount = 1;
    r->maxRecs = r->mRecCount;
    return r;

};



