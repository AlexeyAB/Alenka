#include "hip/hip_runtime.h"
/*
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */


#include "cm.h"

unsigned int findPartitionColumn(CudaSet* c, stack<string> op_v2);
head_flag_predicate<bool> binary_pred_l;


void add(CudaSet* c, CudaSet* b, queue<string> op_v3, stack<string> op_v2)
{
    if (c->columnNames.empty()) {
        // create d_columns and h_columns

        map<string,int>::iterator it;
        map<int,string> columnNames1;
        for ( it=b->columnNames.begin() ; it != b->columnNames.end(); ++it ) {
            c->columnNames[(*it).first] = (*it).second;
            columnNames1[(*it).second] = (*it).first;
        };

        string f_name;
        c->grp_type = new unsigned int[c->mColumnCount];

        for(unsigned int i=0; i < b->mColumnCount; i++) {
            c->cols[i] = b->cols[i];
            c->type[i] = b->type[i];
            c->grp_type[i] = b->grp_type[i];

            if (b->type[i] == 0)
                hipHostMalloc(&(c->h_columns[i]), c->mRecCount*int_size);
            else if (b->type[i] == 1)
                hipHostMalloc(&(c->h_columns[i]), c->mRecCount*float_size);
            else
                c->h_columns[i] = new CudaChar(((CudaChar*) (b->h_columns)[i])->mColumnCount, c->mRecCount);
        };

    }
    // append b to c
    for(unsigned int i=0; i < b->mColumnCount; i++) {
        if (b->type[i] == 0 )
            hipMemcpy((int_type*)c->h_columns[i]+(c->mRecCount-b->mRecCount), b->d_columns[i], b->mRecCount*int_size, hipMemcpyDeviceToHost);
        else if (b->type[i] == 1 )
            hipMemcpy((float_type*)c->h_columns[i]+(c->mRecCount-b->mRecCount), b->d_columns[i], b->mRecCount*float_size, hipMemcpyDeviceToHost);
        else { //CudaChar
            CudaChar *s = (CudaChar*)(b->h_columns)[i];
            CudaChar *d = (CudaChar*)c->h_columns[i];
            for(unsigned int j=0; j < s->mColumnCount; j++)
                hipMemcpy((char*)d->h_columns[j]+(c->mRecCount-b->mRecCount), s->d_columns[j], b->mRecCount, hipMemcpyDeviceToHost);
        };
    };
}


void order_inplace(CudaSet* a, stack<string> exe_type)
{

     // initialize permutation to [0, 1, 2, ... ,N-1]
    thrust::device_ptr<unsigned int> permutation = thrust::device_malloc<unsigned int>(a->mRecCount);
    thrust::sequence(permutation, permutation+(a->mRecCount));
	
    void* temp;
    CUDA_SAFE_CALL(hipMalloc((void **) &temp, a->mRecCount*float_size));
    unsigned int* raw_ptr = thrust::raw_pointer_cast(permutation);	

	
    for(int i=0; !exe_type.empty(); ++i, exe_type.pop()) {
	
	         
		if ((a->columnNames).find(exe_type.top()) ==  a->columnNames.end()) {
		    cout << "Sort couldn't find field " << exe_type.top() << endl;
			exit(1);		
		};
		   
        int colInd = (a->columnNames).find(exe_type.top())->second;


        if(a->d_columns[colInd] == 0 && a->type[colInd] < 2) {
            a->allocColumnOnDevice(colInd,a->mRecCount);
            a->CopyColumnToGpu(colInd, 0, a->mRecCount);
        };
		
	
        if ((a->type)[colInd] == 0) 
            update_permutation((int_type*)(a->d_columns)[colInd], raw_ptr, a->mRecCount, "ASC", (int_type*)temp);
        else if ((a->type)[colInd] == 1) 
            update_permutation((float_type*)(a->d_columns)[colInd], raw_ptr, a->mRecCount,"ASC", (float_type*)temp);
        else {
            CudaChar* c = (CudaChar*)(a->h_columns)[colInd];
            thrust::device_ptr<char> tmp = thrust::device_malloc<char>(a->mRecCount);

            unsigned int curr_pos;
            for(int j=(c->mColumnCount)-1; j>=0 ; j--) {
                hipMalloc((void **) &c->d_columns[j], a->mRecCount);
                curr_pos = 0;
                for(int k = 0; k < a->m_position.size(); k++) {
				    
                    hipMemcpy((void *) (c->d_columns[j] + curr_pos), (void *) (c->h_columns[j] + (a->m_position[a->m_current])[k]), (a->m_size[a->m_current])[k], hipMemcpyHostToDevice);
                    curr_pos = curr_pos + (a->m_size[a->m_current])[k];
                };
                update_permutation_char((c->d_columns)[j], raw_ptr, a->mRecCount, thrust::raw_pointer_cast(tmp), "ASC");
                hipFree(c->d_columns[j]);
                c->d_columns[j] =0;
            };
            thrust::device_free(tmp);
        };
        a->deAllocColumnOnDevice(colInd);
    };
	
	
    for(int i=0; i<(a->mColumnCount); ++i) {
        if(a->d_columns[i] == 0 && a->type[i] < 2) {
            a->allocColumnOnDevice(i,a->mRecCount);
            a->CopyColumnToGpu(i, 0, a->mRecCount);
        };
	
        if (a->type[i] == 0) 
            apply_permutation((int_type*)(a->d_columns)[i], raw_ptr, a->mRecCount, (int_type*)temp);
        else if (a->type[i] == 1)
            apply_permutation((float_type*)(a->d_columns)[i], raw_ptr, a->mRecCount, (float_type*)temp);
        else {
            CudaChar* c = (CudaChar*)(a->h_columns)[i];
            //thrust::device_ptr<char> tmp = thrust::device_malloc<char>(a->mRecCount);

            unsigned int curr_pos;
            for(int j=(c->mColumnCount)-1; j>=0 ; j--) {
                hipMalloc((void **) &c->d_columns[j], a->mRecCount);
                curr_pos = 0;
                for(int k = 0; k < a->m_position.size(); k++) {
                    hipMemcpy((void *) (c->d_columns[j] + curr_pos), (void *) (c->h_columns[j] + (a->m_position[a->m_current])[k]), (a->m_size[a->m_current])[k], hipMemcpyHostToDevice);
                    curr_pos = curr_pos + (a->m_size[a->m_current])[k];
                };

                apply_permutation_char((c->d_columns)[j], raw_ptr, a->mRecCount, (char*)temp);
                curr_pos = 0;
                for(int k = 0; k < a->m_position.size(); k++) {
                    hipMemcpy((void *) (c->h_columns[j] + (a->m_position[a->m_current])[k]), (void *) (c->d_columns[j] + curr_pos), (a->m_size[a->m_current])[k], hipMemcpyDeviceToHost);
                    curr_pos = curr_pos + (a->m_size[a->m_current])[k];
                };

                hipFree(c->d_columns[j]);
                c->d_columns[j] =0;
            };
            //thrust::device_free(tmp);
        };
        if (a->type[i] != 2)
            a->CopyColumnToHost(i);
        a->deAllocColumnOnDevice(i);
    };
    thrust::device_free(permutation);
	hipFree(temp);


}



CudaSet* merge(CudaSet* c, queue<string> op_v3, stack<string> op_v2)
{
    int countIndex;
    int avg_index = -1;
    int pieces = 1;

    for(int i = 0; i < c->mColumnCount; i++) {
        if(c->grp_type[i] == 0) // COUNT
            countIndex = i;
        else if(c->grp_type[i] == 1) // AVG
            avg_index = i;
    };

    unsigned int oldRecCount = c->mRecCount;




    // check if c set needs partitioning

    // c->mRecCount*int_size*4 > getFreeMem()

    if(c->mRecCount*int_size*4 > getFreeMem()) {
        unsigned int col = findPartitionColumn(c, op_v2);
        pieces = c->partitionCudaSet(col);
    }
    else {
        c->m_position.push_back(vector <unsigned int>());
        c->m_size.push_back(vector <unsigned int>());
        c->m_position[0].push_back(0);
        c->m_size[0].push_back(c->mRecCount);
    };


//pieces = c->partitionCudaSet(0);

    CudaSet *r = c->copyStruct(c->mRecCount);
    r->mRecCount = 0;


    for(int i = 0; i < c->m_position.size(); i++) {
        c->mRecCount = 0;
        c->m_current = i;

        for(int j = 0; j < pieces; j++)
            c->mRecCount = c->mRecCount + (c->m_size[i])[j];


        if (c->mRecCount != 0) {
            order_inplace(c,op_v2);
            c->GroupBy(op_v3);

            thrust::device_ptr<bool> d_grp(c->grp);
            //thrust::device_ptr<bool> d_di(c->di);


            for(unsigned int j=0; j < c->mColumnCount; j++) {
                c->allocColumnOnDevice(j, c->mRecCount);
                c->CopyColumnToGpu(j, 0, c->mRecCount);

                if (c->grp_type[j] == 3) {	      	  	  // non-grouped columns
                    if (c->type[j] == 0) {
                        thrust::device_ptr<int_type> ss((int_type*)(c->d_columns)[j]);
                        thrust::device_ptr<int_type> diff = thrust::device_malloc<int_type>(c->grp_count);
                        thrust::copy_if(ss,ss+(c->mRecCount), d_grp, diff, nz<bool>());
                        hipMemcpy((void*)((int_type*)r->h_columns[j] + r->mRecCount), (void*)thrust::raw_pointer_cast(diff) , c->grp_count*int_size, hipMemcpyDeviceToHost);
						//CopyColumnToHost(j, 0, c->grp_count);
                        thrust::device_free(diff);
                    }
                    else if (c->type[j] == 1) {
                        thrust::device_ptr<float_type> ss((float_type*)(c->d_columns)[j]);
                        thrust::device_ptr<float_type> diff = thrust::device_malloc<float_type>(c->grp_count);
                        thrust::copy_if(ss,ss+(c->mRecCount), d_grp, diff, nz<bool>());
                        hipMemcpy((void*)((float_type*)r->h_columns[j] + r->mRecCount), (void*)thrust::raw_pointer_cast(diff) , c->grp_count*float_size, hipMemcpyDeviceToHost);
						//CopyColumnToHost(j, 0, c->grp_count);
                        thrust::device_free(diff);
                    }
                    else if (c->type[j] == 2) {
                        CudaChar *cc = (CudaChar*)(c->h_columns)[j];
                        CudaChar *rr = (CudaChar*)(r->h_columns)[j];
                        thrust::device_ptr<char> diff = thrust::device_malloc<char>(c->grp_count);

                        for(unsigned int k=0; k < (cc->mColumnCount); k++) {
                            thrust::device_ptr<char> sr((cc->d_columns)[k]);
                            thrust::copy_if(sr,sr+(c->mRecCount), d_grp, diff, nz<bool>());
                            hipMemcpy((void*)(rr->h_columns[k] + r->mRecCount), (void*)thrust::raw_pointer_cast(diff) , c->grp_count, hipMemcpyDeviceToHost);
                        };
                        thrust::device_free(diff);
                    }
                }
                else if (c->grp_type[j] == 2 || c->grp_type[j] == 1) {  // sum and avg

                    if (c->type[j] == 0) {
                        thrust::device_ptr<int_type> ss((int_type*)(c->d_columns)[j]);
                        thrust::reduce_by_key(d_grp, d_grp+(c->mRecCount), ss,
                                              thrust::make_discard_iterator(), ss,
                                              binary_pred_l,thrust::plus<int_type>());
                        hipMemcpy((void*)((int_type*)r->h_columns[j] + r->mRecCount), (void*)thrust::raw_pointer_cast(ss) , c->grp_count*int_size, hipMemcpyDeviceToHost);
                    }
                    else if (c->type[j] == 1) {
                        thrust::device_ptr<float_type> ss((float_type*)(c->d_columns)[j]);
                        thrust::reduce_by_key(d_grp, d_grp+(c->mRecCount), ss,
                                              thrust::make_discard_iterator(), ss,
                                              binary_pred_l,thrust::plus<float_type>());
                        hipMemcpy((void*)((float_type*)r->h_columns[j] + r->mRecCount), (void*)thrust::raw_pointer_cast(ss) , c->grp_count*float_size, hipMemcpyDeviceToHost);
                    }
                }
                else if (c->grp_type[j] == 0) {  // count
                    thrust::device_ptr<int_type> ss((int_type*)(c->d_columns)[j]);
                    thrust::reduce_by_key(d_grp, d_grp+(c->mRecCount), ss,
                                          thrust::make_discard_iterator(), ss,
                                          binary_pred_l,thrust::plus<int_type>());
                    hipMemcpy((void*)((int_type*)r->h_columns[j] + r->mRecCount), (void*)thrust::raw_pointer_cast(ss) , c->grp_count*int_size, hipMemcpyDeviceToHost);
                }
                else if(c->grp_type[j] == 4) {  // min
                    if (c->type[j] == 0) {
                        thrust::device_ptr<int_type> ss((int_type*)(c->d_columns)[j]);
                        thrust::reduce_by_key(d_grp, d_grp+(c->mRecCount), ss,
                                              thrust::make_discard_iterator(), ss,
                                              binary_pred_l,thrust::minimum<int_type>());
                        hipMemcpy((void*)((int_type*)r->h_columns[j] + r->mRecCount), (void*)thrust::raw_pointer_cast(ss) , c->grp_count*int_size, hipMemcpyDeviceToHost);
                    }
                    else if (c->type[j] == 1) {
                        thrust::device_ptr<float_type> ss((float_type*)(c->d_columns)[j]);
                        thrust::reduce_by_key(d_grp, d_grp+(c->mRecCount), ss,
                                              thrust::make_discard_iterator(), ss,
                                              binary_pred_l,thrust::minimum<float_type>());
                        hipMemcpy((void*)((float_type*)r->h_columns[j] + r->mRecCount), (void*)thrust::raw_pointer_cast(ss) , c->grp_count*float_size, hipMemcpyDeviceToHost);
                    };
                }
                else if(c->grp_type[j] == 5) {  // max
                    if (c->type[j] == 0) {
                        thrust::device_ptr<int_type> ss((int_type*)(c->d_columns)[j]);
                        thrust::reduce_by_key(d_grp, d_grp+(c->mRecCount), ss,
                                              thrust::make_discard_iterator(), ss,
                                              binary_pred_l,thrust::maximum<int_type>());
                        hipMemcpy((void*)((int_type*)r->h_columns[j] + r->mRecCount), (void*)thrust::raw_pointer_cast(ss) , c->grp_count*int_size, hipMemcpyDeviceToHost);
                    }
                    else if (c->type[j] == 1) {
                        thrust::device_ptr<float_type> ss((float_type*)(c->d_columns)[j]);
                        thrust::reduce_by_key(d_grp, d_grp+(c->mRecCount), ss,
                                              thrust::make_discard_iterator(), ss,
                                              binary_pred_l,thrust::maximum<float_type>());
                        hipMemcpy((void*)((float_type*)r->h_columns[j] + r->mRecCount), (void*)thrust::raw_pointer_cast(ss) , c->grp_count*float_size, hipMemcpyDeviceToHost);
                    };
                };
                c->deAllocColumnOnDevice(j);
            };


            if (avg_index != -1) {
                thrust::device_ptr<float_type> count_d = thrust::device_malloc<float_type>(c->grp_count);

                r->allocColumnOnDevice(countIndex,c->grp_count);
                r->CopyColumnToGpu(countIndex, r->mRecCount, c->grp_count);
                thrust::device_ptr<int_type> src((int_type*)(r->d_columns)[countIndex]);

                thrust::transform(src, src + c->grp_count, count_d, long_to_float_type());
                r->deAllocColumnOnDevice(countIndex);

                for(int k = 0; k < c->mColumnCount; k++)	{
                    if(c->grp_type[k] == 1) {   // AVG

                        r->allocColumnOnDevice(k, c->grp_count);
                        r->CopyColumnToGpu(k, r->mRecCount, c->grp_count);

                        if (c->type[k] == 0 ) { // int
                            // convert int field and count to float
                            thrust::device_ptr<int_type> s((int_type*)(r->d_columns)[k]);
                            thrust::device_ptr<float_type> d = thrust::device_malloc<float_type>(c->grp_count);
                            thrust::transform(s, s + c->grp_count, count_d, d, div_long_to_float_type());
                            hipHostFree(r->h_columns[k]);
                            r->type[k] = 1;
                            hipHostMalloc(&(r->h_columns[k]), c->grp_count*float_size);
                            r->d_columns[k] = thrust::raw_pointer_cast(d);
                        }
                        else  {             // float
                            thrust::device_ptr<float_type> s((float_type*)(r->d_columns)[k]);
                            thrust::transform(s, s + c->grp_count, count_d, s, thrust::divides<float_type>());
                        }
                        r->CopyColumnToHost(k, r->mRecCount, c->grp_count);
                        r->deAllocColumnOnDevice(k);
                    };
                };
                thrust::device_free(count_d);
            };
            r->mRecCount = r->mRecCount + c->grp_count;
        };
    };

    c->m_position.clear();
    c->m_size.clear();
    c->mRecCount = oldRecCount;
	r->segCount = 1;
	r->maxRecs = r->mRecCount;
    return r;

};



unsigned int findPartitionColumn(CudaSet* c, stack<string> op_v2)
{
    int colInd;
    unsigned int selectedColumn = 1000000;
    unsigned int maxDiff = 0, diffCurrent;

    if(c->mRecCount < 11) {
        cout << "FindPartitionColumn error : set is too small - there is no need for partitioning " << endl;
        exit(0);
    };


    while(!op_v2.empty()) {
        colInd = c->columnNames.find(op_v2.top())->second;
        diffCurrent = 0;
        for(int i =0; i < 10; i++)
            if(((int_type*)c->h_columns[colInd])[i] != ((int_type*)c->h_columns[colInd])[i+1])
                diffCurrent++;
        if(diffCurrent > maxDiff) {
            maxDiff = diffCurrent;
            selectedColumn = colInd;
        };
        op_v2.pop();
    };
    return selectedColumn;

}
