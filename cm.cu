#include "hip/hip_runtime.h"
/*
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <cctype>
#include <functional>
#include <numeric>
#include "cm.h"
#include "atof.h"
#include "compress.cu"


#ifdef _WIN64
#define atoll(S) _atoi64(S)
#endif


using namespace std;
using namespace thrust::placeholders;

unsigned long long int total_count = 0;
unsigned int total_segments = 0;
unsigned int total_max;
unsigned int process_count;
map <unsigned int, unsigned int> str_offset;
long long int totalRecs = 0;
bool fact_file_loaded = 0;
char map_check;
void* d_v = NULL;
void* s_v = NULL;
unsigned int oldCount;
queue<string> op_type;
queue<string> op_value;
queue<int_type> op_nums;
queue<float_type> op_nums_f;
queue<string> col_aliases;

void* alloced_tmp;
unsigned int alloced_sz = 0;
bool alloced_switch = 0;

map<string,CudaSet*> varNames; //  STL map to manage CudaSet variables
map<string,string> setMap; //map to keep track of column names and set names


struct is_match
{
    __host__ __device__
    bool operator()(unsigned int x)
    {
        return x != 4294967295;
    }
};



struct f_equal_to
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return (((x-y) < EPSILON) && ((x-y) > -EPSILON));
    }
};


struct f_less
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return ((y-x) > EPSILON);
    }
};

struct f_greater
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return ((x-y) > EPSILON);
    }
};

struct f_greater_equal_to
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return (((x-y) > EPSILON) || (((x-y) < EPSILON) && ((x-y) > -EPSILON)));
    }
};

struct f_less_equal
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return (((y-x) > EPSILON) || (((x-y) < EPSILON) && ((x-y) > -EPSILON)));
    }
};

struct f_not_equal_to
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return !(((x-y) < EPSILON) && ((x-y) > -EPSILON));
    }
};


struct long_to_float_type
{
    __host__ __device__
    float_type operator()(const int_type x)
    {
        return (float_type)x;
    }
};


struct l_to_ui
{
    __host__ __device__
    float_type operator()(const int_type x)
    {
        return (unsigned int)x;
    }
};

struct float_to_decimal
{
    __host__ __device__
    float_type operator()(const float_type x)
    {
        return (int_type)(x*100);
    }
};


struct to_zero
{
    __host__ __device__
    bool operator()(const int_type x)
    {
        if(x == -1)
            return 0;
        else
            return 1;
    }
};



struct div_long_to_float_type
{
    __host__ __device__
    float_type operator()(const int_type x, const float_type y)
    {
        return (float_type)x/y;
    }
};


struct long_to_float
{
    __host__ __device__
    float_type operator()(const long long int x)
    {
        return (((float_type)x)/100.0);
    }
};


// trim from start
static inline std::string &ltrim(std::string &s) {
    s.erase(s.begin(), std::find_if(s.begin(), s.end(), std::not1(std::ptr_fun<int, int>(std::isspace))));
    return s;
}

// trim from end
static inline std::string &rtrim(std::string &s) {
    s.erase(std::find_if(s.rbegin(), s.rend(), std::not1(std::ptr_fun<int, int>(std::isspace))).base(), s.end());
    return s;
}

// trim from both ends
static inline std::string &trim(std::string &s) {
    return ltrim(rtrim(s));
}


void allocColumns(CudaSet* a, queue<string> fields);
void copyColumns(CudaSet* a, queue<string> fields, unsigned int segment, unsigned int& count);
void mygather(unsigned int tindex, unsigned int idx, CudaSet* a, CudaSet* t, unsigned int count, unsigned int g_size);
void mycopy(unsigned int tindex, unsigned int idx, CudaSet* a, CudaSet* t, unsigned int count, unsigned int g_size);
void write_compressed_char(string file_name, unsigned int index, unsigned int mCount);
unsigned int largest_prm(CudaSet* a);
unsigned int max_tmp(CudaSet* a);


unsigned int curr_segment = 10000000;

size_t getFreeMem();
char zone_map_check(queue<string> op_type, queue<string> op_value, queue<int_type> op_nums,queue<float_type> op_nums_f, CudaSet* a, unsigned int segment);




CudaSet::CudaSet(queue<string> &nameRef, queue<string> &typeRef, queue<int> &sizeRef, queue<int> &colsRef, int_type Recs)
    : mColumnCount(0), mRecCount(0)
{
    initialize(nameRef, typeRef, sizeRef, colsRef, Recs);
    keep = false;
    partial_load = 0;
    source = 1;
    text_source = 1;
    grp = NULL;
};

CudaSet::CudaSet(queue<string> &nameRef, queue<string> &typeRef, queue<int> &sizeRef, queue<int> &colsRef, int_type Recs, char* file_name)
    : mColumnCount(0),  mRecCount(0)
{
    initialize(nameRef, typeRef, sizeRef, colsRef, Recs, file_name);
    keep = false;
    partial_load = 1;
    source = 1;
    text_source = 0;
    grp = NULL;
};

CudaSet::CudaSet(unsigned int RecordCount, unsigned int ColumnCount)
{
    initialize(RecordCount, ColumnCount);
    keep = false;
    partial_load = 0;
    source = 0;
    text_source = 0;
    grp = NULL;
};

CudaSet::CudaSet(CudaSet* a, CudaSet* b, int_type Recs, queue<string> op_sel, queue<string> op_sel_as)
{
    initialize(a,b,Recs, op_sel, op_sel_as);
    keep = false;
    partial_load = 0;
    source = 0;
    text_source = 0;
    grp = NULL;
};


CudaSet::~CudaSet()
{
    free();
};


void CudaSet::allocColumnOnDevice(unsigned int colIndex, unsigned int RecordCount)
{
    if (type[colIndex] == 0) {
        d_columns_int[type_index[colIndex]].resize(RecordCount);
    }
    else if (type[colIndex] == 1)
        d_columns_float[type_index[colIndex]].resize(RecordCount);
    else {
        void* d;
        hipMalloc(&d, char_size[type_index[colIndex]]*RecordCount);
        d_columns_char[type_index[colIndex]] = (char*)d;
    };
};


void CudaSet::decompress_char_hash(unsigned int colIndex, unsigned int segment, unsigned int i_cnt)
{

    unsigned int bits_encoded, fit_count, sz, vals_count, real_count, old_count;
    const unsigned int len = char_size[type_index[colIndex]];

    char f1[100];
    strcpy(f1, load_file_name);
    strcat(f1,".");
    char col_pos[3];
    itoaa(cols[colIndex],col_pos);
    strcat(f1,col_pos);

    strcat(f1,".");
    itoaa(segment,col_pos);
    strcat(f1,col_pos);
    FILE* f;
    f = fopen (f1 , "rb" );
    fread(&sz, 4, 1, f);
    char* d_array = new char[sz*len];
    fread((void*)d_array, sz*len, 1, f);

    unsigned long long int* hashes  = new unsigned long long int[sz];

    for(unsigned int i = 0; i < sz ; i++) {
        hashes[i] = MurmurHash64A(&d_array[i*len], len, hash_seed); // divide by 2 so it will fit into a signed long long
    };

    void* d;
    hipMalloc((void **) &d, sz*int_size);
    hipMemcpy( d, (void *) hashes, sz*8, hipMemcpyHostToDevice);

    thrust::device_ptr<unsigned long long int> dd_int((unsigned long long int*)d);

    delete[] d_array;
    delete[] hashes;

    fread(&fit_count, 4, 1, f);
    fread(&bits_encoded, 4, 1, f);
    fread(&vals_count, 4, 1, f);
    fread(&real_count, 4, 1, f);

    unsigned long long int* int_array = new unsigned long long int[vals_count];
    fread((void*)int_array, 1, vals_count*8, f);
    fclose(f);

    void* d_val;
    hipMalloc((void **) &d_val, vals_count*8);
    hipMemcpy(d_val, (void *) int_array, vals_count*8, hipMemcpyHostToDevice);

    thrust::device_ptr<unsigned long long int> mval((unsigned long long int*)d_val);


    delete[] int_array;

    void* d_int;
    hipMalloc((void **) &d_int, real_count*4);

    // convert bits to ints and then do gather

    void* d_v;
    hipMalloc((void **) &d_v, 8);
    thrust::device_ptr<unsigned int> dd_v((unsigned int*)d_v);
    dd_v[1] = fit_count;
    dd_v[0] = bits_encoded;

    thrust::counting_iterator<unsigned int> begin(0);
    decompress_functor_str ff((unsigned long long int*)d_val,(unsigned int*)d_int, (unsigned int*)d_v);
    thrust::for_each(begin, begin + real_count, ff);

    //thrust::device_ptr<long long int> dd_int((long long int*)d);
    thrust::device_ptr<unsigned int> dd_val((unsigned int*)d_int);

    if(!prm.empty()) {
        if(prm_index[segment] == 'R') {
            thrust::device_ptr<int_type> d_tmp = thrust::device_malloc<int_type>(real_count);
            thrust::gather(dd_val, dd_val + real_count, dd_int, d_tmp);

            if(prm_d.size() == 0) // find the largest prm segment
                prm_d.resize(largest_prm(this));
            hipMemcpy((void**)(thrust::raw_pointer_cast(prm_d.data())), (void**)prm[segment],
                       4*prm_count[segment], hipMemcpyHostToDevice);

            old_count = d_columns_int[i_cnt].size();
            d_columns_int[i_cnt].resize(old_count + prm_count[segment]);
            thrust::gather(prm_d.begin(), prm_d.begin() + prm_count[segment], d_tmp, d_columns_int[i_cnt].begin() + old_count);
            thrust::device_free(d_tmp);

        }
        else if(prm_index[segment] == 'A') {
            old_count = d_columns_int[i_cnt].size();
            d_columns_int[i_cnt].resize(old_count + real_count);
            thrust::gather(dd_val, dd_val + real_count, dd_int, d_columns_int[i_cnt].begin() + old_count);
        }
    }
    else {

        old_count = d_columns_int[i_cnt].size();
        d_columns_int[i_cnt].resize(old_count + real_count);
        thrust::gather(dd_val, dd_val + real_count, dd_int, d_columns_int[i_cnt].begin() + old_count);

    };

    hipFree(d);
    hipFree(d_val);
    hipFree(d_v);
    hipFree(d_int);
};




// takes a char column , hashes strings, copies them to a gpu
void CudaSet::add_hashed_strings(string field, unsigned int segment, unsigned int i_cnt)
{
    unsigned int colInd2 = columnNames.find(field)->second;
    CudaSet *t = varNames[setMap[field]];

    if(not_compressed) { // decompressed strings on a host

        unsigned int old_count;
        unsigned long long int* hashes  = new unsigned long long int[t->mRecCount];

        for(unsigned int i = 0; i < t->mRecCount ; i++)
            hashes[i] = MurmurHash64A(t->h_columns_char[t->type_index[colInd2]] + i*t->char_size[t->type_index[colInd2]], t->char_size[t->type_index[colInd2]], hash_seed);

        if(!prm.empty()) {
            if(prm_index[segment] == 'R') {

                thrust::device_ptr<unsigned long long int> d_tmp = thrust::device_malloc<unsigned long long int>(t->mRecCount);
                thrust::copy(hashes, hashes+mRecCount, d_tmp);

                if(prm_d.size() == 0) // find the largest prm segment
                    prm_d.resize(largest_prm(this));

                hipMemcpy((void**)(thrust::raw_pointer_cast(prm_d.data())), (void**)prm[segment],
                           4*prm_count[segment], hipMemcpyHostToDevice);

                old_count = d_columns_int[i_cnt].size();
                d_columns_int[i_cnt].resize(old_count + prm_count[segment]);
                thrust::gather(prm_d.begin(), prm_d.begin() + prm_count[segment], d_tmp, d_columns_int[i_cnt].begin() + old_count);
                thrust::device_free(d_tmp);

            }
            else if(prm_index[segment] == 'A') {
                old_count = d_columns_int[i_cnt].size();
                d_columns_int[i_cnt].resize(old_count + mRecCount);
                thrust::copy(hashes, hashes + mRecCount, d_columns_int[i_cnt].begin() + old_count);
            }
        }
        else {
            old_count = d_columns_int[i_cnt].size();
            d_columns_int[i_cnt].resize(old_count + mRecCount);
            thrust::copy(hashes, hashes + mRecCount, d_columns_int[i_cnt].begin() + old_count);
        }
    }
    else { // hash the dictionary
        decompress_char_hash(colInd2, segment, i_cnt);
    };
};


void CudaSet::resize(unsigned int addRecs)
{    
    mRecCount = mRecCount + addRecs;
    for(unsigned int i=0; i <mColumnCount; i++) {
        if(type[i] == 0) {
            h_columns_int[type_index[i]].resize(mRecCount);
        }
        else if(type[i] == 1) {
            h_columns_float[type_index[i]].resize(mRecCount);
        }
        else {
            if (h_columns_char[type_index[i]]) {
                if (mRecCount > prealloc_char_size) {
                    prealloc_char_size = mRecCount;
                    h_columns_char[type_index[i]] = (char*)realloc(h_columns_char[type_index[i]], (unsigned long long int)mRecCount*(unsigned long long int)char_size[type_index[i]]);
                };
            }
            else {
                h_columns_char[type_index[i]] = new char[(unsigned long long int)mRecCount*(unsigned long long int)char_size[type_index[i]]];
            };
        };

    };
};

void CudaSet::reserve(unsigned int Recs)
{

    for(unsigned int i=0; i <mColumnCount; i++) {
        if(type[i] == 0)
            h_columns_int[type_index[i]].reserve(Recs);
        else if(type[i] == 1)
            h_columns_float[type_index[i]].reserve(Recs);
        else {
		    unsigned long long int sz = (unsigned long long int)Recs*(unsigned long long int)char_size[type_index[i]];
            h_columns_char[type_index[i]] = new char[(unsigned long long int)Recs*(unsigned long long int)char_size[type_index[i]]];
			if(h_columns_char[type_index[i]] == NULL) {
			    cout << "Could not allocate on a host " << Recs << " records of size " << char_size[type_index[i]] << endl;
			    exit(0);
			};
            prealloc_char_size = Recs;
        };

    };
};


void CudaSet::deAllocColumnOnDevice(unsigned int colIndex)
{
    if (type[colIndex] == 0 && !d_columns_int.empty()) {
        d_columns_int[type_index[colIndex]].resize(0);
        d_columns_int[type_index[colIndex]].shrink_to_fit();
    }
    else if (type[colIndex] == 1 && !d_columns_float.empty()) {
        d_columns_float[type_index[colIndex]].resize(0);
        d_columns_float[type_index[colIndex]].shrink_to_fit();
    }
    else if (type[colIndex] == 2 && d_columns_char[type_index[colIndex]] != NULL) {
        hipFree(d_columns_char[type_index[colIndex]]);
        d_columns_char[type_index[colIndex]] = NULL;
    };
};

void CudaSet::allocOnDevice(unsigned int RecordCount)
{
    for(unsigned int i=0; i < mColumnCount; i++)
        allocColumnOnDevice(i, RecordCount);
};

void CudaSet::deAllocOnDevice()
{
    for(unsigned int i=0; i <mColumnCount; i++)
        deAllocColumnOnDevice(i);

    if(!columnGroups.empty() && mRecCount !=0) {
        hipFree(grp);
        grp = NULL;
    };

    if(!prm.empty()) { // free the sources
        string some_field;
        map<string,int>::iterator it=columnNames.begin();
        some_field = (*it).first;

        if(setMap[some_field].compare(name)) {
            CudaSet* t = varNames[setMap[some_field]];
            t->deAllocOnDevice();
        };
    };
};

void CudaSet::resizeDeviceColumn(unsigned int RecCount, unsigned int colIndex)
{
    if (RecCount) {
        if (type[colIndex] == 0)
            d_columns_int[type_index[colIndex]].resize(mRecCount+RecCount);
        else if (type[colIndex] == 1)
            d_columns_float[type_index[colIndex]].resize(mRecCount+RecCount);
        else {
            if (d_columns_char[type_index[colIndex]] != NULL)
                hipFree(d_columns_char[type_index[colIndex]]);
            void *d;
            hipMalloc((void **) &d, (mRecCount+RecCount)*char_size[type_index[colIndex]]);
            d_columns_char[type_index[colIndex]] = (char*)d;
        };
    };
};



void CudaSet::resizeDevice(unsigned int RecCount)
{
    if (RecCount)
        for(unsigned int i=0; i < mColumnCount; i++)
            resizeDeviceColumn(RecCount, i);
};

bool CudaSet::onDevice(unsigned int i)
{
    unsigned j = type_index[i];

    if (type[i] == 0) {
        if (d_columns_int.empty())
            return 0;
        if (d_columns_int[j].size() == 0)
            return 0;
    }
    else if (type[i] == 1) {
        if (d_columns_float.empty())
            return 0;
        if(d_columns_float[j].size() == 0)
            return 0;
    }
    else if  (type[i] == 2) {
        if(d_columns_char.empty())
            return 0;
        if(d_columns_char[j] == NULL)
            return 0;
    };
    return 1;
}



CudaSet* CudaSet::copyDeviceStruct()
{

    CudaSet* a = new CudaSet(mRecCount, mColumnCount);
    a->not_compressed = not_compressed;
    a->segCount = segCount;
    a->maxRecs = maxRecs;

    for ( map<string,int>::iterator it=columnNames.begin() ; it != columnNames.end(); ++it )
        a->columnNames[(*it).first] = (*it).second;

    for(unsigned int i=0; i < mColumnCount; i++) {
        a->cols[i] = cols[i];
        a->type[i] = type[i];

        if(a->type[i] == 0) {
            a->d_columns_int.push_back(thrust::device_vector<int_type>());
            a->h_columns_int.push_back(thrust::host_vector<int_type, uninitialized_host_allocator<int_type> >());
            a->type_index[i] = a->d_columns_int.size()-1;
        }
        else if(a->type[i] == 1) {
            a->d_columns_float.push_back(thrust::device_vector<float_type>());
            a->h_columns_float.push_back(thrust::host_vector<float_type, uninitialized_host_allocator<float_type> >());
            a->type_index[i] = a->d_columns_float.size()-1;
            a->decimal[i] = decimal[i];
        }
        else {
            a->h_columns_char.push_back(NULL);
            a->d_columns_char.push_back(NULL);
            a->type_index[i] = a->d_columns_char.size()-1;
        };
    };
    a->char_size = char_size;
    a->load_file_name = load_file_name;

    a->mRecCount = 0;
    return a;
}



unsigned long long int CudaSet::readSegmentsFromFile(unsigned int segNum, unsigned int colIndex)
{
    char f1[100];
    strcpy(f1, load_file_name);
    strcat(f1,".");
    char col_pos[3];
    itoaa(cols[colIndex],col_pos);
    strcat(f1,col_pos);
    unsigned int cnt;

    strcat(f1,".");
    itoaa(segNum,col_pos);
    strcat(f1,col_pos);

    FILE* f;

    f = fopen (f1 , "rb" );

    if(type[colIndex] == 0) {
        fread(h_columns_int[type_index[colIndex]].data(), 4, 1, f);
        cnt = ((unsigned int*)(h_columns_int[type_index[colIndex]].data()))[0];
        fread((unsigned int*)(h_columns_int[type_index[colIndex]].data()) + 1, (cnt+8)*8 - 4, 1, f);
    }
    else if(type[colIndex] == 1) {
        fread(h_columns_float[type_index[colIndex]].data(), 4, 1, f);
        cnt = ((unsigned int*)(h_columns_float[type_index[colIndex]].data()))[0];
        fread((unsigned int*)(h_columns_float[type_index[colIndex]].data()) + 1, (cnt+8)*8 - 4, 1, f);
    }
    else {
        decompress_char(f, colIndex, segNum);
    };

    fclose(f);
    return 0;
};


void CudaSet::decompress_char(FILE* f, unsigned int colIndex, unsigned int segNum)
{
    unsigned int bits_encoded, fit_count, sz, vals_count, real_count;
    const unsigned int len = char_size[type_index[colIndex]];

    fread(&sz, 4, 1, f);
    char* d_array = new char[sz*len];
    fread((void*)d_array, sz*len, 1, f);

    void* d;
    hipMalloc((void **) &d, sz*len);
    hipMemcpy( d, (void *) d_array, sz*len, hipMemcpyHostToDevice);
    delete[] d_array;

    fread(&fit_count, 4, 1, f);
    fread(&bits_encoded, 4, 1, f);
    fread(&vals_count, 4, 1, f);
    fread(&real_count, 4, 1, f);

    unsigned long long int* int_array = new unsigned long long int[vals_count];
    fread((void*)int_array, 1, vals_count*8, f);
    fclose(f);

    void* d_val;
    hipMalloc((void **) &d_val, vals_count*8);
    hipMemcpy(d_val, (void *) int_array, vals_count*8, hipMemcpyHostToDevice);
    delete[] int_array;

    void* d_int;
    hipMalloc((void **) &d_int, real_count*4);

    // convert bits to ints and then do gather

    void* d_v;
    hipMalloc((void **) &d_v, 8);
    thrust::device_ptr<unsigned int> dd_v((unsigned int*)d_v);
    dd_v[1] = fit_count;
    dd_v[0] = bits_encoded;

    thrust::counting_iterator<unsigned int> begin(0);
    decompress_functor_str ff((unsigned long long int*)d_val,(unsigned int*)d_int, (unsigned int*)d_v);
    thrust::for_each(begin, begin + real_count, ff);

    //thrust::device_ptr<unsigned int> dd_r((unsigned int*)d_int);
    //for(int z = 0 ; z < 3; z++)
    //cout << "DD " << dd_r[z] << endl;

    //void* d_char;
    //hipMalloc((void **) &d_char, real_count*len);
    //hipMemset(d_char, 0, real_count*len);
    //str_gather(d_int, real_count, d, d_char, len);
    if(str_offset.count(colIndex) == 0)
        str_offset[colIndex] = 0;
    //cout << "str off " << str_offset[colIndex] << endl;
    if(!alloced_switch)
        str_gather(d_int, real_count, d, d_columns_char[type_index[colIndex]] + str_offset[colIndex]*len, len);
    else
        str_gather(d_int, real_count, d, alloced_tmp, len);

    if(!prm.empty()) {
        str_offset[colIndex] = str_offset[colIndex] + prm_count[segNum];
    }
    else {
        str_offset[colIndex] = str_offset[colIndex] + real_count;
    };

    //if(d_columns_char[type_index[colIndex]])
    //    hipFree(d_columns_char[type_index[colIndex]]);
    //d_columns_char[type_index[colIndex]] = (char*)d_char;

    mRecCount = real_count;

    hipFree(d);
    hipFree(d_val);
    hipFree(d_v);
    hipFree(d_int);
}




void CudaSet::CopyToGpu(unsigned int offset, unsigned int count)
{
    if (not_compressed) {
        for(unsigned int i = 0; i < mColumnCount; i++) {
            switch(type[i]) {
            case 0 :
                thrust::copy(h_columns_int[type_index[i]].begin() + offset, h_columns_int[type_index[i]].begin() + offset + count, d_columns_int[type_index[i]].begin());
                break;
            case 1 :
                thrust::copy(h_columns_float[type_index[i]].begin() + offset, h_columns_float[type_index[i]].begin() + offset + count, d_columns_float[type_index[i]].begin());
                break;
            default :
                hipMemcpy(d_columns_char[type_index[i]], h_columns_char[type_index[i]], char_size[type_index[i]]*(offset + count), hipMemcpyHostToDevice);
            };
        };
    }
    else
        for(unsigned int i = 0; i < mColumnCount; i++)
            CopyColumnToGpu(i,  offset, count);
};


void CudaSet::CopyColumnToGpu(unsigned int colIndex,  unsigned int segment)
{
    if(not_compressed) {
        switch(type[colIndex]) {
        case 0 :
            if(!alloced_switch)
                thrust::copy(h_columns_int[type_index[colIndex]].begin(), h_columns_int[type_index[colIndex]].begin() + mRecCount, d_columns_int[type_index[colIndex]].begin());
            else {
                thrust::device_ptr<int_type> d_col((int_type*)alloced_tmp);
                thrust::copy(h_columns_int[type_index[colIndex]].begin(), h_columns_int[type_index[colIndex]].begin() + mRecCount, d_col);
            };
            break;
        case 1 :
            if(!alloced_switch)
                thrust::copy(h_columns_float[type_index[colIndex]].begin(), h_columns_float[type_index[colIndex]].begin() + mRecCount, d_columns_float[type_index[colIndex]].begin());
            else {
                thrust::device_ptr<float_type> d_col((float_type*)alloced_tmp);
                thrust::copy(h_columns_float[type_index[colIndex]].begin(), h_columns_float[type_index[colIndex]].begin() + mRecCount, d_col);
            };
            break;
        default :
            if(!alloced_switch)
                hipMemcpy(d_columns_char[type_index[colIndex]], h_columns_char[type_index[colIndex]], char_size[type_index[colIndex]]*mRecCount, hipMemcpyHostToDevice);
            else
                hipMemcpy(alloced_tmp, h_columns_char[type_index[colIndex]], char_size[type_index[colIndex]]*mRecCount, hipMemcpyHostToDevice);
        };
    }
    else {


        unsigned long long int data_offset;
        if (partial_load)
            data_offset = readSegmentsFromFile(segment,colIndex);

        if(type[colIndex] != 2) {
            if(d_v == NULL)
                CUDA_SAFE_CALL(hipMalloc((void **) &d_v, 12));
            if(s_v == NULL);
            CUDA_SAFE_CALL(hipMalloc((void **) &s_v, 8));
        };

        if(type[colIndex] == 0) {
            if(!alloced_switch) {
                mRecCount = pfor_decompress(thrust::raw_pointer_cast(d_columns_int[type_index[colIndex]].data()), h_columns_int[type_index[colIndex]].data() + data_offset, d_v, s_v);
            }
            else {
                mRecCount = pfor_decompress(alloced_tmp, h_columns_int[type_index[colIndex]].data() + data_offset, d_v, s_v);
            };
        }
        else if(type[colIndex] == 1) {
            if(decimal[colIndex]) {
                if(!alloced_switch) {
                    mRecCount = pfor_decompress( thrust::raw_pointer_cast(d_columns_float[type_index[colIndex]].data()) , h_columns_float[type_index[colIndex]].data() + data_offset, d_v, s_v);
                    thrust::device_ptr<long long int> d_col_int((long long int*)thrust::raw_pointer_cast(d_columns_float[type_index[colIndex]].data()));
                    thrust::transform(d_col_int,d_col_int+mRecCount,d_columns_float[type_index[colIndex]].begin(), long_to_float());
                }
                else {
                    mRecCount = pfor_decompress(alloced_tmp, h_columns_float[type_index[colIndex]].data() + data_offset, d_v, s_v);
                    thrust::device_ptr<long long int> d_col_int((long long int*)alloced_tmp);
                    thrust::device_ptr<float_type> d_col_float((float_type*)alloced_tmp);
                    thrust::transform(d_col_int,d_col_int+mRecCount, d_col_float, long_to_float());
                };
            }
            //else // uncompressed float
            //hipMemcpy( d_columns[colIndex], (void *) ((float_type*)h_columns[colIndex] + offset), count*float_size, hipMemcpyHostToDevice);
            // will have to fix it later so uncompressed data will be written by segments too
        }
    };
}



void CudaSet::CopyColumnToGpu(unsigned int colIndex) // copy all segments
{
    if(not_compressed) {
        switch(type[colIndex]) {
        case 0 :
            thrust::copy(h_columns_int[type_index[colIndex]].begin(), h_columns_int[type_index[colIndex]].begin() + mRecCount, d_columns_int[type_index[colIndex]].begin());
            break;
        case 1 :
            thrust::copy(h_columns_float[type_index[colIndex]].begin(), h_columns_float[type_index[colIndex]].begin() + mRecCount, d_columns_float[type_index[colIndex]].begin());
            break;
        default :
            hipMemcpy(d_columns_char[type_index[colIndex]], h_columns_char[type_index[colIndex]], char_size[type_index[colIndex]]*mRecCount, hipMemcpyHostToDevice);
        };
    }
    else {
        long long int data_offset;
        unsigned long long int totalRecs = 0;
        if(d_v == NULL)
            CUDA_SAFE_CALL(hipMalloc((void **) &d_v, 12));
        if(s_v == NULL);
        CUDA_SAFE_CALL(hipMalloc((void **) &s_v, 8));

        str_offset[colIndex] = 0;
        for(unsigned int i = 0; i < segCount; i++) {

            if (partial_load)
                data_offset = readSegmentsFromFile(i,colIndex);


            if(type[colIndex] == 0) {
                mRecCount = pfor_decompress(thrust::raw_pointer_cast(d_columns_int[type_index[colIndex]].data() + totalRecs), h_columns_int[type_index[colIndex]].data() + data_offset, d_v, s_v);
            }
            else if(type[colIndex] == 1) {
                if(decimal[colIndex]) {
                    mRecCount = pfor_decompress( thrust::raw_pointer_cast(d_columns_float[type_index[colIndex]].data() + totalRecs) , h_columns_float[type_index[colIndex]].data() + data_offset, d_v, s_v);
                    thrust::device_ptr<long long int> d_col_int((long long int*)thrust::raw_pointer_cast(d_columns_float[type_index[colIndex]].data() + totalRecs));
                    thrust::transform(d_col_int,d_col_int+mRecCount,d_columns_float[type_index[colIndex]].begin() + totalRecs, long_to_float());
                }
                // else  uncompressed float
                //hipMemcpy( d_columns[colIndex], (void *) ((float_type*)h_columns[colIndex] + offset), count*float_size, hipMemcpyHostToDevice);
                // will have to fix it later so uncompressed data will be written by segments too
            };

            totalRecs = totalRecs + mRecCount;
        };

        mRecCount = totalRecs;
    };
}




void CudaSet::CopyColumnToGpu(unsigned int colIndex,  unsigned int offset, unsigned int count)
{
    if(not_compressed) {
        switch(type[colIndex]) {
        case 0 :
            thrust::copy(h_columns_int[type_index[colIndex]].begin(), h_columns_int[type_index[colIndex]].begin() + offset + count, d_columns_int[type_index[colIndex]].begin());
            break;
        case 1 :
            thrust::copy(h_columns_float[type_index[colIndex]].begin(), h_columns_float[type_index[colIndex]].begin() + offset + count, d_columns_float[type_index[colIndex]].begin());
            break;
        default :
            hipMemcpy(d_columns_char[type_index[colIndex]], h_columns_char[type_index[colIndex]], char_size[type_index[colIndex]]*(offset + count), hipMemcpyHostToDevice);
        };
    }
    else {
    };
}




void CudaSet::CopyColumnToHost(int colIndex, unsigned int offset, unsigned int RecCount)
{

    switch(type[colIndex]) {
    case 0 :
        thrust::copy(d_columns_int[type_index[colIndex]].begin(), d_columns_int[type_index[colIndex]].begin() + RecCount, h_columns_int[type_index[colIndex]].begin() + offset);
        break;
    case 1 :
        thrust::copy(d_columns_float[type_index[colIndex]].begin(), d_columns_float[type_index[colIndex]].begin() + RecCount, h_columns_float[type_index[colIndex]].begin() + offset);
        break;
    default :
        hipMemcpy(h_columns_char[type_index[colIndex]] + offset*char_size[type_index[colIndex]], d_columns_char[type_index[colIndex]], char_size[type_index[colIndex]]*RecCount, hipMemcpyDeviceToHost);
    }
}



void CudaSet::CopyColumnToHost(int colIndex)
{
    CopyColumnToHost(colIndex, 0, mRecCount);
}

void CudaSet::CopyToHost(unsigned int offset, unsigned int count)
{
    for(unsigned int i = 0; i < mColumnCount; i++) {
        CopyColumnToHost(i, offset, count);
    };
}

float_type* CudaSet::get_float_type_by_name(string name)
{
    unsigned int colIndex = columnNames.find(name)->second;
    return thrust::raw_pointer_cast(d_columns_float[type_index[colIndex]].data());
}

int_type* CudaSet::get_int_by_name(string name)
{
    unsigned int colIndex = columnNames.find(name)->second;
    return thrust::raw_pointer_cast(d_columns_int[type_index[colIndex]].data());
}

float_type* CudaSet::get_host_float_by_name(string name)
{
    unsigned int colIndex = columnNames.find(name)->second;
    return thrust::raw_pointer_cast(h_columns_float[type_index[colIndex]].data());
}

int_type* CudaSet::get_host_int_by_name(string name)
{
    unsigned int colIndex = columnNames.find(name)->second;
    return thrust::raw_pointer_cast(h_columns_int[type_index[colIndex]].data());
}



void CudaSet::GroupBy(stack<string> columnRef, unsigned int int_col_count)
{
    int grpInd, colIndex;

    if(grp)
        hipFree(grp);

    CUDA_SAFE_CALL(hipMalloc((void **) &grp, mRecCount * sizeof(bool)));
    thrust::device_ptr<bool> d_grp(grp);

    thrust::sequence(d_grp, d_grp+mRecCount, 0, 0);

    thrust::device_ptr<bool> d_group = thrust::device_malloc<bool>(mRecCount);

    d_group[mRecCount-1] = 1;
    unsigned int i_count = 0;

    for(int i = 0; i < columnRef.size(); columnRef.pop()) {

        columnGroups.push(columnRef.top()); // save for future references
        colIndex = columnNames[columnRef.top()];

        if(!onDevice(colIndex)) {
            allocColumnOnDevice(colIndex,mRecCount);
            CopyColumnToGpu(colIndex,  mRecCount);
            grpInd = 1;
        }
        else
            grpInd = 0;

        if (type[colIndex] == 0) {  // int_type
            thrust::transform(d_columns_int[type_index[colIndex]].begin(), d_columns_int[type_index[colIndex]].begin() + mRecCount - 1,
                              d_columns_int[type_index[colIndex]].begin()+1, d_group, thrust::not_equal_to<int_type>());
        }
        else if (type[colIndex] == 1) {  // float_type
            thrust::transform(d_columns_float[type_index[colIndex]].begin(), d_columns_float[type_index[colIndex]].begin() + mRecCount - 1,
                              d_columns_float[type_index[colIndex]].begin()+1, d_group, f_not_equal_to());
        }
        else  {  // Char
            //str_grp(d_columns_char[type_index[colIndex]], mRecCount, d_group, char_size[type_index[colIndex]]);
            //use int_type

            thrust::transform(d_columns_int[int_col_count+i_count].begin(), d_columns_int[int_col_count+i_count].begin() + mRecCount - 1,
                              d_columns_int[int_col_count+i_count].begin()+1, d_group, thrust::not_equal_to<int_type>());
            i_count++;

        };
        thrust::transform(d_group, d_group+mRecCount, d_grp, d_grp, thrust::logical_or<bool>());

        if (grpInd == 1)
            deAllocColumnOnDevice(colIndex);
    };

    thrust::device_free(d_group);
    grp_count = thrust::count(d_grp, d_grp+mRecCount,1);
};


void CudaSet::addDeviceColumn(int_type* col, int colIndex, string colName, unsigned int recCount)
{
    if (columnNames.find(colName) == columnNames.end()) {
        columnNames[colName] = colIndex;
        type[colIndex] = 0;
        d_columns_int.push_back(thrust::device_vector<int_type>(recCount));
        h_columns_int.push_back(thrust::host_vector<int_type, uninitialized_host_allocator<int_type> >());
        type_index[colIndex] = d_columns_int.size()-1;
    }
    else {  // already exists, my need to resize it
        if(d_columns_int[type_index[colIndex]].size() < recCount) {
            d_columns_int[type_index[colIndex]].resize(recCount);
        };
    };
    // copy data to d columns
    thrust::device_ptr<int_type> d_col((int_type*)col);
    thrust::copy(d_col, d_col+recCount, d_columns_int[type_index[colIndex]].begin());
};

void CudaSet::addDeviceColumn(float_type* col, int colIndex, string colName, unsigned int recCount)
{
    if (columnNames.find(colName) == columnNames.end()) {
        columnNames[colName] = colIndex;
        type[colIndex] = 1;
        d_columns_float.push_back(thrust::device_vector<float_type>(recCount));
        h_columns_float.push_back(thrust::host_vector<float_type, uninitialized_host_allocator<float_type> >());
        type_index[colIndex] = d_columns_float.size()-1;
    }
    else {  // already exists, my need to resize it
        if(d_columns_float[type_index[colIndex]].size() < recCount)
            d_columns_float[type_index[colIndex]].resize(recCount);
    };

    thrust::device_ptr<float_type> d_col((float_type*)col);
    thrust::copy(d_col, d_col+recCount, d_columns_float[type_index[colIndex]].begin());
};


void CudaSet::writeHeader(char* file_name, unsigned int col) {

    char str[100];
    char col_pos[3];

    strcpy(str, file_name);
    strcat(str,".");
    itoaa(col,col_pos);
    strcat(str,col_pos);
    string ff = str;
    strcat(str,".header");

    fstream binary_file(str,ios::out|ios::binary|ios::app);
    binary_file.write((char *)&total_count, 8);
    binary_file.write((char *)&total_segments, 4);
    binary_file.write((char *)&total_max, 4);
    binary_file.write((char *)&cnt_counts[ff], 4);
    binary_file.close();
};

void CudaSet::Store(char* file_name, char* sep, unsigned int limit, bool binary )
{
    if (mRecCount == 0 && binary == 1) { // write tails
        for(unsigned int i = 0; i< mColumnCount; i++) {
            writeHeader(file_name, cols[i]);
        };
        return;
    };

    unsigned int mCount;

    if(limit != 0 && limit < mRecCount)
        mCount = limit;
    else
        mCount = mRecCount;

    if(binary == 0) {

        char buffer [33];
        queue<string> op_vx;
        for ( map<string,int>::iterator it=columnNames.begin() ; it != columnNames.end(); ++it )
            op_vx.push((*it).first);
        curr_segment = 1000000;
        FILE *file_pr = fopen(file_name, "w");
        if (file_pr  == NULL)
            cout << "Could not open file " << file_name << endl;

        if(prm.size() || source)
            allocColumns(this, op_vx);
        unsigned int curr_seg = 0, cnt = 0;
        unsigned curr_count, sum_printed = 0;
        while(sum_printed < mCount) {

//			    cout << "mcount " << mCount << " " << prm.size() << " " << keep << endl;
            if(prm.size() || source)  {
                copyColumns(this, op_vx, curr_seg, cnt);
                // if host arrays are empty
                unsigned int olRecs = mRecCount;
                resize(mRecCount);
                mRecCount = olRecs;
                CopyToHost(0,mRecCount);
                if(sum_printed + mRecCount <= mCount)
                    curr_count = mRecCount;
                else {
                    curr_count = mCount - sum_printed;
                };
            }
            else
                curr_count = mCount;


            sum_printed = sum_printed + mRecCount;
            string ss;

            for(unsigned int i=0; i < curr_count; i++) {
                for(unsigned int j=0; j < mColumnCount; j++) {
                    if (type[j] == 0) {
                        sprintf(buffer, "%lld", (h_columns_int[type_index[j]])[i] );
                        fputs(buffer,file_pr);
                        fputs(sep, file_pr);
                    }
                    else if (type[j] == 1) {
                        sprintf(buffer, "%.2f", (h_columns_float[type_index[j]])[i] );
                        fputs(buffer,file_pr);
                        fputs(sep, file_pr);
                    }
                    else {
                        ss.assign(h_columns_char[type_index[j]] + (i*char_size[type_index[j]]), char_size[type_index[j]]);
                        trim(ss);
                        fputs(ss.c_str(), file_pr);
                        fputs(sep, file_pr);
                    };
                };
                if (i != mCount -1)
                    fputs("\n",file_pr);
            };
            curr_seg++;
        };
        fclose(file_pr);
    }
    else if(text_source) {  //writing a binary file using a text file as a source

        char str[100];
        char col_pos[3];
        total_count = total_count + mCount;
        total_segments = total_segments + 1;
        if (mCount > total_max)
            total_max = mCount;

        void* d;
        CUDA_SAFE_CALL(hipMalloc((void **) &d, mCount*float_size));

        for(unsigned int i = 0; i< mColumnCount; i++) {
            strcpy(str, file_name);
            strcat(str,".");
            itoaa(cols[i],col_pos);
            strcat(str,col_pos);
            curr_file = str;

            strcat(str,".");
            itoaa(total_segments-1,col_pos);
            strcat(str,col_pos);
			cout << "Writing to " << str << endl;


            if(type[i] == 0) {
                thrust::device_ptr<int_type> d_col((int_type*)d);
                thrust::copy(h_columns_int[type_index[i]].begin(), h_columns_int[type_index[i]].begin() + mCount, d_col);
				cout << "Compressing " << endl;
                pfor_compress( d, mCount*int_size, str, h_columns_int[type_index[i]], 0, 0);
				cout << "Compressed " << endl;
            }
            else if(type[i] == 1) {
                if(decimal[i]) {
                    thrust::device_ptr<float_type> d_col((float_type*)d);
                    thrust::copy(h_columns_float[type_index[i]].begin(), h_columns_float[type_index[i]].begin() + mCount, d_col);
                    thrust::device_ptr<long long int> d_col_dec((long long int*)d);
                    thrust::transform(d_col,d_col+mCount,d_col_dec, float_to_long());
					cout << "Compressing " << endl;
                    pfor_compress( d, mCount*float_size, str, h_columns_float[type_index[i]], 1, 0);
					cout << "Compressed " << endl;
                }
                else { // do not compress -- float
                    fstream binary_file(str,ios::out|ios::binary|fstream::app);
                    binary_file.write((char *)&mCount, 4);
                    binary_file.write((char *)(h_columns_float[type_index[i]].data()),mCount*float_size);
                    unsigned int comp_type = 3;
                    binary_file.write((char *)&comp_type, 4);
                    binary_file.close();
                };
            }
            else { //char
			    cout << "Compressing char" << endl;
                compress_char(str, i, mCount);
				cout << "Compressed char " << endl;
            };

            if(fact_file_loaded) {
                writeHeader(file_name, cols[i]);
            };

        };


        for(unsigned int i = 0; i< mColumnCount; i++)
            if(type[i] == 2)
                deAllocColumnOnDevice(i);

        hipFree(d);

    }
    else { //writing a binary file using a binary file as a source
        fact_file_loaded = 1;
        // do it for every segment

        // will add this later
    };
}


void CudaSet::compress_char(string file_name, unsigned int index, unsigned int mCount)
{
    std::vector<string> v1;
    std::map<string,unsigned int> dict;
    std::vector<string> dict_ordered;
    std::vector<unsigned int> dict_val;
    map<string,unsigned int>::iterator iter;
    unsigned int bits_encoded;
    char* field;
    unsigned int len = char_size[type_index[index]];

    field = new char[len];

    for (unsigned int i = 0 ; i < mCount; i++) {

        strncpy(field, h_columns_char[type_index[index]] + i*len, char_size[type_index[index]]);
        v1.push_back(field);

        if((iter = dict.find(field)) != dict.end()) {
            dict_val.push_back(iter->second);
        }
        else {
            string f = field;
            dict[f] = dict.size();
            dict_val.push_back(dict.size()-1);
            dict_ordered.push_back(f);
        };
    };

    bits_encoded = (unsigned int)ceil(log2(double(dict.size()+1)));

    char *cc = new char[len+1];
    unsigned int sz = dict_ordered.size();
    // write to a file
    fstream binary_file(file_name.c_str(),ios::out|ios::binary);
    binary_file.write((char *)&sz, 4);
    for(unsigned int i = 0; i < dict_ordered.size(); i++) {
        memset(&cc[0], 0, len);
        strcpy(cc,dict_ordered[i].c_str());
        binary_file.write(cc, len);
    };

    delete [] cc;
    unsigned int fit_count = 64/bits_encoded;
    unsigned long long int val = 0;
    binary_file.write((char *)&fit_count, 4);
    binary_file.write((char *)&bits_encoded, 4);
    unsigned int curr_cnt = 1;
    unsigned int vals_count = dict_val.size()/fit_count;
    if(!vals_count || dict_val.size()%fit_count)
        vals_count++;
    binary_file.write((char *)&vals_count, 4);
    unsigned int real_count = dict_val.size();
    binary_file.write((char *)&real_count, 4);

    for(unsigned int i = 0; i < dict_val.size(); i++) {

        val = val | dict_val[i];

        if(curr_cnt < fit_count)
            val = val << bits_encoded;

        if( (curr_cnt == fit_count) || (i == (dict_val.size() - 1)) ) {
            if (curr_cnt < fit_count) {
                val = val << ((fit_count-curr_cnt)-1)*bits_encoded;
            };
            curr_cnt = 1;
            binary_file.write((char *)&val, 8);
            val = 0;
        }
        else
            curr_cnt = curr_cnt + 1;
    };
    binary_file.close();
};


void CudaSet::LoadFile(char* file_name, char* sep )
{
    unsigned int count = 0;
    char line[500];
    char* field;
    unsigned int current_column = 1;

    FILE *file_ptr = fopen(file_name, "r");
    if (file_ptr  == NULL)
        cout << "Could not open file " << file_name << endl;

    unsigned int *seq = new unsigned int[mColumnCount];
    thrust::sequence(seq, seq+mColumnCount,0,1);
    thrust::stable_sort_by_key(cols, cols+mColumnCount, seq);


    while (fgets(line, 500, file_ptr) != NULL ) {

        current_column = 1;
        field = strtok(line,sep);

        for(unsigned int i = 0; i< mColumnCount; i++) {

            while(cols[i] > current_column) {
                field = strtok(NULL,sep);
                current_column++;
            };

            if (type[seq[i]] == 0) {
                if (strchr(field,'-') == NULL) {
                    (h_columns_int[type_index[seq[i]]])[count] = atoll(field);
                }
                else {   // handling possible dates
                    strncpy(field+4,field+5,2);
                    strncpy(field+6,field+8,2);
                    field[8] = '\0';
                    (h_columns_int[type_index[seq[i]]])[count] = atoll(field);
                };
            }
            else if (type[seq[i]] == 1)
                (h_columns_float[type_index[seq[i]]])[count] = atoff(field);
            else {
                strcpy(h_columns_char[type_index[seq[i]]] + count*char_size[type_index[seq[i]]], field);
            };
        };
        count++;
        if (count == mRecCount) {
            mRecCount = mRecCount + process_count;
            resize(mRecCount);
        };
    };
    fclose(file_ptr);
    mRecCount = count;
};


int CudaSet::LoadBigFile(const char* file_name, const char* sep )
{
    unsigned int count = 0;
    char line[1000];
    char* field;
    unsigned int current_column = 1;

    if (file_p == NULL)
        file_p = fopen(file_name, "r");
    if (file_p  == NULL)
        cout << "Could not open file " << file_name << endl;

    if (seq == 0) {
        seq = new unsigned int[mColumnCount];
        thrust::sequence(seq, seq+mColumnCount,0,1);
        thrust::stable_sort_by_key(cols, cols+mColumnCount, seq);
    };

    while (count < process_count && fgets(line, 1000, file_p) != NULL) {

        current_column = 1;
        field = strtok(line,sep);

        for(unsigned int i = 0; i< mColumnCount; i++) {

            while(cols[i] > current_column) {
                field = strtok(NULL,sep);
                current_column++;
            };
            if (type[seq[i]] == 0) {
                if (strchr(field,'-') == NULL) {
                    (h_columns_int[type_index[seq[i]]])[count] = atoll(field);
                }
                else {   // handling possible dates
                    strncpy(field+4,field+5,2);
                    strncpy(field+6,field+8,2);
                    field[8] = '\0';
                    (h_columns_int[type_index[seq[i]]])[count] = atoll(field);
                };
            }
            else if (type[seq[i]] == 1)
                (h_columns_float[type_index[seq[i]]])[count] = atoff(field);
            else  {//char
                strcpy(h_columns_char[type_index[seq[i]]] + count*char_size[type_index[seq[i]]], field);
            }
        };
        count++;
    };

    mRecCount = count;

    if(count < process_count)  {
        fclose(file_p);
        return 1;
    }
    else
        return 0;
};


void CudaSet::free()  {

    if (!seq)
        delete seq;

    for(unsigned int i = 0; i < mColumnCount; i++ ) {
        if(type[i] == 2 && h_columns_char[type_index[i]] && prm.empty()) {
            delete [] h_columns_char[type_index[i]];
            h_columns_char[type_index[i]] = NULL;
        };
    };

    if(!prm.empty()) { // free the sources
        string some_field;
        map<string,int>::iterator it=columnNames.begin();
        some_field = (*it).first;
        CudaSet* t = varNames[setMap[some_field]];
        t->deAllocOnDevice();

    };

    delete type;
    delete cols;

    if(!columnGroups.empty() && mRecCount !=0 && grp != NULL)
        hipFree(grp);

    for(unsigned int i = 0; i < prm.size(); i++)
        delete [] prm[i];
};


bool* CudaSet::logical_and(bool* column1, bool* column2)
{
    thrust::device_ptr<bool> dev_ptr1(column1);
    thrust::device_ptr<bool> dev_ptr2(column2);

    thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, dev_ptr1, thrust::logical_and<bool>());

    thrust::device_free(dev_ptr2);
    return column1;
}


bool* CudaSet::logical_or(bool* column1, bool* column2)
{

    thrust::device_ptr<bool> dev_ptr1(column1);
    thrust::device_ptr<bool> dev_ptr2(column2);

    thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, dev_ptr1, thrust::logical_or<bool>());
    thrust::device_free(dev_ptr2);
    return column1;
}



bool* CudaSet::compare(int_type s, int_type d, int_type op_type)
{
    bool res;

    if (op_type == 2) // >
        if(d>s) res = 1;
        else res = 0;
    else if (op_type == 1)  // <
        if(d<s) res = 1;
        else res = 0;
    else if (op_type == 6) // >=
        if(d>=s) res = 1;
        else res = 0;
    else if (op_type == 5)  // <=
        if(d<=s) res = 1;
        else res = 0;
    else if (op_type == 4)// =
        if(d==s) res = 1;
        else res = 0;
    else // !=
        if(d!=s) res = 1;
        else res = 0;

    thrust::device_ptr<bool> p = thrust::device_malloc<bool>(mRecCount);
    thrust::sequence(p, p+mRecCount,res,(bool)0);

    return thrust::raw_pointer_cast(p);
};


bool* CudaSet::compare(float_type s, float_type d, int_type op_type)
{
    bool res;

    if (op_type == 2) // >
        if ((d-s) > EPSILON) res = 1;
        else res = 0;
    else if (op_type == 1)  // <
        if ((s-d) > EPSILON) res = 1;
        else res = 0;
    else if (op_type == 6) // >=
        if (((d-s) > EPSILON) || (((d-s) < EPSILON) && ((d-s) > -EPSILON))) res = 1;
        else res = 0;
    else if (op_type == 5)  // <=
        if (((s-d) > EPSILON) || (((d-s) < EPSILON) && ((d-s) > -EPSILON))) res = 1;
        else res = 0;
    else if (op_type == 4)// =
        if (((d-s) < EPSILON) && ((d-s) > -EPSILON)) res = 1;
        else res = 0;
    else // !=
        if (!(((d-s) < EPSILON) && ((d-s) > -EPSILON))) res = 1;
        else res = 0;

    thrust::device_ptr<bool> p = thrust::device_malloc<bool>(mRecCount);
    thrust::sequence(p, p+mRecCount,res,(bool)0);

    return thrust::raw_pointer_cast(p);
}


bool* CudaSet::compare(int_type* column1, int_type d, int_type op_type)
{
    thrust::device_ptr<bool> temp = thrust::device_malloc<bool>(mRecCount);
    thrust::device_ptr<int_type> dev_ptr(column1);


    if (op_type == 2) // >
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::greater<int_type>());
    else if (op_type == 1)  // <
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::less<int_type>());
    else if (op_type == 6) // >=
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::greater_equal<int_type>());
    else if (op_type == 5)  // <=
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::less_equal<int_type>());
    else if (op_type == 4)// =
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::equal_to<int_type>());
    else // !=
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::not_equal_to<int_type>());

    return thrust::raw_pointer_cast(temp);

}

bool* CudaSet::compare(float_type* column1, float_type d, int_type op_type)
{
    thrust::device_ptr<bool> res = thrust::device_malloc<bool>(mRecCount);
    thrust::device_ptr<float_type> dev_ptr(column1);

    if (op_type == 2) // >
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_greater());
    else if (op_type == 1)  // <
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_less());
    else if (op_type == 6) // >=
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_greater_equal_to());
    else if (op_type == 5)  // <=
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_less_equal());
    else if (op_type == 4)// =
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_equal_to());
    else // !=
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_not_equal_to());

    return thrust::raw_pointer_cast(res);
}


bool* CudaSet::compare(int_type* column1, int_type* column2, int_type op_type)
{
    thrust::device_ptr<int_type> dev_ptr1(column1);
    thrust::device_ptr<int_type> dev_ptr2(column2);
    thrust::device_ptr<bool> temp = thrust::device_malloc<bool>(mRecCount);

    if (op_type == 2) // >
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::greater<int_type>());
    else if (op_type == 1)  // <
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::less<int_type>());
    else if (op_type == 6) // >=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::greater_equal<int_type>());
    else if (op_type == 5)  // <=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::less_equal<int_type>());
    else if (op_type == 4)// =
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::equal_to<int_type>());
    else // !=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::not_equal_to<int_type>());

    return thrust::raw_pointer_cast(temp);
}

bool* CudaSet::compare(float_type* column1, float_type* column2, int_type op_type)
{
    thrust::device_ptr<float_type> dev_ptr1(column1);
    thrust::device_ptr<float_type> dev_ptr2(column2);
    thrust::device_ptr<bool> temp = thrust::device_malloc<bool>(mRecCount);

    if (op_type == 2) // >
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_greater());
    else if (op_type == 1)  // <
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_less());
    else if (op_type == 6) // >=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_greater_equal_to());
    else if (op_type == 5)  // <=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_less_equal());
    else if (op_type == 4)// =
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_equal_to());
    else // !=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_not_equal_to());

    return thrust::raw_pointer_cast(temp);

}


bool* CudaSet::compare(float_type* column1, int_type* column2, int_type op_type)
{
    thrust::device_ptr<float_type> dev_ptr1(column1);
    thrust::device_ptr<int_type> dev_ptr(column2);
    thrust::device_ptr<float_type> dev_ptr2 = thrust::device_malloc<float_type>(mRecCount);
    thrust::device_ptr<bool> temp = thrust::device_malloc<bool>(mRecCount);

    thrust::transform(dev_ptr, dev_ptr + mRecCount, dev_ptr2, long_to_float_type());

    if (op_type == 2) // >
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_greater());
    else if (op_type == 1)  // <
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_less());
    else if (op_type == 6) // >=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_greater_equal_to());
    else if (op_type == 5)  // <=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_less_equal());
    else if (op_type == 4)// =
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_equal_to());
    else // !=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_not_equal_to());

    thrust::device_free(dev_ptr2);
    return thrust::raw_pointer_cast(temp);
}


float_type* CudaSet::op(int_type* column1, float_type* column2, string op_type, int reverse)
{

    thrust::device_ptr<float_type> temp = thrust::device_malloc<float_type>(mRecCount);
    thrust::device_ptr<int_type> dev_ptr(column1);

    thrust::transform(dev_ptr, dev_ptr + mRecCount, temp, long_to_float_type()); // in-place transformation

    thrust::device_ptr<float_type> dev_ptr1(column2);

    if(reverse == 0) {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::minus<float_type>());
        else
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::divides<float_type>());
    }
    else {
        if (op_type.compare("MUL") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::minus<float_type>());
        else
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::divides<float_type>());

    };

    return thrust::raw_pointer_cast(temp);

}




int_type* CudaSet::op(int_type* column1, int_type* column2, string op_type, int reverse)
{

    thrust::device_ptr<int_type> temp = thrust::device_malloc<int_type>(mRecCount);
    thrust::device_ptr<int_type> dev_ptr1(column1);
    thrust::device_ptr<int_type> dev_ptr2(column2);

    if(reverse == 0) {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::multiplies<int_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::plus<int_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::minus<int_type>());
        else
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::divides<int_type>());
    }
    else  {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::multiplies<int_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::plus<int_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::minus<int_type>());
        else
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::divides<int_type>());
    }

    return thrust::raw_pointer_cast(temp);

}

float_type* CudaSet::op(float_type* column1, float_type* column2, string op_type, int reverse)
{

    thrust::device_ptr<float_type> temp = thrust::device_malloc<float_type>(mRecCount);
    thrust::device_ptr<float_type> dev_ptr1(column1);
    thrust::device_ptr<float_type> dev_ptr2(column2);

    if(reverse == 0) {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::minus<float_type>());
        else
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::divides<float_type>());
    }
    else {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::minus<float_type>());
        else
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::divides<float_type>());
    };
    return thrust::raw_pointer_cast(temp);
}

int_type* CudaSet::op(int_type* column1, int_type d, string op_type, int reverse)
{
    thrust::device_ptr<int_type> temp = thrust::device_malloc<int_type>(mRecCount);
    thrust::fill(temp, temp+mRecCount, d);

    thrust::device_ptr<int_type> dev_ptr1(column1);

    if(reverse == 0) {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::multiplies<int_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::plus<int_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::minus<int_type>());
        else
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::divides<int_type>());
    }
    else {
        if (op_type.compare("MUL") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::multiplies<int_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::plus<int_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::minus<int_type>());
        else
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::divides<int_type>());
    };
    return thrust::raw_pointer_cast(temp);
}

float_type* CudaSet::op(int_type* column1, float_type d, string op_type, int reverse)
{
    thrust::device_ptr<float_type> temp = thrust::device_malloc<float_type>(mRecCount);
    thrust::fill(temp, temp+mRecCount, d);

    thrust::device_ptr<int_type> dev_ptr(column1);
    thrust::device_ptr<float_type> dev_ptr1 = thrust::device_malloc<float_type>(mRecCount);
    thrust::transform(dev_ptr, dev_ptr + mRecCount, dev_ptr1, long_to_float_type());

    if(reverse == 0) {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::minus<float_type>());
        else
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::divides<float_type>());
    }
    else  {
        if (op_type.compare("MUL") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::minus<float_type>());
        else
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::divides<float_type>());

    };
    thrust::device_free(dev_ptr1);
    return thrust::raw_pointer_cast(temp);
}


float_type* CudaSet::op(float_type* column1, float_type d, string op_type,int reverse)
{
    thrust::device_ptr<float_type> temp = thrust::device_malloc<float_type>(mRecCount);
    thrust::device_ptr<float_type> dev_ptr1(column1);

    if(reverse == 0) {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_constant_iterator(d), temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_constant_iterator(d), temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_constant_iterator(d), temp, thrust::minus<float_type>());
        else
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_constant_iterator(d), temp, thrust::divides<float_type>());
    }
    else	{
        if (op_type.compare("MUL") == 0)
            thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d)+mRecCount, dev_ptr1, temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d)+mRecCount, dev_ptr1, temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d)+mRecCount, dev_ptr1, temp, thrust::minus<float_type>());
        else
            thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d)+mRecCount, dev_ptr1, temp, thrust::divides<float_type>());

    };

    return thrust::raw_pointer_cast(temp);

}


void CudaSet::initialize(queue<string> &nameRef, queue<string> &typeRef, queue<int> &sizeRef, queue<int> &colsRef, int_type Recs, char* file_name) // compressed data for DIM tables
{
    mColumnCount = nameRef.size();
    type = new unsigned int[mColumnCount];
    cols = new unsigned int[mColumnCount];
    decimal = new bool[mColumnCount];
    unsigned int cnt;
    file_p = NULL;
    FILE* f;
    char f1[100];

    not_compressed = 0;
    mRecCount = Recs;
    load_file_name = file_name;



    for(unsigned int i=0; i < mColumnCount; i++) {

        columnNames[nameRef.front()] = i;
        cols[i] = colsRef.front();
        seq = 0;

        strcpy(f1, file_name);
        strcat(f1,".");
        char col_pos[3];
        itoaa(colsRef.front(),col_pos);
        strcat(f1,col_pos); // read the size of a segment

        strcat(f1, ".header");
        f = fopen (f1 , "rb" );
        for(unsigned int j = 0; j < 5; j++)
            fread((char *)&cnt, 4, 1, f);
        fclose(f);
        //cout << "creating " << f1 << " " << cnt << endl;

        if ((typeRef.front()).compare("int") == 0) {
            type[i] = 0;
            decimal[i] = 0;
            h_columns_int.push_back(thrust::host_vector<int_type, uninitialized_host_allocator<int_type> >(cnt + 9));
            d_columns_int.push_back(thrust::device_vector<int_type>());
            type_index[i] = h_columns_int.size()-1;
        }
        else if ((typeRef.front()).compare("float") == 0) {
            type[i] = 1;
            decimal[i] = 0;
            h_columns_float.push_back(thrust::host_vector<float_type, uninitialized_host_allocator<float_type> >(cnt + 9));
            d_columns_float.push_back(thrust::device_vector<float_type >());
            type_index[i] = h_columns_float.size()-1;
        }
        else if ((typeRef.front()).compare("decimal") == 0) {
            type[i] = 1;
            decimal[i] = 1;
            h_columns_float.push_back(thrust::host_vector<float_type, uninitialized_host_allocator<float_type> >(cnt + 9));
            d_columns_float.push_back(thrust::device_vector<float_type>());
            type_index[i] = h_columns_float.size()-1;
        }
        else {
            type[i] = 2;
            decimal[i] = 0;
            h_columns_char.push_back(NULL);
            d_columns_char.push_back(NULL);
            char_size.push_back(sizeRef.front());
            type_index[i] = h_columns_char.size()-1;
        };

        nameRef.pop();
        typeRef.pop();
        sizeRef.pop();
        colsRef.pop();
    };
};



void CudaSet::initialize(queue<string> &nameRef, queue<string> &typeRef, queue<int> &sizeRef, queue<int> &colsRef, int_type Recs)
{
    mColumnCount = nameRef.size();
    type = new unsigned int[mColumnCount];
    cols = new unsigned int[mColumnCount];
    decimal = new bool[mColumnCount];

    file_p = NULL;

    mRecCount = Recs;
    segCount = 1;

    for(unsigned int i=0; i < mColumnCount; i++) {

        columnNames[nameRef.front()] = i;
        cols[i] = colsRef.front();
        seq = 0;

        if ((typeRef.front()).compare("int") == 0) {
            type[i] = 0;
            decimal[i] = 0;
            h_columns_int.push_back(thrust::host_vector<int_type, uninitialized_host_allocator<int_type> >());
            d_columns_int.push_back(thrust::device_vector<int_type>());
            type_index[i] = h_columns_int.size()-1;
        }
        else if ((typeRef.front()).compare("float") == 0) {
            type[i] = 1;
            decimal[i] = 0;
            h_columns_float.push_back(thrust::host_vector<float_type, uninitialized_host_allocator<float_type> >());
            d_columns_float.push_back(thrust::device_vector<float_type>());
            type_index[i] = h_columns_float.size()-1;
        }
        else if ((typeRef.front()).compare("decimal") == 0) {
            type[i] = 1;
            decimal[i] = 1;
            h_columns_float.push_back(thrust::host_vector<float_type, uninitialized_host_allocator<float_type> >());
            d_columns_float.push_back(thrust::device_vector<float_type>());
            type_index[i] = h_columns_float.size()-1;
        }

        else {
            type[i] = 2;
            decimal[i] = 0;
            h_columns_char.push_back(NULL);
            d_columns_char.push_back(NULL);
            char_size.push_back(sizeRef.front());
            type_index[i] = h_columns_char.size()-1;
        };
        nameRef.pop();
        typeRef.pop();
        sizeRef.pop();
        colsRef.pop();
    };
};

void CudaSet::initialize(unsigned int RecordCount, unsigned int ColumnCount)
{
    mRecCount = RecordCount;
    mColumnCount = ColumnCount;

    type = new unsigned int[mColumnCount];
    cols = new unsigned int[mColumnCount];
    decimal = new bool[mColumnCount];
    seq = 0;

    for(unsigned int i =0; i < mColumnCount; i++) {
        cols[i] = i;
    };


};


void CudaSet::initialize(CudaSet* a, CudaSet* b, int_type Recs, queue<string> op_sel, queue<string> op_sel_as)
{
    mRecCount = Recs;
    mColumnCount = op_sel.size();

    type = new unsigned int[mColumnCount];
    cols = new unsigned int[mColumnCount];
    decimal = new bool[mColumnCount];
    maxRecs = b->maxRecs;

    map<string,int>::iterator it;
    seq = 0;
    unsigned int i = 0;
    segCount = 1;
    not_compressed = 1;

    col_aliases = op_sel_as;
    queue<string> names(op_sel);

    while(!names.empty()) {
        columnNames[names.front()] = i;
        names.pop();
        i++;
    };


    unsigned int index;
    for(unsigned int i=0; i < mColumnCount; i++) {

        if((it = a->columnNames.find(op_sel.front())) !=  a->columnNames.end()) {
            index = it->second;
            cols[i] = i;
            decimal[i] = a->decimal[i];

            if ((a->type)[index] == 0)  {
                d_columns_int.push_back(thrust::device_vector<int_type>());
                h_columns_int.push_back(thrust::host_vector<int_type>());
                type[i] = 0;
                type_index[i] = h_columns_int.size()-1;
            }
            else if ((a->type)[index] == 1) {
                d_columns_float.push_back(thrust::device_vector<float_type>());
                h_columns_float.push_back(thrust::host_vector<float_type>());
                type[i] = 1;
                type_index[i] = h_columns_float.size()-1;
            }
            else {
                h_columns_char.push_back(NULL);
                d_columns_char.push_back(NULL);
                type[i] = 2;
                type_index[i] = h_columns_char.size()-1;
                char_size.push_back(a->char_size[a->type_index[index]]);
                prealloc_char_size = 0;
            };
        }
        else {
            it = b->columnNames.find(op_sel.front());
            index = it->second;

            cols[i] = i;
            decimal[i] = b->decimal[index];

            if ((b->type)[index] == 0) {
                d_columns_int.push_back(thrust::device_vector<int_type>());
                h_columns_int.push_back(thrust::host_vector<int_type, uninitialized_host_allocator<int_type> >());
                type[i] = 0;
                type_index[i] = h_columns_int.size()-1;
            }
            else if ((b->type)[index] == 1) {
                d_columns_float.push_back(thrust::device_vector<float_type>());
                h_columns_float.push_back(thrust::host_vector<float_type, uninitialized_host_allocator<float_type> >());
                type[i] = 1;
                type_index[i] = h_columns_float.size()-1;
            }
            else {
                h_columns_char.push_back(NULL);
                d_columns_char.push_back(NULL);
                type[i] = 2;
                type_index[i] = h_columns_char.size()-1;
                char_size.push_back(b->char_size[b->type_index[index]]);
                prealloc_char_size = 0;
            };
        }
        op_sel.pop();
    };
};



int_type reverse_op(int_type op_type)
{
    if (op_type == 2) // >
        return 5;
    else if (op_type == 1)  // <
        return 6;
    else if (op_type == 6) // >=
        return 1;
    else if (op_type == 5)  // <=
        return 2;
    else return op_type;
}


size_t getFreeMem()
{
    size_t available, total;
    hipMemGetInfo(&available, &total);
    return available;
} ;



void allocColumns(CudaSet* a, queue<string> fields)
{
    if(!a->prm.empty()) {
        unsigned int max_sz = max_tmp(a) ;
        CudaSet* t = varNames[setMap[fields.front()]];
        if(max_sz*t->maxRecs > alloced_sz) {
            if(alloced_sz) {
                hipFree(alloced_tmp);
            };
            hipMalloc((void **) &alloced_tmp, max_sz*t->maxRecs);
            alloced_sz = max_sz*t->maxRecs;
        }
    }
    else {

        while(!fields.empty()) {
            if(setMap.count(fields.front()) > 0) {

                unsigned int idx = a->columnNames[fields.front()];
                bool onDevice = 0;

                if(a->type[idx] == 0) {
                    if(a->d_columns_int[a->type_index[idx]].size() > 0) {
                        onDevice = 1;
                    }
                }
                else if(a->type[idx] == 1) {
                    if(a->d_columns_float[a->type_index[idx]].size() > 0) {
                        onDevice = 1;
                    };
                }
                else {
                    if((a->d_columns_char[a->type_index[idx]]) != NULL) {
                        onDevice = 1;
                    };
                };

                if (!onDevice) {
                    if(a->prm.empty()) {
                        a->allocColumnOnDevice(idx, a->maxRecs);
                    }
                    else {
                        a->allocColumnOnDevice(idx, largest_prm(a));
                    };
                };
            };
            fields.pop();
        };
    };
}

unsigned int largest_prm(CudaSet* a)
{
    unsigned int maxx = 0;

    for(unsigned int i = 0; i < a->prm_count.size(); i++)
        if(maxx < a->prm_count[i])
            maxx = a->prm_count[i];
    if(maxx == 0)
        maxx = a->maxRecs;
    return maxx;
};


void gatherColumns(CudaSet* a, CudaSet* t, string field, unsigned int segment, unsigned int& count)
{

    unsigned int tindex = t->columnNames[field];
    unsigned int idx = a->columnNames[field];

    //find the largest possible size of a gathered segment
    if(!a->onDevice(idx)) {
        unsigned int max_count = 0;

        for(unsigned int i = 0; i < a->prm.size(); i++)
            if (a->prm_count[i] > max_count)
                max_count = a->prm_count[i];
        a->allocColumnOnDevice(idx, max_count);
    };



    unsigned int g_size = a->prm_count[segment];

    if(a->prm_index[segment] == 'R') {

        if(a->prm_d.size() == 0) // find the largest prm segment
            a->prm_d.resize(largest_prm(a));

        if(curr_segment != segment) {
            hipMemcpy((void**)(thrust::raw_pointer_cast(a->prm_d.data())), (void**)a->prm[segment],
                       4*g_size, hipMemcpyHostToDevice);
            curr_segment = segment;
        };

        mygather(tindex, idx, a, t, count, g_size);
    }
    else {
        mycopy(tindex, idx, a, t, count, g_size);
    };

    a->mRecCount = g_size;
}

unsigned int getSegmentRecCount(CudaSet* a, unsigned int segment) {
    if (segment == a->segCount-1) {
        return oldCount - a->maxRecs*segment;
    }
    else
        return 	a->maxRecs;
}



void copyColumns(CudaSet* a, queue<string> fields, unsigned int segment, unsigned int& count)
{
    set<string> uniques;
    CudaSet *t;

    while(!fields.empty()) {
        if (uniques.count(fields.front()) == 0 && setMap.count(fields.front()) > 0)	{
            if(!a->prm.empty()) {
                t = varNames[setMap[fields.front()]];
                if(a->prm_count[segment]) {

                    alloced_switch = 1;
                    //cout << "copy " << fields.front() << " " << alloced_switch << endl;
                    t->CopyColumnToGpu(t->columnNames[fields.front()], segment); // segment i
                    //cout << "gather " << fields.front() << endl;
                    gatherColumns(a, t, fields.front(), segment, count);
                    //cout << "end " << endl;
                    alloced_switch = 0;
                }
                else
                    a->mRecCount = 0;
            }
            else {
                a->CopyColumnToGpu(a->columnNames[fields.front()], segment); // segment i
            };
            uniques.insert(fields.front());
        };
        fields.pop();
    };
}



void setPrm(CudaSet* a, CudaSet* b, char val, unsigned int segment) {

    b->prm.push_back(NULL);
    b->prm_index.push_back(val);

    if (val == 'A') {
        b->mRecCount = b->mRecCount + getSegmentRecCount(a,segment);
        b->prm_count.push_back(getSegmentRecCount(a, segment));
    }
    else {
        b->prm_count.push_back(0);
    };
}



void mygather(unsigned int tindex, unsigned int idx, CudaSet* a, CudaSet* t, unsigned int offset, unsigned int g_size)
{
    if(t->type[tindex] == 0) {
        if(!alloced_switch) {
            thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size,
                           t->d_columns_int[t->type_index[tindex]].begin(), a->d_columns_int[a->type_index[idx]].begin() + offset);
        }
        else {
            thrust::device_ptr<int_type> d_col((int_type*)alloced_tmp);
            thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size,
                           d_col, a->d_columns_int[a->type_index[idx]].begin() + offset);
        };
    }
    else if(t->type[tindex] == 1) {
        if(!alloced_switch) {
            thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size,
                           t->d_columns_float[t->type_index[tindex]].begin(), a->d_columns_float[a->type_index[idx]].begin() + offset);
        }
        else {
            thrust::device_ptr<float_type> d_col((float_type*)alloced_tmp);
            thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size,
                           d_col, a->d_columns_float[a->type_index[idx]].begin() + offset);
        };
    }
    else {
        if(!alloced_switch) {
            str_gather((void*)thrust::raw_pointer_cast(a->prm_d.data()), g_size,
                       (void*)t->d_columns_char[t->type_index[tindex]], (void*)a->d_columns_char[a->type_index[idx]], a->char_size[a->type_index[idx]] );
        }
        else {
            str_gather((void*)thrust::raw_pointer_cast(a->prm_d.data()), g_size,
                       alloced_tmp, (void*)a->d_columns_char[a->type_index[idx]], a->char_size[a->type_index[idx]] );
        };
    }
};

void mycopy(unsigned int tindex, unsigned int idx, CudaSet* a, CudaSet* t, unsigned int offset, unsigned int g_size)
{
    if(t->type[tindex] == 0) {
        if(!alloced_switch) {
            thrust::copy(t->d_columns_int[t->type_index[tindex]].begin(), t->d_columns_int[t->type_index[tindex]].begin() + g_size,
                         a->d_columns_int[a->type_index[idx]].begin() + offset);
        }
        else {
            thrust::device_ptr<int_type> d_col((int_type*)alloced_tmp);
            thrust::copy(d_col, d_col + g_size, a->d_columns_int[a->type_index[idx]].begin() + offset);

        };
    }
    else if(t->type[tindex] == 1) {
        if(!alloced_switch) {
            thrust::copy(t->d_columns_float[t->type_index[tindex]].begin(), t->d_columns_float[t->type_index[tindex]].begin() + g_size,
                         a->d_columns_float[a->type_index[idx]].begin() + offset);
        }
        else {
            thrust::device_ptr<float_type> d_col((float_type*)alloced_tmp);
            thrust::copy(d_col, d_col + g_size,	a->d_columns_float[a->type_index[idx]].begin() + offset);
        };
    }
    else {
        if(!alloced_switch) {
            hipMemcpy((void**)(a->d_columns_char[a->type_index[idx]] + offset*a->char_size[a->type_index[idx]]), (void**)t->d_columns_char[t->type_index[tindex]],
                       g_size*t->char_size[t->type_index[tindex]], hipMemcpyDeviceToDevice);
        }
        else {
            hipMemcpy((void**)(a->d_columns_char[a->type_index[idx]] + offset*a->char_size[a->type_index[idx]]), alloced_tmp,
                       g_size*t->char_size[t->type_index[tindex]], hipMemcpyDeviceToDevice);
        };
    };
};



unsigned int load_queue(queue<string> c1, CudaSet* right, bool str_join, string f2, unsigned int &rcount)
{
    queue<string> cc;
    while(!c1.empty()) {
        if(right->columnNames.find(c1.front()) !=  right->columnNames.end()) {
            if(f2 != c1.front() || str_join) {
                cc.push(c1.front());
            };
        };
        c1.pop();
    };
    if(!str_join) {
        cc.push(f2);
    };

    unsigned int cnt_r = 0;
    if(!right->prm.empty()) {
        allocColumns(right, cc);
        rcount = std::accumulate(right->prm_count.begin(), right->prm_count.end(), 0 );
    }
    else
        rcount = right->mRecCount;

    queue<string> ct(cc);
    reset_offsets();

    while(!ct.empty()) {
        right->allocColumnOnDevice(right->columnNames[ct.front()], rcount);
        ct.pop();
    };


    ct = cc;
    if(right->prm.empty()) {
        //copy all records
        while(!ct.empty()) {
            right->CopyColumnToGpu(right->columnNames[ct.front()]);
            ct.pop();
        };
        cnt_r = right->mRecCount;
    }
    else {
        //copy and gather all records
        for(unsigned int i = 0; i < right->segCount; i++) {
            copyColumns(right, cc, i, cnt_r);
            cnt_r = cnt_r + right->prm_count[i];
        };
    };
    return cnt_r;

}

unsigned int max_char(CudaSet* a)
{
    unsigned int max_char = 0;
    for(unsigned int i = 0; i < a->char_size.size(); i++)
        if (a->char_size[i] > max_char)
            max_char = a->char_size[i];

    return max_char;
};

unsigned int max_char(CudaSet* a, set<string> field_names)
{
    unsigned int max_char = 0;
    for (set<string>::iterator it=field_names.begin(); it!=field_names.end(); ++it) {
        int i = a->columnNames[*it];	
		if (a->type[i] == 2) {
			if (a->char_size[a->type_index[i]] > max_char)
				max_char = a->char_size[a->type_index[i]];
		};
	};	
    return max_char;
};


unsigned int max_tmp(CudaSet* a)
{
    unsigned int max_sz = 0;
    for(unsigned int i = 0; i < a->mColumnCount; i++) {
        if(a->type[i] == 0) {
            if(int_size > max_sz)
                max_sz = int_size;
        }
        else if(a->type[i] == 1) {
            if(float_size > max_sz)
                max_sz = float_size;
        };
    };
    unsigned int m_char = max_char(a);
    if(m_char > max_sz)
        return m_char;
    else
        return max_sz;

};


void reset_offsets() {
    map<unsigned int, unsigned int>::iterator iter;

    for (iter = str_offset.begin(); iter != str_offset.end(); ++iter) {
        iter->second = 0;
    };

};

void setSegments(CudaSet* a, queue<string> cols)
{
	size_t mem_available = getFreeMem();
	unsigned int tot_sz = 0, idx;
	while(!cols.empty()) {
	    idx = a->columnNames[cols.front()];
	    if(a->type[idx] != 2)
			tot_sz = tot_sz + int_size;
		else
            tot_sz = tot_sz + a->char_size[a->type_index[idx]];
        cols.pop();		
	};
	cout << "tot " << tot_sz << endl;
	if(a->mRecCount*tot_sz > mem_available/2) {
	    a->segCount = (a->mRecCount*tot_sz)/(mem_available/2) + 1;	
	};

};
