/*
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <thrust/copy.h>

using namespace std;


	struct Str1 {
	    char A[1];
			
        __host__ __device__
         bool operator<(const Str1& other) const
         {
		    		    
		    for(unsigned int i = 0; i < 1 ; i++) {
			    if(A[i] > other.A[i]) {
				    return 0;
				};	
			    if(A[i] < other.A[i]) {
                    return 1;
				};	
				
			};	
            return 0;   
         }				 
		 
        __host__ __device__
         bool operator>(const Str1& other) const
         {
		    for(unsigned int i = 0; i < 1 ; i++) {
			    if(A[i] > other.A[i]) {
				    return 1;
				};	
			    if(A[i] < other.A[i]) {
				    return 0;
				};					
			};	
            return 0;   
         }			
		 
		 
         __host__ __device__
         bool operator!=(const Str1& other) const
         {
		    for(unsigned int i = 0; i < 1 ; i++) {
			    if(A[i] != other.A[i]) {
				    return 1;
				};					
			};	
            return 0;   
         }					 
		 
		 
		 
	};

    
	struct Str2 {
	    char A[2];
			
        __host__ __device__
         bool operator<(const Str2& other) const
         {
		    		    
		    for(unsigned int i = 0; i < 2 ; i++) {
			    if(A[i] > other.A[i]) {
				    return 0;
				};	
			    if(A[i] < other.A[i]) {
                    return 1;
				};	
				
			};	
            return 0;   
         }				 
		 
        __host__ __device__
         bool operator>(const Str2& other) const
         {
		    for(unsigned int i = 0; i < 2 ; i++) {
			    if(A[i] > other.A[i]) {
				    return 1;
				};	
			    if(A[i] < other.A[i]) {
				    return 0;
				};					
			};	
            return 0;   
         }			
		 
		 
         __host__ __device__
         bool operator!=(const Str2& other) const
         {
		    for(unsigned int i = 0; i < 2 ; i++) {
			    if(A[i] != other.A[i]) {
				    return 1;
				};					
			};	
            return 0;   
         }					 
		 
		 
		 
	};

	struct Str3 {
	    char A[3];
			
        __host__ __device__
         bool operator<(const Str3& other) const
         {
		    		    
		    for(unsigned int i = 0; i < 3 ; i++) {
			    if(A[i] > other.A[i]) {
				    return 0;
				};	
			    if(A[i] < other.A[i]) {
                    return 1;
				};	
				
			};	
            return 0;   
         }				 
		 
        __host__ __device__
         bool operator>(const Str3& other) const
         {
		    for(unsigned int i = 0; i < 3 ; i++) {
			    if(A[i] > other.A[i]) {
				    return 1;
				};	
			    if(A[i] < other.A[i]) {
				    return 0;
				};					
			};	
            return 0;   
         }			
		 
		 
         __host__ __device__
         bool operator!=(const Str3& other) const
         {
		    for(unsigned int i = 0; i < 3 ; i++) {
			    if(A[i] != other.A[i]) {
				    return 1;
				};					
			};	
            return 0;   
         }					 
		 
		 
		 
	};

	struct Str4 {
	    char A[4];
			
        __host__ __device__
         bool operator<(const Str4& other) const
         {
		    		    
		    for(unsigned int i = 0; i < 4 ; i++) {
			    if(A[i] > other.A[i]) {
				    return 0;
				};	
			    if(A[i] < other.A[i]) {
                    return 1;
				};	
				
			};	
            return 0;   
         }				 
		 
        __host__ __device__
         bool operator>(const Str4& other) const
         {
		    for(unsigned int i = 0; i < 4 ; i++) {
			    if(A[i] > other.A[i]) {
				    return 1;
				};	
			    if(A[i] < other.A[i]) {
				    return 0;
				};					
			};	
            return 0;   
         }			
		 
		 
         __host__ __device__
         bool operator!=(const Str4& other) const
         {
		    for(unsigned int i = 0; i < 4 ; i++) {
			    if(A[i] != other.A[i]) {
				    return 1;
				};					
			};	
            return 0;   
         }					 
		 
		 
		 
	};

	struct Str5 {
	    char A[5];
			
        __host__ __device__
         bool operator<(const Str5& other) const
         {
		    		    
		    for(unsigned int i = 0; i < 5 ; i++) {
			    if(A[i] > other.A[i]) {
				    return 0;
				};	
			    if(A[i] < other.A[i]) {
                    return 1;
				};	
				
			};	
            return 0;   
         }				 
		 
        __host__ __device__
         bool operator>(const Str5& other) const
         {
		    for(unsigned int i = 0; i < 5 ; i++) {
			    if(A[i] > other.A[i]) {
				    return 1;
				};	
			    if(A[i] < other.A[i]) {
				    return 0;
				};					
			};	
            return 0;   
         }			
		 
		 
         __host__ __device__
         bool operator!=(const Str5& other) const
         {
		    for(unsigned int i = 0; i < 5 ; i++) {
			    if(A[i] != other.A[i]) {
				    return 1;
				};					
			};	
            return 0;   
         }					 
		 
		 
		 
	};

	struct Str6 {
	    char A[6];
			
        __host__ __device__
         bool operator<(const Str6& other) const
         {
		    		    
		    for(unsigned int i = 0; i < 6 ; i++) {
			    if(A[i] > other.A[i]) {
				    return 0;
				};	
			    if(A[i] < other.A[i]) {
                    return 1;
				};	
				
			};	
            return 0;   
         }				 
		 
        __host__ __device__
         bool operator>(const Str6& other) const
         {
		    for(unsigned int i = 0; i < 6 ; i++) {
			    if(A[i] > other.A[i]) {
				    return 1;
				};	
			    if(A[i] < other.A[i]) {
				    return 0;
				};					
			};	
            return 0;   
         }			
		 
		 
         __host__ __device__
         bool operator!=(const Str6& other) const
         {
		    for(unsigned int i = 0; i < 6 ; i++) {
			    if(A[i] != other.A[i]) {
				    return 1;
				};					
			};	
            return 0;   
         }					 

		 
		 
	};

	struct Str7 {
	    char A[7];
			
        __host__ __device__
         bool operator<(const Str7& other) const
         {
		    		    
		    for(unsigned int i = 0; i < 7 ; i++) {
			    if(A[i] > other.A[i]) {
				    return 0;
				};	
			    if(A[i] < other.A[i]) {
                    return 1;
				};	
				
			};	
            return 0;   
         }				 
		 
        __host__ __device__
         bool operator>(const Str7& other) const
         {
		    for(unsigned int i = 0; i < 7 ; i++) {
			    if(A[i] > other.A[i]) {
				    return 1;
				};	
			    if(A[i] < other.A[i]) {
				    return 0;
				};					
			};	
            return 0;   
         }			
		 
		 
         __host__ __device__
         bool operator!=(const Str7& other) const
         {
		    for(unsigned int i = 0; i < 7 ; i++) {
			    if(A[i] != other.A[i]) {
				    return 1;
				};					
			};	
            return 0;   
         }					 
		 
		 
		 
	};

	struct Str8 {
	    char A[8];
			
        __host__ __device__
         bool operator<(const Str8& other) const
         {
		    		    
		    for(unsigned int i = 0; i < 8 ; i++) {
			    if(A[i] > other.A[i]) {
				    return 0;
				};	
			    if(A[i] < other.A[i]) {
                    return 1;
				};	
				
			};	
            return 0;   
         }				 
		 
        __host__ __device__
         bool operator>(const Str8& other) const
         {
		    for(unsigned int i = 0; i < 8 ; i++) {
			    if(A[i] > other.A[i]) {
				    return 1;
				};	
			    if(A[i] < other.A[i]) {
				    return 0;
				};					
			};	
            return 0;   
         }			
		 
		 
         __host__ __device__
         bool operator!=(const Str8& other) const
         {
		    for(unsigned int i = 0; i < 8 ; i++) {
			    if(A[i] != other.A[i]) {
				    return 1;
				};					
			};	
            return 0;   
         }					 
		 
		 
	};

	struct Str9 {
	    char A[9];
			
        __host__ __device__
         bool operator<(const Str9& other) const
         {
		    		    
		    for(unsigned int i = 0; i < 9 ; i++) {
			    if(A[i] > other.A[i]) {
				    return 0;
				};	
			    if(A[i] < other.A[i]) {
                    return 1;
				};	
				
			};	
            return 0;   
         }				 
		 
        __host__ __device__
         bool operator>(const Str9& other) const
         {
		    for(unsigned int i = 0; i < 9 ; i++) {
			    if(A[i] > other.A[i]) {
				    return 1;
				};	
			    if(A[i] < other.A[i]) {
				    return 0;
				};					
			};	
            return 0;   
         }			
		 
		 
         __host__ __device__
         bool operator!=(const Str9& other) const
         {
		    for(unsigned int i = 0; i < 9 ; i++) {
			    if(A[i] != other.A[i]) {
				    return 1;
				};					
			};	
            return 0;   
         }					 
		 
		 
		 
	};
	
	
	struct   Str10 {
	    char A[10];
			
        __host__ __device__
         bool operator<(const Str10& other) const
         {
		    		    
		    for(unsigned int i = 0; i < 10 ; i++) {
			    //printf("CMP %s %s \n ", A, other.A);
			    if(A[i] > other.A[i]) {
				    return 0;
				};	
			    if(A[i] < other.A[i]) {
                    return 1;
				};	
				
			};	
            return 0;   
         }				 
		 
        __host__ __device__
         bool operator>(const Str10& other) const
         {
		    for(unsigned int i = 0; i < 10 ; i++) {
			    if(A[i] > other.A[i]) {
				    return 1;
				};	
			    if(A[i] < other.A[i]) {
				    return 0;
				};					
			};	
            return 0;   
         }			
		 
		 
         __host__ __device__
         bool operator!=(const Str10& other) const
         {
		    for(unsigned int i = 0; i < 10 ; i++) {
			    if(A[i] != other.A[i]) {
				    return 1;
				};					
			};	
            return 0;   
         }					 
		 
		
		 
	};
	
	
 
		
void str_gather_host(unsigned int* d_int, unsigned int real_count, void* d, void* d_char, unsigned int len)
{
		if(len  == 1) {	
	        thrust::gather(d_int, d_int + real_count, (Str1*)d, (Str1*)d_char);       
		}
		else if(len  == 2) {	
   	        thrust::gather(d_int, d_int + real_count, (Str2*)d, (Str2*)d_char);       
		}
		else if(len  == 3) {	
	        thrust::gather(d_int, d_int + real_count, (Str3*)d, (Str3*)d_char);       
		}
		else if(len  == 4) {	
	        thrust::gather(d_int, d_int + real_count, (Str4*)d, (Str4*)d_char);       
		}
		else if(len  == 5) {	
	        thrust::gather(d_int, d_int + real_count, (Str5*)d, (Str5*)d_char);       
		}
		else if(len  == 6) {	
	        thrust::gather(d_int, d_int + real_count, (Str6*)d, (Str6*)d_char);       
		}
		else if(len  == 7) {	
	        thrust::gather(d_int, d_int + real_count, (Str7*)d, (Str7*)d_char);       
		}
		else if(len  == 8) {	
	        thrust::gather(d_int, d_int + real_count, (Str8*)d, (Str8*)d_char);       
		}
		else if(len  == 9) {	
	        thrust::gather(d_int, d_int + real_count, (Str9*)d, (Str9*)d_char);       
		}
		else if(len  == 10) {	
	        thrust::gather(d_int, d_int + real_count, (Str10*)d, (Str10*)d_char);       
		};		

}		 
	

void str_gather(void* d_int, unsigned int real_count, void* d, void* d_char, unsigned int len)
{

    thrust::device_ptr<unsigned int> res((unsigned int*)d_int);

		if(len  == 1) {	
	        thrust::device_ptr<Str1> dev_ptr_char((Str1*)d_char);
		    thrust::device_ptr<Str1> dev_ptr((Str1*)d);			
	        thrust::gather(res, res + real_count, dev_ptr, dev_ptr_char);       
		}
		else if(len  == 2) {	
	        thrust::device_ptr<Str2> dev_ptr_char((Str2*)d_char);
		    thrust::device_ptr<Str2> dev_ptr((Str2*)d);			
	        thrust::gather(res, res + real_count, dev_ptr, dev_ptr_char);       
		}
		else if(len  == 3) {	
	        thrust::device_ptr<Str3> dev_ptr_char((Str3*)d_char);
		    thrust::device_ptr<Str3> dev_ptr((Str3*)d);			
	        thrust::gather(res, res + real_count, dev_ptr, dev_ptr_char);       
		}
		else if(len  == 4) {	
	        thrust::device_ptr<Str4> dev_ptr_char((Str4*)d_char);
		    thrust::device_ptr<Str4> dev_ptr((Str4*)d);			
	        thrust::gather(res, res + real_count, dev_ptr, dev_ptr_char);       
		}
		else if(len  == 5) {	
	        thrust::device_ptr<Str5> dev_ptr_char((Str5*)d_char);
		    thrust::device_ptr<Str5> dev_ptr((Str5*)d);			
	        thrust::gather(res, res + real_count, dev_ptr, dev_ptr_char);       
		}
		else if(len  == 6) {	
	        thrust::device_ptr<Str6> dev_ptr_char((Str6*)d_char);
		    thrust::device_ptr<Str6> dev_ptr((Str6*)d);			
	        thrust::gather(res, res + real_count, dev_ptr, dev_ptr_char);       
		}
		else if(len  == 7) {	
	        thrust::device_ptr<Str7> dev_ptr_char((Str7*)d_char);
		    thrust::device_ptr<Str7> dev_ptr((Str7*)d);			
	        thrust::gather(res, res + real_count, dev_ptr, dev_ptr_char);       
		}
		else if(len  == 8) {	
	        thrust::device_ptr<Str8> dev_ptr_char((Str8*)d_char);
		    thrust::device_ptr<Str8> dev_ptr((Str8*)d);			
	        thrust::gather(res, res + real_count, dev_ptr, dev_ptr_char);       
		}
		else if(len  == 9) {	
	        thrust::device_ptr<Str9> dev_ptr_char((Str9*)d_char);
		    thrust::device_ptr<Str9> dev_ptr((Str9*)d);			
	        thrust::gather(res, res + real_count, dev_ptr, dev_ptr_char);       
		}
		else if(len  == 10) {	
	        thrust::device_ptr<Str10> dev_ptr_char((Str10*)d_char);
		    thrust::device_ptr<Str10> dev_ptr((Str10*)d);			
	        thrust::gather(res, res + real_count, dev_ptr, dev_ptr_char);     
		};


}


void str_sort_host(char* tmp, unsigned int RecCount, unsigned int* permutation, bool srt, unsigned int len)
{
	if(len  == 1) {	
        if(srt)
            thrust::stable_sort_by_key((Str1*)tmp, (Str1*)tmp+RecCount, permutation, thrust::greater<Str1>());			
		else
            thrust::stable_sort_by_key((Str1*)tmp, (Str1*)tmp+RecCount, permutation);				        
	}
	if(len  == 2) {	
        if(srt)
            thrust::stable_sort_by_key((Str2*)tmp, (Str2*)tmp+RecCount, permutation, thrust::greater<Str2>());			
		else
            thrust::stable_sort_by_key((Str2*)tmp, (Str2*)tmp+RecCount, permutation);				        
	}
	if(len  == 3) {	
        if(srt)
            thrust::stable_sort_by_key((Str3*)tmp, (Str3*)tmp+RecCount, permutation, thrust::greater<Str3>());			
		else
            thrust::stable_sort_by_key((Str3*)tmp, (Str3*)tmp+RecCount, permutation);				        
	}
	if(len  == 4) {	
        if(srt)
            thrust::stable_sort_by_key((Str4*)tmp, (Str4*)tmp+RecCount, permutation, thrust::greater<Str4>());			
		else
            thrust::stable_sort_by_key((Str4*)tmp, (Str4*)tmp+RecCount, permutation);				        
	}
	if(len  == 5) {	
        if(srt)
            thrust::stable_sort_by_key((Str5*)tmp, (Str5*)tmp+RecCount, permutation, thrust::greater<Str5>());			
		else
            thrust::stable_sort_by_key((Str5*)tmp, (Str5*)tmp+RecCount, permutation);				        
	}
	if(len  == 6) {	
        if(srt)
            thrust::stable_sort_by_key((Str6*)tmp, (Str6*)tmp+RecCount, permutation, thrust::greater<Str6>());			
		else
            thrust::stable_sort_by_key((Str6*)tmp, (Str6*)tmp+RecCount, permutation);				        
	}
	if(len  == 7) {	
        if(srt)
            thrust::stable_sort_by_key((Str7*)tmp, (Str7*)tmp+RecCount, permutation, thrust::greater<Str7>());			
		else
            thrust::stable_sort_by_key((Str7*)tmp, (Str7*)tmp+RecCount, permutation);				        
	}
	if(len  == 8) {	
        if(srt)
            thrust::stable_sort_by_key((Str8*)tmp, (Str8*)tmp+RecCount, permutation, thrust::greater<Str8>());			
		else
            thrust::stable_sort_by_key((Str8*)tmp, (Str8*)tmp+RecCount, permutation);				        
	}
	if(len  == 9) {	
        if(srt)
            thrust::stable_sort_by_key((Str9*)tmp, (Str9*)tmp+RecCount, permutation, thrust::greater<Str9>());			
		else
            thrust::stable_sort_by_key((Str9*)tmp, (Str9*)tmp+RecCount, permutation);				        
	}
	if(len  == 10) {	
        if(srt)
            thrust::stable_sort_by_key((Str10*)tmp, (Str10*)tmp+RecCount, permutation, thrust::greater<Str10>());			
		else
            thrust::stable_sort_by_key((Str10*)tmp, (Str10*)tmp+RecCount, permutation);				        
	}
	

}



void str_sort(char* tmp, unsigned int RecCount, unsigned int* permutation, bool srt, unsigned int len)
{
	thrust::device_ptr<unsigned int> dev_per((unsigned int*)permutation);

		if(len  == 1) {	
	        thrust::device_ptr<Str1> temp((Str1*)tmp);
            if(srt)
                thrust::stable_sort_by_key(temp, temp+RecCount, dev_per, thrust::greater<Str1>());			
			else
                thrust::stable_sort_by_key(temp, temp+RecCount, dev_per);				        
		}
		else if(len  == 2) {	
	        thrust::device_ptr<Str2> temp((Str2*)tmp);
            if(srt)
                thrust::stable_sort_by_key(temp, temp+RecCount, dev_per, thrust::greater<Str2>());			
			else
                thrust::stable_sort_by_key(temp, temp+RecCount, dev_per);				        
		}
		else if(len  == 3) {	
	        thrust::device_ptr<Str3> temp((Str3*)tmp);
            if(srt)
                thrust::stable_sort_by_key(temp, temp+RecCount, dev_per, thrust::greater<Str3>());			
			else
                thrust::stable_sort_by_key(temp, temp+RecCount, dev_per);				        
		}
		else if(len  == 4) {	
	        thrust::device_ptr<Str4> temp((Str4*)tmp);
            if(srt)
                thrust::stable_sort_by_key(temp, temp+RecCount, dev_per, thrust::greater<Str4>());			
			else
                thrust::stable_sort_by_key(temp, temp+RecCount, dev_per);				        
		}
		else if(len  == 5) {	
	        thrust::device_ptr<Str5> temp((Str5*)tmp);
            if(srt)
                thrust::stable_sort_by_key(temp, temp+RecCount, dev_per, thrust::greater<Str5>());			
			else
                thrust::stable_sort_by_key(temp, temp+RecCount, dev_per);				        
		}
		else if(len  == 6) {	
	        thrust::device_ptr<Str6> temp((Str6*)tmp);
            if(srt)
                thrust::stable_sort_by_key(temp, temp+RecCount, dev_per, thrust::greater<Str6>());			
			else
                thrust::stable_sort_by_key(temp, temp+RecCount, dev_per);				        
		}
		else if(len  == 7) {	
	        thrust::device_ptr<Str7> temp((Str7*)tmp);
            if(srt)
                thrust::stable_sort_by_key(temp, temp+RecCount, dev_per, thrust::greater<Str7>());			
			else
                thrust::stable_sort_by_key(temp, temp+RecCount, dev_per);				        
		}
		else if(len  == 8) {	
	        thrust::device_ptr<Str8> temp((Str8*)tmp);
            if(srt)
                thrust::stable_sort_by_key(temp, temp+RecCount, dev_per, thrust::greater<Str8>());			
			else
                thrust::stable_sort_by_key(temp, temp+RecCount, dev_per);				        
		}
		else if(len  == 9) {	
	        thrust::device_ptr<Str9> temp((Str9*)tmp);
            if(srt)
                thrust::stable_sort_by_key(temp, temp+RecCount, dev_per, thrust::greater<Str9>());			
			else
                thrust::stable_sort_by_key(temp, temp+RecCount, dev_per);				        
		}
		else if(len  == 10) {	
			thrust::device_ptr<Str10> temp((Str10*)tmp);
			
			
            if(srt) {
                thrust::stable_sort_by_key(temp, temp+RecCount, dev_per, thrust::greater<Str10>());			
			}	
			else {
//			    char* c = new char[10*RecCount];                
	//			char* dd = new char[10];
				
		//	    hipMemcpy( c, (void *) tmp, 10*RecCount, hipMemcpyDeviceToHost);
			//	cout << "Printing " << RecCount << endl;
			  //  for(int z = 0; z < RecCount; z++) {
                //   std::cout << "before " << z << " " << strncpy(dd,&c[z*10],10) << std::endl;				
			    //};				
				
                thrust::stable_sort_by_key(temp, temp+RecCount, dev_per);				        
				
			};	
		}	

}


void str_grp(char* d_char, unsigned int real_count, thrust::device_ptr<bool>& d_group, unsigned int len)
{


		if(len  == 1) {	
	        thrust::device_ptr<Str1> d_str((Str1*)d_char);
            thrust::transform(d_str, d_str + real_count -1 , d_str+1, d_group, thrust::not_equal_to<Str1>());			
		}
		else if(len  == 2) {	
	        thrust::device_ptr<Str2> d_str((Str2*)d_char);
            thrust::transform(d_str, d_str + real_count -1, d_str+1, d_group, thrust::not_equal_to<Str2>());			
		}
		else if(len  == 3) {	
	        thrust::device_ptr<Str3> d_str((Str3*)d_char);
            thrust::transform(d_str, d_str + real_count -1, d_str+1, d_group, thrust::not_equal_to<Str3>());			
		}
		else if(len  == 4) {	
	        thrust::device_ptr<Str4> d_str((Str4*)d_char);
            thrust::transform(d_str, d_str + real_count -1, d_str+1, d_group, thrust::not_equal_to<Str4>());			
		}
		else if(len  == 5) {	
	        thrust::device_ptr<Str5> d_str((Str5*)d_char);
            thrust::transform(d_str, d_str + real_count -1, d_str+1, d_group, thrust::not_equal_to<Str5>());			
		}
		else if(len  == 6) {	
	        thrust::device_ptr<Str6> d_str((Str6*)d_char);
            thrust::transform(d_str, d_str + real_count -1, d_str+1, d_group, thrust::not_equal_to<Str6>());			
		}
		else if(len  == 7) {	
	        thrust::device_ptr<Str7> d_str((Str7*)d_char);
            thrust::transform(d_str, d_str + real_count -1, d_str+1, d_group, thrust::not_equal_to<Str7>());			
		}
		else if(len  == 8) {	
	        thrust::device_ptr<Str8> d_str((Str8*)d_char);
            thrust::transform(d_str, d_str + real_count -1, d_str+1, d_group, thrust::not_equal_to<Str8>());			
		}
		else if(len  == 9) {	
	        thrust::device_ptr<Str9> d_str((Str9*)d_char);
            thrust::transform(d_str, d_str + real_count -1, d_str+1, d_group, thrust::not_equal_to<Str9>());			
		}
		else if(len  == 10) {	
	        thrust::device_ptr<Str10> d_str((Str10*)d_char);
            thrust::transform(d_str, d_str + real_count -1, d_str+1, d_group, thrust::not_equal_to<Str10>());			
		}
		

}


void str_copy_if(char* source, unsigned int mRecCount, char* dest, thrust::device_ptr<bool>& d_grp, unsigned int len)
{
		if(len  == 1) {	
	        thrust::device_ptr<Str1> d_str((Str1*)source);
			thrust::device_ptr<Str1> d_dest((Str1*)dest);			
			
            thrust::copy_if(d_str, d_str + mRecCount, d_grp, d_dest, thrust::identity<bool>());
		}
		else if(len  == 2) {	
	        thrust::device_ptr<Str2> d_str((Str2*)source);
			thrust::device_ptr<Str2> d_dest((Str2*)dest);
			
            thrust::copy_if(d_str, d_str + mRecCount, d_grp, d_dest, thrust::identity<bool>());
		}
		if(len  == 3) {	
	        thrust::device_ptr<Str3> d_str((Str3*)source);
			thrust::device_ptr<Str3> d_dest((Str3*)dest);
			
            thrust::copy_if(d_str, d_str + mRecCount, d_grp, d_dest, thrust::identity<bool>());
		}
		if(len  == 4) {	
	        thrust::device_ptr<Str4> d_str((Str4*)source);
			thrust::device_ptr<Str4> d_dest((Str4*)dest);
			
            thrust::copy_if(d_str, d_str + mRecCount, d_grp, d_dest, thrust::identity<bool>());
		}
		if(len  == 5) {	
	        thrust::device_ptr<Str5> d_str((Str5*)source);
			thrust::device_ptr<Str5> d_dest((Str5*)dest);
			
            thrust::copy_if(d_str, d_str + mRecCount, d_grp, d_dest, thrust::identity<bool>());
		}
		if(len  == 6) {	
	        thrust::device_ptr<Str6> d_str((Str6*)source);
			thrust::device_ptr<Str6> d_dest((Str6*)dest);
			
            thrust::copy_if(d_str, d_str + mRecCount, d_grp, d_dest, thrust::identity<bool>());
		}
		if(len  == 7) {	
	        thrust::device_ptr<Str7> d_str((Str7*)source);
			thrust::device_ptr<Str7> d_dest((Str7*)dest);
			
            thrust::copy_if(d_str, d_str + mRecCount, d_grp, d_dest, thrust::identity<bool>());
		}
		if(len  == 8) {	
	        thrust::device_ptr<Str8> d_str((Str8*)source);
			thrust::device_ptr<Str8> d_dest((Str8*)dest);
			
            thrust::copy_if(d_str, d_str + mRecCount, d_grp, d_dest, thrust::identity<bool>());
		}
		if(len  == 9) {	
	        thrust::device_ptr<Str9> d_str((Str9*)source);
			thrust::device_ptr<Str9> d_dest((Str9*)dest);
			
            thrust::copy_if(d_str, d_str + mRecCount, d_grp, d_dest, thrust::identity<bool>());
		}
		if(len  == 10) {	
	        thrust::device_ptr<Str10> d_str((Str10*)source);
			thrust::device_ptr<Str10> d_dest((Str10*)dest);
			
            thrust::copy_if(d_str, d_str + mRecCount, d_grp, d_dest, thrust::identity<bool>());
		}

}






       		
	
	