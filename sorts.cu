#include <thrust/device_vector.h>
#include "strings.cu"

using namespace std;

template <typename KeyType>
void update_permutation(thrust::device_vector<KeyType>& key, unsigned int* permutation, unsigned int RecCount, string SortType, KeyType* tmp)
{
    thrust::device_ptr<unsigned int> dev_per(permutation);
    // temporary storage for keys
    thrust::device_ptr<KeyType> temp(tmp);
    // permute the keys with the current reordering
    thrust::gather(dev_per, dev_per+RecCount, key.begin(), temp);

    // stable_sort the permuted keys and update the permutation
    if (SortType.compare("DESC") == 0 )
        thrust::stable_sort_by_key(temp, temp+RecCount, dev_per, thrust::greater<KeyType>());
    else
        thrust::stable_sort_by_key(temp, temp+RecCount, dev_per);
}

template <typename KeyType>
void update_permutation_host(KeyType* key, unsigned int* permutation, unsigned int RecCount, string SortType, KeyType* tmp)
{
	thrust::gather(permutation, permutation+RecCount, key, tmp);

    if (SortType.compare("DESC") == 0 )
        thrust::stable_sort_by_key(tmp, tmp+RecCount, permutation, thrust::greater<KeyType>());
    else
        thrust::stable_sort_by_key(tmp, tmp+RecCount, permutation);
}



template <typename KeyType>
void apply_permutation(thrust::device_vector<KeyType>& key, unsigned int* permutation, unsigned int RecCount, KeyType* tmp)
{
    thrust::device_ptr<unsigned int> dev_per(permutation);
    thrust::device_ptr<KeyType> temp(tmp);
    // copy keys to temporary vector
    thrust::copy(key.begin(), key.begin() + RecCount, temp);
    // permute the keys
    thrust::gather(dev_per, dev_per+RecCount, temp, key.begin());
}

template <typename KeyType>
void apply_permutation_host(KeyType* key, unsigned int* permutation, unsigned int RecCount, KeyType* res)
{
    //thrust::copy(key, key + RecCount, tmp);
    thrust::gather(permutation, permutation + RecCount, key, res);
}



void update_permutation_char(char* key, unsigned int* permutation, unsigned int RecCount, string SortType, char* tmp, unsigned int len)
{
    //thrust::device_ptr<unsigned int> dev_per(permutation);
    // temporary storage for keys
    //thrust::device_ptr<char> temp(tmp);
    // permute the keys with the current reordering
	
	str_gather((void*)permutation, RecCount, (void*)key, (void*)tmp, len);	

    // stable_sort the permuted keys and update the permutation
    if (SortType.compare("DESC") == 0 )
		str_sort(tmp, RecCount, permutation, 1, len);
    else
		str_sort(tmp, RecCount, permutation, 0, len);
}

void update_permutation_char_host(char* key, unsigned int* permutation, unsigned int RecCount, string SortType, char* tmp, unsigned int len)
{ 	
	str_gather_host(permutation, RecCount, (void*)key, (void*)tmp, len);	
	
    if (SortType.compare("DESC") == 0 )
		str_sort_host(tmp, RecCount, permutation, 1, len);
    else
		str_sort_host(tmp, RecCount, permutation, 0, len);		
		
}



void apply_permutation_char(char* key, unsigned int* permutation, unsigned int RecCount, char* tmp, unsigned int len)
{
     // copy keys to temporary vector    
	hipMemcpy( (void*)tmp, (void*) key, RecCount*len, hipMemcpyDeviceToDevice);		            
    // permute the keys
	str_gather((void*)permutation, RecCount, (void*)tmp, (void*)key, len);
}


void apply_permutation_char_host(char* key, unsigned int* permutation, unsigned int RecCount, char* res, unsigned int len)
{    
	//hipMemcpy( (void*)tmp, (void*) key, RecCount*len, hipMemcpyDeviceToDevice);		            
 	str_gather_host(permutation, RecCount, (void*)key, (void*)res, len);
}




