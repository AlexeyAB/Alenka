#include <string>

using namespace std;

template <typename KeyType>
void update_permutation(KeyType* key, unsigned int* permutation, unsigned int RecCount, string SortType, KeyType* tmp)
{
    thrust::device_ptr<KeyType> dev_key(key);
    thrust::device_ptr<unsigned int> dev_per(permutation);
    // temporary storage for keys
	
	thrust::device_ptr<KeyType> temp(tmp);
    // permute the keys with the current reordering
    thrust::gather(dev_per, dev_per+RecCount, dev_key, temp);

    // stable_sort the permuted keys and update the permutation
    if (SortType.compare("DESC") == 0 )
        thrust::stable_sort_by_key(temp, temp+RecCount, dev_per, thrust::greater<KeyType>());
    else 
        thrust::stable_sort_by_key(temp, temp+RecCount, dev_per);    
}


template <typename KeyType>
void apply_permutation(KeyType* key, unsigned int* permutation, unsigned int RecCount, KeyType* tmp)
{
    thrust::device_ptr<KeyType> dev_key(key);
    thrust::device_ptr<unsigned int> dev_per(permutation);
	
	thrust::device_ptr<KeyType> temp(tmp);
    // copy keys to temporary vector
    //thrust::copy(dev_key, dev_key+RecCount, temp);
	hipMemcpy((void *) tmp, (void *) key, RecCount*sizeof(KeyType), hipMemcpyDeviceToDevice);
	
    // permute the keys
    thrust::gather(dev_per, dev_per+RecCount, temp, dev_key);	
}

void apply_permutation_char(char* key, unsigned int* permutation, unsigned int RecCount, char* tmp)
{
    thrust::device_ptr<char> dev_key(key);
    thrust::device_ptr<unsigned int> dev_per(permutation);
    thrust::device_ptr<char> temp(tmp);
    // copy keys to temporary vector
    thrust::copy(dev_key, dev_key+RecCount, temp);

    // permute the keys
    thrust::gather(dev_per, dev_per+RecCount, temp, dev_key);
}


void update_permutation_char(char* key, unsigned int* permutation, unsigned int RecCount, char* tmp, string SortType)
{
    thrust::device_ptr<char> dev_key(key);
    thrust::device_ptr<unsigned int> dev_per(permutation);
    // temporary storage for keys
    thrust::device_ptr<char> temp(tmp);

    // permute the keys with the current reordering
    thrust::gather(dev_per, dev_per+RecCount, dev_key, temp);
	
	    // stable_sort the permuted keys and update the permutation
    if (SortType.compare("DESC") == 0 )
        thrust::stable_sort_by_key(temp, temp+RecCount, dev_per,  thrust::greater<char>());
    else
        thrust::stable_sort_by_key(temp, temp+RecCount, dev_per);
	}
