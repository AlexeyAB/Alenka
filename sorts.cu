/*
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */
 
#include <thrust/device_vector.h>
#include "strings.cu"

using namespace std;

template <typename KeyType>
void update_permutation(thrust::device_vector<KeyType>& key, unsigned int* permutation, unsigned int RecCount, string SortType, KeyType* tmp)
{
    thrust::device_ptr<unsigned int> dev_per(permutation);
    // temporary storage for keys
    thrust::device_ptr<KeyType> temp(tmp);
    // permute the keys with the current reordering
    thrust::gather(dev_per, dev_per+RecCount, key.begin(), temp);

    // stable_sort the permuted keys and update the permutation
    if (SortType.compare("DESC") == 0 )
        thrust::stable_sort_by_key(temp, temp+RecCount, dev_per, thrust::greater<KeyType>());
    else
        thrust::stable_sort_by_key(temp, temp+RecCount, dev_per);
}

template <typename KeyType>
void update_permutation_host(KeyType* key, unsigned int* permutation, unsigned int RecCount, string SortType, KeyType* tmp)
{
	thrust::gather(permutation, permutation+RecCount, key, tmp);

    if (SortType.compare("DESC") == 0 )
        thrust::stable_sort_by_key(tmp, tmp+RecCount, permutation, thrust::greater<KeyType>());
    else
        thrust::stable_sort_by_key(tmp, tmp+RecCount, permutation);
}



template <typename KeyType>
void apply_permutation(thrust::device_vector<KeyType>& key, unsigned int* permutation, unsigned int RecCount, KeyType* tmp)
{
    thrust::device_ptr<unsigned int> dev_per(permutation);
    thrust::device_ptr<KeyType> temp(tmp);
    // copy keys to temporary vector
    thrust::copy(key.begin(), key.begin() + RecCount, temp);
    // permute the keys
    thrust::gather(dev_per, dev_per+RecCount, temp, key.begin());
}

template <typename KeyType>
void apply_permutation_host(KeyType* key, unsigned int* permutation, unsigned int RecCount, KeyType* res)
{
    thrust::gather(permutation, permutation + RecCount, key, res);
}



void update_permutation_char(char* key, unsigned int* permutation, unsigned int RecCount, string SortType, char* tmp, unsigned int len)
{
	
	str_gather((void*)permutation, RecCount, (void*)key, (void*)tmp, len);	

    // stable_sort the permuted keys and update the permutation
    if (SortType.compare("DESC") == 0 )
		str_sort(tmp, RecCount, permutation, 1, len);
    else
		str_sort(tmp, RecCount, permutation, 0, len);
}

void update_permutation_char_host(char* key, unsigned int* permutation, unsigned int RecCount, string SortType, char* tmp, unsigned int len)
{ 	
	str_gather_host(permutation, RecCount, (void*)key, (void*)tmp, len);	
	
    if (SortType.compare("DESC") == 0 )
		str_sort_host(tmp, RecCount, permutation, 1, len);
    else
		str_sort_host(tmp, RecCount, permutation, 0, len);		
		
}



void apply_permutation_char(char* key, unsigned int* permutation, unsigned int RecCount, char* tmp, unsigned int len)
{
     // copy keys to temporary vector    
	hipMemcpy( (void*)tmp, (void*) key, RecCount*len, hipMemcpyDeviceToDevice);		            
    // permute the keys
	str_gather((void*)permutation, RecCount, (void*)tmp, (void*)key, len);
}


void apply_permutation_char_host(char* key, unsigned int* permutation, unsigned int RecCount, char* res, unsigned int len)
{    
 	str_gather_host(permutation, RecCount, (void*)key, (void*)res, len);
}




