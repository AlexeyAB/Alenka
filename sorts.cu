#include <string>
#include <thrust/device_vector.h>

using namespace std;

template <typename KeyType>
void update_permutation(KeyType* key, unsigned int* permutation, unsigned int RecCount, string SortType, KeyType* tmp)
{
    thrust::device_ptr<KeyType> dev_key(key);
    thrust::device_ptr<unsigned int> dev_per(permutation);
    // temporary storage for keys

    thrust::device_ptr<KeyType> temp(tmp);
    // permute the keys with the current reordering
    thrust::gather(dev_per, dev_per+RecCount, dev_key, temp);

    // stable_sort the permuted keys and update the permutation
    if (SortType.compare("DESC") == 0 )
        thrust::stable_sort_by_key(temp, temp+RecCount, dev_per, thrust::greater<KeyType>());
    else
        thrust::stable_sort_by_key(temp, temp+RecCount, dev_per);
}


template <typename KeyType>
void apply_permutation(KeyType* key, unsigned int* permutation, unsigned int RecCount, KeyType* tmp)
{
    thrust::device_ptr<KeyType> dev_key(key);
    thrust::device_ptr<unsigned int> dev_per(permutation);

    thrust::device_ptr<KeyType> temp(tmp);
    // copy keys to temporary vector
    //thrust::copy(dev_key, dev_key+RecCount, temp);
    hipMemcpy((void *) tmp, (void *) key, RecCount*sizeof(KeyType), hipMemcpyDeviceToDevice);

    // permute the keys
    thrust::gather(dev_per, dev_per+RecCount, temp, dev_key);
}

void apply_permutation_char(thrust::device_vector<char>& key, unsigned int* permutation, unsigned int RecCount, char* tmp)
{
    thrust::device_ptr<unsigned int> dev_per(permutation);
    thrust::device_ptr<char> temp(tmp);
    // copy keys to temporary vector
    thrust::copy(key.begin(), key.begin()+RecCount, temp);

    // permute the keys
    thrust::gather(dev_per, dev_per+RecCount, temp, key.begin());
}


void update_permutation_char(thrust::device_vector<char>& key, unsigned int* permutation, unsigned int RecCount, char* tmp, string SortType)
{
    thrust::device_ptr<unsigned int> dev_per(permutation);
    // temporary storage for keys
    thrust::device_ptr<char> temp(tmp);

    // permute the keys with the current reordering
    thrust::gather(dev_per, dev_per+RecCount, key.begin(), temp);

    // stable_sort the permuted keys and update the permutation
    if (SortType.compare("DESC") == 0 )
        thrust::stable_sort_by_key(temp, temp+RecCount, dev_per,  thrust::greater<char>());
    else
        thrust::stable_sort_by_key(temp, temp+RecCount, dev_per);
}
